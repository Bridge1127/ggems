#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

#ifndef GLOBAL_CU
#define GLOBAL_CU
#include "global.cuh"

// Some usefull functions

// Reset the GPU
void reset_gpu_device()
{
    printf ( "[\033[32;01mok\033[00m] Reset device .. \n" );
    hipDeviceReset();
}

// comes from "cuda by example" book
void HandleError ( hipError_t err,
                   const char *file,
                   int line )
{
    if ( err != hipSuccess )
    {
        printf ( "%s in %s at line %d\n", hipGetErrorString ( err ),
                 file, line );
        exit ( EXIT_FAILURE );
    }
}

// comes from "cuda programming" book
__host__ void cuda_error_check ( const char * prefix, const char * postfix )
{
    if ( hipPeekAtLastError() != hipSuccess )
    {
        printf ( "\n%s%s%s\n",prefix, hipGetErrorString ( hipGetLastError() ),postfix );
        hipDeviceReset();
        exit ( EXIT_FAILURE );
    }
}

// Set a GPU device
void set_gpu_device ( int deviceChoice, f32 minversion )
{

    int deviceCount = 0;
    hipGetDeviceCount ( &deviceCount );

    if ( deviceCount == 0 )
    {
        printf ( "[\033[31;03mWARNING\033[00m] There is no device supporting CUDA\n" );
        exit ( EXIT_FAILURE );
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties ( &prop, deviceChoice%deviceCount );

    if ( prop.major<minversion )
    {
        printf ( "[\033[31;03mWARNING\033[00m] Your device is not compatible with %1.1f version\n",minversion );
        exit ( EXIT_FAILURE );
    }

    hipSetDevice ( deviceChoice%deviceCount );
    printf ( "[\033[32;01mok\033[00m] \033[32;01m%s\033[00m found\n", prop.name );

}



// Print out for error
void print_error ( std::string msg )
{
    printf ( "\033[31;03m[ERROR] - %s\033[00m", msg.c_str() );
}

// Print out for warning
void print_warning ( std::string msg )
{
    printf ( "\033[33;03m[WARNING] - %s\033[00m", msg.c_str() );
}

// Print out run time
void print_time ( std::string txt, f64 t )
{

    f64 res;
    ui32 time_h = ( ui32 ) ( t / 3600.0 );
    res = t - ( time_h*3600.0 );
    ui32 time_m = ( ui32 ) ( res / 60.0 );
    res -= ( time_m * 60.0 );
    ui32 time_s = ( ui32 ) ( res );
    res -= time_s;
    ui32 time_ms = ( ui32 ) ( res*1000.0 );

    printf ( "[\033[32;01mRun time\033[00m] %s: ", txt.c_str() );

    if ( time_h != 0 ) printf ( "%i h ", time_h );
    if ( time_m != 0 ) printf ( "%i m ", time_m );
    if ( time_s != 0 ) printf ( "%i s ", time_s );
    printf ( "%i ms\n", time_ms );

}

// Print out memory usage
void print_memory ( std::string txt, ui32 t )
{

    std::vector<std::string> pref;
    pref.push_back ( "B" );
    pref.push_back ( "kB" );
    pref.push_back ( "MB" );
    pref.push_back ( "GB" );

    ui32 iemem = ( ui32 ) ( log ( t ) / log ( 1000 ) );
    f32 mem = f32 ( t ) / ( pow ( 1000, iemem ) );

    printf ( "[\033[34;01mMemory usage\033[00m] %s: %5.2f %s\n", txt.c_str(), mem, pref[iemem].c_str() );

}

// Print GGEMS banner
void print_banner(std::string institution, std::string exp_day, std::string exp_month, std::string exp_year, std::string version) {

    printf("      \033[32;01m____\033[00m                  \n");
    printf(".--. \033[32;01m/\\__/\\\033[00m .--.            \n");
    printf("`\033[33;01mO\033[00m  \033[32;01m/ /  \\ \\\033[00m  .`     GGEMS %s  \n", version.c_str());
    printf("  `-\033[32;01m| |  | |\033[00m\033[33;01mO\033[00m`              \n");
    printf("   -\033[32;01m|\033[00m`\033[32;01m|\033[00m..\033[32;01m|\033[00m`\033[32;01m|\033[00m-        License:  \n");
    printf(" .` \033[32;01m\\\033[00m.\033[32;01m\\__/\033[00m.\033[32;01m/\033[00m `.        %s       \n", institution.c_str());
    printf("'.-` \033[32;01m\\/__\\/\033[00m `-.'       %s-%s-%s \n", exp_day.c_str(), exp_month.c_str(), exp_year.c_str());
    printf("\n");

}

// Abort the current simulation
void exit_simulation()
{
    printf ( "\n[\033[31;03mSimulation aborded\033[00m]\n" );
    exit ( EXIT_FAILURE );
}

/*
// Create a color
Color make_color(f32 r, f32 g, f32 b) {
    Color c;
    c.r = r;
    c.g = g;
    c.b = b;
    return c;
}
*/

// Get time
f64 get_time()
{
    timeval tv;
    gettimeofday ( &tv, NULL );
    return tv.tv_sec + tv.tv_usec / 1000000.0;
}







#endif
