#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

#ifndef GLOBAL_CU
#define GLOBAL_CU
#include "global.cuh"

// Some usefull functions

// Reset the GPU
void reset_gpu_device()
{
    hipDeviceReset();
}

// comes from "cuda by example" book
void HandleError ( hipError_t err,
                   const char *file,
                   int line )
{
    if ( err != hipSuccess )
    {
        printf ( "%s in %s at line %d\n", hipGetErrorString ( err ),
                 file, line );
        exit ( EXIT_FAILURE );
    }
}

// comes from "cuda programming" book
__host__ void cuda_error_check ( const char * prefix, const char * postfix )
{
    if ( hipPeekAtLastError() != hipSuccess )
    {
        printf ( "\n%s%s%s\n",prefix, hipGetErrorString ( hipGetLastError() ),postfix );
        hipDeviceReset();
        exit ( EXIT_FAILURE );
    }
}

// Set a GPU device
void set_gpu_device ( int deviceChoice )
{

    f32 minversion = 3.0;

    i32 deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount( &deviceCount );

    if (error_id != hipSuccess)
    {
        GGcerr << "hipGetDeviceCount returned " << ( i32 ) error_id
               << " " << hipGetErrorString(error_id) << GGendl;
        exit_simulation();
    }

    if ( deviceCount == 0 )
    {
        GGcerr << "There are no available device(s) that support CUDA" << GGendl;
        exit_simulation();
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties ( &prop, deviceChoice%deviceCount );

    if ( prop.major < minversion )
    {
        GGcerr << "Your device is not compatible with " << minversion << " version" << GGendl;
        exit_simulation();
    }

    hipSetDevice ( deviceChoice%deviceCount );
    GGcout << "GPU found: " << prop.name << " (id: " << deviceChoice%deviceCount << ") " << GGendl;
    GGnewline();

    
/*    // Debug, pour calculer automatiquement le nbre de threads  et blocks en fonction du GPU
    int numBlocks;        // Occupancy in terms of active blocks
    int blockSize = 32;

    // These variables are used to convert occupancy to warps

    int activeWarps;
    int maxWarps;

    
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        MyKernel,
        blockSize,
        0);

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
    
    std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;*/
    
}



// Print out for error
void print_error ( std::string msg )
{
    printf ( "\033[31;03m[ERROR] - %s\033[00m", msg.c_str() );
}

// Print out for warning
void print_warning ( std::string msg )
{
    printf ( "\033[33;03m[WARNING] - %s\033[00m", msg.c_str() );
}

// Print out run time
void GGcout_time ( std::string txt, f64 t )
{

    f64 res;
    ui32 time_h = ( ui32 ) ( t / 3600.0 );
    res = t - ( time_h*3600.0 );
    ui32 time_m = ( ui32 ) ( res / 60.0 );
    res -= ( time_m * 60.0 );
    ui32 time_s = ( ui32 ) ( res );
    res -= time_s;
    ui32 time_ms = ( ui32 ) ( res*1000.0 );

    printf ( "[GGEMS] %s: ", txt.c_str() );

    if ( time_h != 0 ) printf ( "%i h ", time_h );
    if ( time_m != 0 ) printf ( "%i m ", time_m );
    if ( time_s != 0 ) printf ( "%i s ", time_s );
    printf ( "%i ms\n", time_ms );

}

// Print date and time
void GGcout_timestamp ()
{
    time_t t = time(NULL);
    struct tm tm = *localtime(&t);

    printf("[GGEMS] %d-%d-%d %02d:%02d:%02d\n", tm.tm_year + 1900, tm.tm_mon + 1, tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);
}

// Print some version information
void GGcout_version ()
{
    i32 Version = 0;
    //i32 DrvVersion = 0;

    hipRuntimeGetVersion( &Version );
    //hipRuntimeGetVersion( &DrvVersion );

    std::string VerTxt = (Version) ? std::to_string( Version/1000.0 ) : "Unknown";
    //std::string DrvTxt = (DrvVersion) ? std::to_string( DrvVersion ) : "Unknown";

    GGcout << "GCC: " << __GNUC__ << "." << __GNUC_MINOR__
           << " NVCC: " << VerTxt
           << GGendl;
}

// Print out memory usage
void GGcout_mem ( std::string txt, ui64 valmem )
{

    std::vector<std::string> pref;
    pref.push_back ( "B" );
    pref.push_back ( "kB" );
    pref.push_back ( "MB" );
    pref.push_back ( "GB" );

    ui32 iemem = ( ui32 ) ( log ( valmem ) / log ( 1000 ) );
    f32 mem = f32 ( f64 ( valmem ) / ( pow ( 1000, iemem ) ) );

    printf ( "[GGEMS] %s: %5.2f %s\n", txt.c_str(), mem, pref[iemem].c_str() );
}


std::string Green_str( std::string txt )
{
    return "\033[32;01m" + txt + "\033[00m";
}

std::string Check_str( std::string txt )
{
    return "[X] " + txt;
}

std::string Red_str( std::string txt )
{
    return "\033[31;03m" + txt + "\033[00m";
}

std::string NoCheck_str( std::string txt )
{
    return "[ ] " + txt;
}

std::string Energy_str( f32 E )
{
    E /= eV;

    std::vector<std::string> pref;
    pref.push_back ( "eV" );
    pref.push_back ( "keV" );
    pref.push_back ( "MeV" );
    pref.push_back ( "GeV" );

    ui32 exp = ( ui32 ) ( log ( E ) / log ( 1000 ) );
    f32 val = f32 ( E ) / ( pow ( 1000, exp ) );

    char tmp[ 100 ];
    sprintf( tmp, "%5.2f %s", val, pref[ exp ].c_str());

    return std::string( tmp );
}

std::string Range_str( f32 range )
{
    range /= nm;

    std::vector<std::string> pref;
    pref.push_back ( "nm" );
    pref.push_back ( "um" );
    pref.push_back ( "mm" );
    pref.push_back ( "m" );

    ui32 exp = ( ui32 ) ( log ( range ) / log ( 1000 ) );
    f32 val = f32 ( range ) / ( pow ( 1000, exp ) );

    char tmp[ 100 ];
    sprintf( tmp, "%5.2f %s", val, pref[ exp ].c_str());

    return std::string( tmp );
}

// Print params
void GGcout_params( GlobalSimulationParametersData params )
{

    if ( params.display_in_color )
    {
        printf("\n");
        printf("[GGEMS] Physics list:\n");
        printf("[GGEMS]    Gamma: %s   %s   %s\n", ( params.physics_list[ PHOTON_COMPTON ] ) ? Green_str("Compton").c_str() : Red_str("Compton").c_str(),
                             ( params.physics_list[ PHOTON_PHOTOELECTRIC ] ) ? Green_str("Photoelectric").c_str() : Red_str("Photoelectric").c_str(),
                                            ( params.physics_list[ PHOTON_RAYLEIGH ] ) ? Green_str("Rayleigh").c_str() : Red_str("Rayleigh").c_str() );

        printf("[GGEMS]    Electron: %s   %s   %s\n", ( params.physics_list[ ELECTRON_IONISATION ] ) ? Green_str("Ionisation").c_str() : Red_str("Ionisation").c_str(),
                                      ( params.physics_list[ ELECTRON_BREMSSTRAHLUNG ] ) ? Green_str("Bremsstrahlung").c_str() : Red_str("Bremsstrahlung").c_str(),
                                       ( params.physics_list[ ELECTRON_MSC ] ) ? Green_str("Multiple scattering").c_str() : Red_str("Multiple scattering").c_str() );

        printf("[GGEMS]    Tables: MinE %s   MaxE %s   Nb of energy bin %i\n", Energy_str( params.cs_table_min_E ).c_str(),
                                                                           Energy_str( params.cs_table_max_E ).c_str(),
                                                                           params.cs_table_nbins );
        printf("[GGEMS]    Range cuts: Gamma %s   Electron %s\n", Range_str( params.photon_cut ).c_str(),
                                                                 Range_str( params.electron_cut ).c_str() );

        printf("[GGEMS] Secondary particles:\n");
        printf("[GGEMS]    Particles: %s   %s\n", ( params.secondaries_list[ PHOTON ] ) ? Green_str("Gamma").c_str() : Red_str("Gamma").c_str(),
                                                ( params.secondaries_list[ ELECTRON ] ) ? Green_str("Electron").c_str() : Red_str("Electron").c_str() );

        printf("[GGEMS]    Levels: %i\n", params.nb_of_secondaries);

        printf("[GGEMS] Geometry tolerance:\n");
        printf("[GGEMS]    Range: %s\n", Range_str( params.geom_tolerance ).c_str() );
        printf("[GGEMS] Simulation:\n");
        printf("[GGEMS]    Device target: %s\n", ( params.device_target == CPU_DEVICE ) ? "CPU" : "GPU" );
        printf("[GGEMS]    Total Nb of particles: %i\n", params.nb_of_particles);
        printf("[GGEMS]    Size of batch: %i\n", params.size_of_particles_batch);
        printf("[GGEMS]    Nb of batches: %i\n", params.nb_of_batches);
        printf("[GGEMS]    Seed value %i\n", params.seed);

        printf("\n");
    }
    else
    {
        printf("\n");
        printf("[GGEMS] Physics list:\n");
        printf("[GGEMS]    Gamma: %s   %s   %s\n", ( params.physics_list[ PHOTON_COMPTON ] ) ? Check_str("Compton").c_str() : NoCheck_str("Compton").c_str(),
                             ( params.physics_list[ PHOTON_PHOTOELECTRIC ] ) ? Check_str("Photoelectric").c_str() : NoCheck_str("Photoelectric").c_str(),
                                            ( params.physics_list[ PHOTON_RAYLEIGH ] ) ? Check_str("Rayleigh").c_str() : NoCheck_str("Rayleigh").c_str() );

        printf("[GGEMS]    Electron: %s   %s   %s\n", ( params.physics_list[ ELECTRON_IONISATION ] ) ? Check_str("Ionisation").c_str() : NoCheck_str("Ionisation").c_str(),
                                      ( params.physics_list[ ELECTRON_BREMSSTRAHLUNG ] ) ? Check_str("Bremsstrahlung").c_str() : NoCheck_str("Bremsstrahlung").c_str(),
                                       ( params.physics_list[ ELECTRON_MSC ] ) ? Check_str("Multiple scattering").c_str() : NoCheck_str("Multiple scattering").c_str() );

        printf("[GGEMS]    Tables: MinE %s   MaxE %s   Nb of energy bin %i\n", Energy_str( params.cs_table_min_E ).c_str(),
                                                                           Energy_str( params.cs_table_max_E ).c_str(),
                                                                           params.cs_table_nbins );
        printf("[GGEMS]    Range cuts: Gamma %s   Electron %s\n", Range_str( params.photon_cut ).c_str(),
                                                                 Range_str( params.electron_cut ).c_str() );

        printf("[GGEMS] Secondary particles:\n");
        printf("[GGEMS]    Particles: %s   %s\n", ( params.secondaries_list[ PHOTON ] ) ? Check_str("Gamma").c_str() : NoCheck_str("Gamma").c_str(),
                                                ( params.secondaries_list[ ELECTRON ] ) ? Check_str("Electron").c_str() : NoCheck_str("Electron").c_str() );

        printf("[GGEMS]    Levels: %i\n", params.nb_of_secondaries);

        printf("[GGEMS] Geometry tolerance:\n");
        printf("[GGEMS]    Range: %s\n", Range_str( params.geom_tolerance ).c_str() );
        printf("[GGEMS] Simulation:\n");
        printf("[GGEMS]    Device target: %s\n", ( params.device_target == CPU_DEVICE ) ? "CPU" : "GPU" );
        printf("[GGEMS]    Total Nb of particles: %i\n", params.nb_of_particles);
        printf("[GGEMS]    Size of batch: %i\n", params.size_of_particles_batch);
        printf("[GGEMS]    Nb of batches: %i\n", params.nb_of_batches);
        printf("[GGEMS]    Seed value %i\n", params.seed);

        printf("\n");
    }



}

// Empty line
void GGnewline( )
{
    printf("\n");
}

//ui64 nb_of_particles;
//ui64 size_of_particles_batch;
//ui32 nb_of_batches;

//ui8 device_target;
//ui32 gpu_id;
//ui32 gpu_block_size;
//ui32 gpu_grid_size;
//f32 time;
//ui32 seed;




// Print GGEMS banner
void print_banner( std::string institution, std::string exp_day, std::string exp_month, std::string exp_year,
                   std::string version, GlobalSimulationParametersData params )
{
    if ( params.display_in_color )
    {
        printf("      \033[32;01m____\033[00m                  \n");
        printf(".--. \033[32;01m/\\__/\\\033[00m .--.            \n");
        printf("`\033[33;01mO\033[00m  \033[32;01m/ /  \\ \\\033[00m  .`     GGEMS %s  \n", version.c_str());
        printf("  `-\033[32;01m| |  | |\033[00m\033[33;01mO\033[00m`              \n");
        printf("   -\033[32;01m|\033[00m`\033[32;01m|\033[00m..\033[32;01m|\033[00m`\033[32;01m|\033[00m-        License:  \n");
        printf(" .` \033[32;01m\\\033[00m.\033[32;01m\\__/\033[00m.\033[32;01m/\033[00m `.        %s       \n", institution.c_str());
        printf("'.-` \033[32;01m\\/__\\/\033[00m `-.'       %s-%s-%s \n", exp_day.c_str(), exp_month.c_str(), exp_year.c_str());
        printf("\n");
    }
    else
    {
        printf("      ____                  \n");
        printf(".--. /\\__/\\ .--.            \n");
        printf("`O  / /  \\ \\  .`     GGEMS %s  \n", version.c_str());
        printf("  `-| |  | |O`              \n");
        printf("   -|`|..|`|-        License:  \n");
        printf(" .` \\.\\__/./ `.        %s       \n", institution.c_str());
        printf("'.-` \\/__\\/ `-.'       %s-%s-%s \n", exp_day.c_str(), exp_month.c_str(), exp_year.c_str());
        printf("\n");
    }
}

// Abort the current simulation
void exit_simulation()
{
    printf ( "\n[\033[31;03mSimulation aborded\033[00m]\n" );
    exit ( EXIT_FAILURE );
}

/*
// Create a color
Color make_color(f32 r, f32 g, f32 b) {
    Color c;
    c.r = r;
    c.g = g;
    c.b = b;
    return c;
}
*/

// Get time
f64 get_time()
{
    timeval tv;
    gettimeofday ( &tv, NULL );
    return tv.tv_sec + tv.tv_usec / 1000000.0;
}







#endif
