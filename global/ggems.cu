// GGEMS Copyright (C) 2015

/*!
 * \file ggems.cuh
 * \brief Main header of GGEMS lib
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 13 novembre 2015
 *
 * Header of the main GGEMS lib
 *
 */

#ifndef GGEMS_CU
#define GGEMS_CU

#include "ggems.cuh"

////// :: GGEMS Const/Dest ::

GGEMS::GGEMS() {

    // Init physics list and secondaries list
    m_parameters_h.physics_list = (bool*)malloc(NB_PROCESSES*sizeof(bool));
    m_parameters_h.secondaries_list = (bool*)malloc(NB_PARTICLES*sizeof(bool));
    
    ui32 i = 0;
    while (i < NB_PROCESSES) {
        m_parameters_h.physics_list[i] = DISABLED;
        ++i;
    }
    i = 0;
    while (i < NB_PARTICLES) {
        m_parameters_h.secondaries_list[i] = DISABLED;
        ++i;
    }

    // Parameters
    m_parameters_h.nb_of_particles = 0;
    m_parameters_h.size_of_particles_batch = 0;
    m_parameters_h.nb_of_batches = 0;
    m_parameters_h.time = 0;
    m_parameters_h.seed = 0;
    m_parameters_h.cs_table_nbins = 0;
    m_parameters_h.cs_table_min_E = 0;
    m_parameters_h.cs_table_max_E = 0;

    // Init by default others parameters
    m_parameters_h.device_target = CPU_DEVICE;
    m_parameters_h.gpu_id = 0;
    m_parameters_h.gpu_block_size = 512;

    // Others parameters
    m_parameters_h.display_run_time = DISABLED;
    m_parameters_h.display_memory_usage = DISABLED;

}

GGEMS::~GGEMS() {
    //delete m_parameters_h;
    //delete m_source;
}

////// :: Setting ::

/// Params

// Set the hardware used for the simulation CPU or GPU (CPU by default)
void GGEMS::set_hardware_target(std::string value) {
    if (value == "GPU") {
        m_parameters_h.device_target = GPU_DEVICE;
    } else {
        m_parameters_h.device_target = CPU_DEVICE;
    }
}

// Set the GPU id
void GGEMS::set_GPU_ID(ui32 valid) {
    m_parameters_h.gpu_id = valid;
}

// Set the GPU block size
void GGEMS::set_GPU_block_size(ui32 val) {
    m_parameters_h.gpu_block_size = val;
}

// Add a process to the physics list
void GGEMS::set_process(std::string process_name) {

    if (process_name == "Compton") {
        m_parameters_h.physics_list[PHOTON_COMPTON] = ENABLED;

    } else if (process_name == "PhotoElectric") {
        m_parameters_h.physics_list[PHOTON_PHOTOELECTRIC] = ENABLED;

    } else if (process_name == "Rayleigh") {
        m_parameters_h.physics_list[PHOTON_RAYLEIGH] = ENABLED;

    } else if (process_name == "eIonisation") {
        m_parameters_h.physics_list[ELECTRON_IONISATION] = ENABLED;

    } else if (process_name == "eBremsstrahlung") {
        m_parameters_h.physics_list[ELECTRON_BREMSSTRAHLUNG] = ENABLED;

    } else if (process_name == "eMultipleScattering") {
        m_parameters_h.physics_list[ELECTRON_MSC] = ENABLED;

    } else {
        print_warning("This process is unknown!!\n");
        printf("     -> %s\n", process_name.c_str());
        exit_simulation();
    }
}

// Enable the simulation of a particular secondary particle
void GGEMS::set_secondary(std::string pname) {

    if (pname == "Photon") {
        m_parameters_h.secondaries_list[PHOTON] = ENABLED;
    } else if (pname == "Electron") {
        m_parameters_h.secondaries_list[ELECTRON] = ENABLED;
    } else {
        print_warning("Secondary particle type is unknow!!");
        printf("     -> %s\n", pname.c_str());
        exit_simulation();
    }
}

// Set the number of particles required for the simulation
void GGEMS::set_number_of_particles(ui64 nb) {
    m_parameters_h.nb_of_particles = nb;
}

// Set the size of particles batch
void GGEMS::set_size_of_particles_batch(ui64 nb) {
    m_parameters_h.size_of_particles_batch = nb;
}

// Set parameters to generate cross sections table
void GGEMS::set_CS_table_nbins(ui32 valbin) {m_parameters_h.cs_table_nbins = valbin;}
void GGEMS::set_CS_table_E_min(f32 valE) {m_parameters_h.cs_table_min_E = valE;}
void GGEMS::set_CS_table_E_max(f32 valE) {m_parameters_h.cs_table_max_E = valE;}

// Set the seed number
void GGEMS::set_seed(ui32 vseed) {
    m_parameters_h.seed = vseed;
}

/// Sources
void GGEMS::set_source(PointSource *aSource) {
    m_sources.set_source(aSource);
}

/// Utils

// Display run time
void GGEMS::set_display_run_time() {
    m_parameters_h.display_run_time = ENABLED;
}

// Display memory usage
void GGEMS::set_display_memory_usage() {
    m_parameters_h.display_memory_usage = ENABLED;
}

////// :: Private functions ::

// Copy the global simulation parameters to the GPU
void GGEMS::m_copy_parameters_cpu2gpu() {

    // Mem allocation
    HANDLE_ERROR( hipMalloc((void**) &m_parameters_d.physics_list, NB_PROCESSES*sizeof(bool)) );
    HANDLE_ERROR( hipMalloc((void**) &m_parameters_d.secondaries_list, NB_PARTICLES*sizeof(bool)) );

    // Copy data
    HANDLE_ERROR( hipMemcpy(m_parameters_d.physics_list, m_parameters_h.physics_list,
                         sizeof(bool)*NB_PROCESSES, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(m_parameters_d.secondaries_list, m_parameters_h.secondaries_list,
                         sizeof(bool)*NB_PARTICLES, hipMemcpyHostToDevice) );

    m_parameters_d.nb_of_particles = m_parameters_h.nb_of_particles;
    m_parameters_d.size_of_particles_batch = m_parameters_h.size_of_particles_batch;
    m_parameters_d.nb_of_batches = m_parameters_h.nb_of_batches;

    m_parameters_d.device_target = m_parameters_h.device_target;
    m_parameters_d.gpu_id = m_parameters_h.gpu_id;
    m_parameters_d.gpu_block_size = m_parameters_h.gpu_block_size;

    m_parameters_d.time = m_parameters_h.time;
    m_parameters_d.seed = m_parameters_h.seed;

    m_parameters_d.display_run_time = m_parameters_h.display_run_time;
    m_parameters_d.display_memory_usage = m_parameters_h.display_memory_usage;

    m_parameters_d.cs_table_nbins = m_parameters_h.cs_table_nbins;
    m_parameters_d.cs_table_min_E = m_parameters_h.cs_table_min_E;
    m_parameters_d.cs_table_max_E = m_parameters_h.cs_table_max_E;
}

////// :: Main functions ::

// Init simualtion
void GGEMS::init_simulation() {

    // Run time
    f64 t_start = 0;
    if (m_parameters_h.display_run_time) {
        t_start = get_time();
    }

    // Memory usage
    ui32 mem = 0;

    // CPU PRNG
    srand(m_parameters_h.seed);

    // Get the number of batch required
    m_parameters_h.nb_of_batches = m_parameters_h.nb_of_particles / m_parameters_h.size_of_particles_batch;

    // Init the GPU if need
    if (m_parameters_h.device_target == GPU_DEVICE) {
        // Reset device
        reset_gpu_device();

        // Set the gpu id
        set_gpu_device(m_parameters_h.gpu_id);
    }

    // Copy global params to GPU
    if (m_parameters_h.device_target == GPU_DEVICE) {
        m_copy_parameters_cpu2gpu();
    }

    /// Init Sources ///////////////
    m_sources.initialize(m_parameters_h);

    /// Material handling ////////////////////////////

    // Load data
    // TODO

    // Build data based on geometry
    // TODO

    // Copy data to GPU
    if (m_parameters_h.device_target == GPU_DEVICE) {
        m_materials.copy_materials_table_cpu2gpu();
    }

    /// Stack handling ///////////////////////////////

    // Init stack params
    m_particles.set_stack_size(m_parameters_h.size_of_particles_batch);
    m_particles.set_seed(m_parameters_h.seed);
    // Init stack (CPU)
    m_particles.cpu_malloc_stack();
    m_particles.cpu_init_stack_seed();
    // If GPU, init
    if (m_parameters_h.device_target == GPU_DEVICE) {
        m_particles.gpu_malloc_stack();
        m_particles.copy_seed_cpu2gpu();
    }

    // Mem usage
    if (m_parameters_h.display_memory_usage) {
        ui32 mem_part = 91*m_particles.stack_h.size + 4;
        mem += mem_part;
        print_memory("Particles stack", mem_part);
    }

    /// Cross sections /////////////////////////////

    // Init Cross sections and physics table
    m_cross_sections.build_table(m_materials.mat_table_h, m_parameters_h);

    // Mem usage
    if (m_parameters_h.display_memory_usage) {
        ui32 n = m_cross_sections.photon_CS_table_h.nb_bins;
        ui32 k = m_cross_sections.photon_CS_table_h.nb_mat;
        ui32 mem_cs = 4*n + 12*n*k + 12*n*101 + 16;
        mem += mem_cs;
        print_memory("Cross sections", mem_cs);

        // Add CS from others particles
    }

    // If GPU
    if (m_parameters_h.device_target == GPU_DEVICE) {
        m_cross_sections.copy_cs_table_cpu2gpu();
    }
    //cs_tables.print();

    // Mem usage
    if (m_parameters_h.display_memory_usage) {
        // Parameters
        ui32 mem_params = NB_PROCESSES+NB_PARTICLES+30;
        mem += mem_params;
        print_memory("Parameters", mem_params);

        /*
        // Geometry
        ui32 mem_geom = 4*geometry.world.ptr_objects_dim + 4*geometry.world.size_of_objects_dim +
                4*geometry.world.data_objects_dim + 4*geometry.world.ptr_nodes_dim +
                4*geometry.world.size_of_nodes_dim + 4*geometry.world.child_nodes_dim +
                4*geometry.world.mother_node_dim + 32;
        mem += mem_geom;
        print_memory("Geometry", mem_geom);
        */

        // Materials
        ui32 n = m_materials.mat_table_h.nb_materials;
        ui32 k = m_materials.mat_table_h.nb_elements_total;
        ui32 mem_mat = 10*k + 80*n + 8;
        mem += mem_mat;
        print_memory("Materials", mem_mat);

        /*
        // Sources
        ui32 mem_src = 4*sources.sources.ptr_sources_dim + 4*sources.sources.data_sources_dim +
                4*sources.sources.seeds_dim + 16;
        mem += mem_src;
        print_memory("Sources", mem_src);
        */
    }

    /*
    /// Digitizer /////////////////////////////////

    // init Digitizer
    if (parameters.digitizer_flag) {
        digitizer.cpu_init_pulses(particles.stack.size);

        if (target == GPU_DEVICE) {
            digitizer.gpu_init_pulses(particles.stack.size);
        }

        // If projection acquisition
        if (digitizer.flag_projection || digitizer.flag_spect_proj) {
            digitizer.init_projection();
        }

        // Mem usage
        if (display_memory_usage_flag) {
            ui32 mem_singles = 64*digitizer.pulses.size + 4;
            mem += mem_singles;
            print_memory("Singles", mem_singles);
        }
    }
    */

    // Run time
    if (m_parameters_h.display_run_time) {
        print_time("Initialization", get_time()-t_start);
    }

    // Mem usage
    if (m_parameters_h.display_memory_usage) {
        print_memory("Total memory usage", mem);
    }
}

/*

////// :: Main functions ::

// Generate particle based on the sources (CPU version)
void SimulationBuilder::primaries_generator() {

    /// CPU ///////////////////////////////////
    if (target == CPU_DEVICE) {
//#ifdef DEBUG
        printf("CPU: primaries generator\n");
//#endif

        f64 t_start;
        if (display_run_time_flag) t_start = get_time();

        // Loop over particle slot
        ui32 id = 0;
        ui32 is = 0;
        printf("particles stack size %d \n",particles.stack.size);
        while (id < particles.stack.size) {

            // TODO - Generic and multi-sources
            //      Read CDF sources
            //      Rnd sources
            is = 0; // first source

            // Get a new particle
            get_primaries(sources.sources, particles.stack, is, id);

            // Next particle
            ++id;

        } // id

        // History record (use only for VRML view)
        if (history.record_flag == ENABLED) {
            id=0; while (id < particles.stack.size) {
                // Record the first position for the tracking history
                history.cpu_new_particle_track(PHOTON);
                history.cpu_record_a_step(particles.stack, id);
                ++id;
            }
        }

        if (display_run_time_flag) {
            print_time("Primaries generator", get_time()-t_start);
        }

    /// GPU /////////////////////////////////////
    } else {

#ifdef DEBUG
        printf("GPU: primaries generator\n");
#endif

        hipEvent_t t_start, t_stop;
        if (display_run_time_flag) {
            hipEventCreate(&t_start);
            hipEventCreate(&t_stop);
            hipEventRecord(t_start);
        }

        // TODO - Generic and multi-sources
        //      Read CDF sources
        //      Rnd sources
        ui32 is = 0; // first source

        // Kernel
        dim3 threads, grid;
        threads.x = gpu_block_size;
        grid.x = (particles.dstack.size + gpu_block_size - 1) / gpu_block_size;
        
        kernel_get_primaries<<<grid, threads>>>(sources.dsources, particles.dstack, is);
        cuda_error_check("Error ", " Kernel_primaries_generator");

        if (display_run_time_flag) {
            hipEventRecord(t_stop);
            hipEventSynchronize(t_stop);
            f32 time_ms = 0;
            hipEventElapsedTime(&time_ms, t_start, t_stop);
            print_time("Primaries generator", time_ms/1000.0); // in s
        }

    }

}

// Main navigation
void SimulationBuilder::main_navigator() {

    printf("target %d \n",target);

    /// CPU ///////////////////////////////
    if (target == CPU_DEVICE) {

#ifdef DEBUG
        printf("CPU: main navigator\n");
#endif

        f64 t_start;
        if (display_run_time_flag) t_start = get_time();

        cpu_main_navigator(particles.stack, geometry.world,
                           materials.materials_table, cs_tables.photon_CS_table, parameters,
                           digitizer.pulses, history);

        if (display_run_time_flag) {
            print_time("Main navigation", get_time()-t_start);
        }

    /// GPU ///////////////////////////////
    } else {

#ifdef DEBUG
        printf("GPU: main navigator\n");
#endif

        hipEvent_t t_start, t_stop;
        if (display_run_time_flag) {
            hipEventCreate(&t_start);
            hipEventCreate(&t_stop);
            hipEventRecord(t_start);
        }
        
        gpu_main_navigator(particles.dstack, geometry.dworld,
                           materials.dmaterials_table, cs_tables.dphoton_CS_table, dparameters,
                           digitizer.dpulses, gpu_block_size);
        
                           
        if (display_run_time_flag) {
            hipEventRecord(t_stop);
            hipEventSynchronize(t_stop);
            f32 time_ms = 0;
            hipEventElapsedTime(&time_ms, t_start, t_stop);
            print_time("Main navigation", time_ms/1000.0); // in s
        }

    }

}



////// :: Getting ::

ParticleBuilder SimulationBuilder::get_particles() {
    return particles;
}

////// :: Command ::



// Start the simulation
void SimulationBuilder::start_simulation() {

    ui32 iter = 0;

    // Main loop
    printf("nb of iterations %d \n", nb_of_iterations);
    while (iter < nb_of_iterations) {

            // If history is required
            if (target == CPU_DEVICE && history.record_flag == ENABLED) history.cur_iter = iter;
            
            printf("primaries_generator \n");
            
            // Sources
            primaries_generator();
            
            // Clear gpu pulses
            if (target == GPU_DEVICE)
                digitizer.clear_gpu_pulses();
                
            digitizer.clear_cpu_pulses();
            
            printf("main_navigator \n");
            // Navigation
            main_navigator();
            
            // Process and store singles on CPU
            if (parameters.digitizer_flag) {
                f64 t_start = get_time();
                if (target == GPU_DEVICE) {
                    digitizer.copy_pulses_gpu2cpu();
                }

                // The complete chain
                digitizer.process_chain(iter, sources.tot_activity, geometry.world);

                // Run time
                if (display_run_time_flag) {
                    print_time("Process singles", get_time()-t_start);
                }
            }
            
        // iter
        ++iter;
        
        printf(">> Iter %i / %i\n", iter, nb_of_iterations);

    } // main loop
    
    // Test if one more iteration is needed
    if (nb_of_particles % particles.stack.size) {
    
        particles.stack.size = nb_of_particles - (nb_of_iterations * particles.stack.size);
        particles.dstack.size = particles.stack.size;
        
        // If history is required
        if (target == CPU_DEVICE && history.record_flag == ENABLED) history.cur_iter = iter;
        
        printf("primaries_generator \n");
        
        // Sources
        primaries_generator();
        
        // Clear gpu pulses
        if (target == GPU_DEVICE)
            digitizer.clear_gpu_pulses();
            
        digitizer.clear_cpu_pulses();
        
        printf("main_navigator \n");
        // Navigation
        main_navigator();
        
        // Process and store singles on CPU
        if (parameters.digitizer_flag) {
            f64 t_start = get_time();

            if (target == GPU_DEVICE) {
                digitizer.copy_pulses_gpu2cpu();
            }

            // The complete chain
            digitizer.process_chain(iter, sources.tot_activity, geometry.world);

            // Run time
            if (display_run_time_flag) {
                print_time("Process singles", get_time()-t_start);
            }
        }
    }
    
    // Free cpu pulses
    digitizer.free_cpu_pulses();
    
    // Free particles stack
    particles.cpu_free_stack();

    // Free materials table
    materials.free_materials_table();
    
    free(parameters.physics_list);
    free(parameters.secondaries_list); 
    
}


////// :: Utils ::





// Set the geometry of the simulation
void SimulationBuilder::set_geometry(GeometryBuilder obj) {
    geometry = obj;
}

// Set the materials definition associated to the geometry
void SimulationBuilder::set_materials(MaterialBuilder tab) {
    materials = tab;
}

// Set the particles stack
void SimulationBuilder::set_particles(ParticleBuilder p) {
    particles = p;
}

// Set the list of sources
void SimulationBuilder::set_sources(SourceBuilder src) {
    sources = src;
}


















*/

#endif
