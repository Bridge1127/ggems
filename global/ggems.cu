#include "hip/hip_runtime.h"
 // This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef GGEMS_CU
#define GGEMS_CU

#include "ggems.cuh"

///////// Simulation Builder class ////////////////////////////////////////////////

SimulationBuilder::SimulationBuilder() {
    target = CPU_DEVICE;

    // Init physics list and secondaries list
    ui32 i = 0;
    while (i < NB_PROCESSES) {
        parameters.physics_list[i] = DISABLED;
        ++i;
    }
    i = 0;
    while (i < NB_PARTICLES) {
        parameters.secondaries_list[i] = DISABLED;
        ++i;
    }

    parameters.record_dose_flag = DISABLED;
    history.record_flag = DISABLED;

    // Optional object (FIXME)
    detector.panel_detector.data = NULL;
    detector_set = false;
}

////// :: Main functions ::

// Generate particle based on the sources (CPU version)
void SimulationBuilder::cpu_primaries_generator() {

    // Loop over particle slot
    ui32 id = 0;
    ui32 is = 0;
    while (id < particles.stack.size) {

        // TODO - Generic and multi-sources
        //      Read CDF sources
        //      Rnd sources
        is = 0; // first source

        // Read the address source
        ui32 adr = sources.sources.ptr_sources[is];

        // Read the kind of sources
        ui32 type = (ui32)(sources.sources.data_sources[adr+ADR_SRC_TYPE]);
        ui32 geom_id = (ui32)(sources.sources.data_sources[adr+ADR_SRC_GEOM_ID]);

        // Point Source
        if (type == POINT_SOURCE) {
            f32 px = sources.sources.data_sources[adr+ADR_POINT_SRC_PX];
            f32 py = sources.sources.data_sources[adr+ADR_POINT_SRC_PY];
            f32 pz = sources.sources.data_sources[adr+ADR_POINT_SRC_PZ];
            f32 energy = sources.sources.data_sources[adr+ADR_POINT_SRC_ENERGY];

            point_source_primary_generator(particles.stack, id, px, py, pz, energy, PHOTON, geom_id);

        } else if (type == CONE_BEAM_SOURCE) {
            f32 px = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_PX];
            f32 py = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_PY];
            f32 pz = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_PZ];
            f32 phi = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_PHI];
            f32 theta = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_THETA];
            f32 psi = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_PSI];
            f32 aperture = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_APERTURE];
            f32 energy = sources.sources.data_sources[adr+ADR_CONE_BEAM_SRC_ENERGY];

            cone_beam_source_primary_generator(particles.stack, id, px, py, pz,
                                               phi, theta, psi, aperture, energy, PHOTON, geom_id);
        }

        // If need record the first position for the tracking history
        if (history.record_flag == ENABLED) {
            history.cpu_new_particle_track(PHOTON);
            history.cpu_record_a_step(particles.stack, id);

        }

        // Next particle
        ++id;

    } // i

}

// Main navigation on CPU
void SimulationBuilder::cpu_main_navigation() {

    cpu_main_navigator(particles.stack, geometry.world,
                       materials.materials_table, cs_tables.photon_CS_table, parameters,
                       detector.panel_detector, history);

}

////// :: Setting ::


// Set the geometry of the simulation
void SimulationBuilder::set_geometry(GeometryBuilder obj) {
    geometry = obj;
}

// Set the materials definition associated to the geometry
void SimulationBuilder::set_materials(MaterialBuilder tab) {
    materials = tab;
}

// Set the particles stack
void SimulationBuilder::set_particles(ParticleBuilder p) {
    particles = p;
}

// Set the list of sources
void SimulationBuilder::set_sources(SourceBuilder src) {
    sources = src;
}

// Set a detector // FIXME
void SimulationBuilder::set_detector(FlatPanelDetector vdetector) {
    detector = vdetector;
    detector_set = true;
}

// Set the hardware used for the simulation CPU or GPU (CPU by default)
void SimulationBuilder::set_hardware_target(std::string value) {
    if (value == "GPU") {
        target = GPU_DEVICE;
    } else {
        target = CPU_DEVICE;
    }
}

// Add a process to the physics list
void SimulationBuilder::set_process(std::string process_name) {

    if (process_name == "Compton") {
        parameters.physics_list[PHOTON_COMPTON] = ENABLED;
        // printf("add Compton\n");
    } else if (process_name == "PhotoElectric") {
        parameters.physics_list[PHOTON_PHOTOELECTRIC] = ENABLED;
        // printf("add photoelectric\n");
    } else if (process_name == "Rayleigh") {
        parameters.physics_list[PHOTON_RAYLEIGH] = ENABLED;
        // printf("add Rayleigh\n");
    } else if (process_name == "eIonisation") {
        parameters.physics_list[ELECTRON_IONISATION] = ENABLED;
        // printf("add photoelectric\n");
    } else if (process_name == "eBremsstrahlung") {
        parameters.physics_list[ELECTRON_BREMSSTRAHLUNG] = ENABLED;
        // printf("add photoelectric\n");
    } else if (process_name == "eMultipleScattering") {
        parameters.physics_list[ELECTRON_MSC] = ENABLED;
        // printf("add photoelectric\n");
    } else {
        print_warning("This process is unknow!!\n");
        printf("     -> %s\n", process_name.c_str());
        exit_simulation();
    }
}

// Set parameters to generate cross sections table
void SimulationBuilder::set_CS_table_nbins(ui32 valbin) {parameters.cs_table_nbins = valbin;}
void SimulationBuilder::set_CS_table_E_min(f32 valE) {parameters.cs_table_min_E = valE;}
void SimulationBuilder::set_CS_table_E_max(f32 valE) {parameters.cs_table_max_E = valE;}

// Enable the simulation of a particular secondary particle
void SimulationBuilder::set_secondary(std::string pname) {

    if (pname == "Photon") {
        parameters.secondaries_list[PHOTON] = ENABLED;
        // printf("add Compton\n");
    } else if (pname == "Electron") {
        parameters.secondaries_list[ELECTRON] = ENABLED;
        // printf("add photoelectric\n");
    } else {
        print_warning("Secondary particle type is unknow!!");
        printf("     -> %s\n", pname.c_str());
        exit_simulation();
    }
}

// Set the number of particles required for the simulation
void SimulationBuilder::set_number_of_particles(ui32 nb) {
    nb_of_particles = nb;
}

// Set the maximum number of iterations (watchdog)
void SimulationBuilder::set_max_number_of_iterations(ui32 nb) {
    max_iteration = nb;
}

// Set to record the history of some particles (only for CPU version)
void SimulationBuilder::set_record_history(ui32 nb_particles) {
    history.record_flag = ENABLED;
    history.max_nb_particles = std::min(nb_particles, nb_of_particles);
    history.stack_size = particles.stack.size;
}

////// :: Getting ::

ParticleBuilder SimulationBuilder::get_particles() {
    return particles;
}

FlatPanelDetector SimulationBuilder::get_detector() {
    return detector;
}

////// :: Command ::


// Init simualtion
void SimulationBuilder::init_simulation() {

    // First compute the number of iterations and the size of a stack // TODO Can be improved - JB
    if (nb_of_particles % particles.stack.size) {
        nb_of_iterations = (nb_of_particles / particles.stack.size) + 1;
    } else {
        nb_of_iterations = nb_of_particles / particles.stack.size;
    }
    particles.stack.size = nb_of_particles / nb_of_iterations;
    nb_of_particles = particles.stack.size * nb_of_iterations;


//    // Reset and set GPU ID and compute grid size
//    wrap_reset_device();
//    wrap_set_device(m_gpu_id);
//    m_grid_size = (m_stack_size + m_block_size - 1) / m_block_size;

//    // copy data to the device
//    wrap_copy_phantom_to_device(h_phantom, d_phantom);
//    wrap_copy_materials_to_device(h_materials, d_materials);

//    // init particle stack
//    wrap_init_particle_stack(d_particles, m_stack_size);

//    // init particle seeds
//    wrap_init_particle_seeds(d_particles, m_seed);

//    // copy the physics list to the device
//    wrap_copy_physics_list_to_device(m_physics_list);

//    // copy the secondaries list to the device
//    wrap_copy_secondaries_list_to_device(m_secondaries_list);


    if (target == CPU_DEVICE) {

        // Init the particle stack
        particles.cpu_malloc_stack();
        particles.init_stack_seed();

    }

    // Init Cross sections and physics table
    cs_tables.build_table(materials.materials_table, parameters);
    //cs_tables.print();

    // Init detector if setting up
    if (detector_set) {

        ui32 adr_geom = geometry.world.ptr_objects[detector.panel_detector.geometry_id];

        // Read first the bounding box
        f32 xmin = geometry.world.data_objects[adr_geom+ADR_AABB_XMIN];
        f32 xmax = geometry.world.data_objects[adr_geom+ADR_AABB_XMAX];
        f32 ymin = geometry.world.data_objects[adr_geom+ADR_AABB_YMIN];
        f32 ymax = geometry.world.data_objects[adr_geom+ADR_AABB_YMAX];
        f32 zmin = geometry.world.data_objects[adr_geom+ADR_AABB_ZMIN];
        f32 zmax = geometry.world.data_objects[adr_geom+ADR_AABB_ZMAX];

        // Init and allocate the image of the flat panel detector
        detector.init(xmin, xmax, ymin, ymax, zmin, zmax);

    }

}

// Start the simulation
void SimulationBuilder::start_simulation() {

    ui32 iter = 0;

    if (target == CPU_DEVICE) {

        // Main loop
        while (iter < nb_of_iterations) {
            // If history is required
            if (history.record_flag == ENABLED) history.cur_iter = iter;

            // Sources
            cpu_primaries_generator();

            // Locate the first particle position within the geometry

            // Navigation
            cpu_main_navigation();

            // iter
            ++iter;
            printf(">> Iter %i / %i\n", iter, nb_of_iterations);
        } // main loop

    }

}


////// :: Utils ::

#endif
