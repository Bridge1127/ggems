#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file vox_phan_img_nav.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 18 novembre 2015
 *
 *
 *
 */

#ifndef VOX_PHAN_DOSI_NAV_CU
#define VOX_PHAN_DOSI_NAV_CU

#include "vox_phan_dosi_nav.cuh"

#define DEBUG 1

////:: GPU Codes

__host__ __device__ void VPDN::track_electron_to_out ( ParticlesData &particles,
                                                       VoxVolumeData vol,
                                                       MaterialsTable materials,
                                                       ElectronsCrossSectionTable electron_CS_table,
                                                       GlobalSimulationParametersData parameters,
                                                       DoseData &dosi,
                                                       f32 &randomnumbereIoni,
                                                       f32 &randomnumbereBrem,
                                                       f32 freeLength,
                                                       ui32 part_id )
{


//    // DEBUG
//    if (part_id == 21246) {
//        printf("   :: Istep %i - E %e pos %e %e %e\n ", 0, particles.E[part_id],
//               particles.px[part_id], particles.py[part_id], particles.pz[part_id]);

//        // Stop simulation if out of the phantom
//        f32xyz p;
//        p.x = particles.px[part_id];
//        p.y = particles.py[part_id];
//        p.z = particles.pz[part_id];
//        if ( !test_point_AABB_with_tolerance (p, vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
//        {
//            printf("  => OUT\n");
//        }

//    }

    // Parameters values need to be stored for every e-step
    f32 alongStepLength = 0.;               // Distance from the last physics interaction.
    bool lastStepisaPhysicEffect = TRUE;    // To store last random number
    bool bool_loop = true;                  // If it is not the last step in the same voxel
    bool secondaryParticleCreated = FALSE;  // If a secondary particle is created

    alongStepLength = freeLength;
    if ( freeLength>0.0 ) lastStepisaPhysicEffect = FALSE; // Changement de voxel sans effet physique

    // Parameters
    f32 trueStepLength = FLT_MAX;
    f32 totalLength = 0.;
    f32 par1, par2;
    f32xyz pos, dir;    // particle state
    f32 energy;

    // Some other params
    f32 lengthtoVertex;               // Value to store the distance from the last physics interaction.
    ui8 next_discrete_process ;
    ui32 table_index;                 // indice de lecture de table de sections efficaces
    f32 next_interaction_distance = FLT_MAX;
    f32 dedxeIoni = 0;
    f32 dedxeBrem = 0;
    f32 erange = 0;
    f32 lambda = 0;
    bool significant_loss;
    f32 edep;
    f32 trueGeomLength;

    f32 electronEcut;

    // DEBUG
    ui32 istep = 0;

    //if ( part_id == 794983 ) printf(":: Istep 0 - pos %e %e %e\n", particles.px[part_id], particles.py[part_id], particles.pz[part_id]);

    do
    {

        // Stop simulation if out of the phantom
        if ( !test_point_AABB_with_tolerance ( make_f32xyz( particles.px[ part_id ], particles.py[ part_id ], particles.pz[ part_id ] ),
                                               vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
        {
            particles.endsimu[ part_id ] = PARTICLE_FREEZE;
            //printf("  ID %i  e- out\n", part_id);
            //printf("   ID %i - istep %i - Electron out\n", part_id, istep );

            return;
        }


        //if ( part_id == 794983 ) printf("--> LoopEntry\n");

        // Get Random number stored until a physic interaction
        if ( lastStepisaPhysicEffect == TRUE )
        {
            randomnumbereBrem = -logf ( prng_uniform( &(particles.prng[part_id]) ) );
            randomnumbereIoni = -logf ( prng_uniform( &(particles.prng[part_id]) ) );
            alongStepLength = 0.f;
            lastStepisaPhysicEffect = FALSE;
        }

        // Read position
        pos.x = particles.px[part_id];
        pos.y = particles.py[part_id];
        pos.z = particles.pz[part_id];

        // Read direction
        dir.x = particles.dx[part_id];
        dir.y = particles.dy[part_id];
        dir.z = particles.dz[part_id];

        // Read energy
        energy = particles.E[part_id];

        // Defined index phantom
        f32xyz ivoxsize;
        ivoxsize.x = 1.0 / vol.spacing_x;
        ivoxsize.y = 1.0 / vol.spacing_y;
        ivoxsize.z = 1.0 / vol.spacing_z;
        ui32xyzw index_phantom;
        index_phantom.x = ui32 ( ( pos.x-vol.off_x ) * ivoxsize.x );
        index_phantom.y = ui32 ( ( pos.y-vol.off_y ) * ivoxsize.y );
        index_phantom.z = ui32 ( ( pos.z-vol.off_z ) * ivoxsize.z );

        index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                + index_phantom.y*vol.nb_vox_x
                + index_phantom.x; // linear index

#ifdef DEBUG
        if ( index_phantom.w < 0 || index_phantom.w >= vol.number_of_voxels )
        {
            printf( "[ERROR] track_electron_to_out: index phantom %i\n", index_phantom.w );
            particles.endsimu[part_id] = PARTICLE_DEAD;
            return;
        }
#endif


        // Get the material that compose this volume
        ui16 mat_id = vol.values[ index_phantom.w ];
        electronEcut = materials.electron_energy_cut[ mat_id ];
              

        // Read the different CS, dE/dx tables
        e_read_CS_table ( mat_id, energy, electron_CS_table, next_discrete_process, table_index,
                          next_interaction_distance, dedxeIoni, dedxeBrem, erange, lambda, randomnumbereBrem, randomnumbereIoni, parameters );

        //if ( part_id == 794983 ) printf("--> ReadCS\n");

        // Vertex length
        lengthtoVertex = ( alongStepLength > next_interaction_distance ) ? 0. : next_interaction_distance - alongStepLength;

        //Get cut step
        f32 cutstep = StepFunction ( erange );

        if ( lengthtoVertex > cutstep )
        {
            significant_loss = true;
            trueStepLength = cutstep;
        }
        else
        {
            significant_loss = false;
            trueStepLength = lengthtoVertex;
        }

        //printf("trueStepLength %e   lengthtoVertex %e   cutstep %e - alongStepLength %e   next_interaction_distance %e\n",
        //       trueStepLength, lengthtoVertex, cutstep, alongStepLength, next_interaction_distance);

        //// Get the next distance boundary volume /////////////////////////////////

        // get voxel params
        f32 vox_xmin = index_phantom.x*vol.spacing_x + vol.off_x;
        f32 vox_ymin = index_phantom.y*vol.spacing_y + vol.off_y;
        f32 vox_zmin = index_phantom.z*vol.spacing_z + vol.off_z;
        f32 vox_xmax = vox_xmin + vol.spacing_x;
        f32 vox_ymax = vox_ymin + vol.spacing_y;
        f32 vox_zmax = vox_zmin + vol.spacing_z;

        //if ( part_id == 794983 ) printf("--> BeforeSafety\n");

        // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
        // TODO: In theory this have to be applied just at the entry of the particle within the volume
        //       in order to avoid particle entry between voxels. Then, computing improvement can be made
        //       by calling this function only once, just for the particle step=0.    - JB
        pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                                vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

        //if ( part_id == 794983 ) printf("--> Safety\n");

        // compute the next distance boundary
        f32 boundary_distance = hit_ray_AABB ( pos, dir, vox_xmin, vox_xmax,
                                               vox_ymin, vox_ymax, vox_zmin, vox_zmax );

#ifdef DEBUG
        if ( boundary_distance == FLT_MAX )
        {
            printf( "[ERROR] track_electron_to_out: boundary distance inf\n" );
            particles.endsimu[part_id] = PARTICLE_DEAD;
            return;
        }
#endif

        //if ( part_id == 794983 ) printf("--> Boundary\n");

        if ( boundary_distance < trueStepLength )
        {
            if ( parameters.physics_list[ELECTRON_MSC] == ENABLED )
            {
                trueGeomLength = gTransformToGeom ( trueStepLength, erange, lambda, energy,
                                                    par1, par2, electron_CS_table, mat_id );

                if ( trueGeomLength > boundary_distance )
                {
                    bool_loop=false;
                }
            }
            else
            {
                bool_loop = false;
            }
        }

        if ( bool_loop==true )
        {
            
            if ( significant_loss == true )
            {
                // Energy loss (call eFluctuation)
                edep = eLoss ( trueStepLength, particles.E[ part_id ], dedxeIoni, dedxeBrem, erange,
                               electron_CS_table, mat_id, materials, particles, parameters, part_id );

                //printf("eLoss %e   stepl %e   E %e   dedxIoni %e   dedxeBrem %e   erange %e\n", edep,
                //       trueStepLength, particles.E[ part_id ], dedxeIoni, dedxeBrem, erange);

    #ifdef DEBUG
                if ( edep > particles.E[ part_id ] )
                {
                    printf( "[ERROR] track_electron_to_out: edep > particle energy\n" );
                    particles.endsimu[part_id] = PARTICLE_DEAD;
                    return;
                }
    #endif
                GlobalMscScattering ( trueStepLength, cutstep, erange, energy, lambda, dedxeIoni,
                                      dedxeBrem,  electron_CS_table,  mat_id, particles, part_id, par1, par2,    // HERE particle move
                                      materials, dosi, index_phantom, vol, parameters );
                
                dose_record_standard ( dosi, edep, particles.px[ part_id ], particles.py[ part_id ], particles.pz[ part_id ] );

                alongStepLength += trueStepLength;
                totalLength += trueStepLength;
                lastStepisaPhysicEffect = FALSE;

                //if ( part_id == 794983 ) printf("--> SignificantLoss\n");

            }
            else
            {

                // Energy loss (call eFluctuation)
                edep = eLoss ( trueStepLength, particles.E[ part_id ], dedxeIoni, dedxeBrem, erange,
                               electron_CS_table, mat_id, materials, particles, parameters, part_id );

                //printf("eloss %e   Ekin %e\n", edep, particles.E[ part_id ]);

                //printf("eLoss %e   stepl %e   E %e   dedxIoni %e   dedxeBrem %e   erange %e\n", edep,
                //       trueStepLength, particles.E[ part_id ], dedxeIoni, dedxeBrem, erange);

    #ifdef DEBUG
                if ( edep > particles.E[ part_id ] )
                {
                    printf( "[ERROR] track_electron_to_out: edep > particle energy\n" );
                    particles.endsimu[part_id] = PARTICLE_DEAD;
                    return;
                }
    #endif
                GlobalMscScattering ( trueStepLength, lengthtoVertex, erange, energy, lambda,   dedxeIoni,
                                      dedxeBrem,   electron_CS_table,  mat_id, particles,  part_id, par1, par2,     // HERE particle move
                                      materials, dosi, index_phantom, vol, parameters );

                dose_record_standard ( dosi, edep, particles.px[part_id], particles.py[part_id], particles.pz[part_id] );

                SecParticle secondary_part;
                secondary_part.E = 0.;
                secondary_part.endsimu = PARTICLE_DEAD;

                if ( next_discrete_process == ELECTRON_IONISATION )
                {

                    secondary_part = eSampleSecondarieElectron ( electronEcut, particles,  part_id );
                    lastStepisaPhysicEffect = TRUE;
                    //secondaryParticleCreated = TRUE;

                }
                else if ( next_discrete_process == ELECTRON_BREMSSTRAHLUNG )
                {
                    // DEBUG
                    //printf("===== BREM =====\n");
                    /// TODO return a photon - JB
                    eSampleSecondarieGamma ( parameters.cs_table_min_E, parameters.cs_table_max_E, particles, part_id, materials, mat_id );
                    lastStepisaPhysicEffect = TRUE;
                    //secondaryParticleCreated = TRUE;    ???? - JB

                }

                /// If there is a secondary particle, push the primary into buffer and track this new particle

                /// Handle secondary //////////////////////

                if ( secondary_part.endsimu == PARTICLE_ALIVE &&
                     particles.level[ part_id ] < parameters.nb_of_secondaries && parameters.secondaries_list[ELECTRON] )
                {

                    //if ( part_id == 794983 ) printf("--> Push\n");

                    // Get the absolute index into secondary buffer
                    ui32 index_level = part_id * parameters.nb_of_secondaries + ( ui32 ) particles.level[ part_id ];

                    //printf("   ID %i - istep %i - Electron - level %i - E %f keV\n", part_id, istep, particles.level[part_id], particles.E[ part_id ]/keV);

                    // If primary is still alive
                    if ( particles.endsimu[ part_id ] == PARTICLE_ALIVE )
                    {
                        // Store the current particle
                        particles.sec_E[ index_level ]  =  particles.E[ part_id ];
                        particles.sec_px[ index_level ] = particles.px[ part_id ];
                        particles.sec_py[ index_level ] = particles.py[ part_id ];
                        particles.sec_pz[ index_level ] = particles.pz[ part_id ];
                        particles.sec_dx[ index_level ] = particles.dx[ part_id ];
                        particles.sec_dy[ index_level ] = particles.dy[ part_id ];
                        particles.sec_dz[ index_level ] = particles.dz[ part_id ];
                        particles.sec_pname[ index_level ] = particles.pname[ part_id ];
                        // Lose a level in the hierarchy
                        particles.level[ part_id ] += 1;
                    }

                    // Fill the main buffer with the new secondary particle
                    particles.E[ part_id ]  = secondary_part.E;
                    particles.dx[ part_id ] = secondary_part.dir.x;
                    particles.dy[ part_id ] = secondary_part.dir.y;
                    particles.dz[ part_id ] = secondary_part.dir.z;
                    particles.pname[ part_id ] = secondary_part.pname;
                    particles.endsimu[ part_id ] = secondary_part.endsimu;

                    return;

                }
                else
                {
                    // This secondary particle is not used, so drop its energy
                    if ( secondary_part.E != 0.0f )
                    {
                        dose_record_standard( dosi, secondary_part.E, particles.px[ part_id ],
                                              particles.py[ part_id ], particles.pz[ part_id ] );
                    }


#ifdef DEBUG
                    if ( particles.level[ part_id ] == parameters.nb_of_secondaries )
                    {
                        printf( "[ERROR] track_electron_to_out: reach max secondary level\n");
                    }
#endif
                }

                alongStepLength = 0;
                freeLength = 0.;
                totalLength += trueStepLength;
                //                 Troncature(particles, id);
            } // significant_loss == false

        } // bool_loop == true

#ifdef DEBUG
        if ( istep > 1000 )
        {
            printf( "[ERROR] track_electron_to_out: e- reach 1000 steps\n" );
            printf("         E %e keV - level %i\n", particles.E[part_id]/keV, particles.level[part_id]);
            particles.endsimu[part_id] = PARTICLE_DEAD;
            return;
        }

        ++istep;
#endif

        // PostStep - DEBUG //////////////////////////////////////////////////////////
        f32xyz poststep;
        poststep = make_f32xyz(particles.px[part_id], particles.py[part_id], particles.pz[part_id]);
        f32xyz deltapos = fxyz_sub( poststep, pos );
        f32 stepl = sqrtf( fxyz_dot( deltapos, deltapos ) );
        //printf("ID %i   dE %e    StepL %e   eloss %e   erange %e\n", part_id, energy-particles.E[part_id], stepl, edep, erange);
        particles.endsimu[ part_id ] = PARTICLE_DEAD;
        return;

        //printf("   ID %i - istep %i - Electron - level %i - E %f keV\n", part_id, istep, particles.level[part_id], particles.E[part_id]/keV);

    }
    while ( ( particles.E[ part_id ] > electronEcut ) && ( bool_loop ) );
    //while ( ( particles.E[ part_id ] > EKINELIMIT ) && ( bool_loop ) );


    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance ( make_f32xyz( particles.px[ part_id ], particles.py[ part_id ], particles.pz[ part_id ] ),
                                           vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
    {
        particles.endsimu[ part_id ] = PARTICLE_FREEZE;
        //printf("  ID %i  e- out\n", part_id);
        //printf("   ID %i - istep %i - Electron out\n", part_id, istep );
        return;
    }




    //if ( part_id == 794983 ) printf(":: Istep %i\n", istep);

    //printf("   ID %i - Electron istep %i - E %f kev\n", part_id, istep, particles.E[part_id]/keV);

    ////////////////////////////////////
    //                            EKINELIMIT
    if ( ( particles.E[part_id] > electronEcut ) /*&&(secondaryParticleCreated == FALSE)*/ ) //>1eV
    {

        ui8 next_discrete_process ;
        ui32 table_index; // index of cross section table
        f32 next_interaction_distance = FLT_MAX;
        f32 dedxeIoni = 0;
        f32 dedxeBrem = 0;
        f32 erange = 0;
        f32 lambda = 0;
        //         bool significant_loss;
        //         f32 edep;
        //         f32 trueGeomLength;
        //         f32 safety;

        // Read position
        f32xyz pos; // mm
        pos.x = particles.px[part_id];
        pos.y = particles.py[part_id];
        pos.z = particles.pz[part_id];

        // Read direction
        f32xyz dir;
        dir.x = particles.dx[part_id];
        dir.y = particles.dy[part_id];
        dir.z = particles.dz[part_id];

        // Get energy
        f32 energy = particles.E[part_id];

        // Defined index phantom
        f32xyz ivoxsize;
        ivoxsize.x = 1.0 / vol.spacing_x;
        ivoxsize.y = 1.0 / vol.spacing_y;
        ivoxsize.z = 1.0 / vol.spacing_z;
        ui32xyzw index_phantom;
        index_phantom.x = ui32 ( ( pos.x-vol.off_x ) * ivoxsize.x );
        index_phantom.y = ui32 ( ( pos.y-vol.off_y ) * ivoxsize.y );
        index_phantom.z = ui32 ( ( pos.z-vol.off_z ) * ivoxsize.z );
        index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                + index_phantom.y*vol.nb_vox_x
                + index_phantom.x; // linear index

#ifdef DEBUG
        if ( index_phantom.w < 0 || index_phantom.w >= vol.number_of_voxels )
        {
            printf( "[ERROR] track_electron_to_out (final): index phantom %i\n", index_phantom.w );
            particles.endsimu[part_id] = PARTICLE_DEAD;
            return;
        }
#endif


        //Get mat index
        //         int mat = (int)(vol.data[index_phantom.w]);
        ui16 mat_id = vol.values[index_phantom.w];

        //// Get the next distance boundary volume /////////////////////////////////

        // get voxel params
        f32 vox_xmin = index_phantom.x*vol.spacing_x + vol.off_x;
        f32 vox_ymin = index_phantom.y*vol.spacing_y + vol.off_y;
        f32 vox_zmin = index_phantom.z*vol.spacing_z + vol.off_z;
        f32 vox_xmax = vox_xmin + vol.spacing_x;
        f32 vox_ymax = vox_ymin + vol.spacing_y;
        f32 vox_zmax = vox_zmin + vol.spacing_z;

        // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
        // TODO: In theory this have to be applied just at the entry of the particle within the volume
        //       in order to avoid particle entry between voxels. Then, computing improvement can be made
        //       by calling this function only once, just for the particle step=0.    - JB
        pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                                vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

        // Get distance to edge of voxel
        f32 fragment = hit_ray_AABB ( pos, dir, vox_xmin, vox_xmax,
                                      vox_ymin, vox_ymax, vox_zmin, vox_zmax );

#ifdef DEBUG
        if ( fragment == FLT_MAX )
        {
            printf( "[ERROR] track_electron_to_out: fragment distance inf\n" );
            particles.endsimu[part_id] = PARTICLE_DEAD;
            return;
        }
#endif


        // fragment += 1.E-2*mm;  ?? - JB
        fragment += parameters.geom_tolerance;

        // Read Cross section table to get dedx, erange, lambda
        e_read_CS_table ( mat_id, energy, electron_CS_table, next_discrete_process, table_index, next_interaction_distance,
                          dedxeIoni,dedxeBrem,erange, lambda, randomnumbereBrem, randomnumbereIoni, parameters );

        f32 cutstep = StepFunction ( erange );

        trueStepLength = GlobalMscScattering ( fragment, cutstep, erange, energy, lambda,   dedxeIoni,
                                               dedxeBrem,  electron_CS_table,  mat_id, particles,  part_id,     // HERE the particle move
                                               par1, par2, materials, dosi, index_phantom, vol, parameters );

        freeLength = alongStepLength + trueStepLength;
        totalLength += trueStepLength;

/*
        /// Need to check, I add energy cut here - JB /////////////////////////////
        if ( particles.E[ part_id ] <= materials.electron_energy_cut[ mat_id ] )
        {
            particles.endsimu[ part_id ] = PARTICLE_DEAD;
            dose_record_standard( dosi, particles.E[ part_id ], particles.px[ part_id ],
                                  particles.py[ part_id ], particles.pz[ part_id ] );

            //printf("  ID %i  Sec last cutE\n", part_id);

            return;
        }
*/
        ///////////////////////////////////////////////////////////////////////////

    }
    else
    {
        // Kill the particle
        particles.endsimu[ part_id ] = PARTICLE_DEAD;

        /// HERE, energy is not droppping ?   - JB   // TO BE CHECKED ////////////
        dose_record_standard( dosi, particles.E[ part_id ], particles.px[ part_id ],
                              particles.py[ part_id ], particles.pz[ part_id ] );
        //////////////////////////////////////////////////////////////////////////


        //printf("   ID %i - Electron kill - E %f kev\n", part_id, particles.E[part_id]/keV);

        return;
    }


    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance ( make_f32xyz( particles.px[ part_id ], particles.py[ part_id ], particles.pz[ part_id ] ),
                                           vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
    {
        particles.endsimu[ part_id ] = PARTICLE_FREEZE;

        //printf("  ID %i  Sec outbound\n", part_id);
        //printf("   ID %i - istep %i - Electron out\n", part_id, istep );
    }


}


__host__ __device__ void VPDN::track_photon_to_out ( ParticlesData &particles,
                                                     VoxVolumeData vol,
                                                     MaterialsTable materials,
                                                     PhotonCrossSectionTable photon_CS_table,
                                                     GlobalSimulationParametersData parameters,
                                                     DoseData dosi,
                                                     ui32 part_id )
{        
    // Read position
    f32xyz pos;
    pos.x = particles.px[part_id];
    pos.y = particles.py[part_id];
    pos.z = particles.pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles.dx[part_id];
    dir.y = particles.dy[part_id];
    dir.z = particles.dz[part_id];

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol.spacing_x;
    ivoxsize.y = 1.0 / vol.spacing_y;
    ivoxsize.z = 1.0 / vol.spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( pos.x-vol.off_x ) * ivoxsize.x );
    index_phantom.y = ui32 ( ( pos.y-vol.off_y ) * ivoxsize.y );
    index_phantom.z = ui32 ( ( pos.z-vol.off_z ) * ivoxsize.z );

    index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                      + index_phantom.y*vol.nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol.values[ index_phantom.w ];

    //// Find next discrete interaction ///////////////////////////////////////

    photon_get_next_interaction ( particles, parameters, photon_CS_table, mat_id, part_id );

    f32 next_interaction_distance = particles.next_interaction_distance[part_id];
    ui8 next_discrete_process = particles.next_discrete_process[part_id];

    //// Get the next distance boundary volume /////////////////////////////////

    f32 vox_xmin = index_phantom.x*vol.spacing_x+vol.off_x;
    f32 vox_ymin = index_phantom.y*vol.spacing_y+vol.off_y;
    f32 vox_zmin = index_phantom.z*vol.spacing_z+vol.off_z;
    f32 vox_xmax = vox_xmin + vol.spacing_x;
    f32 vox_ymax = vox_ymin + vol.spacing_y;
    f32 vox_zmax = vox_zmin + vol.spacing_z;

    // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
    // TODO: In theory this have to be applied just at the entry of the particle within the volume
    //       in order to avoid particle entry between voxels. Then, computing improvement can be made
    //       by calling this function only once, just for the particle step=0.    - JB
    pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                            vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

    f32 boundary_distance = hit_ray_AABB ( pos, dir, vox_xmin, vox_xmax,
                                           vox_ymin, vox_ymax, vox_zmin, vox_zmax );

    if ( boundary_distance <= next_interaction_distance )
    {
        next_interaction_distance = boundary_distance + parameters.geom_tolerance; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;
    }

    //// Move particle //////////////////////////////////////////////////////

    // get the new position
    pos = fxyz_add ( pos, fxyz_scale ( dir, next_interaction_distance ) );

    // get safety position (outside the current voxel)
    pos = transport_get_safety_outside_AABB( pos, vox_xmin, vox_xmax,
                                             vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

    // store new position
    particles.px[part_id] = pos.x;
    particles.py[part_id] = pos.y;
    particles.pz[part_id] = pos.z;

    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance (pos, vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
    {
        particles.endsimu[part_id] = PARTICLE_FREEZE;
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    if ( next_discrete_process != GEOMETRY_BOUNDARY )
    {
        // Resolve discrete process
        SecParticle electron = photon_resolve_discrete_process ( particles, parameters, photon_CS_table,
                                                                 materials, mat_id, part_id );

        /// Energy cut /////////////

        // If gamma particle not enough energy (Energy cut)
        if ( particles.E[ part_id ] <= materials.photon_energy_cut[ mat_id ] )
        {
            // Kill without mercy
            particles.endsimu[ part_id ] = PARTICLE_DEAD;
        }

        // If electron particle not enough energy (Energy cut)
        if ( electron.E <= materials.electron_energy_cut[ mat_id ] )
        {
            // Kill without mercy
            electron.endsimu = PARTICLE_DEAD;
        }

        /// Drope energy ////////////

        // If gamma particle is dead (PE, Compton or energy cut)
        if ( particles.endsimu[ part_id ] == PARTICLE_DEAD &&  particles.E[ part_id ] != 0.0f )
        {
            dose_record_standard( dosi, particles.E[ part_id ], particles.px[ part_id ],
                                  particles.py[ part_id ], particles.pz[ part_id ] );
        }

        // If electron particle is dead (PE, Compton or energy cut)
        if ( electron.endsimu == PARTICLE_DEAD &&  electron.E != 0.0f )
        {
            dose_record_standard( dosi, electron.E, particles.px[ part_id ],
                                  particles.py[ part_id ], particles.pz[ part_id ] );
        }


        /// Handle secondary

        if ( electron.endsimu == PARTICLE_ALIVE )
        {

            // If secondary enable and enough level space
            if ( particles.level[ part_id ] < parameters.nb_of_secondaries && parameters.secondaries_list[ELECTRON] )
            {
                // Get the absolute index into secondary buffer
                ui32 index_level = part_id * parameters.nb_of_secondaries + ( ui32 ) particles.level[ part_id ];

                // If the current gamma is still alive, store it into the buffer
                if ( particles.endsimu[ part_id ] == PARTICLE_ALIVE )
                {
                    particles.sec_E[ index_level ]  =  particles.E[ part_id ];
                    particles.sec_px[ index_level ] = particles.px[ part_id ];
                    particles.sec_py[ index_level ] = particles.py[ part_id ];
                    particles.sec_pz[ index_level ] = particles.pz[ part_id ];
                    particles.sec_dx[ index_level ] = particles.dx[ part_id ];
                    particles.sec_dy[ index_level ] = particles.dy[ part_id ];
                    particles.sec_dz[ index_level ] = particles.dz[ part_id ];
                    particles.sec_pname[ index_level ] = particles.pname[ part_id ];
                    // Lose a level in the hierarchy
                    particles.level[ part_id ] += 1;
                }

                // Fill the main buffer with the new secondary particle
                particles.E[ part_id ]  = electron.E;
                particles.dx[ part_id ] = electron.dir.x;
                particles.dy[ part_id ] = electron.dir.y;
                particles.dz[ part_id ] = electron.dir.z;
                particles.pname[ part_id ] = electron.pname;
                particles.endsimu[ part_id ] = electron.endsimu;


//                printf("ID %i - Sec level %i (push gtrack) pos %e %e %e dir %e %e %e\n", part_id, particles.level[ part_id ], particles.px[ part_id ],
//                       particles.py[ part_id ],particles.pz[ part_id ],particles.dx[ part_id ],
//                       particles.dy[ part_id ],particles.dz[ part_id ]);


            }
            else
            {
                // This secondary is not used, then drop its energy
                dose_record_standard( dosi, electron.E, particles.px[ part_id ],
                                      particles.py[ part_id ], particles.pz[ part_id ] );

//                printf("ID %i -Sec level %i (Not gtrack) pos %e %e %e dir %e %e %e\n", part_id, particles.level[ part_id ], particles.px[ part_id ],
//                       particles.py[ part_id ],particles.pz[ part_id ],particles.dx[ part_id ],
//                       particles.dy[ part_id ],particles.dz[ part_id ]);

            }
        }


    } // discrete process


}

// Device Kernel that move particles to the voxelized volume boundary
__global__ void VPDN::kernel_device_track_to_in ( ParticlesData particles, f32 xmin, f32 xmax,
                                                  f32 ymin, f32 ymax, f32 zmin, f32 zmax, f32 tolerance )
{  
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;    
    transport_track_to_in_AABB( particles, xmin, xmax, ymin, ymax, zmin, zmax, tolerance, id);
}


// Host Kernel that move particles to the voxelized volume boundary
void VPDN::kernel_host_track_to_in ( ParticlesData particles, f32 xmin, f32 xmax,
                                     f32 ymin, f32 ymax, f32 zmin, f32 zmax, f32 tolerance, ui32 part_id )
{       
    transport_track_to_in_AABB( particles, xmin, xmax, ymin, ymax, zmin, zmax, tolerance, part_id);
}

// Device kernel that track particles within the voxelized volume until boundary
__global__ void VPDN::kernel_device_track_to_out ( ParticlesData particles,
                                                   VoxVolumeData vol,
                                                   MaterialsTable materials,
                                                   PhotonCrossSectionTable photon_CS_table,
                                                   ElectronsCrossSectionTable electron_CS_table,
                                                   GlobalSimulationParametersData parameters,
                                                   DoseData dosi )
{   
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;    

    // For multivoxels navigation
    f32 randomnumbereIoni= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
    f32 randomnumbereBrem= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
    f32 freeLength = 0.0*mm;

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE )
    {

        if ( particles.pname[id] == PHOTON )
        {
            /// DEBUG
            //printf("GPU tracking photon\n");
            VPDN::track_photon_to_out ( particles, vol, materials, photon_CS_table, parameters, dosi, id );

        }
        else if ( particles.pname[id] == ELECTRON )
        {

            /// DEBUG
            //printf("GPU tracking electron\n");
            VPDN::track_electron_to_out ( particles, vol, materials, electron_CS_table, parameters, dosi,
                                          randomnumbereIoni, randomnumbereBrem, freeLength, id );


        }

        // Condition if particle is dead and if it was a secondary
        if ( ( ( particles.endsimu[id]==PARTICLE_DEAD ) || ( particles.endsimu[id]==PARTICLE_FREEZE ) ) && ( particles.level[id]>PRIMARY ) )
        {

            /// Pull back the particle stored in the secondary buffer to the main one

            // DEBUG
            //printf(" PULL e-  ID %i Level %i\n", id, particles.level[id]);


            // Wake up the particle
            particles.endsimu[id] = PARTICLE_ALIVE;
            // Earn a higher level
            particles.level[id]  -= 1;
            // Get the absolute index into secondary buffer
            ui32 index_level = id * parameters.nb_of_secondaries + ( ui32 ) particles.level[id];

            // FreeLength must be reinitialized due to voxels navigation (diff mats)
            freeLength = 0.0*mm;
            randomnumbereIoni= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
            randomnumbereBrem= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)

            // Get back the stored particle into the primary buffer
            particles.E[ id ]     = particles.sec_E[ index_level ]    ;
            particles.px[ id ]    = particles.sec_px[ index_level ]   ;
            particles.py[ id ]    = particles.sec_py[ index_level ]   ;
            particles.pz[ id ]    = particles.sec_pz[ index_level ]   ;
            particles.dx[ id ]    = particles.sec_dx[ index_level ]   ;
            particles.dy[ id ]    = particles.sec_dy[ index_level ]   ;
            particles.dz[ id ]    = particles.sec_dz[ index_level ]   ;
            particles.pname[ id ] = particles.sec_pname[ index_level ];

//            printf("ID %i - Sec level %i (pull device) pos %e %e %e dir %e %e %e INDEX %i \n", id, particles.level[ id ], particles.px[ id ],
//                   particles.py[ id ],particles.pz[ id ], particles.dx[ id ], particles.dy[ id ], particles.dz[ id ], index_level );

        }

        /// DEBUG

    }

    /// DEBUG
    //if ( step > 1 ) printf("ID %i Step %i\n", id, step);

}

// Host kernel that track particles within the voxelized volume until boundary
void VPDN::kernel_host_track_to_out ( ParticlesData particles,
                                      VoxVolumeData vol,
                                      MaterialsTable materials,
                                      PhotonCrossSectionTable photon_CS_table,
                                      ElectronsCrossSectionTable electron_CS_table,
                                      GlobalSimulationParametersData parameters,
                                      DoseData dosi,
                                      ui32 id )
{
    // For multivoxels navigation
    f32 randomnumbereIoni= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
    f32 randomnumbereBrem= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
    f32 freeLength = 0.0*mm;

    ui32 step = 0;

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE )
    {

        if ( particles.pname[id] == PHOTON )
        { 
            //if ( id == 794983 ) printf("Photon tracking - Level %i\n", particles.level[ id ]);
            /// DEBUG
            //printf("CPU tracking photon\n");
            printf("ID %i - Gamma - track %i - level %i - E %f keV\n", id, step, particles.level[id], particles.E[id]/keV);
            VPDN::track_photon_to_out ( particles, vol, materials, photon_CS_table, parameters, dosi, id );

        }
        else if ( particles.pname[id] == ELECTRON )
        {            
            //if ( id == 794983 ) printf("Electron tracking - Level %i\n", particles.level[ id ]);
            /// DEBUG
            //printf("CPU tracking electron\n");
            if ( particles.level[id] == 1)
            {
                printf("ID %i - Electron - track %i - level %i - E %f keV\n", id, step, particles.level[id], particles.E[id]/keV);
            }
            VPDN::track_electron_to_out ( particles, vol, materials, electron_CS_table, parameters, dosi,
                                          randomnumbereIoni, randomnumbereBrem, freeLength, id );
        }

        // Condition if particle is dead and if it was a secondary
        if ( ( ( particles.endsimu[id]==PARTICLE_DEAD ) || ( particles.endsimu[id]==PARTICLE_FREEZE ) ) && ( particles.level[id]>PRIMARY ) )
        {

            /// Pull back the particle stored in the secondary buffer to the main one            

            // Wake up the particle
            particles.endsimu[id] = PARTICLE_ALIVE;
            // Earn a higher level
            particles.level[id]  -= 1;
            // Get the absolute index into secondary buffer
            ui32 index_level = id * parameters.nb_of_secondaries + ( ui32 ) particles.level[id];

            // FreeLength must be reinitialized due to voxels navigation (diff mats)
            freeLength = 0.0*mm;
            randomnumbereIoni= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)
            randomnumbereBrem= -logf ( prng_uniform( &(particles.prng[id]) ) ); // -log(RN)

            // Get back the stored particle into the primary buffer
            particles.E[ id ]     = particles.sec_E[ index_level ]    ;
            particles.px[ id ]    = particles.sec_px[ index_level ]   ;
            particles.py[ id ]    = particles.sec_py[ index_level ]   ;
            particles.pz[ id ]    = particles.sec_pz[ index_level ]   ;
            particles.dx[ id ]    = particles.sec_dx[ index_level ]   ;
            particles.dy[ id ]    = particles.sec_dy[ index_level ]   ;
            particles.dz[ id ]    = particles.sec_dz[ index_level ]   ;
            particles.pname[ id ] = particles.sec_pname[ index_level ];

//            printf("ID %i - Sec level %i (pull host) pos %e %e %e dir %e %e %e INDEX %i\n", id, particles.level[ id ], particles.px[ id ],
//                   particles.py[ id ],particles.pz[ id ], particles.dx[ id ], particles.dy[ id ], particles.dz[ id ], index_level);


        }

        ++step;


    }


}

////:: Privates

bool VoxPhanDosiNav::m_check_mandatory()
{

    if ( m_phantom.data_h.nb_vox_x == 0 || m_phantom.data_h.nb_vox_y == 0 || m_phantom.data_h.nb_vox_z == 0 ||
            m_phantom.data_h.spacing_x == 0 || m_phantom.data_h.spacing_y == 0 || m_phantom.data_h.spacing_z == 0 ||
            m_phantom.list_of_materials.size() == 0 || m_materials_filename.empty() )
    {
        return false;
    }
    else
    {
        return true;
    }

}

// return memory usage
ui64 VoxPhanDosiNav::m_get_memory_usage()
{
    ui64 mem = 0;

    // First the voxelized phantom
    mem += ( m_phantom.data_h.number_of_voxels * sizeof( ui16 ) );
    // Then material data
    mem += ( ( 2 * m_materials.data_h.nb_elements_total + 23 * m_materials.data_h.nb_materials ) * sizeof( f32 ) );
    // Then cross sections (gamma)
    ui64 n = m_cross_sections.photon_CS.data_h.nb_bins;
    ui64 k = m_cross_sections.photon_CS.data_h.nb_mat;
    mem += ( ( n + 3*n*k + 3*101*n ) * sizeof( f32 ) );
    // Cross section (electron)
    mem += ( n*k*7*sizeof( f32 ) );
    // Finally the dose map
    n = m_dose_calculator.dose.data_h.nb_of_voxels;
    mem += ( 4*n*sizeof( f64 ) + n*sizeof( ui32 ) );

    return mem;
}

////:: Main functions

VoxPhanDosiNav::VoxPhanDosiNav ()
{
    // Default doxel size (if 0 = same size to the phantom)
    m_doxel_size_x = 0;
    m_doxel_size_y = 0;
    m_doxel_size_z = 0;

    m_materials_filename = "";
}

void VoxPhanDosiNav::track_to_in ( Particles particles )
{

    if ( m_params.data_h.device_target == CPU_DEVICE )
    {
        ui32 id=0;
        while ( id<particles.size )
        {
            VPDN::kernel_host_track_to_in ( particles.data_h, m_phantom.data_h.xmin, m_phantom.data_h.xmax,
                                            m_phantom.data_h.ymin, m_phantom.data_h.ymax,
                                            m_phantom.data_h.zmin, m_phantom.data_h.zmax,
                                            m_params.data_h.geom_tolerance,
                                            id );
            ++id;
        }
    }
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

        VPDN::kernel_device_track_to_in<<<grid, threads>>> ( particles.data_d, m_phantom.data_d.xmin, m_phantom.data_d.xmax,
                                                                               m_phantom.data_d.ymin, m_phantom.data_d.ymax,
                                                                               m_phantom.data_d.zmin, m_phantom.data_d.zmax,
                                                                               m_params.data_d.geom_tolerance );
        cuda_error_check ( "Error ", " Kernel_VoxPhanDosi (track to in)" );
        hipDeviceSynchronize();
    }

    // DEBUG
    //printf("TrackToIn  ok\n");

}

void VoxPhanDosiNav::track_to_out ( Particles particles )
{
    //
    if ( m_params.data_h.device_target == CPU_DEVICE )
    {

        ui32 id=0;
        while ( id<particles.size )
        {

            // DEBUG
            //printf("TrackToOut id %i\n", id);
            VPDN::kernel_host_track_to_out ( particles.data_h, m_phantom.data_h,
                                             m_materials.data_h, m_cross_sections.photon_CS.data_h, m_cross_sections.electron_CS.data_h,
                                             m_params.data_h, m_dose_calculator.dose.data_h, id );

            ++id;
        }
    }
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {       
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;//
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;
        hipDeviceSynchronize();
        VPDN::kernel_device_track_to_out<<<grid, threads>>> ( particles.data_d, m_phantom.data_d, m_materials.data_d,
                                                              m_cross_sections.photon_CS.data_d,
                                                              m_cross_sections.electron_CS.data_d,
                                                              m_params.data_d, m_dose_calculator.dose.data_d );
        cuda_error_check ( "Error ", " Kernel_VoxPhanDosi (track to out)" );
        
        hipDeviceSynchronize();
    }
    
    
}

void VoxPhanDosiNav::load_phantom_from_mhd ( std::string filename, std::string range_mat_name )
{
    m_phantom.load_from_mhd ( filename, range_mat_name );
}

void VoxPhanDosiNav::write ( std::string filename )
{
//     m_dose_calculator.m_copy_dose_gpu2cpu();

    m_dose_calculator.write ( filename );
}


void VoxPhanDosiNav::initialize ( GlobalSimulationParameters params )
{
    // Check params
    if ( !m_check_mandatory() )
    {
        print_error ( "VoxPhanDosi: missing parameters." );
        exit_simulation();
    }

    // Params
    m_params = params;

    // Phantom
    m_phantom.set_name ( "VoxPhanDosiNav" );
    m_phantom.initialize ( params );

    // Materials table
    m_materials.load_materials_database( m_materials_filename );
    m_materials.initialize ( m_phantom.list_of_materials, params );

    // Cross Sections
    m_cross_sections.initialize ( m_materials, params );

    // Init dose map
    if ( m_doxel_size_x != 0 && m_doxel_size_y != 0 && m_doxel_size_z != 0 )
    {
        f32 sizex = m_phantom.data_h.nb_vox_x*m_phantom.data_h.spacing_x;
        f32 sizey = m_phantom.data_h.nb_vox_y*m_phantom.data_h.spacing_y;
        f32 sizez = m_phantom.data_h.nb_vox_z*m_phantom.data_h.spacing_z;

        m_dose_calculator.set_size_in_voxel ( ( ui32 ) ( sizex / m_doxel_size_x ),
                                              ( ui32 ) ( sizey / m_doxel_size_y ),
                                              ( ui32 ) ( sizez / m_doxel_size_z ) );
        m_dose_calculator.set_voxel_size ( m_doxel_size_x,
                                           m_doxel_size_y,
                                           m_doxel_size_z );
    }
    else
    {
        m_dose_calculator.set_size_in_voxel ( m_phantom.data_h.nb_vox_x,
                                              m_phantom.data_h.nb_vox_y,
                                              m_phantom.data_h.nb_vox_z );
        m_dose_calculator.set_voxel_size ( m_phantom.data_h.spacing_x,
                                           m_phantom.data_h.spacing_y,
                                           m_phantom.data_h.spacing_z );
    }

    m_dose_calculator.set_offset ( m_phantom.data_h.off_x,
                                   m_phantom.data_h.off_y,
                                   m_phantom.data_h.off_z );
    m_dose_calculator.initialize ( m_params ); // CPU&GPU
    
    m_dose_calculator.set_voxelized_phantom(m_phantom);
    m_dose_calculator.set_materials(m_materials);

    // Some verbose if required
    if ( params.data_h.display_memory_usage )
    {
        ui64 mem = m_get_memory_usage();
        GGcout_mem("VoxPhanDosiNav", mem);
    }

}

void VoxPhanDosiNav::calculate_dose_to_water(){

    m_dose_calculator.calculate_dose_to_water();

}

void VoxPhanDosiNav::calculate_dose_to_phantom(){

    m_dose_calculator.calculate_dose_to_phantom();

}

void VoxPhanDosiNav::set_materials( std::string filename )
{
    m_materials_filename = filename;
}

void VoxPhanDosiNav::set_doxel_size( f32 sizex, f32 sizey, f32 sizez )
{
    m_doxel_size_x = sizex;
    m_doxel_size_y = sizey;
    m_doxel_size_z = sizez;
}


#undef DEBUG

#endif
