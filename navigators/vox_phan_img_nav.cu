#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file vox_phan_img_nav.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 18 novembre 2015
 *
 *
 *
 */

#ifndef VOX_PHAN_IMG_NAV_CU
#define VOX_PHAN_IMG_NAV_CU

#include "vox_phan_img_nav.cuh"

////:: GPU Codes

// Move particles to the voxelized volume
__host__ __device__ void vox_phan_track_to_in(ParticlesData &particles, f32 xmin, f32 xmax,
                                              f32 ymin, f32 ymax, f32 zmin, f32 zmax,
                                              ui32 id) {

    // Read position
    f64xyz pos;
    pos.x = particles.px[id];
    pos.y = particles.py[id];
    pos.z = particles.pz[id];

    // Read direction
    f64xyz dir;
    dir.x = particles.dx[id];
    dir.y = particles.dy[id];
    dir.z = particles.dz[id];

    f32 dist = hit_ray_AABB(pos, dir, xmin, xmax, ymin, ymax, zmin, zmax);

    // the particle not hitting the voxelized volume
    if (dist == FLT_MAX) {                            // TODO: Don't know why F32_MAX doesn't work...
        particles.endsimu[id] = PARTICLE_FREEZE;
        return;
    } else {
        // Check if the path of the particle cross the volume sufficiently
        f32 cross = dist_overlap_ray_AABB(pos, dir, xmin, xmax, ymin, ymax, zmin, zmax);
        if (cross < EPSILON3) {
            particles.endsimu[id] = PARTICLE_FREEZE;
            return;
        }
        // move the particle slightly inside the volume
        pos = fxyz_add(pos, fxyz_scale(pos, dist+EPSILON6));

        // TODO update tof
        // ...
    }

    // set photons
    particles.px[id] = pos.x;
    particles.py[id] = pos.y;
    particles.pz[id] = pos.z;

}


__host__ __device__ void vox_phan_track_to_out(ParticlesData &particles,
                                               VoxVolumeData vol,
                                               MaterialsTable materials,
                                               PhotonCrossSectionTable photon_CS_table,
                                               GlobalSimulationParametersData parameters,
                                               ui32 part_id) {


    // Read position
    f32xyz pos;
    pos.x = particles.px[part_id];
    pos.y = particles.py[part_id];
    pos.z = particles.pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles.dx[part_id];
    dir.y = particles.dy[part_id];
    dir.z = particles.dz[part_id];

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol.spacing_x;
    ivoxsize.y = 1.0 / vol.spacing_y;
    ivoxsize.z = 1.0 / vol.spacing_z;
    ui16xyzw index_phantom;
    index_phantom.x = ui16( (pos.x+vol.org_x) * ivoxsize.x );
    index_phantom.y = ui16( (pos.y+vol.org_y) * ivoxsize.y );
    index_phantom.z = ui16( (pos.z+vol.org_z) * ivoxsize.z );
    index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                      + index_phantom.y*vol.nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol.values[index_phantom.w];

    //// Find next discrete interaction ///////////////////////////////////////

float distance_next_i = 100000000.0;
float distance = 100000000.0f;
//for( i = 0; i < processListActivated.size();++i)
//for( i = 0; i < nProcessActivated;++i)
//for( std::list<AbstractProcess>::iterator iter = processListActivated.begin();
    //iter != processListActivated.end(); ++iter)
{
    // Compton and Photoelectric activated
    photon_get_next_interaction( particles, *iter, photon_CS_table[*iter], mat_id, part_id, &distance );
    distance = distance < distance_next_i ? distance : distance_next_i;
}

std::cout <<;
 
    photon_get_next_interaction(particles, parameters, photon_CS_table, mat_id, part_id);




    f32 next_interaction_distance = particles.next_interaction_distance[part_id];
    ui8 next_discrete_process = particles.next_discrete_process[part_id];

    //// Get the next distance boundary volume /////////////////////////////////

    f32 vox_xmin = index_phantom.x*vol.spacing_x;
    f32 vox_ymin = index_phantom.y*vol.spacing_y;
    f32 vox_zmin = index_phantom.z*vol.spacing_z;
    f32 vox_xmax = vox_xmin + vol.spacing_x;
    f32 vox_ymax = vox_ymin + vol.spacing_y;
    f32 vox_zmax = vox_zmin + vol.spacing_z;

    f32 boundary_distance = hit_ray_AABB(pos, dir, vox_xmin, vox_xmax,
                                         vox_ymin, vox_ymax, vox_zmin, vox_zmax);

    if (boundary_distance <= next_interaction_distance) {
        next_interaction_distance = boundary_distance + EPSILON3; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;
    }

    //// Move particle //////////////////////////////////////////////////////

    pos = fxyz_add(pos, fxyz_scale(dir, next_interaction_distance));

    // Update TOF - TODO
    //particles.tof[part_id] += c_light * next_interaction_distance;

    particles.px[part_id] = pos.x;
    particles.py[part_id] = pos.y;
    particles.pz[part_id] = pos.z;

    // Stop simulation if out of the phantom
    if (!test_point_AABB(pos, vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax)) {
        particles.endsimu[part_id] = PARTICLE_FREEZE;
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    if (next_discrete_process != GEOMETRY_BOUNDARY) {
        // Resolve discrete process
        SecParticle electron = photon_resolve_discrete_process(particles, parameters, photon_CS_table,
                                                               materials, mat_id, part_id);

        //// Here e- are not tracked, and lost energy not drop

    }

    //// Energy cut
    if (particles.E[part_id] <= materials.electron_energy_cut[mat_id]) {
        particles.endsimu[part_id] = PARTICLE_DEAD;
        return;
    }

}

// Device Kernel that move particles to the voxelized volume boundary
__global__ void kernel_device_track_to_in(ParticlesData particles, f32 xmin, f32 xmax,
                                            f32 ymin, f32 ymax, f32 zmin, f32 zmax) {

    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particles.size) return;

    vox_phan_track_to_in(particles, xmin, xmax, ymin, ymax, zmin, zmax, id);

}

// Host Kernel that move particles to the voxelized volume boundary
void kernel_host_track_to_in(ParticlesData particles, f32 xmin, f32 xmax,
                             f32 ymin, f32 ymax, f32 zmin, f32 zmax, ui32 id) {

    vox_phan_track_to_in(particles, xmin, xmax, ymin, ymax, zmin, zmax, id);

}

// Device kernel that track particles within the voxelized volume until boundary
__global__ void kernel_device_track_to_out(ParticlesData particles,
                                           VoxVolumeData vol,
                                           MaterialsTable materials,
                                           PhotonCrossSectionTable photon_CS_table,
                                           GlobalSimulationParametersData parameters) {

    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particles.size) return;

    // Stepping loop
    while (particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE) {
        vox_phan_track_to_out(particles, vol, materials, photon_CS_table, parameters, id);
    }

}

// Host kernel that track particles within the voxelized volume until boundary
void kernel_host_track_to_out(ParticlesData particles,
                              VoxVolumeData vol,
                              MaterialsTable materials,
                              PhotonCrossSectionTable photon_CS_table,
                              GlobalSimulationParametersData parameters, ui32 id) {

    // Stepping loop
    while (particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE) {
        vox_phan_track_to_out(particles, vol, materials, photon_CS_table, parameters, id);
    }
}

////:: Privates

// Copy the phantom to the GPU
void VoxPhanImgNav::m_copy_phantom_cpu2gpu() {

    // Mem allocation
    HANDLE_ERROR( hipMalloc((void**) &phantom.volume.data_d.values, phantom.volume.data_h.number_of_voxels*sizeof(ui16)) );
    // Copy data
    HANDLE_ERROR( hipMemcpy(phantom.volume.data_d.values, phantom.volume.data_h.values,
                  phantom.volume.data_h.number_of_voxels*sizeof(ui16), hipMemcpyHostToDevice) );

    phantom.volume.data_d.nb_vox_x = phantom.volume.data_h.nb_vox_x;
    phantom.volume.data_d.nb_vox_y = phantom.volume.data_h.nb_vox_y;
    phantom.volume.data_d.nb_vox_z = phantom.volume.data_h.nb_vox_z;

    phantom.volume.data_d.spacing_x = phantom.volume.data_h.spacing_x;
    phantom.volume.data_d.spacing_y = phantom.volume.data_h.spacing_y;
    phantom.volume.data_d.spacing_z = phantom.volume.data_h.spacing_z;

    phantom.volume.data_d.org_x = phantom.volume.data_h.org_x;
    phantom.volume.data_d.org_y = phantom.volume.data_h.org_y;
    phantom.volume.data_d.org_z = phantom.volume.data_h.org_z;

    phantom.volume.data_d.number_of_voxels = phantom.volume.data_h.number_of_voxels;
}

bool VoxPhanImgNav::m_check_mandatory() {

    if (phantom.volume.data_h.nb_vox_x == 0 || phantom.volume.data_h.nb_vox_y == 0 || phantom.volume.data_h.nb_vox_z == 0 ||
        phantom.volume.data_h.spacing_x == 0 || phantom.volume.data_h.spacing_y == 0 || phantom.volume.data_h.spacing_z == 0 ||
        phantom.list_of_materials.size() == 0) {
        return false;
    } else {
        return true;
    }

}

////:: Main functions

void VoxPhanImgNav::track_to_in(Particles particles) {

    if (m_params.data_h.device_target == CPU_DEVICE) {
        ui32 id=0; while (id<particles.size) {
            kernel_host_track_to_in(particles.data_h, phantom.volume.data_h.xmin, phantom.volume.data_h.xmax,
                                                   phantom.volume.data_h.ymin, phantom.volume.data_h.ymax,
                                                   phantom.volume.data_h.zmin, phantom.volume.data_h.zmax,
                                                   id);
            ++id;
        }
    } else if (m_params.data_h.device_target == GPU_DEVICE) {

        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = (particles.size + m_params.data_h.gpu_block_size - 1) / m_params.data_h.gpu_block_size;

        kernel_device_track_to_in<<<grid, threads>>>(particles.data_d, phantom.volume.data_h.xmin, phantom.volume.data_h.xmax,
                                                     phantom.volume.data_h.ymin, phantom.volume.data_h.ymax,
                                                     phantom.volume.data_h.zmin, phantom.volume.data_h.zmax);
        cuda_error_check("Error ", " Kernel_VoxPhanImgNav (track to in)");

    }


}

void VoxPhanImgNav::track_to_out(Particles particles, Materials materials, PhotonCrossSection photon_CS) {

    if (m_params.data_h.device_target == CPU_DEVICE) {

        ui32 id=0; while (id<particles.size) {
           
            kernel_host_track_to_out(particles.data_h, phantom.volume.data_h,
                                     materials.data_h, photon_CS.data_h, m_params.data_h, id);
            ++id;
        }
    } else if (m_params.data_h.device_target == GPU_DEVICE) {

        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = (particles.size + m_params.data_h.gpu_block_size - 1) / m_params.data_h.gpu_block_size;

        kernel_device_track_to_out<<<grid, threads>>>(particles.data_d, phantom.volume.data_d, materials.data_d,
                                                      photon_CS.data_d, m_params.data_d);
        cuda_error_check("Error ", " Kernel_VoxPhanImgNav (track to out)");

    }

}


void VoxPhanImgNav::load_phantom(std::string file, std::string matfile)
{

    if (ImageReader::get_format(file) == "mhd") 
    {
    
        load_phantom_from_mhd(file, matfile);
        
    }
    else
    {
        
        print_error("Unknown phantom format ... \n");
        exit_simulation();
        
    }

}

void VoxPhanImgNav::load_phantom_from_mhd(std::string mhdfile, std::string matfile)
{

    phantom.load_from_mhd(mhdfile, matfile);

}

void VoxPhanImgNav::initialize(GlobalSimulationParameters params) {
    // Check params
    if (!m_check_mandatory()) {
        print_error("VoxPhanImgNav: missing parameters.");
        exit_simulation();
    }

    // Params
    m_params = params;

    // Phantom name
    phantom.set_name("VoxPhanImgNav");

    // Copy data to GPU
    if (m_params.data_h.device_target == GPU_DEVICE) {
        m_copy_phantom_cpu2gpu();
    }

}

// Get list of materials
std::vector<std::string> VoxPhanImgNav::get_materials_list() {
    return phantom.list_of_materials;
}

// Get data that contains materials index
ui16* VoxPhanImgNav::get_data_materials_indices() {
    return phantom.volume.data_h.values;
}

// Get the size of data
ui32 VoxPhanImgNav::get_data_size() {
    return phantom.volume.data_h.number_of_voxels;
}
























#endif
