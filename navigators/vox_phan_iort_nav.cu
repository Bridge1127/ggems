#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file vox_phan_iort_nav.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 23/03/2016
 *
 *
 *
 */

#ifndef VOX_PHAN_IORT_NAV_CU
#define VOX_PHAN_IORT_NAV_CU

#include "vox_phan_iort_nav.cuh"

////// HOST-DEVICE GPU Codes ////////////////////////////////////////////

__host__ __device__ void VPIORTN::track_to_out( ParticlesData &particles,
                                                VoxVolumeData<ui16> vol,
                                                MaterialsTable materials,
                                                PhotonCrossSectionTable photon_CS_table,
                                                GlobalSimulationParametersData parameters,
                                                DoseData dosi,
                                                Mu_MuEn_Table mu_table,
                                                HistoryMap hist_map,
                                                ui32 part_id )
{        
    // Read position
    f32xyz pos;
    pos.x = particles.px[part_id];
    pos.y = particles.py[part_id];
    pos.z = particles.pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles.dx[part_id];
    dir.y = particles.dy[part_id];
    dir.z = particles.dz[part_id];

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol.spacing_x;
    ivoxsize.y = 1.0 / vol.spacing_y;
    ivoxsize.z = 1.0 / vol.spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( pos.x + vol.off_x ) * ivoxsize.x );
    index_phantom.y = ui32 ( ( pos.y + vol.off_y ) * ivoxsize.y );
    index_phantom.z = ui32 ( ( pos.z + vol.off_z ) * ivoxsize.z );

    index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                      + index_phantom.y*vol.nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol.values[ index_phantom.w ];

    // Vars
    f32 next_interaction_distance;
    ui8 next_discrete_process;

    //// Find next discrete interaction ///////////////////////////////////////

#ifdef SKIP_VOXEL
    if ( mat_id != particles.geometry_id[ part_id ] )
    {
        // Store the material id
        particles.geometry_id[ part_id ] = mat_id;

        // Get next interaction and update values
        photon_get_next_interaction ( particles, parameters, photon_CS_table, mat_id, part_id );
    }

    // Read values
    next_interaction_distance = particles.next_interaction_distance[part_id];
    next_discrete_process = particles.next_discrete_process[part_id];

#else
    photon_get_next_interaction ( particles, parameters, photon_CS_table, mat_id, part_id );

    next_interaction_distance = particles.next_interaction_distance[part_id];
    next_discrete_process = particles.next_discrete_process[part_id];

#endif

    //// Get the next distance boundary volume /////////////////////////////////

    f32 vox_xmin = index_phantom.x*vol.spacing_x - vol.off_x;
    f32 vox_ymin = index_phantom.y*vol.spacing_y - vol.off_y;
    f32 vox_zmin = index_phantom.z*vol.spacing_z - vol.off_z;
    f32 vox_xmax = vox_xmin + vol.spacing_x;
    f32 vox_ymax = vox_ymin + vol.spacing_y;
    f32 vox_zmax = vox_zmin + vol.spacing_z;

    // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
    // TODO: In theory this have to be applied just at the entry of the particle within the volume
    //       in order to avoid particle entry between voxels. Then, computing improvement can be made
    //       by calling this function only once, just for the particle step=0.    - JB
    pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                            vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

    f32 boundary_distance = hit_ray_AABB ( pos, dir, vox_xmin, vox_xmax,
                                           vox_ymin, vox_ymax, vox_zmin, vox_zmax );

    if ( boundary_distance <= next_interaction_distance )
    {
        next_interaction_distance = boundary_distance + parameters.geom_tolerance; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;

        #ifdef SKIP_VOXEL
            // Update the the next distance interaction based on the length particle step
            particles.next_interaction_distance[ part_id ] -= ( boundary_distance + parameters.geom_tolerance );
        #endif

    }

    //// Move particle //////////////////////////////////////////////////////

    // get the new position
    pos = fxyz_add ( pos, fxyz_scale ( dir, next_interaction_distance ) );

    // get safety position (outside the current voxel)
    pos = transport_get_safety_outside_AABB( pos, vox_xmin, vox_xmax,
                                             vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters.geom_tolerance );

    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance (pos, vol.xmin, vol.xmax, vol.ymin, vol.ymax, vol.zmin, vol.zmax, parameters.geom_tolerance ) )
    {
        particles.endsimu[part_id] = PARTICLE_FREEZE;
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    f32 energy = particles.E[ part_id ];

    // If TLE
    if ( mu_table.flag == TLE ) {

        if ( next_discrete_process != GEOMETRY_BOUNDARY )
        {
            // Resolve discrete process
            SecParticle electron = photon_resolve_discrete_process ( particles, parameters, photon_CS_table,
                                                                     materials, mat_id, part_id );

            #ifdef SKIP_VOXEL
                particles.geometry_id[ part_id ] = 999999999; // Init with unknow material id to force process update
            #endif

        } // discrete process

        /// Drop energy ////////////

        // Get the mu_en for the current E
        ui32 E_index = binary_search ( energy, mu_table.E_bins, mu_table.nb_bins );

        f32 mu_en;

        if ( E_index == 0 )
        {
            mu_en = mu_table.mu_en[ mat_id*mu_table.nb_bins ];
        }
        else
        {
            mu_en = linear_interpolation( mu_table.E_bins[E_index-1],  mu_table.mu_en[mat_id*mu_table.nb_bins + E_index-1],
                                          mu_table.E_bins[E_index],    mu_table.mu_en[mat_id*mu_table.nb_bins + E_index],
                                          energy );
        }

        //                             record to the old position (current voxel)
        dose_record_TLE( dosi, energy, particles.px[ part_id ], particles.py[ part_id ],
                         particles.pz[ part_id ], next_interaction_distance,  mu_en );

        /// Energy cut /////////////

        // If gamma particle not enough energy (Energy cut)
        if ( particles.E[ part_id ] <= materials.photon_energy_cut[ mat_id ] )
        {
            // Kill without mercy
            particles.endsimu[ part_id ] = PARTICLE_DEAD;
        }

    }
    else // Else Analog or seTLE
    {

        // Resolve process
        SecParticle electron;
        if ( next_discrete_process != GEOMETRY_BOUNDARY )
        {
            // Resolve discrete process
            electron = photon_resolve_discrete_process ( particles, parameters, photon_CS_table,
                                                                     materials, mat_id, part_id );

            #ifdef SKIP_VOXEL
                particles.geometry_id[ part_id ] = 999999999; // Init with unknow material id to force process update
            #endif

            /// Energy cut /////////////

            // If gamma particle not enough energy (Energy cut)
            if ( particles.E[ part_id ] <= materials.photon_energy_cut[ mat_id ] )
            {
                // Kill without mercy
                particles.endsimu[ part_id ] = PARTICLE_DEAD;
            }
        }

        // seTLE
        if ( mu_table.flag == seTLE )
        {
            /// seTLE /////////////////

            if ( next_discrete_process == PHOTON_COMPTON || next_discrete_process == PHOTON_RAYLEIGH )
            {
                // Increment the number of interactions at this position
                index_phantom.x = ui32 ( ( pos.x + vol.off_x ) * ivoxsize.x );
                index_phantom.y = ui32 ( ( pos.y + vol.off_y ) * ivoxsize.y );
                index_phantom.z = ui32 ( ( pos.z + vol.off_z ) * ivoxsize.z );

                index_phantom.w = index_phantom.z*vol.nb_vox_x*vol.nb_vox_y
                        + index_phantom.y*vol.nb_vox_x
                        + index_phantom.x; // linear index

                ggems_atomic_add( hist_map.interaction, index_phantom.w, ui32 ( 1 ) );

                // Add the particle energy in memory
                ggems_atomic_add( hist_map.energy, index_phantom.w, particles.E[ part_id ] );
            }

        }
        else // Analog
        {
            if ( next_discrete_process != GEOMETRY_BOUNDARY )
            {
                /// Drop energy ////////////

                // If gamma particle is dead (PE, Compton or energy cut)
                if ( particles.endsimu[ part_id ] == PARTICLE_DEAD &&  particles.E[ part_id ] != 0.0f )
                {
                    dose_record_standard( dosi, particles.E[ part_id ], pos.x,
                                          pos.y, pos.z );
                }

                // If electron particle has energy
                if ( electron.E != 0.0f )
                {
                    dose_record_standard( dosi, electron.E, pos.x,
                                          pos.y, pos.z );
                }
            } // geom boundary

        } // Analog

    } // TLE

    // store the new position
    particles.px[part_id] = pos.x;
    particles.py[part_id] = pos.y;
    particles.pz[part_id] = pos.z;
}


// Se TLE function
__host__ __device__ void VPIORTN::track_seTLE( ParticlesData &particles, VoxVolumeData<ui16> vol, COOHistoryMap coo_hist_map,
                                               DoseData dose, Mu_MuEn_Table mu_table,
                                               ui32 nb_of_rays, f32 edep_th, ui32 id )
{
    // Read an interaction position
    ui16 vox_x = coo_hist_map.x[ id ];
    ui16 vox_y = coo_hist_map.y[ id ];
    ui16 vox_z = coo_hist_map.z[ id ];

    // Nb of interaction and total energy
    ui32 nb_int = coo_hist_map.interaction[ id ];
    f32 mean_energy = coo_hist_map.energy[ id ] / f32( nb_int );

    // Total nb of rays is given by the ponderation of the nb of interactions
    nb_of_rays *= nb_int;

    // Weight in
    f32 win_init = 1 / f32( nb_of_rays );

    // vars DDA
    ui32 n;
    f32 length;

    f32 flength;
    f32 lx, ly, lz;
    f32 fxinc, fyinc, fzinc, fx, fy, fz;
    ui32 ix, iy, iz;
    f32 diffx, diffy, diffz;

    ui32 step = vol.nb_vox_x * vol.nb_vox_y;
    ui32 ind;

    // Rnd ray
    f32 phi, theta;
    f32xyz ray_p, ray_q, ray_d;
    f32 aabb_dist;

    // seTLE
    ui16 mat_id;
    ui32 E_index;
    f32 mu, mu_en, path_length;
    f32 win, wout, edep;

    // Pre-compute the energy index to access to the mu and mu_en tables
    E_index = binary_search ( mean_energy, mu_table.E_bins, mu_table.nb_bins );

    // Loop over raycasting
    ui32 iray=0; while ( iray < nb_of_rays )
    {
        // Generate a ray
        ray_p.x = f32(vox_x) + 0.5f;  // Center of the voxel
        ray_p.y = f32(vox_y) + 0.5f;  // Center of the voxel
        ray_p.z = f32(vox_z) + 0.5f;  // Center of the voxel

        phi = prng_uniform( particles, id );
        theta = prng_uniform( particles, id );
        phi  *= gpu_twopi;
        theta = acosf ( 1.0f - 2.0f*theta );
        ray_d.x = cosf( phi ) * sinf( theta );
        ray_d.y = sinf( phi ) * sinf( theta );
        ray_d.z = cosf( theta );

        // Get the second voxel point for the ray
        aabb_dist = hit_ray_AABB(ray_p, ray_d, 0, vol.nb_vox_x, 0, vol.nb_vox_y, 0, vol.nb_vox_z);
        ray_q = fxyz_add ( ray_p, fxyz_scale ( ray_d, aabb_dist ) );

        // DDA params
        diffx = floorf( ray_q.x ) - vox_x;
        diffy = floorf( ray_q.y ) - vox_y;
        diffz = floorf( ray_q.z ) - vox_z;

        lx = fabsf( diffx );
        ly = fabsf( diffy );
        lz = fabsf( diffz );

        length = fmaxf( ly, fmaxf( lx, lz ) );
        flength = 1.0 / length;

        fxinc = diffx * flength;
        fyinc = diffy * flength;
        fzinc = diffz * flength;

        fx = ray_p.x;
        fy = ray_p.y;
        fz = ray_p.z;

        // Path length of the increment step in mm
        path_length = powf( (fxinc*vol.spacing_x)*(fxinc*vol.spacing_x) +
                            (fyinc*vol.spacing_y)*(fyinc*vol.spacing_y) +
                            (fzinc*vol.spacing_z)*(fzinc*vol.spacing_z), 0.5f );

        // Init the particle weigth
        win = win_init;

        // DDA loop
        n = 0; while ( n < length )
        {
            ix = (ui32)fx; iy = (ui32)fy; iz = (ui32)fz;

            // if inside the volume
            if (fx >= 0 && fy >= 0 && fz >= 0 &&
                ix < vol.nb_vox_x && iy < vol.nb_vox_y && iz < vol.nb_vox_z
                && n != 0 )
            {

                // get index and accumulate
                ind = iz*step + iy*vol.nb_vox_x + ix;

                // Read material
                mat_id = vol.values[ ind ];

                // Get mu and mu_en for the current E
                if ( E_index == 0 )
                {
                    mu = mu_table.mu[ mat_id*mu_table.nb_bins ];
                    mu_en = mu_table.mu_en[ mat_id*mu_table.nb_bins ];
                }
                else
                {
                    mu = linear_interpolation( mu_table.E_bins[E_index-1],  mu_table.mu[mat_id*mu_table.nb_bins + E_index-1],
                                               mu_table.E_bins[E_index],    mu_table.mu[mat_id*mu_table.nb_bins + E_index],
                                               mean_energy );

                    mu_en = linear_interpolation( mu_table.E_bins[E_index-1],  mu_table.mu_en[mat_id*mu_table.nb_bins + E_index-1],
                                                  mu_table.E_bins[E_index],    mu_table.mu_en[mat_id*mu_table.nb_bins + E_index],
                                                  mean_energy );
                }

                // Compute the weight out
                wout = win * expf( -mu * path_length / 10.0 ); // Factor from GATE?? - JB

                // Compute the energy to drop
                edep = mean_energy * mu_en/mu * ( win - wout );

                // Drop energy
                ggems_atomic_add_f64( dose.edep, ind, f64( edep ) );
                ggems_atomic_add_f64( dose.edep_squared, ind, f64( edep) * f64( edep ) );
                ggems_atomic_add( dose.number_of_hits, ind, ui32 ( 1 ) );

                // Update the weight
                win = wout;

                // Energy cut
                if (edep <= edep_th) break;

            }

            // step the line
            fx += fxinc;
            fy += fyinc;
            fz += fzinc;
            ++n;
        }

        ++iray;
    } // Rays

}


/// KERNELS /////////////////////////////////


// Device Kernel that move particles to the voxelized volume boundary
__global__ void VPIORTN::kernel_device_track_to_in( ParticlesData particles, f32 xmin, f32 xmax,
                                                    f32 ymin, f32 ymax, f32 zmin, f32 zmax, f32 tolerance )
{  
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;    
    transport_track_to_in_AABB( particles, xmin, xmax, ymin, ymax, zmin, zmax, tolerance, id);
}


// Host Kernel that move particles to the voxelized volume boundary
void VPIORTN::kernel_host_track_to_in( ParticlesData particles, f32 xmin, f32 xmax,
                                     f32 ymin, f32 ymax, f32 zmin, f32 zmax, f32 tolerance, ui32 part_id )
{       
    transport_track_to_in_AABB( particles, xmin, xmax, ymin, ymax, zmin, zmax, tolerance, part_id);
}

// Device kernel that track particles within the voxelized volume until boundary
__global__ void VPIORTN::kernel_device_track_to_out( ParticlesData particles,
                                                     VoxVolumeData<ui16> vol,
                                                     MaterialsTable materials,
                                                     PhotonCrossSectionTable photon_CS_table,
                                                     GlobalSimulationParametersData parameters,
                                                     DoseData dosi,
                                                     Mu_MuEn_Table mu_table,
                                                     HistoryMap hist_map )
{   
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;    

#ifdef SKIP_VOXEL
    particles.geometry_id[ id ] = 999999999; // Init with unknow material id
#endif

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE )
    {
        VPIORTN::track_to_out( particles, vol, materials, photon_CS_table, parameters, dosi, mu_table, hist_map, id );
    }

}

// Host kernel that track particles within the voxelized volume until boundary
void VPIORTN::kernel_host_track_to_out( ParticlesData particles,
                                       VoxVolumeData<ui16> vol,
                                       MaterialsTable materials,
                                       PhotonCrossSectionTable photon_CS_table,
                                       GlobalSimulationParametersData parameters,
                                       DoseData dosi,
                                       Mu_MuEn_Table mu_table,
                                       HistoryMap hist_map )
{

    ui32 id=0;
    while ( id < particles.size )
    {
        #ifdef SKIP_VOXEL
            particles.geometry_id[ id ] = 999999999; // Init with unknow material id
        #endif

        // Stepping loop - Get out of loop only if the particle was dead and it was a primary
        while ( particles.endsimu[id] != PARTICLE_DEAD && particles.endsimu[id] != PARTICLE_FREEZE )
        {
            VPIORTN::track_to_out( particles, vol, materials, photon_CS_table, parameters, dosi, mu_table, hist_map, id );
        }
        ++id;
    }
}


// Device kernel that perform seTLE
__global__ void VPIORTN::kernel_device_seTLE( ParticlesData particles, VoxVolumeData<ui16> vol,
                                              COOHistoryMap coo_hist_map,
                                              DoseData dosi,
                                              Mu_MuEn_Table mu_table , ui32 nb_of_rays , f32 edep_th )
{
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= coo_hist_map.nb_data ) return;

    VPIORTN::track_seTLE( particles, vol, coo_hist_map, dosi, mu_table, nb_of_rays, edep_th, id );
}

// Host kernel that perform seTLE
void VPIORTN::kernel_host_seTLE( ParticlesData particles, VoxVolumeData<ui16> vol,
                                 COOHistoryMap coo_hist_map,
                                 DoseData dosi,
                                 Mu_MuEn_Table mu_table , ui32 nb_of_rays , f32 edep_th )
{
    ui32 id = 0;
    while ( id < coo_hist_map.nb_data )
    {
        VPIORTN::track_seTLE( particles, vol, coo_hist_map, dosi, mu_table, nb_of_rays, edep_th, id );
        ++id;
    }
}

////:: Privates

bool VoxPhanIORTNav::m_check_mandatory()
{

    if ( m_phantom.data_h.nb_vox_x == 0 || m_phantom.data_h.nb_vox_y == 0 || m_phantom.data_h.nb_vox_z == 0 ||
         m_phantom.data_h.spacing_x == 0 || m_phantom.data_h.spacing_y == 0 || m_phantom.data_h.spacing_z == 0 ||
         m_phantom.list_of_materials.size() == 0 || m_materials_filename.empty() )
    {
        return false;
    }
    else
    {
        return true;
    }

}

// Init mu and mu_en table
void VoxPhanIORTNav::m_init_mu_table()
{
    // Load mu data
    f32 *energies  = new f32[mu_nb_energies];
    f32 *mu        = new f32[mu_nb_energies];
    f32 *mu_en     = new f32[mu_nb_energies];
    ui32 *mu_index = new ui32[mu_nb_elements];

    ui32 index_table = 0;
    ui32 index_data = 0;

    for (ui32 i= 0; i < mu_nb_elements; i++)
    {
        ui32 nb_energies = mu_nb_energy_bin[ i ];
        mu_index[ i ] = index_table;

        for (ui32 j = 0; j < nb_energies; j++)
        {
            energies[ index_table ] = mu_data[ index_data++ ];
            mu[ index_table ]       = mu_data[ index_data++ ];
            mu_en[ index_table ]    = mu_data[ index_data++ ];
            index_table++;
        }
    }

    // Build mu and mu_en according material
    m_mu_table.nb_mat = m_materials.data_h.nb_materials;
    m_mu_table.E_max = m_params.data_h.cs_table_max_E;
    m_mu_table.E_min = m_params.data_h.cs_table_min_E;
    m_mu_table.nb_bins = m_params.data_h.cs_table_nbins;

    HANDLE_ERROR( hipMallocManaged( &(m_mu_table.E_bins), m_mu_table.nb_bins*sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_mu_table.mu), m_mu_table.nb_mat*m_mu_table.nb_bins*sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_mu_table.mu_en), m_mu_table.nb_mat*m_mu_table.nb_bins*sizeof( f32 ) ) );

    // Fill energy table with log scale
    f32 slope = log(m_mu_table.E_max / m_mu_table.E_min);
    ui32 i = 0;
    while (i < m_mu_table.nb_bins) {
        m_mu_table.E_bins[ i ] = m_mu_table.E_min * exp( slope * ( (f32)i / ( (f32)m_mu_table.nb_bins-1 ) ) ) * MeV;
        ++i;
    }

    // For each material and energy bin compute mu and muen
    ui32 imat = 0;
    ui32 abs_index, E_index, mu_index_E;
    ui32 iZ, Z;
    f32 energy, mu_over_rho, mu_en_over_rho, frac;
    while (imat < m_mu_table.nb_mat) {

        // for each energy bin
        i=0; while (i < m_mu_table.nb_bins) {

            // absolute index to store data within the table
            abs_index = imat*m_mu_table.nb_bins + i;

            // Energy value
            energy = m_mu_table.E_bins[ i ];

            // For each element of the material
            mu_over_rho = 0.0f; mu_en_over_rho = 0.0f;
            iZ=0; while (iZ < m_materials.data_h.nb_elements[ imat ]) {

                // Get Z and mass fraction
                Z = m_materials.data_h.mixture[ m_materials.data_h.index[ imat ] + iZ ];
                frac = m_materials.data_h.mass_fraction[ m_materials.data_h.index[ imat ] + iZ ];

                // Get energy index
                mu_index_E = mu_index_energy[ Z ];
                E_index = binary_search ( energy, energies, mu_index_E+mu_nb_energy_bin[ Z ], mu_index_E );

                // Get mu an mu_en from interpolation
                if ( E_index == mu_index_E )
                {
                    mu_over_rho += mu[ E_index ];
                    mu_en_over_rho += mu_en[ E_index ];
                }
                else
                {
                    mu_over_rho += frac * linear_interpolation(energies[E_index-1],  mu[E_index-1],
                                                               energies[E_index],    mu[E_index],
                                                               energy);
                    mu_en_over_rho += frac * linear_interpolation(energies[E_index-1],  mu_en[E_index-1],
                                                                  energies[E_index],    mu_en[E_index],
                                                                  energy);
                }
                ++iZ;
            }

            // Store values
            m_mu_table.mu[ abs_index ] = mu_over_rho * m_materials.data_h.density[ imat ] / (g/cm3);
            m_mu_table.mu_en[ abs_index ] = mu_en_over_rho * m_materials.data_h.density[ imat ] / (g/cm3);

            ++i;



        } // E bin

        ++imat;


    } // Mat

}

// Compress history map to be process by the GPU (in a non-sparse way)
void VoxPhanIORTNav::m_compress_history_map()
{
    // 1. count the number of non-zero
    ui32 ct = 0; ui32 i = 0; ui32 val_int;
    while ( i < m_phantom.data_h.number_of_voxels )
    {
        val_int = m_hist_map.interaction[ i++ ];
        if ( val_int ) ++ct;
    }
    m_coo_hist_map.nb_data = ct;
    GGcout << "Coo History map has " << ct << " non-zeros" << GGendl;

    // 2. init memory
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.x), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.y), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.z), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.energy), ct * sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.interaction), ct * sizeof( ui32 ) ) );

    // 3. compressed data
    ui16 x, y, z;
    z = i = ct = 0;

    while ( z < m_phantom.data_h.nb_vox_z )
    {
        y = 0;
        while ( y < m_phantom.data_h.nb_vox_y )
        {
            x = 0;
            while ( x < m_phantom.data_h.nb_vox_x )
            {
                val_int = m_hist_map.interaction[ i ];
                if ( val_int )
                {
                    m_coo_hist_map.x[ ct ] = x;
                    m_coo_hist_map.y[ ct ] = y;
                    m_coo_hist_map.z[ ct ] = z;
                    m_coo_hist_map.interaction[ ct ] = val_int;
                    m_coo_hist_map.energy[ ct ] = m_hist_map.energy[ i ];
                    ++ct;
                }
                ++i;
                ++x;
            }
            ++y;
        }
        ++z;
    }

}

// return memory usage
ui64 VoxPhanIORTNav::m_get_memory_usage()
{
    ui64 mem = 0;

    // First the voxelized phantom
    mem += ( m_phantom.data_h.number_of_voxels * sizeof( ui16 ) );
    // Then material data
    mem += ( ( 3 * m_materials.data_h.nb_elements_total + 23 * m_materials.data_h.nb_materials ) * sizeof( f32 ) );
    // Then cross sections (gamma)
    ui64 n = m_cross_sections.photon_CS.data_h.nb_bins;
    ui64 k = m_cross_sections.photon_CS.data_h.nb_mat;
    mem += ( ( n + 3*n*k + 3*101*n ) * sizeof( f32 ) );
    // Cross section (electron)
    mem += ( n*k*7*sizeof( f32 ) );
    // Finally the dose map
    n = m_dose_calculator.dose.tot_nb_dosels;
    mem += ( 2*n*sizeof( f64 ) + n*sizeof( ui32 ) );
    mem += ( 20 * sizeof( f32 ) );

    // If TLE
    if ( m_flag_TLE )
    {
        n = m_mu_table.nb_bins;
        mem += ( n*k*2 * sizeof( f32 ) ); // mu and mu_en
        mem += ( n*sizeof( f32 ) );       // energies
    }

    // If seTLE
    if ( m_flag_TLE == seTLE )
    {
        mem += ( m_phantom.data_h.number_of_voxels * ( sizeof( ui32 ) + sizeof( f32 ) ) );
    }

    return mem;
}

////:: Main functions

VoxPhanIORTNav::VoxPhanIORTNav ()
{
    // Default doxel size (if 0 = same size to the phantom)
    m_dosel_size_x = 0;
    m_dosel_size_y = 0;
    m_dosel_size_z = 0;

    m_xmin = 0.0; m_xmax = 0.0;
    m_ymin = 0.0; m_ymax = 0.0;
    m_zmin = 0.0; m_zmax = 0.0;

    m_flag_TLE = analog;

    m_materials_filename = "";

    // Mu table
    m_mu_table.nb_mat = 0;
    m_mu_table.nb_bins = 0;
    m_mu_table.E_max = 0;
    m_mu_table.E_min = 0;

    m_mu_table.E_bins = NULL;
    m_mu_table.mu = NULL;
    m_mu_table.mu_en = NULL;

    m_hist_map.interaction = NULL;
    m_hist_map.energy = NULL;

    m_coo_hist_map.x = NULL;
    m_coo_hist_map.y = NULL;
    m_coo_hist_map.z = NULL;
    m_coo_hist_map.energy = NULL;
    m_coo_hist_map.interaction = NULL;
    m_coo_hist_map.nb_data = 0;

    m_mu_table.flag = analog; // Not used

    set_name( "VoxPhanIORTNav" );
}

void VoxPhanIORTNav::track_to_in( Particles particles )
{

    if ( m_params.data_h.device_target == CPU_DEVICE )
    {
        ui32 id=0;
        while ( id<particles.size )
        {
            VPDN::kernel_host_track_to_in ( particles.data_h, m_phantom.data_h.xmin, m_phantom.data_h.xmax,
                                            m_phantom.data_h.ymin, m_phantom.data_h.ymax,
                                            m_phantom.data_h.zmin, m_phantom.data_h.zmax,
                                            m_params.data_h.geom_tolerance,
                                            id );
            ++id;
        }
    }
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

        VPIORTN::kernel_device_track_to_in<<<grid, threads>>> ( particles.data_d, m_phantom.data_d.xmin, m_phantom.data_d.xmax,
                                                                               m_phantom.data_d.ymin, m_phantom.data_d.ymax,
                                                                               m_phantom.data_d.zmin, m_phantom.data_d.zmax,
                                                                               m_params.data_d.geom_tolerance );
        cuda_error_check ( "Error ", " Kernel_VoxPhanIORT (track to in)" );
        hipDeviceSynchronize();
    }

}

void VoxPhanIORTNav::track_to_out ( Particles particles )
{
    //
    if ( m_params.data_h.device_target == CPU_DEVICE )
    {
        VPIORTN::kernel_host_track_to_out( particles.data_h, m_phantom.data_h,
                                           m_materials.data_h, m_cross_sections.photon_CS.data_h,
                                           m_params.data_h, m_dose_calculator.dose,
                                           m_mu_table, m_hist_map );

        // Apply seTLE: splitting and determinstic raycasting
        if( m_flag_TLE == seTLE )
        {
            f64 t_start = get_time();
            m_compress_history_map();
            GGcout_time ( "Compress history map", get_time()-t_start );

            t_start = get_time();
            VPIORTN::kernel_host_seTLE( particles.data_h, m_phantom.data_h,
                                        m_coo_hist_map, m_dose_calculator.dose,
                                        m_mu_table, 100, 0.0 *eV );
            GGcout_time ( "Raycast", get_time()-t_start );
            GGnewline();

        }

    }
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {       
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;
        VPIORTN::kernel_device_track_to_out<<<grid, threads>>> ( particles.data_d, m_phantom.data_d, m_materials.data_d,
                                                              m_cross_sections.photon_CS.data_d,
                                                              m_params.data_d, m_dose_calculator.dose,
                                                              m_mu_table, m_hist_map );
        cuda_error_check ( "Error ", " Kernel_VoxPhanDosi (track to out)" );             
        hipDeviceSynchronize();

        // Apply seTLE: splitting and determinstic raycasting
        if( m_flag_TLE == seTLE )
        {
            f64 t_start = get_time();
            m_compress_history_map();
            GGcout_time ( "Compress history map", get_time()-t_start );

            threads.x = m_params.data_h.gpu_block_size;//
            grid.x = ( m_coo_hist_map.nb_data + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

            t_start = get_time();
            VPIORTN::kernel_device_seTLE<<<grid, threads>>> ( particles.data_d, m_phantom.data_d,
                                                              m_coo_hist_map, m_dose_calculator.dose,
                                                              m_mu_table, 1000, 0.0 *eV );
            cuda_error_check ( "Error ", " Kernel_device_seTLE" );
            hipDeviceSynchronize();
            GGcout_time ( "Raycast", get_time()-t_start );
            GGnewline();
        }
    }
        
}

void VoxPhanIORTNav::load_phantom_from_mhd ( std::string filename, std::string range_mat_name )
{
    m_phantom.load_from_mhd ( filename, range_mat_name );
}

void VoxPhanIORTNav::write ( std::string filename )
{
//     m_dose_calculator.m_copy_dose_gpu2cpu();

    m_dose_calculator.write ( filename );
}

// Export density values of the phantom
void VoxPhanIORTNav::export_density_map( std::string filename )
{
    ui32 N = m_phantom.data_h.number_of_voxels;
    f32 *density = new f32[ N ];
    ui32 i = 0;
    while (i < N)
    {
        density[ i ] = m_materials.data_h.density[ m_phantom.data_h.values[ i ] ];
        ++i;
    }

    f32xyz offset = make_f32xyz( m_phantom.data_h.off_x, m_phantom.data_h.off_y, m_phantom.data_h.off_z );
    f32xyz voxsize = make_f32xyz( m_phantom.data_h.spacing_x, m_phantom.data_h.spacing_y, m_phantom.data_h.spacing_z );
    ui32xyz nbvox = make_ui32xyz( m_phantom.data_h.nb_vox_x, m_phantom.data_h.nb_vox_y, m_phantom.data_h.nb_vox_z );

    ImageIO *im_io = new ImageIO;
    im_io->write_3D( filename, density, nbvox, offset, voxsize );
    delete im_io;
}

// Export materials index of the phantom
void VoxPhanIORTNav::export_materials_map( std::string filename )
{
    f32xyz offset = make_f32xyz( m_phantom.data_h.off_x, m_phantom.data_h.off_y, m_phantom.data_h.off_z );
    f32xyz voxsize = make_f32xyz( m_phantom.data_h.spacing_x, m_phantom.data_h.spacing_y, m_phantom.data_h.spacing_z );
    ui32xyz nbvox = make_ui32xyz( m_phantom.data_h.nb_vox_x, m_phantom.data_h.nb_vox_y, m_phantom.data_h.nb_vox_z );

    ImageIO *im_io = new ImageIO;
    im_io->write_3D( filename, m_phantom.data_h.values, nbvox, offset, voxsize );
    delete im_io;
}

// Export history map from seTLE
void VoxPhanIORTNav::export_history_map( std::string filename )
{
    if ( m_flag_TLE == seTLE )
    {
        f32xyz offset = make_f32xyz( m_phantom.data_h.off_x, m_phantom.data_h.off_y, m_phantom.data_h.off_z );
        f32xyz voxsize = make_f32xyz( m_phantom.data_h.spacing_x, m_phantom.data_h.spacing_y, m_phantom.data_h.spacing_z );
        ui32xyz nbvox = make_ui32xyz( m_phantom.data_h.nb_vox_x, m_phantom.data_h.nb_vox_y, m_phantom.data_h.nb_vox_z );


        // Create an IO object
        ImageIO *im_io = new ImageIO;

        std::string format = im_io->get_extension( filename );
        filename = im_io->get_filename_without_extension( filename );

        // Convert Edep from f64 to f32
        ui32 tot = m_dose_calculator.dose.nb_dosels.x * m_dose_calculator.dose.nb_dosels.y * m_dose_calculator.dose.nb_dosels.z;
        f32 *f32edep = new f32[ tot ];
        ui32 i=0; while ( i < tot )
        {
            f32edep[ i ] = (f32)m_dose_calculator.dose.edep[ i ];
            ++i;
        }

        // Get output name
        std::string int_out( filename + "-Interaction." + format );
        std::string energy_out( filename + "-Energies." + format );

        // Export
        im_io->write_3D( int_out, m_hist_map.interaction, nbvox, offset, voxsize );
        im_io->write_3D( energy_out, m_hist_map.energy, nbvox, offset, voxsize );
    }
    else
    {
        GGwarn << "History map export is only available while using seTLE option!" << GGendl;
    }
}

void VoxPhanIORTNav::initialize ( GlobalSimulationParameters params )
{
    // Check params
    if ( !m_check_mandatory() )
    {
        print_error ( "VoxPhanIORT: missing parameters." );
        exit_simulation();
    }

    // Params
    m_params = params;

    // Phantom
    m_phantom.set_name( "VoxPhanIORTNav" );
    m_phantom.initialize( params );

    // Materials table
    m_materials.load_materials_database( m_materials_filename );
    m_materials.initialize( m_phantom.list_of_materials, params );    

    // Cross Sections
    m_cross_sections.initialize( m_materials, params );

    // Init dose map
    m_dose_calculator.set_voxelized_phantom( m_phantom );
    m_dose_calculator.set_materials( m_materials );
    m_dose_calculator.set_dosel_size( m_dosel_size_x, m_dosel_size_y, m_dosel_size_z );
    m_dose_calculator.set_voi( m_xmin, m_xmax, m_ymin, m_ymax, m_zmin, m_zmax );
    m_dose_calculator.initialize( m_params ); // CPU&GPU

    // If TLE init mu and mu_en table
    if ( m_flag_TLE )
    {
        m_init_mu_table();
    }

    // if seTLE init history map
    if ( m_flag_TLE == seTLE )
    {
        HANDLE_ERROR( hipMallocManaged( &(m_hist_map.interaction), m_phantom.data_h.number_of_voxels * sizeof( ui32 ) ) );
        HANDLE_ERROR( hipMallocManaged( &(m_hist_map.energy), m_phantom.data_h.number_of_voxels * sizeof( f32 ) ) );

        ui32 i=0; while (i < m_phantom.data_h.number_of_voxels )
        {
            m_hist_map.interaction[ i ] = 0;
            m_hist_map.energy[ i ] = 0.0;
            ++i;
        }
    }

    // Some verbose if required
    if ( params.data_h.display_memory_usage )
    {
        ui64 mem = m_get_memory_usage();
        GGcout_mem("VoxPhanIORTNav", mem);
    }

}

void VoxPhanIORTNav::calculate_dose_to_water()
{
    m_dose_calculator.calculate_dose_to_water();

}

void VoxPhanIORTNav::calculate_dose_to_phantom()
{
    m_dose_calculator.calculate_dose_to_phantom();

}

void VoxPhanIORTNav::set_materials( std::string filename )
{
    m_materials_filename = filename;
}

/*
void VoxPhanIORTNav::set_doxel_size( f32 sizex, f32 sizey, f32 sizez )
{
    m_doxel_size_x = sizex;
    m_doxel_size_y = sizey;
    m_doxel_size_z = sizez;
}
*/

/*
void VoxPhanIORTNav::set_volume_of_interest( f32 xmin, f32 xmax, f32 ymin, f32 ymax, f32 zmin, f32 zmax )
{
    m_xmin = xmin; m_xmax = xmax;
    m_ymin = ymin; m_ymax = ymax;
    m_zmin = zmin; m_zmax = zmax;
}
*/

void VoxPhanIORTNav::set_kerma_estimator( std::string kind )
{
    // Transform the name of the process in small letter
    std::transform( kind.begin(), kind.end(), kind.begin(), ::tolower );

    if ( kind == "tle" )
    {
        m_flag_TLE = TLE;
        m_mu_table.flag = TLE; // Use TLE
    }
    else if ( kind == "setle" )
    {
        m_flag_TLE = seTLE;
        m_mu_table.flag = seTLE; // Use seTLE
    }
    else if ( kind == "analog" )
    {
        m_flag_TLE = analog;
        m_mu_table.flag = analog; // Use analog
    }
    else
    {
        GGcerr << "Track length estimator not recognized: '" << kind << "'!" << GGendl;
        exit_simulation();
    }
}

VoxVolumeData<f32> * VoxPhanIORTNav::get_dose_map()
{
    return m_dose_calculator.get_dose_map();
}

AabbData VoxPhanIORTNav::get_bounding_box()
{
    AabbData box;

    box.xmin = m_phantom.data_h.xmin;
    box.xmax = m_phantom.data_h.xmax;
    box.ymin = m_phantom.data_h.ymin;
    box.ymax = m_phantom.data_h.ymax;
    box.zmin = m_phantom.data_h.zmin;
    box.zmax = m_phantom.data_h.zmax;

    return box;
}


#undef DEBUG

#endif
