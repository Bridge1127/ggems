#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2017

/*!
 * \file vox_phan_iort_nav.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.2
 * \date 23/03/2016
 *
 * v0.2: JB - Change all structs and remove CPU exec
 *
 */

#ifndef VOX_PHAN_IORT_NAV_CU
#define VOX_PHAN_IORT_NAV_CU

#include "vox_phan_iort_nav.cuh"

////// HOST-DEVICE GPU Codes ////////////////////////////////////////////

__host__ __device__ void VPIORTN::track_to_out_analog( ParticlesData *particles,
                                                       const VoxVolumeData<ui16> *vol,
                                                       const MaterialsData *materials,
                                                       const PhotonCrossSectionData *photon_CS_table,
                                                       const GlobalSimulationParametersData *parameters,
                                                       DoseData *dosi,
                                                       ui32 part_id )
{
    // Read position
    f32xyz pos;
    pos.x = particles->px[part_id];
    pos.y = particles->py[part_id];
    pos.z = particles->pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles->dx[part_id];
    dir.y = particles->dy[part_id];
    dir.z = particles->dz[part_id];

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol->spacing_x;
    ivoxsize.y = 1.0 / vol->spacing_y;
    ivoxsize.z = 1.0 / vol->spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( pos.x + vol->off_x ) * ivoxsize.x );
    index_phantom.y = ui32 ( ( pos.y + vol->off_y ) * ivoxsize.y );
    index_phantom.z = ui32 ( ( pos.z + vol->off_z ) * ivoxsize.z );

    index_phantom.w = index_phantom.z*vol->nb_vox_x*vol->nb_vox_y
                      + index_phantom.y*vol->nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol->values[ index_phantom.w ];

    // Vars
    f32 next_interaction_distance;
    ui8 next_discrete_process;

    //// Find next discrete interaction ///////////////////////////////////////

    photon_get_next_interaction( particles, parameters, photon_CS_table, mat_id, part_id );

    next_interaction_distance = particles->next_interaction_distance[part_id];
    next_discrete_process = particles->next_discrete_process[part_id];

    //// Get the next distance boundary volume /////////////////////////////////

    f32 vox_xmin = index_phantom.x*vol->spacing_x - vol->off_x;
    f32 vox_ymin = index_phantom.y*vol->spacing_y - vol->off_y;
    f32 vox_zmin = index_phantom.z*vol->spacing_z - vol->off_z;
    f32 vox_xmax = vox_xmin + vol->spacing_x;
    f32 vox_ymax = vox_ymin + vol->spacing_y;
    f32 vox_zmax = vox_zmin + vol->spacing_z;

    // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
    // TODO: In theory this have to be applied just at the entry of the particle within the volume
    //       in order to avoid particle entry between voxels. Then, computing improvement can be made
    //       by calling this function only once, just for the particle step=0.    - JB
    pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                            vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters->geom_tolerance );

    f32 boundary_distance = hit_ray_AABB( pos, dir, vox_xmin, vox_xmax,
                                          vox_ymin, vox_ymax, vox_zmin, vox_zmax );

    if ( boundary_distance <= next_interaction_distance )
    {
        next_interaction_distance = boundary_distance + parameters->geom_tolerance; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;
    }

    //// Move particle //////////////////////////////////////////////////////

    // get the new position
    pos = fxyz_add( pos, fxyz_scale( dir, next_interaction_distance ) );

    // get safety position (outside the current voxel)
    pos = transport_get_safety_outside_AABB( pos, vox_xmin, vox_xmax,
                                             vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters->geom_tolerance );

    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance ( pos, vol->xmin, vol->xmax, vol->ymin, vol->ymax,
                                           vol->zmin, vol->zmax, parameters->geom_tolerance ) )
    {
        particles->status[part_id] = PARTICLE_FREEZE;
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    // Resolve process
    if ( next_discrete_process != GEOMETRY_BOUNDARY )
    {
        // Resolve discrete process
        SecParticle electron = photon_resolve_discrete_process ( particles, parameters, photon_CS_table,
                                                     materials, mat_id, part_id );
        /// Energy cut /////////////

        // If gamma particle not enough energy (Energy cut)
        if ( particles->E[ part_id ] <= materials->photon_energy_cut[ mat_id ] )
        {
            // Kill without mercy
            particles->status[ part_id ] = PARTICLE_DEAD;
        }

        /// Drop energy ////////////

        // If gamma particle is dead (PE, Compton or energy cut)
        if ( particles->status[ part_id ] == PARTICLE_DEAD &&  particles->E[ part_id ] != 0.0f )
        {
            dose_record_standard( dosi, particles->E[ part_id ], pos.x,
                                  pos.y, pos.z );
        }

        // If electron particle has energy
        if ( electron.E != 0.0f )
        {
            dose_record_standard( dosi, electron.E, pos.x,
                                  pos.y, pos.z );
        }
    } // geom boundary

    // store the new position
    particles->px[part_id] = pos.x;
    particles->py[part_id] = pos.y;
    particles->pz[part_id] = pos.z;

}

__host__ __device__ void VPIORTN::track_to_out_tle( ParticlesData *particles,
                                                    const VoxVolumeData<ui16> *vol,
                                                    const MaterialsData *materials,
                                                    const PhotonCrossSectionData *photon_CS_table,
                                                    const GlobalSimulationParametersData *parameters,
                                                    DoseData *dosi,
                                                    const Mu_MuEn_Data *mu_table,
                                                    ui32 part_id )
{
    // Read position
    f32xyz pos;
    pos.x = particles->px[part_id];
    pos.y = particles->py[part_id];
    pos.z = particles->pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles->dx[part_id];
    dir.y = particles->dy[part_id];
    dir.z = particles->dz[part_id];

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol->spacing_x;
    ivoxsize.y = 1.0 / vol->spacing_y;
    ivoxsize.z = 1.0 / vol->spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( pos.x + vol->off_x ) * ivoxsize.x );
    index_phantom.y = ui32 ( ( pos.y + vol->off_y ) * ivoxsize.y );
    index_phantom.z = ui32 ( ( pos.z + vol->off_z ) * ivoxsize.z );

    index_phantom.w = index_phantom.z*vol->nb_vox_x*vol->nb_vox_y
                      + index_phantom.y*vol->nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol->values[ index_phantom.w ];

    // Vars
    f32 next_interaction_distance;
    ui8 next_discrete_process;

    //// Find next discrete interaction ///////////////////////////////////////

    photon_get_next_interaction( particles, parameters, photon_CS_table, mat_id, part_id );

    next_interaction_distance = particles->next_interaction_distance[part_id];
    next_discrete_process = particles->next_discrete_process[part_id];

    //// Get the next distance boundary volume /////////////////////////////////

    f32 vox_xmin = index_phantom.x*vol->spacing_x - vol->off_x;
    f32 vox_ymin = index_phantom.y*vol->spacing_y - vol->off_y;
    f32 vox_zmin = index_phantom.z*vol->spacing_z - vol->off_z;
    f32 vox_xmax = vox_xmin + vol->spacing_x;
    f32 vox_ymax = vox_ymin + vol->spacing_y;
    f32 vox_zmax = vox_zmin + vol->spacing_z;

    // get a safety position for the particle within this voxel (sometime a particle can be right between two voxels)
    // TODO: In theory this have to be applied just at the entry of the particle within the volume
    //       in order to avoid particle entry between voxels. Then, computing improvement can be made
    //       by calling this function only once, just for the particle step=0.    - JB
    pos = transport_get_safety_inside_AABB( pos, vox_xmin, vox_xmax,
                                            vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters->geom_tolerance );

    f32 boundary_distance = hit_ray_AABB( pos, dir, vox_xmin, vox_xmax,
                                          vox_ymin, vox_ymax, vox_zmin, vox_zmax );

    if ( boundary_distance <= next_interaction_distance )
    {
        next_interaction_distance = boundary_distance + parameters->geom_tolerance; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;
    }

    //// Move particle //////////////////////////////////////////////////////

    // get the new position
    pos = fxyz_add( pos, fxyz_scale( dir, next_interaction_distance ) );

    // get safety position (outside the current voxel)
    pos = transport_get_safety_outside_AABB( pos, vox_xmin, vox_xmax,
                                             vox_ymin, vox_ymax, vox_zmin, vox_zmax, parameters->geom_tolerance );

    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance ( pos, vol->xmin, vol->xmax, vol->ymin, vol->ymax,
                                           vol->zmin, vol->zmax, parameters->geom_tolerance ) )
    {
        particles->status[part_id] = PARTICLE_FREEZE;
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    f32 energy = particles->E[ part_id ];

    if ( next_discrete_process != GEOMETRY_BOUNDARY )
    {
        // Resolve discrete process
        SecParticle electron = photon_resolve_discrete_process ( particles, parameters, photon_CS_table,
                                                                 materials, mat_id, part_id );
    } // discrete process

    /// Drop energy ////////////

    // Get the mu_en for the current E
    ui32 E_index = binary_search ( energy, mu_table->E_bins, mu_table->nb_bins );

    f32 mu_en;

    if ( E_index == 0 )
    {
        mu_en = mu_table->mu_en[ mat_id*mu_table->nb_bins ];
    }
    else
    {
        mu_en = linear_interpolation( mu_table->E_bins[E_index-1],  mu_table->mu_en[mat_id*mu_table->nb_bins + E_index-1],
                                      mu_table->E_bins[E_index],    mu_table->mu_en[mat_id*mu_table->nb_bins + E_index],
                                      energy );
    }

    //                             record to the old position (current voxel)
    dose_record_TLE( dosi, energy, particles->px[ part_id ], particles->py[ part_id ],
                     particles->pz[ part_id ], next_interaction_distance,  mu_en );

    /// Energy cut /////////////

    // If gamma particle not enough energy (Energy cut)
    if ( particles->E[ part_id ] <= materials->photon_energy_cut[ mat_id ] )
    {
        // Kill without mercy
        particles->status[ part_id ] = PARTICLE_DEAD;
    }

    // store the new position
    particles->px[part_id] = pos.x;
    particles->py[part_id] = pos.y;
    particles->pz[part_id] = pos.z;

}


/// Experimental ///////////////////////////////////////////////

__host__ __device__ void VPIORTN::track_to_out_woodcock( ParticlesData *particles,
                                                         const VoxVolumeData<ui16> *vol,
                                                         const MaterialsData *materials,
                                                         const PhotonCrossSectionData *photon_CS_table,
                                                         const GlobalSimulationParametersData *parameters,
                                                         DoseData *dosi,
                                                         f32* mumax_table,
                                                         ui32 part_id )
{
    // Read position
    f32xyz pos;
    pos.x = particles->px[part_id];
    pos.y = particles->py[part_id];
    pos.z = particles->pz[part_id];

    // Read direction
    f32xyz dir;
    dir.x = particles->dx[part_id];
    dir.y = particles->dy[part_id];
    dir.z = particles->dz[part_id];

    // Vars
    f32 next_interaction_distance;
    f32 interaction_distance;

    //// Find next discrete interaction ///////////////////////////////////////

    // Search the energy index to read CS
    f32 energy = particles->E[part_id];
    ui32 E_index = binary_search( energy, photon_CS_table->E_bins,
                                  photon_CS_table->nb_bins );

    // Get index CS table (considering mat id)
    f32 CS_max = get_CS_from_table( photon_CS_table->E_bins, mumax_table,
                                    energy, E_index, E_index );

    // Woodcock tracking
    next_interaction_distance = -log( prng_uniform( particles, part_id ) ) * CS_max;

    //// Move particle //////////////////////////////////////////////////////

    // get the new position
    pos = fxyz_add ( pos, fxyz_scale ( dir, next_interaction_distance ) );

    // Stop simulation if out of the phantom
    if ( !test_point_AABB_with_tolerance( pos, vol->xmin, vol->xmax, vol->ymin, vol->ymax,
                                          vol->zmin, vol->zmax, parameters->geom_tolerance ) )
    {
        particles->status[part_id] = PARTICLE_FREEZE;
        return;
    }

    // store the new position
    particles->px[part_id] = pos.x;
    particles->py[part_id] = pos.y;
    particles->pz[part_id] = pos.z;

    //// Real or fictif process /////////////////////////////////////////////////

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / vol->spacing_x;
    ivoxsize.y = 1.0 / vol->spacing_y;
    ivoxsize.z = 1.0 / vol->spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32( ( pos.x + vol->off_x ) * ivoxsize.x );
    index_phantom.y = ui32( ( pos.y + vol->off_y ) * ivoxsize.y );
    index_phantom.z = ui32( ( pos.z + vol->off_z ) * ivoxsize.z );

    index_phantom.w = index_phantom.z*vol->nb_vox_x*vol->nb_vox_y
                      + index_phantom.y*vol->nb_vox_x
                      + index_phantom.x; // linear index

    // Get the material that compose this volume
    ui16 mat_id = vol->values[ index_phantom.w ];

    // Get index CS table (considering mat id)
    ui32 CS_index = mat_id*photon_CS_table->nb_bins + E_index;
    f32 sum_CS = 0.0;
    f32 CS_PE = 0.0;
    f32 CS_CPT = 0.0;
    f32 CS_RAY = 0.0;
    next_interaction_distance = F32_MAX;
    ui8 next_discrete_process = 0;

    if ( parameters->physics_list[PHOTON_PHOTOELECTRIC] )
    {
        CS_PE = get_CS_from_table( photon_CS_table->E_bins, photon_CS_table->Photoelectric_Std_CS,
                                   energy, E_index, CS_index );
        sum_CS += CS_PE;
    }

    if ( parameters->physics_list[PHOTON_COMPTON] )
    {
        CS_CPT = get_CS_from_table( photon_CS_table->E_bins, photon_CS_table->Compton_Std_CS,
                                    energy, E_index, CS_index );
        sum_CS += CS_CPT;
    }

    if ( parameters->physics_list[PHOTON_RAYLEIGH] )
    {
        CS_RAY = get_CS_from_table( photon_CS_table->E_bins, photon_CS_table->Rayleigh_Lv_CS,
                                    energy, E_index, CS_index );
        sum_CS += CS_RAY;
    }

    f32 rnd = prng_uniform( particles, part_id );

    if ( rnd > sum_CS * CS_max  )
    {        
        // Fictive interaction, keep going!
        return;
    }

    //// Apply discrete process //////////////////////////////////////////////////

    // Resolve process
    if ( parameters->physics_list[PHOTON_PHOTOELECTRIC] )
    {
        rnd = prng_uniform( particles, part_id );
        interaction_distance = -log( rnd ) / CS_PE;
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_PHOTOELECTRIC;
        }
    }

    if ( parameters->physics_list[PHOTON_COMPTON] )
    {
        rnd = prng_uniform( particles, part_id );
        interaction_distance = -log( rnd ) / CS_CPT;
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_COMPTON;
        }
    }

    if ( parameters->physics_list[PHOTON_RAYLEIGH] )
    {
        rnd = prng_uniform( particles, part_id );
        interaction_distance = -log( rnd ) / CS_RAY;
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_RAYLEIGH;
        }
    }

    // Apply discrete process
    SecParticle electron;
    electron.endsimu = PARTICLE_DEAD;
    electron.dir.x = 0.;
    electron.dir.y = 0.;
    electron.dir.z = 1.;
    electron.E = 0.;    

    if ( next_discrete_process == PHOTON_COMPTON )
    {
        electron = Compton_SampleSecondaries_standard( particles, materials->electron_energy_cut[mat_id],
                                                       parameters->secondaries_list[ELECTRON], part_id );
    }

    if ( next_discrete_process == PHOTON_PHOTOELECTRIC )
    {
        electron = Photoelec_SampleSecondaries_standard( particles, materials, photon_CS_table,
                                                         E_index, materials->electron_energy_cut[mat_id],
                                                         mat_id, parameters->secondaries_list[ELECTRON], part_id );
    }

    if ( next_discrete_process == PHOTON_RAYLEIGH )
    {
        Rayleigh_SampleSecondaries_Livermore( particles, materials, photon_CS_table, E_index, mat_id, part_id );
    }

    /// Energy cut /////////////

    // If gamma particle not enough energy (Energy cut)
    if ( particles->E[ part_id ] <= materials->photon_energy_cut[ mat_id ] )
    {
        // Kill without mercy
        particles->status[ part_id ] = PARTICLE_DEAD;
    }

    /// Drop energy ////////////

    // If gamma particle is dead (PE, Compton or energy cut)
    if ( particles->status[ part_id ] == PARTICLE_DEAD &&  particles->E[ part_id ] != 0.0f )
    {
        dose_record_standard( dosi, particles->E[ part_id ], pos.x,
                              pos.y, pos.z );
    }

    // If electron particle has energy
    if ( electron.E != 0.0f )
    {
        dose_record_standard( dosi, electron.E, pos.x,
                              pos.y, pos.z );
    }

}

//////////////////////////////////////////////////////////////////////

/*
// Se TLE function
__host__ __device__ void VPIORTN::track_seTLE( ParticlesData particles,
                                               VoxVolumeData<ui16> vol,
                                               COOHistoryMap coo_hist_map,
                                               DoseData dose,
                                               Mu_MuEn_Table mu_table,
                                               ui32 nb_of_rays, f32 edep_th, ui32 id )
{
    // Read an interaction position
    ui16 vox_x = coo_hist_map.x[ id ];
    ui16 vox_y = coo_hist_map.y[ id ];
    ui16 vox_z = coo_hist_map.z[ id ];

    // Nb of interaction and total energy
    ui32 nb_int = coo_hist_map.interaction[ id ];
    f32 mean_energy = coo_hist_map.energy[ id ] / f32( nb_int );

    // Total nb of rays is given by the ponderation of the nb of interactions
    nb_of_rays *= nb_int;

    // Weight in
    f32 win_init = 1 / f32( nb_of_rays );

    // vars DDA
    ui32 n;
    f32 length;

    f32 flength;
    f32 lx, ly, lz;
    f32 fxinc, fyinc, fzinc, fx, fy, fz;
    ui32 ix, iy, iz;
    f32 diffx, diffy, diffz;

    ui32 step = vol.nb_vox_x * vol.nb_vox_y;
    ui32 ind;

    // Rnd ray
    f32 phi, theta;
    f32xyz ray_p, ray_q, ray_d;
    f32 aabb_dist;

    // seTLE
    ui16 mat_id;
    ui32 E_index;
    f32 mu, mu_en, path_length;
    f32 win, wout, edep;

    // Pre-compute the energy index to access to the mu and mu_en tables
    E_index = binary_search ( mean_energy, mu_table.E_bins, mu_table.nb_bins );

    // Loop over raycasting
    ui32 iray=0; while ( iray < nb_of_rays )
    {
        // Generate a ray
        ray_p.x = f32(vox_x) + 0.5f;  // Center of the voxel
        ray_p.y = f32(vox_y) + 0.5f;  // Center of the voxel
        ray_p.z = f32(vox_z) + 0.5f;  // Center of the voxel

        phi = prng_uniform( particles, id );
        theta = prng_uniform( particles, id );
        phi  *= gpu_twopi;
        theta = acosf ( 1.0f - 2.0f*theta );
        ray_d.x = cosf( phi ) * sinf( theta );
        ray_d.y = sinf( phi ) * sinf( theta );
        ray_d.z = cosf( theta );

        // Get the second voxel point for the ray
        aabb_dist = hit_ray_AABB(ray_p, ray_d, 0, vol.nb_vox_x, 0, vol.nb_vox_y, 0, vol.nb_vox_z);
        ray_q = fxyz_add ( ray_p, fxyz_scale ( ray_d, aabb_dist ) );

        // DDA params
        diffx = floorf( ray_q.x ) - vox_x;
        diffy = floorf( ray_q.y ) - vox_y;
        diffz = floorf( ray_q.z ) - vox_z;

        lx = fabsf( diffx );
        ly = fabsf( diffy );
        lz = fabsf( diffz );

        length = fmaxf( ly, fmaxf( lx, lz ) );
        flength = 1.0 / length;

        fxinc = diffx * flength;
        fyinc = diffy * flength;
        fzinc = diffz * flength;

        fx = ray_p.x;
        fy = ray_p.y;
        fz = ray_p.z;

        // Path length of the increment step in mm
        path_length = powf( (fxinc*vol.spacing_x)*(fxinc*vol.spacing_x) +
                            (fyinc*vol.spacing_y)*(fyinc*vol.spacing_y) +
                            (fzinc*vol.spacing_z)*(fzinc*vol.spacing_z), 0.5f );

        // Init the particle weigth
        win = win_init;

        // DDA loop
        n = 0; while ( n < length )
        {
            ix = (ui32)fx; iy = (ui32)fy; iz = (ui32)fz;

            // if inside the volume
            if (fx >= 0 && fy >= 0 && fz >= 0 &&
                ix < vol.nb_vox_x && iy < vol.nb_vox_y && iz < vol.nb_vox_z
                && n != 0 )
            {

                // get index and accumulate
                ind = iz*step + iy*vol.nb_vox_x + ix;

                // Read material
                mat_id = vol.values[ ind ];

                // Get mu and mu_en for the current E
                if ( E_index == 0 )
                {
                    mu = mu_table.mu[ mat_id*mu_table.nb_bins ];
                    mu_en = mu_table.mu_en[ mat_id*mu_table.nb_bins ];
                }
                else
                {
                    mu = linear_interpolation( mu_table.E_bins[E_index-1],  mu_table.mu[mat_id*mu_table.nb_bins + E_index-1],
                                               mu_table.E_bins[E_index],    mu_table.mu[mat_id*mu_table.nb_bins + E_index],
                                               mean_energy );

                    mu_en = linear_interpolation( mu_table.E_bins[E_index-1],  mu_table.mu_en[mat_id*mu_table.nb_bins + E_index-1],
                                                  mu_table.E_bins[E_index],    mu_table.mu_en[mat_id*mu_table.nb_bins + E_index],
                                                  mean_energy );
                }

                // Compute the weight out
                wout = win * expf( -mu * path_length / 10.0 ); // Factor from GATE?? - JB

                // Compute the energy to drop
                edep = mean_energy * mu_en/mu * ( win - wout );

                // Drop energy
                ggems_atomic_add_f64( dose.edep, ind, f64( edep ) );
                ggems_atomic_add_f64( dose.edep_squared, ind, f64( edep) * f64( edep ) );
                ggems_atomic_add( dose.number_of_hits, ind, ui32 ( 1 ) );

                // Update the weight
                win = wout;

                // Energy cut
                if (edep <= edep_th) break;

            }

            // step the line
            fx += fxinc;
            fy += fyinc;
            fz += fzinc;
            ++n;
        }

        ++iray;
    } // Rays

}
*/

/// KERNELS /////////////////////////////////


// Device Kernel that move particles to the voxelized volume boundary
__global__ void VPIORTN::kernel_device_track_to_in( ParticlesData *particles, f32 xmin, f32 xmax,
                                                    f32 ymin, f32 ymax, f32 zmin, f32 zmax, f32 tolerance )
{  
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles->size ) return;
    transport_track_to_in_AABB( particles, xmin, xmax, ymin, ymax, zmin, zmax, tolerance, id);
}

// Device kernel that track particles within the voxelized volume until boundary
__global__ void VPIORTN::kernel_device_track_to_out_analog( ParticlesData *particles,
                                                            const VoxVolumeData<ui16> *vol,
                                                            const MaterialsData *materials,
                                                            const PhotonCrossSectionData *photon_CS_table,
                                                            const GlobalSimulationParametersData *parameters,
                                                            DoseData *dosi  )
{
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles->size ) return;

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles->status[id] != PARTICLE_DEAD && particles->status[id] != PARTICLE_FREEZE )
    {
        VPIORTN::track_to_out_analog( particles, vol, materials, photon_CS_table, parameters, dosi, id );
    }
}

// Device kernel that track particles within the voxelized volume until boundary
__global__ void VPIORTN::kernel_device_track_to_out_tle( ParticlesData *particles,
                                                         const VoxVolumeData<ui16> *vol,
                                                         const MaterialsData *materials,
                                                         const PhotonCrossSectionData *photon_CS_table,
                                                         const GlobalSimulationParametersData *parameters,
                                                         DoseData *dosi,
                                                         const Mu_MuEn_Data *mu_table )
{
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles->size ) return;

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles->status[id] != PARTICLE_DEAD && particles->status[id] != PARTICLE_FREEZE )
    {
        VPIORTN::track_to_out_tle( particles, vol, materials, photon_CS_table,
                                   parameters, dosi, mu_table, id );
    }
}

/// Experimental

// Device kernel that track particles within the voxelized volume until boundary
__global__ void VPIORTN::kernel_device_track_to_out_woodcock( ParticlesData *particles,
                                                              const VoxVolumeData<ui16> *vol,
                                                              const MaterialsData *materials,
                                                              const PhotonCrossSectionData *photon_CS_table,
                                                              const GlobalSimulationParametersData *parameters,
                                                              DoseData *dosi,
                                                              f32* mumax_table )
{
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles->size ) return;

    // Stepping loop - Get out of loop only if the particle was dead and it was a primary
    while ( particles->status[id] != PARTICLE_DEAD && particles->status[id] != PARTICLE_FREEZE )
    {
        VPIORTN::track_to_out_woodcock( particles, vol, materials, photon_CS_table,
                                        parameters, dosi, mumax_table, id );
    }
}

/*
// Device kernel that perform seTLE
__global__ void VPIORTN::kernel_device_seTLE( ParticlesData particles,
                                              VoxVolumeData<ui16> vol,
                                              COOHistoryMap coo_hist_map,
                                              DoseData dosi,
                                              Mu_MuEn_Table mu_table , ui32 nb_of_rays , f32 edep_th )
{
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= coo_hist_map.nb_data ) return;

    VPIORTN::track_seTLE( particles, vol, coo_hist_map, dosi, mu_table, nb_of_rays, edep_th, id );
}

// Host kernel that perform seTLE
void VPIORTN::kernel_host_seTLE( ParticlesData particles,
                                 VoxVolumeData<ui16> vol,
                                 COOHistoryMap coo_hist_map,
                                 DoseData dosi,
                                 Mu_MuEn_Table mu_table , ui32 nb_of_rays , f32 edep_th )
{
    ui32 id = 0;
    while ( id < coo_hist_map.nb_data )
    {
        VPIORTN::track_seTLE( particles, vol, coo_hist_map, dosi, mu_table, nb_of_rays, edep_th, id );
        ++id;
    }
}
*/

///////////////////// Privates

bool VoxPhanIORTNav::m_check_mandatory()
{

    if ( m_phantom.h_volume->nb_vox_x == 0 || m_phantom.h_volume->nb_vox_y == 0 || m_phantom.h_volume->nb_vox_z == 0 ||
         m_phantom.h_volume->spacing_x == 0 || m_phantom.h_volume->spacing_y == 0 || m_phantom.h_volume->spacing_z == 0 ||
         m_phantom.list_of_materials.size() == 0 || m_materials_filename.empty() )
    {
        return false;
    }
    else
    {
        return true;
    }

}

// Init mu and mu_en table
void VoxPhanIORTNav::m_init_mu_table()
{
    // Load mu data
    f32 *energies  = new f32[mu_nb_energies];
    f32 *mu        = new f32[mu_nb_energies];
    f32 *mu_en     = new f32[mu_nb_energies];
    ui32 *mu_index = new ui32[mu_nb_elements];

    ui32 index_table = 0;
    ui32 index_data = 0;

    for (ui32 i= 0; i < mu_nb_elements; i++)
    {
        ui32 nb_energies = mu_nb_energy_bin[ i ];
        mu_index[ i ] = index_table;

        for (ui32 j = 0; j < nb_energies; j++)
        {
            energies[ index_table ] = mu_data[ index_data++ ];
            mu[ index_table ]       = mu_data[ index_data++ ];
            mu_en[ index_table ]    = mu_data[ index_data++ ];
            index_table++;
        }
    }

    // Build mu and mu_en according material
    ui32 n = mh_params->cs_table_nbins;
    ui32 k = m_materials.h_materials->nb_materials;

    mh_mu_table->E_bins = (f32*)malloc( n*sizeof(f32) );
    mh_mu_table->mu = (f32*)malloc( n*k*sizeof(f32) );
    mh_mu_table->mu_en = (f32*)malloc( n*k*sizeof(f32) );

    mh_mu_table->nb_mat = k;
    mh_mu_table->E_max = mh_params->cs_table_max_E;
    mh_mu_table->E_min = mh_params->cs_table_min_E;
    mh_mu_table->nb_bins = n;

    // Fill energy table with log scale
    f32 slope = log(mh_mu_table->E_max / mh_mu_table->E_min);
    ui32 i = 0;
    while (i < mh_mu_table->nb_bins) {
        mh_mu_table->E_bins[ i ] = mh_mu_table->E_min * exp( slope * ( (f32)i / ( (f32)mh_mu_table->nb_bins-1 ) ) ) * MeV;
        ++i;
    }

    // For each material and energy bin compute mu and muen
    ui32 imat = 0;
    ui32 abs_index, E_index, mu_index_E;
    ui32 iZ, Z;
    f32 energy, mu_over_rho, mu_en_over_rho, frac;
    while (imat < mh_mu_table->nb_mat) {

        // for each energy bin
        i=0; while (i < mh_mu_table->nb_bins) {

            // absolute index to store data within the table
            abs_index = imat*mh_mu_table->nb_bins + i;

            // Energy value
            energy = mh_mu_table->E_bins[ i ];

            // For each element of the material
            mu_over_rho = 0.0f; mu_en_over_rho = 0.0f;
            iZ=0; while (iZ < m_materials.h_materials->nb_elements[ imat ]) {

                // Get Z and mass fraction
                Z = m_materials.h_materials->mixture[ m_materials.h_materials->index[ imat ] + iZ ];
                frac = m_materials.h_materials->mass_fraction[ m_materials.h_materials->index[ imat ] + iZ ];

                // Get energy index
                mu_index_E = mu_index_energy[ Z ];
                E_index = binary_search ( energy, energies, mu_index_E+mu_nb_energy_bin[ Z ], mu_index_E );

                // Get mu an mu_en from interpolation
                if ( E_index == mu_index_E )
                {
                    mu_over_rho += mu[ E_index ];
                    mu_en_over_rho += mu_en[ E_index ];
                }
                else
                {
                    mu_over_rho += frac * linear_interpolation(energies[E_index-1],  mu[E_index-1],
                                                               energies[E_index],    mu[E_index],
                                                               energy);
                    mu_en_over_rho += frac * linear_interpolation(energies[E_index-1],  mu_en[E_index-1],
                                                                  energies[E_index],    mu_en[E_index],
                                                                  energy);
                }
                ++iZ;
            }

            // Store values
            mh_mu_table->mu[ abs_index ] = mu_over_rho * m_materials.h_materials->density[ imat ] / (g/cm3);
            mh_mu_table->mu_en[ abs_index ] = mu_en_over_rho * m_materials.h_materials->density[ imat ] / (g/cm3);

            ++i;

        } // E bin

        ++imat;


    } // Mat


    ////  GPU copy handling  ////////////////////////:


    /// First, struct allocation
    HANDLE_ERROR( hipMalloc( (void**) &md_mu_table, sizeof( Mu_MuEn_Data ) ) );

    /// Device pointers allocation
    f32* d_E_bins;      // n
    HANDLE_ERROR( hipMalloc((void**) &d_E_bins, n*sizeof(f32)) );
    f32* d_mu;          // n*k
    HANDLE_ERROR( hipMalloc((void**) &d_mu, n*k*sizeof(f32)) );
    f32* d_mu_en;       // n*k
    HANDLE_ERROR( hipMalloc((void**) &d_mu_en, n*k*sizeof(f32)) );

    /// Copy host data to device
    HANDLE_ERROR( hipMemcpy( d_E_bins, mh_mu_table->E_bins,
                              n*sizeof(f32), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( d_mu, mh_mu_table->mu,
                              n*k*sizeof(f32), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( d_mu_en, mh_mu_table->mu_en,
                              n*k*sizeof(f32), hipMemcpyHostToDevice ) );

    /// Bind data to the struct
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->E_bins), &d_E_bins,
                              sizeof(md_mu_table->E_bins), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->mu), &d_mu,
                              sizeof(md_mu_table->mu), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->mu_en), &d_mu_en,
                              sizeof(md_mu_table->mu_en), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->nb_mat), &k,
                              sizeof(md_mu_table->nb_mat), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->nb_bins), &n,
                              sizeof(md_mu_table->nb_bins), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->E_min), &(mh_mu_table->E_min),
                              sizeof(md_mu_table->E_min), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(md_mu_table->E_max), &(mh_mu_table->E_max),
                              sizeof(md_mu_table->E_max), hipMemcpyHostToDevice ) );

}

/*
// Compress history map to be process by the GPU (in a non-sparse way)
void VoxPhanIORTNav::m_compress_history_map()
{
    // 1. count the number of non-zero
    ui32 ct = 0; ui32 i = 0; ui32 val_int;
    while ( i < m_phantom.h_volume->number_of_voxels )
    {
        val_int = m_hist_map.interaction[ i++ ];
        if ( val_int ) ++ct;
    }
    m_coo_hist_map.nb_data = ct;
    GGcout << "Coo History map has " << ct << " non-zeros" << GGendl;

    // 2. init memory
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.x), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.y), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.z), ct * sizeof( ui16 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.energy), ct * sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(m_coo_hist_map.interaction), ct * sizeof( ui32 ) ) );

    // 3. compressed data
    ui16 x, y, z;
    z = i = ct = 0;

    while ( z < m_phantom.h_volume->nb_vox_z )
    {
        y = 0;
        while ( y < m_phantom.h_volume->nb_vox_y )
        {
            x = 0;
            while ( x < m_phantom.h_volume->nb_vox_x )
            {
                val_int = m_hist_map.interaction[ i ];
                if ( val_int )
                {
                    m_coo_hist_map.x[ ct ] = x;
                    m_coo_hist_map.y[ ct ] = y;
                    m_coo_hist_map.z[ ct ] = z;
                    m_coo_hist_map.interaction[ ct ] = val_int;
                    m_coo_hist_map.energy[ ct ] = m_hist_map.energy[ i ];
                    ++ct;
                }
                ++i;
                ++x;
            }
            ++y;
        }
        ++z;
    }

}
*/

// return memory usage
ui64 VoxPhanIORTNav::m_get_memory_usage()
{
    ui64 mem = 0;

    // First the voxelized phantom
    mem += ( m_phantom.h_volume->number_of_voxels * sizeof( ui16 ) );

    // Then material data
    mem += ( ( 3 * m_materials.h_materials->nb_elements_total + 23 * m_materials.h_materials->nb_materials ) * sizeof( f32 ) );

    // Then cross sections (gamma)
    ui64 n = m_cross_sections.h_photon_CS->nb_bins;
    ui64 k = m_cross_sections.h_photon_CS->nb_mat;
    mem += ( ( n + 3*n*k + 3*101*n ) * sizeof( f32 ) );
    // Cross section (electron)
    mem += ( n*k*7*sizeof( f32 ) );

    // Finally the dose map
    n = m_dose_calculator.h_dose->tot_nb_dosels;
    mem += ( 2*n*sizeof( f64 ) + n*sizeof( ui32 ) );
    mem += ( 20 * sizeof( f32 ) );

    // If TLE
    if ( m_flag_vrt == VRT_TLE || m_flag_vrt == VRT_SETLE )
    {
        n = mh_mu_table->nb_bins;
        mem += ( n*k*2 * sizeof( f32 ) ); // mu and mu_en
        mem += ( n*sizeof( f32 ) );       // energies
    }

    // If seTLE
    if ( m_flag_vrt == VRT_SETLE )
    {
        mem += ( m_phantom.h_volume->number_of_voxels * ( sizeof( ui32 ) + sizeof( f32 ) ) );
    }

    // If Woodcock
    if ( m_flag_vrt == VRT_WOODCOCK )
    {
        mem += m_cross_sections.h_photon_CS->nb_bins * sizeof(ui32);
    }

    return mem;
}

////:: Experimental

// Use for woodcock navigation
void VoxPhanIORTNav::m_build_mumax_table()
{
    // Init mumax table vector
    ui32 nb_bins_E = m_cross_sections.h_photon_CS->nb_bins;
    HANDLE_ERROR( hipMallocManaged( &(m_mumax_table), nb_bins_E * sizeof( ui32 ) ) );

    // Find the most attenuate material
    f32 max_dens = 0.0;
    ui32 ind_mat = 0;
    ui32 i = 0; while ( i < m_materials.h_materials->nb_materials )
    {
        if ( m_materials.h_materials->density[i] > max_dens )
        {
            max_dens = m_materials.h_materials->density[ i ];
            ind_mat = i;
        }
        ++i;
    }

    // Build table using max density  [ 1 / Sum( CS ) ]
    i=0; while ( i < nb_bins_E )
    {
        ui32 index = ind_mat * nb_bins_E + i;
        f32 sum_CS = 0.0;

        if ( mh_params->physics_list[PHOTON_PHOTOELECTRIC] )
        {
            sum_CS += m_cross_sections.h_photon_CS->Photoelectric_Std_CS[ index ];
        }

        if ( mh_params->physics_list[PHOTON_COMPTON] )
        {
            sum_CS += m_cross_sections.h_photon_CS->Compton_Std_CS[ index ];
        }

        if ( mh_params->physics_list[PHOTON_RAYLEIGH] )
        {
            sum_CS += m_cross_sections.h_photon_CS->Rayleigh_Lv_CS[ index ];
        }

        m_mumax_table[ i ] = 1.0 / sum_CS;
        ++i;
    }

}


////:: Main functions

VoxPhanIORTNav::VoxPhanIORTNav ()
{
    // Default doxel size (if 0 = same size to the phantom)
    m_dosel_size_x = 0;
    m_dosel_size_y = 0;
    m_dosel_size_z = 0;

    m_xmin = 0.0; m_xmax = 0.0;
    m_ymin = 0.0; m_ymax = 0.0;
    m_zmin = 0.0; m_zmax = 0.0;

    m_flag_vrt = VRT_ANALOG;

    m_materials_filename = "";

    // Init Mu table struct
    mh_mu_table = (Mu_MuEn_Data*)malloc( sizeof(Mu_MuEn_Data) );

    mh_mu_table->nb_mat = 0;
    mh_mu_table->nb_bins = 0;
    mh_mu_table->E_max = 0;
    mh_mu_table->E_min = 0;

    mh_mu_table->E_bins = nullptr;
    mh_mu_table->mu = nullptr;
    mh_mu_table->mu_en = nullptr;
/*
    m_hist_map.interaction = NULL;
    m_hist_map.energy = NULL;

    m_coo_hist_map.x = NULL;
    m_coo_hist_map.y = NULL;
    m_coo_hist_map.z = NULL;
    m_coo_hist_map.energy = NULL;
    m_coo_hist_map.interaction = NULL;
    m_coo_hist_map.nb_data = 0;
*/

    // experimental (Woodcock tracking)
    m_mumax_table = nullptr;

    mh_params = nullptr;
    md_params = nullptr;

    set_name( "VoxPhanIORTNav" );
}

void VoxPhanIORTNav::track_to_in(ParticlesData *d_particles )
{    
    dim3 threads, grid;
    threads.x = mh_params->gpu_block_size;
    grid.x = ( mh_params->size_of_particles_batch + mh_params->gpu_block_size - 1 ) / mh_params->gpu_block_size;

    VPIORTN::kernel_device_track_to_in<<<grid, threads>>> ( d_particles, m_phantom.h_volume->xmin, m_phantom.h_volume->xmax,
                                                            m_phantom.h_volume->ymin, m_phantom.h_volume->ymax,
                                                            m_phantom.h_volume->zmin, m_phantom.h_volume->zmax,
                                                            mh_params->geom_tolerance );
    hipDeviceSynchronize();
    cuda_error_check ( "Error ", " Kernel_VoxPhanIORT (track to in)" );    

}

void VoxPhanIORTNav::track_to_out(ParticlesData *d_particles )
{

    dim3 threads, grid;
    threads.x = mh_params->gpu_block_size;
    grid.x = ( mh_params->size_of_particles_batch + mh_params->gpu_block_size - 1 ) / mh_params->gpu_block_size;

    if ( m_flag_vrt == VRT_ANALOG )
    {
        VPIORTN::kernel_device_track_to_out_analog<<<grid, threads>>>( d_particles,
                                                                       m_phantom.d_volume,
                                                                       m_materials.d_materials,
                                                                       m_cross_sections.d_photon_CS,
                                                                       md_params,
                                                                       m_dose_calculator.d_dose );
    }
    else if ( m_flag_vrt == VRT_TLE )
    {
        VPIORTN::kernel_device_track_to_out_tle<<<grid, threads>>>( d_particles,
                                                                    m_phantom.d_volume,
                                                                    m_materials.d_materials,
                                                                    m_cross_sections.d_photon_CS,
                                                                    md_params,
                                                                    m_dose_calculator.d_dose,
                                                                    md_mu_table );
    }
    else if ( m_flag_vrt == VRT_WOODCOCK )
    {
        VPIORTN::kernel_device_track_to_out_woodcock<<<grid, threads>>>( d_particles,
                                                                    m_phantom.d_volume,
                                                                    m_materials.d_materials,
                                                                    m_cross_sections.d_photon_CS,
                                                                    md_params,
                                                                    m_dose_calculator.d_dose,
                                                                    m_mumax_table );
    }
    hipDeviceSynchronize();
    cuda_error_check ( "Error ", " Kernel_VoxPhanIORT" );    


/*
        // Apply seTLE: splitting and determinstic raycasting
        if( m_flag_TLE == seTLE )
        {
            f64 t_start = get_time();
            m_compress_history_map();
            GGcout_time ( "Compress history map", get_time()-t_start );

            threads.x = m_params.data_h.gpu_block_size;//
            grid.x = ( m_coo_hist_map.nb_data + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

            t_start = get_time();
            VPIORTN::kernel_device_seTLE<<<grid, threads>>> ( d_particles.data_d, m_phantom.data_d,
                                                              m_coo_hist_map, m_dose_calculator.dose,
                                                              m_mu_table, 1000, 0.0 *eV );
            cuda_error_check ( "Error ", " Kernel_device_seTLE" );
            hipDeviceSynchronize();
            GGcout_time ( "Raycast", get_time()-t_start );
            GGnewline();
        }
*/

}

void VoxPhanIORTNav::load_phantom_from_mhd( std::string filename, std::string range_mat_name )
{
    m_phantom.load_from_mhd( filename, range_mat_name );
}

void VoxPhanIORTNav::write( std::string filename )
{
    m_dose_calculator.write( filename );
}

// Export density values of the phantom
void VoxPhanIORTNav::export_density_map( std::string filename )
{
    ui32 N = m_phantom.h_volume->number_of_voxels;
    f32 *density = new f32[ N ];
    ui32 i = 0;
    while (i < N)
    {
        density[ i ] = m_materials.h_materials->density[ m_phantom.h_volume->values[ i ] ];
        ++i;
    }

    f32xyz offset = make_f32xyz( m_phantom.h_volume->off_x, m_phantom.h_volume->off_y, m_phantom.h_volume->off_z );
    f32xyz voxsize = make_f32xyz( m_phantom.h_volume->spacing_x, m_phantom.h_volume->spacing_y, m_phantom.h_volume->spacing_z );
    ui32xyz nbvox = make_ui32xyz( m_phantom.h_volume->nb_vox_x, m_phantom.h_volume->nb_vox_y, m_phantom.h_volume->nb_vox_z );

    ImageIO *im_io = new ImageIO;
    im_io->write_3D( filename, density, nbvox, offset, voxsize );
    delete im_io;
}

// Export materials index of the phantom
void VoxPhanIORTNav::export_materials_map( std::string filename )
{
    f32xyz offset = make_f32xyz( m_phantom.h_volume->off_x, m_phantom.h_volume->off_y, m_phantom.h_volume->off_z );
    f32xyz voxsize = make_f32xyz( m_phantom.h_volume->spacing_x, m_phantom.h_volume->spacing_y, m_phantom.h_volume->spacing_z );
    ui32xyz nbvox = make_ui32xyz( m_phantom.h_volume->nb_vox_x, m_phantom.h_volume->nb_vox_y, m_phantom.h_volume->nb_vox_z );

    ImageIO *im_io = new ImageIO;
    im_io->write_3D( filename, m_phantom.h_volume->values, nbvox, offset, voxsize );
    delete im_io;
}

/*
// Export history map from seTLE
void VoxPhanIORTNav::export_history_map( std::string filename )
{
    if ( m_flag_TLE == seTLE )
    {
        f32xyz offset = make_f32xyz( m_phantom.h_volume->off_x, m_phantom.h_volume->off_y, m_phantom.h_volume->off_z );
        f32xyz voxsize = make_f32xyz( m_phantom.h_volume->spacing_x, m_phantom.h_volume->spacing_y, m_phantom.h_volume->spacing_z );
        ui32xyz nbvox = make_ui32xyz( m_phantom.h_volume->nb_vox_x, m_phantom.h_volume->nb_vox_y, m_phantom.h_volume->nb_vox_z );


        // Create an IO object
        ImageIO *im_io = new ImageIO;

        std::string format = im_io->get_extension( filename );
        filename = im_io->get_filename_without_extension( filename );

        // Convert Edep from f64 to f32
        ui32 tot = m_dose_calculator.dose.nb_dosels.x * m_dose_calculator.dose.nb_dosels.y * m_dose_calculator.dose.nb_dosels.z;
        f32 *f32edep = new f32[ tot ];
        ui32 i=0; while ( i < tot )
        {
            f32edep[ i ] = (f32)m_dose_calculator.dose.edep[ i ];
            ++i;
        }

        // Get output name
        std::string int_out( filename + "-Interaction." + format );
        std::string energy_out( filename + "-Energies." + format );

        // Export
        im_io->write_3D( int_out, m_hist_map.interaction, nbvox, offset, voxsize );
        im_io->write_3D( energy_out, m_hist_map.energy, nbvox, offset, voxsize );
    }
    else
    {
        GGwarn << "History map export is only available while using seTLE option!" << GGendl;
    }
}
*/

void VoxPhanIORTNav::initialize (GlobalSimulationParametersData *h_params , GlobalSimulationParametersData *d_params)
{   

    // Check params
    if ( !m_check_mandatory() )
    {
        print_error ( "VoxPhanIORT: missing parameters." );
        exit_simulation();
    }

    // Params
    mh_params = h_params;
    md_params = d_params;

    // Phantom
    m_phantom.set_name( "VoxPhanIORTNav" );
    m_phantom.initialize();

    // Materials table
    m_materials.load_materials_database( m_materials_filename );
    m_materials.initialize( m_phantom.list_of_materials, mh_params );

    // Cross Sections
    m_cross_sections.initialize( m_materials.h_materials, mh_params );

    // Init dose map
    m_dose_calculator.set_voxelized_phantom( m_phantom );
    m_dose_calculator.set_materials( m_materials );
    m_dose_calculator.set_dosel_size( m_dosel_size_x, m_dosel_size_y, m_dosel_size_z );
    m_dose_calculator.set_voi( m_xmin, m_xmax, m_ymin, m_ymax, m_zmin, m_zmax );
    m_dose_calculator.initialize( mh_params );

    // If TLE init mu and mu_en table
    if ( m_flag_vrt == VRT_TLE || m_flag_vrt == VRT_SETLE )
    {
        m_init_mu_table();
    }

    // If Woodcock init mumax table
    if ( m_flag_vrt == VRT_WOODCOCK )
    {
        m_build_mumax_table();
    }


/*
    // if seTLE init history map
    if ( m_flag_TLE == seTLE )
    {
        HANDLE_ERROR( hipMallocManaged( &(m_hist_map.interaction), m_phantom.h_volume->number_of_voxels * sizeof( ui32 ) ) );
        HANDLE_ERROR( hipMallocManaged( &(m_hist_map.energy), m_phantom.h_volume->number_of_voxels * sizeof( f32 ) ) );

        ui32 i=0; while (i < m_phantom.h_volume->number_of_voxels )
        {
            m_hist_map.interaction[ i ] = 0;
            m_hist_map.energy[ i ] = 0.0;
            ++i;
        }
    }
*/

    // Some verbose if required
    if ( mh_params->display_memory_usage )
    {
        ui64 mem = m_get_memory_usage();
        GGcout_mem("VoxPhanIORTNav", mem);
    }

}

void VoxPhanIORTNav::calculate_dose_to_water()
{
    m_dose_calculator.calculate_dose_to_water();

}

void VoxPhanIORTNav::calculate_dose_to_medium()
{
    m_dose_calculator.calculate_dose_to_medium();

}

void VoxPhanIORTNav::set_materials( std::string filename )
{
    m_materials_filename = filename;
}

void VoxPhanIORTNav::set_vrt( std::string kind )
{
    // Transform the name of the process in small letter
    std::transform( kind.begin(), kind.end(), kind.begin(), ::tolower );

    if ( kind == "tle" )
    {
        m_flag_vrt = VRT_TLE;
    }
    else if ( kind == "setle" )
    {
        m_flag_vrt = VRT_SETLE;
    }
    else if ( kind == "analog" )
    {
        m_flag_vrt = VRT_ANALOG;
    }
    else if ( kind == "woodcock" )
    {
        m_flag_vrt = VRT_WOODCOCK;
    }
    else
    {
        GGcerr << "Variance reduction technique not recognized: '" << kind << "'!" << GGendl;
        exit_simulation();
    }
}

VoxVolumeData<f32> * VoxPhanIORTNav::get_dose_map()
{
    return m_dose_calculator.get_dose_map();
}

AabbData VoxPhanIORTNav::get_bounding_box()
{
    AabbData box;

    box.xmin = m_phantom.h_volume->xmin;
    box.xmax = m_phantom.h_volume->xmax;
    box.ymin = m_phantom.h_volume->ymin;
    box.ymax = m_phantom.h_volume->ymax;
    box.zmin = m_phantom.h_volume->zmin;
    box.zmax = m_phantom.h_volume->zmax;

    return box;
}


#undef DEBUG

#endif
