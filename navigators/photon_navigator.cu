#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file photon_navigator.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 20 novembre 2015
 *
 *
 *
 */

#ifndef PHOTON_NAVIGATOR_CU
#define PHOTON_NAVIGATOR_CU

#include "photon_navigator.cuh"

__host__ __device__ void photon_get_next_interaction ( ParticlesData particles,
                                                       GlobalSimulationParametersData parameters,
                                                       PhotonCrossSectionTable photon_CS_table,
                                                       ui16 mat_id, ui32 part_id )
{
    f32 next_interaction_distance = F32_MAX;
    ui8 next_discrete_process = 0;
    f32 interaction_distance;
    f32 cross_section;

    // Search the energy index to read CS
    f32 energy = particles.E[part_id];
    ui32 E_index = binary_search ( energy, photon_CS_table.E_bins,
                                   photon_CS_table.nb_bins );

    // Get index CS table (considering mat id)
    ui32 CS_index = mat_id*photon_CS_table.nb_bins + E_index;

    // If photoelectric
    if ( parameters.physics_list[PHOTON_PHOTOELECTRIC] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Photoelectric_Std_CS,
                                            energy, E_index, CS_index );
        interaction_distance = -log ( prng_uniform( particles, part_id ) ) / cross_section;

        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_PHOTOELECTRIC;
        }
    }

    // If Compton
    if ( parameters.physics_list[PHOTON_COMPTON] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Compton_Std_CS,
                                            energy, E_index, CS_index );
        interaction_distance = -log ( prng_uniform( particles, part_id ) ) / cross_section;

        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_COMPTON;
        }
    }

    // If Rayleigh
    if ( parameters.physics_list[PHOTON_RAYLEIGH] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Rayleigh_Lv_CS,
                                            energy, E_index, CS_index );
        interaction_distance = -log ( prng_uniform( particles, part_id ) ) / cross_section;

        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_RAYLEIGH;
        }
    }
    // Store results
    particles.next_interaction_distance[part_id] = next_interaction_distance;
    particles.next_discrete_process[part_id] = next_discrete_process;
    particles.E_index[part_id] = E_index;

}



__host__ __device__ SecParticle photon_resolve_discrete_process ( ParticlesData particles,
                                                                  GlobalSimulationParametersData parameters,
                                                                  PhotonCrossSectionTable photon_CS_table,
                                                                  MaterialsTable materials,
                                                                  ui16 mat_id, ui32 part_id )
{

    SecParticle electron;
    electron.endsimu = PARTICLE_DEAD;
    electron.dir.x = 0.;
    electron.dir.y = 0.;
    electron.dir.z = 1.;
    electron.E = 0.;
    ui8 next_discrete_process = particles.next_discrete_process[part_id];    

    if ( next_discrete_process == PHOTON_COMPTON )
    {        
        electron = Compton_SampleSecondaries_standard ( particles, materials.electron_energy_cut[mat_id],
                   part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_PHOTOELECTRIC )
    {        
        electron = Photoelec_SampleSecondaries_standard ( particles, materials, photon_CS_table,
                   particles.E_index[part_id], materials.electron_energy_cut[mat_id],
                   mat_id, part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_RAYLEIGH )
    {        
        Rayleigh_SampleSecondaries_Livermore ( particles, materials, photon_CS_table, particles.E_index[part_id], mat_id, part_id );
    }

    return electron;

}
/*
__host__ __device__ SecParticle _photon_resolve_discrete_process ( ParticlesData particles,
                                                                  GlobalSimulationParametersData parameters,
                                                                  PhotonCrossSectionTable photon_CS_table,
                                                                  MaterialsTable &materials,
                                                                  ui16 mat_id, ui32 part_id )
{

    SecParticle electron;
    electron.endsimu = PARTICLE_DEAD;
    electron.dir.x = 0.;
    electron.dir.y = 0.;
    electron.dir.z = 1.;
    electron.E = 0.;

    ui8 next_discrete_process = particles.next_discrete_process[part_id];

    //printf("id %i resolve  matid %i nb elts %i\n", part_id, mat_id, materials.nb_elements[0]);

    //printf("id %i matindex %i nb elts %i nb mat %i\n", part_id, mat_id, materials.nb_elements[0], materials.nb_materials);

    if ( next_discrete_process == PHOTON_COMPTON )
    {
        //printf("id %i Compton\n", part_id);
        electron = Compton_SampleSecondaries_standard ( particles, materials.electron_energy_cut[mat_id],
                   part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_PHOTOELECTRIC )
    {
        //printf("id %i pe\n", part_id);
        electron = Photoelec_SampleSecondaries_standard ( particles, materials, photon_CS_table,
                   particles.E_index[part_id], materials.electron_energy_cut[mat_id],
                   mat_id, part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_RAYLEIGH )
    {
        //printf("id %i ray\n", part_id);
        _Rayleigh_SampleSecondaries_Livermore( particles, materials, photon_CS_table, particles.E_index[part_id], mat_id, part_id );
        //Rayleigh_SampleSecondaries_Livermore( particles, materials, photon_CS_table, particles.E_index[part_id], mat_id, part_id );
    }

    return electron;

}

*/

















#endif
