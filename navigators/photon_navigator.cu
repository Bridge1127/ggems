#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file photon_navigator.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 20 novembre 2015
 *
 *
 *
 */

#ifndef PHOTON_NAVIGATOR_CU
#define PHOTON_NAVIGATOR_CU

#include "photon_navigator.cuh"

__host__ __device__ void photon_get_next_interaction ( ParticlesData &particles,
        GlobalSimulationParametersData parameters,
        PhotonCrossSectionTable photon_CS_table,
        ui16 mat_id, ui32 part_id )
{
//     printf("    ---> Distances : ");
    f32 next_interaction_distance = F32_MAX;
    ui8 next_discrete_process = 0;
    f32 interaction_distance;
    f32 cross_section;

    // Search the energy index to read CS
    f32 energy = particles.E[part_id];
    ui32 E_index = binary_search ( energy, photon_CS_table.E_bins,
                                   photon_CS_table.nb_bins );

    // If photoelectric
    if ( parameters.physics_list[PHOTON_PHOTOELECTRIC] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Photoelectric_Std_CS,
                                            energy, E_index, mat_id, photon_CS_table.nb_bins );
        f32 alea = JKISS32 ( particles, part_id );
        interaction_distance = -log ( alea ) / cross_section;
//         printf("Energy : %g, PE %g %g %g ",energy,interaction_distance, alea, cross_section);
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_PHOTOELECTRIC;
        }
    }

    // If Compton
    if ( parameters.physics_list[PHOTON_COMPTON] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Compton_Std_CS,
                                            energy, E_index, mat_id, photon_CS_table.nb_bins );
        interaction_distance = -log ( JKISS32 ( particles, part_id ) ) / cross_section;
//         printf("CPT %g ",interaction_distance);
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_COMPTON;
        }
    }

    // If Rayleigh
    if ( parameters.physics_list[PHOTON_RAYLEIGH] )
    {
        cross_section = get_CS_from_table ( photon_CS_table.E_bins, photon_CS_table.Rayleigh_Lv_CS,
                                            energy, E_index, mat_id, photon_CS_table.nb_bins );
        interaction_distance = -log ( JKISS32 ( particles, part_id ) ) / cross_section;
//         printf("R %g ",interaction_distance);
        if ( interaction_distance < next_interaction_distance )
        {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_RAYLEIGH;
        }
    }
//     printf("\n");
    // Store results
    particles.next_interaction_distance[part_id] = next_interaction_distance;
    particles.next_discrete_process[part_id] = next_discrete_process;
    particles.E_index[part_id] = E_index;

}



__host__ __device__ SecParticle photon_resolve_discrete_process ( ParticlesData &particles,
        GlobalSimulationParametersData parameters,
        PhotonCrossSectionTable photon_CS_table,
        MaterialsTable materials,
        ui16 mat_id, ui32 part_id )
{

    SecParticle electron;
    electron.endsimu = PARTICLE_DEAD;
    electron.dir.x = 0.;
    electron.dir.y = 0.;
    electron.dir.z = 1.;
    electron.E = 0.;
    ui8 next_discrete_process = particles.next_discrete_process[part_id];

    if ( next_discrete_process == PHOTON_COMPTON )
    {
        electron = Compton_SampleSecondaries_standard ( particles, materials.electron_energy_cut[mat_id],
                   part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_PHOTOELECTRIC )
    {
        electron = Photoelec_SampleSecondaries_standard ( particles, materials, photon_CS_table,
                   particles.E_index[part_id], materials.electron_energy_cut[mat_id],
                   mat_id, part_id, parameters );
    }

    if ( next_discrete_process == PHOTON_RAYLEIGH )
    {
        Rayleigh_SampleSecondaries_Livermore ( particles, materials, photon_CS_table, particles.E_index[part_id], mat_id, part_id );

    }

    return electron;

}





















#endif
