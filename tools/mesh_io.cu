#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file mesh_io.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date Tuesday June 7, 2016
 *
 * v0.1: JB - First code
 *
 */

#ifndef MESH_IO_CU
#define MESH_IO_CU

#include "mesh_io.cuh"

/////// Main functions

MeshIO::MeshIO()
{
    m_filename   = "";
}

// Read a phasespace file
MeshData MeshIO::read_mesh_file( std::string filename )
{

    MeshData meshes;
    m_filename = filename;

    std::string ext = filename.substr( filename.find_last_of( "." ) + 1 );
    if ( ext == "raw" )
    {
        meshes = m_read_raw_data();
    }
    else if ( ext == "obj" )
    {
        GGcout << "MeshIO: read obj mesh" << GGendl;
        meshes = m_read_obj_data();
    }
    else
    {
        GGcerr << "MeshIO can only read data in raw format (.raw) or Wavefront format (.obj)!" << GGendl;
        exit_simulation();
    }

    return meshes;
}

/////// Private functions

// Skip comment starting with "#"
void MeshIO::m_skip_comment(std::istream & is) {
    i8 c;
    i8 line[1024];
    if (is.eof()) return;
    is >> c;
    while (is && (c=='#')) {
        is.getline(line, 1024);
        is >> c;
        if (is.eof()) return;
    }
    is.unget();
}

// Read the list of tokens in a txt line
std::vector< std::string > MeshIO::m_split_txt( std::string line ) {

    std::istringstream iss(line);
    std::vector<std::string> tokens;
    std::copy(std::istream_iterator<std::string>(iss),
         std::istream_iterator<std::string>(),
         std::back_inserter(tokens));

    return tokens;

}

// Read the list of tokens in a txt line
std::vector< std::string > MeshIO::m_split_slash_txt( std::string line ) {

    std::stringstream ss(line);
    std::string tok;
    std::vector<std::string> tokens;
    char delimiter = '/';

    while ( std::getline( ss, tok, delimiter) )
    {
        tokens.push_back( tok );
    }

    return tokens;

}

// Read the list of tokens in a txt line
std::vector< std::string > MeshIO::m_split_txt_with( std::string line, i8 delimiter ) {

    std::stringstream ss(line);
    std::string tok;
    std::vector<std::string> tokens;

    while ( std::getline( ss, tok, delimiter) )
    {
        tokens.push_back( tok );
    }

    return tokens;

}

// Raw Format
// v1x v1y v1z v2x v2y v2z v3x v3y v3z

// Read data from raw data. Raw data contains only one mesh
MeshData MeshIO::m_read_raw_data()
{   

    // Open the mesh file
    std::ifstream input( m_filename.c_str(), std::ios::in );
    if( !input )
    {
        GGcerr << "Error to open the Mesh file'" << m_filename << "'!" << GGendl;
        exit_simulation();
    }

    // Compute number of triangles
    std::string line;
    ui32 N;
    while( std::getline( input, line ) ) ++N;

    // Returning to beginning of the file to read it again
    input.clear();
    input.seekg( 0, std::ios::beg );

    // Mem allocation
    MeshData mesh;
    HANDLE_ERROR( hipMallocManaged( &(mesh.v1), N * sizeof( f32xyz ) ) );
    HANDLE_ERROR( hipMallocManaged( &(mesh.v2), N * sizeof( f32xyz ) ) );
    HANDLE_ERROR( hipMallocManaged( &(mesh.v3), N * sizeof( f32xyz ) ) );

    HANDLE_ERROR( hipMallocManaged( &(mesh.mesh_index), sizeof( ui32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &(mesh.nb_triangles), sizeof( ui32 ) ) );

    HANDLE_ERROR( hipMallocManaged( &(mesh.aabb), sizeof( AabbData ) ) );

    mesh.mesh_names.push_back( "NoName" );

    // Store data from file
    size_t idx = 0;
    f32xyz v1, v2, v3;
    f32 xmin = FLT_MAX; f32 xmax = -FLT_MAX;
    f32 ymin = FLT_MAX; f32 ymax = -FLT_MAX;
    f32 zmin = FLT_MAX; f32 zmax = -FLT_MAX;
    while( std::getline( input, line ) )
    {
        std::istringstream iss( line );
        iss >> v1.x >> v1.y >> v1.z >> v2.x >> v2.y >> v2.z >> v3.x >> v3.y >> v3.z;

        // Get min and max from every dimension (bounding box)
        if ( v1.x > xmax ) xmax = v1.x;
        if ( v2.x > xmax ) xmax = v2.x;
        if ( v3.x > xmax ) xmax = v3.x;

        if ( v1.y > ymax ) ymax = v1.y;
        if ( v2.y > ymax ) ymax = v2.y;
        if ( v3.y > ymax ) ymax = v3.y;

        if ( v1.z > zmax ) zmax = v1.z;
        if ( v2.z > zmax ) zmax = v2.z;
        if ( v3.z > zmax ) zmax = v3.z;

        if ( v1.x < xmin ) xmin = v1.x;
        if ( v2.x < xmin ) xmin = v2.x;
        if ( v3.x < xmin ) xmin = v3.x;

        if ( v1.y < ymin ) ymin = v1.y;
        if ( v2.y < ymin ) ymin = v2.y;
        if ( v3.y < ymin ) ymin = v3.y;

        if ( v1.z < zmin ) zmin = v1.z;
        if ( v2.z < zmin ) zmin = v2.z;
        if ( v3.z < zmin ) zmin = v3.z;

        // Save data
        mesh.v1[ idx ] = v1;
        mesh.v2[ idx ] = v2;
        mesh.v3[ idx ] = v3;

        ++idx;
    }

    // Save AABB
    mesh.aabb[ 0 ].xmin = xmin;
    mesh.aabb[ 0 ].xmax = xmax;
    mesh.aabb[ 0 ].ymin = ymin;
    mesh.aabb[ 0 ].ymax = ymax;
    mesh.aabb[ 0 ].zmin = zmin;
    mesh.aabb[ 0 ].zmax = zmax;

    // Close the file
    input.close();

    return mesh;

}

//  obj format (Wavefront)
//  o SolidName
//  v x y z              // Geometric vertices
//  vt x y z             // Texture coordinates
//  vn x y z             // Vertex normals
//  s off                // Smooth shading
//  f i j k              // Face indices
//  f i/a j/b k/c        // Face and texture indices
//  f i/a/u j/b/v k/c/w  // Face, texture and normal indices

// Read data from MHD format
MeshData MeshIO::m_read_obj_data()
{   

    // Open the mesh file
    std::ifstream file( m_filename.c_str(), std::ios::in );
    if( !file )
    {
        GGcerr << "Error to open the Mesh file'" << m_filename << "'!" << GGendl;
        exit_simulation();
    }

    MeshData meshes;

    std::string line;
    std::vector< std::string > keys;
    std::vector< std::string > elts;

    // Obj data
    std::map< std::string, std::vector< f32xyz > >  vertices;
    std::map< std::string, std::vector< ui32xyz > >  faces;

    std::vector< f32xyz > buf_vertices;
    std::vector< ui32xyz > buf_faces;

    f32 x, y, z;
    ui32 i, j, k;
    std::string solid_name;

    // Empty the key for the beginning
    keys.clear();
    keys.push_back("");

    // Loop that read the complete file
    while ( file )
    {
        m_skip_comment( file );

        /// Search object //////////////////////////////////
        while ( keys[ 0 ] != "o" && file )
        {
            // Read a line
            std::getline( file, line );

            if ( file )
            {
                keys = m_split_txt( line );
            }
        }

        // Get the solid index ( xxx_Mesh )
        elts = m_split_txt_with( keys[ 1 ], '_' );
        solid_name = elts[ 0 ];
        meshes.mesh_names.push_back( solid_name );

        /// Then read all vertices ///////////////////////

        buf_vertices.clear();

        // Read next line
        std::getline( file, line );
        if ( file )
        {
            keys = m_split_txt( line );
        }

        //GGcout << "Find vertices: " << keys[ 0 ] << GGendl;

        // watch dog
        if ( keys[ 0 ] != "v" )
        {
            GGcout << "Mesh file (.obj): Vertices are not stored right after the solide name!" << GGendl;
            exit_simulation();
        }

        while ( keys[ 0 ] == "v" && file )
        {
            // Read coordinates
            std::stringstream( keys[ 1 ] ) >> x;
            std::stringstream( keys[ 2 ] ) >> y;
            std::stringstream( keys[ 3 ] ) >> z;

            // Store data
            buf_vertices.push_back( make_f32xyz( x, y, z ) );

            //GGcout << "Find vertices: " << x << " " << y << " " << z << GGendl;

            // Read new line
            std::getline( file, line );
            if ( file )
            {
                keys = m_split_txt( line );
            }
        }

        //GGcout << "Cur line: " << line << GGendl;

        /// Searching for faces /////////////////////////

        while ( keys[ 0 ] != "f" && file )
        {
            // Read a line
            std::getline( file, line );

            if ( file )
            {
                keys = m_split_txt( line );
            }
        }

        //GGcout << "Find face: " << line << GGendl;

        /// Read all faces //////////////////////////////

        buf_faces.clear();

        // Check if faces data describe triangles and not polygons ( f x y z w )
        if ( keys.size() > 4 )
        {
            GGcerr << "Mesh file data must contains triangle mesh and not polygon mesh!" << GGendl;
            exit_simulation();
        }

        // read all faces (start with the line already readed)
        while ( keys[ 0 ] == "f" && file )
        {
            // faces can be: a/b/c or a/b or a//c or a
            // we are only interested on the first index ( vertex )
            elts = m_split_slash_txt( keys[ 1 ] );
            std::stringstream( elts[ 0 ] ) >> i;
            elts = m_split_slash_txt( keys[ 2 ] );
            std::stringstream( elts[ 0 ] ) >> j;
            elts = m_split_slash_txt( keys[ 3 ] );
            std::stringstream( elts[ 0 ] ) >> k;

            // Store data
            buf_faces.push_back( make_ui32xyz( i, j, k ) );

            //GGcout << "Find face: " << i << " " << j << " " << k << GGendl;

            // Read new line
            std::getline( file, line );
            if ( file )
            {
                keys = m_split_txt( line );
            }

        }

        // Store the complete object
        vertices[ solid_name ] = buf_vertices;
        faces[ solid_name ] = buf_faces;

        GGcout << "Find solid name: " << solid_name << " with " << buf_vertices.size()
               << " vertices and " << buf_faces.size() << " faces" << GGendl;

    } // complete file


    // Convert data into mesh

    return meshes;

}


#endif












