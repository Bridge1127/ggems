#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef CONE_BEAM_SOURCE_CU
#define CONE_BEAM_SOURCE_CU

#include "cone_beam_source.cuh"

// External function
__host__ __device__ void cone_beam_source_primary_generator(ParticleStack particles, unsigned int id,
                                                            float px, float py, float pz,
                                                            float rphi, float rtheta, float rpsi,
                                                            float aperture, float energy,
                                                            unsigned char pname, unsigned int geom_id) {

    // Get direction
    float phi = JKISS32(particles, id);
    float theta = JKISS32(particles, id);
    float val_aper = 1.0f - cosf(aperture);
    phi  *= gpu_twopi;
    theta = acosf(1.0f - val_aper*theta);

    float dx = cosf(phi)*sinf(theta);
    float dy = sinf(phi)*sinf(theta);
    float dz = cosf(theta);

    // Apply rotation
    float3 d = f3_rotate(make_float3(dx, dy, dz), make_float3(rphi, rtheta, rpsi));

    // set photons
    particles.E[id] = energy;
    particles.dx[id] = d.x;
    particles.dy[id] = d.y;
    particles.dz[id] = d.z;
    particles.px[id] = px;
    particles.py[id] = py;
    particles.pz[id] = pz;
    particles.tof[id] = 0.0f;
    particles.endsimu[id] = PARTICLE_ALIVE;
    particles.level[id] = PRIMARY;
    particles.pname[id] = pname;
    particles.geometry_id[id] = geom_id;
}


ConeBeamSource::ConeBeamSource() {

    // Default parameters
    px = 0.0f; py = 0.0f; pz = 0.0f;
    phi = 0.0f; theta = 0.0f; psi = 0.0f;
    aperture = 8.0f*deg;
    energy = 60.0*keV;
    source_name = "Source01";
    seed = 10;
    geometry_id = 0;

}

// Setting function

void ConeBeamSource::set_position(float vpx, float vpy, float vpz) {
    px=vpx; py=vpy; pz=vpz;
}

void ConeBeamSource::set_rotation(float vphi, float vtheta, float vpsi) {
    phi=vphi; theta=vtheta; psi=vpsi;
}

void ConeBeamSource::set_aperture(float vaperture) {
    aperture=vaperture;
}

void ConeBeamSource::set_energy(float venergy) {
    energy=venergy;
}

void ConeBeamSource::set_seed(unsigned int vseed) {
    seed=vseed;
}

void ConeBeamSource::set_in_geometry(unsigned int vgeometry_id) {
    geometry_id=vgeometry_id;
}

void ConeBeamSource::set_source_name(std::string vsource_name) {
    source_name=vsource_name;
}







#endif
