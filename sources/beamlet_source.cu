#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file beamlet_source.cu
 * \brief Beamlet source
 * \author Julien Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date Thursday May 19, 2016
*/

#ifndef BEAMLET_SOURCE_CU
#define BEAMLET_SOURCE_CU

#include "beamlet_source.cuh"

///////// GPU code ////////////////////////////////////////////////////

// Internal function that create a new particle to the buffer at the slot id
__host__ __device__ void beamlet_source ( ParticlesData particles, f32xyz pos, f32xyz src, f32xyz size, f32matrix44 trans,
                                          f32 *spectrum_E, f32 *spectrum_CDF,
                                          ui32 nb_of_energy_bins, ui8 ptype, ui32 id )
{

    // 1. First choose a local position within the 2D beamlet (one of the corrdinate should be 0)
    f32xyz part_pos = { 0.0, 0.0, 0.0 };
    if ( size.x != 0.0 )
    {
        part_pos.x = size.x*prng_uniform( particles, id ) - 0.5f*size.x;
        part_pos.x = pos.x + part_pos.x;
    }
    if ( size.y != 0.0 )
    {
        part_pos.y = size.y*prng_uniform( particles, id ) - 0.5f*size.y;
        part_pos.y = pos.y + part_pos.y;
    }
    if ( size.z != 0.0 )
    {
        part_pos.z = size.z*prng_uniform( particles, id ) - 0.5f*size.z;
        part_pos.z = pos.z + part_pos.z;
    }

    // 2. Transform the beamlet and the source position from local to the global frame
    part_pos = fxyz_local_to_global_position( trans, part_pos );
    src = fxyz_local_to_global_position( trans, src );

    // 3. Get the direction
    f32xyz part_dir = fxyz_sub( part_pos, src );
    part_dir = fxyz_unit( part_dir );

    // 4. Get energy
    if( nb_of_energy_bins == 1 ) // mono energy
    {
        particles.E[ id ] = spectrum_E[ 0 ];
    }
    else // poly
    {
        f32 rndm = prng_uniform( particles, id );
        ui32 pos = binary_search_left( rndm, spectrum_CDF, nb_of_energy_bins );
        if ( pos == ( nb_of_energy_bins - 1 ) )
        {
            particles.E[ id ] = spectrum_E[ pos ];
        }
        else
        {
            particles.E[ id ] = linear_interpolation ( spectrum_CDF[ pos ],     spectrum_E[ pos ],
                                                       spectrum_CDF[ pos + 1 ], spectrum_E[ pos + 1 ], rndm );
        }

    }

    // 5. Then set the mandatory field to create a new particle
    particles.px[id] = part_pos.x;                        // Position in mm
    particles.py[id] = part_pos.y;                        //
    particles.pz[id] = part_pos.z;                        //

    particles.dx[id] = part_dir.x;                        // Direction (unit vector)
    particles.dy[id] = part_dir.y;                        //
    particles.dz[id] = part_dir.z;                        //

    particles.tof[id] = 0.0f;                             // Time of flight
    particles.endsimu[id] = PARTICLE_ALIVE;               // Status of the particle

    particles.level[id] = PRIMARY;                        // It is a primary particle
    particles.pname[id] = ptype;                          // a photon or an electron

    particles.geometry_id[id] = 0;                        // Some internal variables
    particles.next_discrete_process[id] = NO_PROCESS;     //
    particles.next_interaction_distance[id] = 0.0;        //
    particles.scatter_order[ id ] = 0;                    //


//    printf("src id %i p %f %f %f d %f %f %f E %f\n", id, part_pos.x, part_pos.y, part_pos.z,
//                                                         part_dir.x, part_dir.y, part_dir.z, particles.E[ id ]);

}

// Kernel to create new particles. This kernel will only call the host/device function
// beamlet source in order to get one new particle.
__global__ void kernel_beamlet_source ( ParticlesData particles, f32xyz pos, f32xyz src, f32xyz size, f32matrix44 trans,
                                        f32 *spectrum_E, f32 *spectrum_CDF,
                                        ui32 nb_of_energy_bins, ui8 particle_type )
{
    // Get thread id
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;    

    // Get a new particle
    beamlet_source( particles, pos, src, size, trans, spectrum_E, spectrum_CDF, nb_of_energy_bins,
                    particle_type, id );
}

//////// Class //////////////////////////////////////////////////////////

// Constructor
BeamletSource::BeamletSource() : GGEMSSource()
{
    // Set the name of the source
    set_name( "BeamletSource" );

    // Init vars
    m_pos = make_f32xyz( 0.0, 0.0, 0.0 );
    m_org = make_f32xyz( 0.0, 0.0, 0.0 );
    m_src = make_f32xyz( 0.0, 0.0, 0.0 );
    m_axis_trans = make_f32matrix33( 1, 0, 0,
                                     0, 1, 0,
                                     0, 0, 1 );
    m_angle = make_f32xyz( 0.0, 0.0, 0.0 );
    m_size = make_f32xyz( 0.0, 0.0, 0.0 );
    m_particle_type = PHOTON;
    m_spectrum_E = NULL;
    m_spectrum_CDF = NULL;
    m_nb_of_energy_bins = 0;
    m_energy = 0;
    m_spectrum_filename = "";
}

// Destructor
BeamletSource::~BeamletSource() {}

//========== Private ===============================================

void BeamletSource::m_load_spectrum()
{
    // Open the histogram file
    std::ifstream input( m_spectrum_filename.c_str(), std::ios::in );
    if( !input )
    {
        GGcerr << "Error to open the file'" << m_spectrum_filename << "'!" << GGendl;
        exit_simulation();
    }

    // Compute number of energy bins
    std::string line;
    while( std::getline( input, line ) ) ++m_nb_of_energy_bins;

    // Returning to beginning of the file to read it again
    input.clear();
    input.seekg( 0, std::ios::beg );

    // Allocating buffers to store data
    HANDLE_ERROR( hipMallocManaged( &m_spectrum_E, m_nb_of_energy_bins * sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &m_spectrum_CDF, m_nb_of_energy_bins * sizeof( f32 ) ) );

    // Store data from file
    size_t idx = 0;
    f64 sum = 0.0;
    while( std::getline( input, line ) )
    {
        std::istringstream iss( line );
        iss >> m_spectrum_E[ idx ] >> m_spectrum_CDF[ idx ];
        sum += m_spectrum_CDF[ idx ];
        ++idx;
    }

    // Compute CDF and normalized in same time by security
    m_spectrum_CDF[ 0 ] /= sum;
    for( ui32 i = 1; i < m_nb_of_energy_bins; ++i )
    {
        m_spectrum_CDF[ i ] = m_spectrum_CDF[ i ] / sum
                              + m_spectrum_CDF[ i - 1 ];
    }

    // Watch dog
    m_spectrum_CDF[ m_nb_of_energy_bins - 1 ] = 1.0;

    // Close the file
    input.close();
}

//========== Setting ===============================================

// Setting position of the beamlet
void BeamletSource::set_local_beamlet_position( f32 posx, f32 posy, f32 posz )
{
    m_pos = make_f32xyz( posx, posy, posz );
}

// Setting the distance between the beamlet plane and the isocenter
void BeamletSource::set_frame_position( f32 posx, f32 posy, f32 posz )
{
    m_org = make_f32xyz( posx, posy, posz );
}

// Setting the axis transformation matrix
void BeamletSource::set_frame_axis( f32 m00, f32 m01, f32 m02,
                                    f32 m10, f32 m11, f32 m12,
                                    f32 m20, f32 m21, f32 m22 )
{
    m_axis_trans.m00 = m00;
    m_axis_trans.m01 = m01;
    m_axis_trans.m02 = m02;
    m_axis_trans.m10 = m10;
    m_axis_trans.m11 = m11;
    m_axis_trans.m12 = m12;
    m_axis_trans.m20 = m20;
    m_axis_trans.m21 = m21;
    m_axis_trans.m22 = m22;
}

// Setting position of the focal beamlet
void BeamletSource::set_local_source_position( f32 posx, f32 posy, f32 posz )
{
    m_src = make_f32xyz( posx, posy, posz );
}

// Setting beamlet size
void BeamletSource::set_local_size( f32 sizex, f32 sizey, f32 sizez )
{
    m_size = make_f32xyz( sizex, sizey, sizez );
}

// Setting orientation of the beamlet
void BeamletSource::set_rotation( f32 agx, f32 agy, f32 agz )
{
    m_angle = make_f32xyz( agx, agy, agz );
}

// Setting energy
void BeamletSource::set_mono_energy( f32 energy )
{
    m_energy = energy;
}

// Setting spectrum
void BeamletSource::set_energy_spectrum( std::string filename )
{
    m_spectrum_filename = filename;
    // Watchdog (avoid to set the two option mono energy and spectrum)
    m_energy = 0;
}

// Setting particle type (photon or electron)
void BeamletSource::set_particle_type( std::string pname )
{
    // Transform the name of the particle in small letter
    std::transform( pname.begin(), pname.end(), pname.begin(), ::tolower );

    if( pname == "photon" )
    {
        m_particle_type = PHOTON;
    }
    else if( pname == "electron" )
    {
        m_particle_type = ELECTRON;
    }
    else
    {
        GGcerr << "Particle '" << pname << "' not recognized!!!" << GGendl;
        exit_simulation();
    }
}

//========== Getting ===============================================

f32xyz BeamletSource::get_local_source_position()
{
    return m_src;
}

f32xyz BeamletSource::get_local_beamlet_position()
{
    return m_pos;
}

f32xyz BeamletSource::get_local_size()
{
    return m_size;
}

f32matrix44 BeamletSource::get_transformation_matrix()
{
    return m_transform;
}

//========= Main function ============================================

// Mandatory function, abstract from GGEMSSource. This function is called
// by GGEMS to initialize and load all necessary data into the graphic card
void BeamletSource::initialize ( GlobalSimulationParameters params )
{
    // Check if everything was set properly
    if ( m_energy == 0 && m_spectrum_filename == "" )
    {
        GGcerr << "No energy or spectrum file specified!" << GGendl;
        exit_simulation();
    }
    if ( m_size.x == 0 && m_size.y == 0 && m_size.z == 0 )
    {
        GGcerr << "Size of the beamlet was not defined!" << GGendl;
        exit_simulation();
    }

    // If mono energy
    if ( m_energy != 0 )
    {
        HANDLE_ERROR( hipMallocManaged( &m_spectrum_E, sizeof( f32 ) ) );
        HANDLE_ERROR( hipMallocManaged( &m_spectrum_CDF, sizeof( f32 ) ) );
        m_spectrum_E[ 0 ] = m_energy;
        m_spectrum_CDF[ 0 ] = 1.0;
        m_nb_of_energy_bins = 1;
    }
    else // else load a spectrum
    {
        m_load_spectrum();
    }

    // Store global parameters: params are provided by GGEMS and are used to
    // know different information about the simulation. For example if the targeted
    // device is a CPU or a GPU.
    m_params = params;

    // Compute the transformation matrix (Beamlet plane is set along the x-axis (angle 0))
    TransformCalculator *trans = new TransformCalculator;
    trans->set_translation( m_org );
    trans->set_rotation( m_angle );
    trans->set_axis_transformation( m_axis_trans );
    m_transform = trans->get_transformation_matrix();
    delete trans;

    // Get distance between the isocenter and the source
    f32 dist_src = fxyz_mag( fxyz_local_to_global_position( m_transform, m_src ) );

    // Convert the global source position to a relative position
    //m_src = fxyz_global_to_local_position( m_transform, m_src );

    // Get the distance between the beamlet origin and the source
    f32 dist_src_beamlet = fxyz_mag( m_src );

    // Get the ratio and compute the size of beamlet at the beamlet position (and not at the isocenter position)
    f32 ratio = dist_src_beamlet / dist_src;
    m_size.x *= ratio;
    m_size.y *= ratio;
    m_size.z *= ratio;

    // Some verbose if required
    if ( params.data_h.display_memory_usage )
    {
        ui32 mem = 8 * m_nb_of_energy_bins;
        GGcout_mem("Beamlet source", mem);
    }

}

// Mandatory function, abstract from GGEMSSource. This function is called
// by GGEMS to fill particle buffer of new fresh particles, which is the role
// of any source.
void BeamletSource::get_primaries_generator ( Particles particles )
{

    // If CPU running, do it on CPU
    if ( m_params.data_h.device_target == CPU_DEVICE )        
    {

        // Loop over the particle buffer
        ui32 id=0;
        while( id < particles.size )
        {
            // Call a point source that get a new particle at a time. In this case data from host (CPU)
            // is passed to the function (particles.data_h).
            beamlet_source( particles.data_h, m_pos, m_src, m_size, m_transform,
                            m_spectrum_E, m_spectrum_CDF, m_nb_of_energy_bins,
                            m_particle_type, id );
            ++id;
        }

    }
    // If GPU running, do it on GPU
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {        

        // Defined threads and grid
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

        // Call GPU kernel of a point source that get fill the complete particle buffer. In this case data
        // from device (GPU) is passed to the kernel (particles.data_d).
        kernel_beamlet_source<<<grid, threads>>>( particles.data_d, m_pos, m_src, m_size, m_transform,
                                                  m_spectrum_E, m_spectrum_CDF, m_nb_of_energy_bins,
                                                  m_particle_type );
        cuda_error_check( "Error ", " Kernel_beamlet_source" );
        hipDeviceSynchronize();
    }

}

#endif

















