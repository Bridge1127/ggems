#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file beamlet_source.cu
 * \brief Beamlet source
 * \author Julien Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date Thursday November 19, 2015
*/

#ifndef BEAMLET_SOURCE_CU
#define BEAMLET_SOURCE_CU

#include "beamlet_source.cuh"

///////// GPU code ////////////////////////////////////////////////////

// Internal function that create a new particle to the buffer at the slot id
__host__ __device__ void beamlet_source ( ParticlesData particles, f32xyz pos, f32xyz foc_pos, f32xy size, f32xyz angle,
                                          f32 *spectrum_E, f32 *spectrum_CDF,
                                          ui32 nb_of_energy_bins, ui8 ptype, ui32 id)
{
    // 1. First chose a position within the beamlet
    f32xyz part_pos;
    part_pos.x = size.x*prng_uniform( particles, id ) - 0.5*size.x;
    part_pos.y = size.y*prng_uniform( particles, id ) - 0.5*size.y;
    part_pos.x = pos.x + part_pos.x;
    part_pos.y = pos.y + part_pos.y;

    // 3. Then compute direction
    f32xyz part_dir = fxyz_sub( foc_pos, part_pos );
    part_dir = fxyz_unit( part_dir );

    // 4. First rotation (Carousel)
    part_pos = fxyz_rotate_y_axis(part_pos, angle.y);  // angle
    part_dir = fxyz_rotate_y_axis(part_dir, angle.y);  // angle

    // 5. Then, beam rotation (z-axis = granty angle, x-axis = non-coplanar beam)
    part_pos = fxyz_rotate_z_axis(part_pos, angle.z);  // angle
    part_dir = fxyz_rotate_z_axis(part_dir, angle.z);  // angle

    part_pos = fxyz_rotate_x_axis(part_pos, angle.x);  // angle
    part_dir = fxyz_rotate_x_axis(part_dir, angle.x);  // angle

    part_dir = fxyz_unit( part_dir ); // to be sure

    // 6. Get energy
    if( nb_of_energy_bins == 1 ) // mono energy
    {
        particles.E[ id ] = spectrum_E[ 0 ];
    }
    else // poly
    {
        f32 rndm = prng_uniform( particles, id );
        ui32 pos = binary_search( rndm, spectrum_CDF, nb_of_energy_bins );
        if ( pos == ( nb_of_energy_bins - 1 ) )
        {
            particles.E[ id ] = spectrum_E[ pos ];
        }
        else
        {
            particles.E[ id ] = linear_interpolation ( spectrum_CDF[ pos ],     spectrum_E[ pos ],
                                                       spectrum_CDF[ pos + 1 ], spectrum_E[ pos + 1 ], rndm );
        }

    }

    // 7. Then set the mandatory field to create a new particle
    particles.px[id] = part_pos.x;                        // Position in mm
    particles.py[id] = part_pos.y;                        //
    particles.pz[id] = part_pos.z;                        //

    particles.dx[id] = part_dir.x;                        // Direction (unit vector)
    particles.dy[id] = part_dir.y;                        //
    particles.dz[id] = part_dir.z;                        //

    particles.tof[id] = 0.0f;                             // Time of flight
    particles.endsimu[id] = PARTICLE_ALIVE;               // Status of the particle

    particles.level[id] = PRIMARY;                        // It is a primary particle
    particles.pname[id] = ptype;                          // a photon or an electron

    particles.geometry_id[id] = 0;                        // Some internal variables
    particles.next_discrete_process[id] = NO_PROCESS;     //
    particles.next_interaction_distance[id] = 0.0;        //

}

// Kernel to create new particles. This kernel will only call the host/device function
// beamlet source in order to get one new particle.
__global__ void kernel_beamlet_source ( ParticlesData particles, f32xyz pos, f32xyz foc_pos, f32xy size, f32xyz angle,
                                        f32 *spectrum_E, f32 *spectrum_CDF,
                                        ui32 nb_of_energy_bins, ui8 particle_type )
{
    // Get thread id
    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= particles.size ) return;

    // Get a new particle
    beamlet_source( particles, pos, foc_pos, size, angle, spectrum_E, spectrum_CDF, nb_of_energy_bins,
                    particle_type, id );
}

//////// Class //////////////////////////////////////////////////////////

// Constructor
BeamletSource::BeamletSource() : GGEMSSource()
{
    // Set the name of the source
    set_name( "point_source" );

    // Init vars
    m_pos = make_f32xyz( 0.0, 0.0, 0.0 );
    m_foc_pos = make_f32xyz( 0.0, 0.0, 0.0 );
    m_angle = make_f32xyz( 0.0, 0.0, 0.0 );
    m_beamlet_size = make_f32xy( 0.0, 0.0 );
    m_particle_type = PHOTON;
    m_spectrum_E = NULL;
    m_spectrum_CDF = NULL;
    m_nb_of_energy_bins = 0;
    m_energy = 0;
    m_spectrum_filename = "";
}

// Destructor
BeamletSource::~BeamletSource() {}

//========== Private ===============================================

void BeamletSource::m_load_spectrum()
{
    // Open the histogram file
    std::ifstream input( m_spectrum_filename.c_str(), std::ios::in );
    if( !input )
    {
        GGcerr << "Error to open the file'" << m_spectrum_filename << "'!" << GGendl;
        exit_simulation();
    }

    // Compute number of energy bins
    std::string line;
    while( std::getline( input, line ) ) ++m_nb_of_energy_bins;

    // Returning to beginning of the file to read it again
    input.clear();
    input.seekg( 0, std::ios::beg );

    // Allocating buffers to store data
    HANDLE_ERROR( hipMallocManaged( &m_spectrum_E, m_nb_of_energy_bins * sizeof( f32 ) ) );
    HANDLE_ERROR( hipMallocManaged( &m_spectrum_CDF, m_nb_of_energy_bins * sizeof( f32 ) ) );

    // Store data from file
    size_t idx = 0;
    f64 sum = 0.0;
    while( std::getline( input, line ) )
    {
        std::istringstream iss( line );
        iss >> m_spectrum_E[ idx ] >> m_spectrum_CDF[ idx ];
        sum += m_spectrum_CDF[ idx ];
        ++idx;
    }

    // Compute CDF and normalized in same time by security
    m_spectrum_CDF[ 0 ] /= sum;
    for( ui32 i = 1; i < m_nb_of_energy_bins; ++i )
    {
        m_spectrum_CDF[ i ] = m_spectrum_CDF[ i ] / sum
                              + m_spectrum_CDF[ i - 1 ];
    }

    // Watch dog
    m_spectrum_CDF[ m_nb_of_energy_bins - 1 ] = 1.0;

    // Close the file
    input.close();
}

//========== Setting ===============================================

// Setting position of the beamlet
void BeamletSource::set_position( f32 posx, f32 posy, f32 posz )
{
    m_pos = make_f32xyz( posx, posy, posz );
}

// Setting position of the focal beamlet
void BeamletSource::set_beamlet_focal_point( f32 posx, f32 posy, f32 posz )
{
    m_foc_pos = make_f32xyz( posx, posy, posz );
}

// Setting orientation of the beamlet
void BeamletSource::set_rotation( f32 agx, f32 agy, f32 agz )
{
    m_angle = make_f32xyz( agx, agy, agz );
}

// Setting energy
void BeamletSource::set_mono_energy( f32 energy )
{
    m_energy = energy;
}

// Setting spectrum
void BeamletSource::set_energy_spectrum( std::string filename )
{
    m_spectrum_filename = filename;
    // Watchdog (avoid to set the two option mono energy and spectrum)
    m_energy = 0;
}

// Setting particle type (photon or electron)
void BeamletSource::set_particle_type( std::string pname )
{
    // Transform the name of the particle in small letter
    std::transform( pname.begin(), pname.end(), pname.begin(), ::tolower );

    if( pname == "photon" )
    {
        m_particle_type = PHOTON;
    }
    else if( pname == "electron" )
    {
        m_particle_type = ELECTRON;
    }
    else
    {
        GGcerr << "Particle '" << pname << "' not recognized!!!" << GGendl;
        exit_simulation();
    }
}

//========= Main function ============================================

// Mandatory function, abstract from GGEMSSource. This function is called
// by GGEMS to initialize and load all necessary data into the graphic card
void BeamletSource::initialize ( GlobalSimulationParameters params )
{
    // Check if everything was set properly
    if ( m_energy == 0 && m_spectrum_filename == "" )
    {
        GGcerr << "No energy or spectrum file specified!" << GGendl;
        exit_simulation();
    }
    if ( m_beamlet_size.x == 0 || m_beamlet_size.y == 0 )
    {
        GGcerr << "Size of the beamlet was not defined!" << GGendl;
        exit_simulation();
    }

    // If mono energy
    if ( m_energy != 0 )
    {
        HANDLE_ERROR( hipMallocManaged( &m_spectrum_E, sizeof( f32 ) ) );
        HANDLE_ERROR( hipMallocManaged( &m_spectrum_CDF, sizeof( f32 ) ) );
        m_spectrum_E[ 0 ] = m_energy;
        m_spectrum_CDF[ 0 ] = 1.0;
        m_nb_of_energy_bins = 1;
    }
    else // else load a spectrum
    {
        m_load_spectrum();
    }

    // Store global parameters: params are provided by GGEMS and are used to
    // know different information about the simulation. For example if the targeted
    // device is a CPU or a GPU.
    m_params = params;   

}

// Mandatory function, abstract from GGEMSSource. This function is called
// by GGEMS to fill particle buffer of new fresh particles, which is the role
// of any source.
void BeamletSource::get_primaries_generator ( Particles particles )
{

    // If CPU running, do it on CPU
    if ( m_params.data_h.device_target == CPU_DEVICE )
    {
        // Loop over the particle buffer
        ui32 id=0;
        while( id < particles.size )
        {
            // Call a point source that get a new particle at a time. In this case data from host (CPU)
            // is passed to the function (particles.data_h).
            beamlet_source( particles.data_h, m_pos, m_foc_pos, m_beamlet_size, m_angle,
                            m_spectrum_E, m_spectrum_CDF, m_nb_of_energy_bins,
                            m_particle_type, id );
            ++id;
        }

    }
    // If GPU running, do it on GPU
    else if ( m_params.data_h.device_target == GPU_DEVICE )
    {
        // Defined threads and grid
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 ) / m_params.data_h.gpu_block_size;

        // Call GPU kernel of a point source that get fill the complete particle buffer. In this case data
        // from device (GPU) is passed to the kernel (particles.data_d).
        kernel_beamlet_source<<<grid, threads>>>( particles.data_h, m_pos, m_foc_pos, m_beamlet_size, m_angle,
                                                  m_spectrum_E, m_spectrum_CDF, m_nb_of_energy_bins,
                                                  m_particle_type );
        cuda_error_check( "Error ", " Kernel_beamlet_source" );
    }

}

#endif

















