#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file point_source.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 13 novembre 2015
 *
 *
 *
 */

#ifndef POINT_SOURCE_CU
#define POINT_SOURCE_CU

#include "point_source.cuh"

///////// GPU code ////////////////////////////////////////////////////

// Internal function
__host__ __device__ void point_source(ParticlesData particles_data, ui32 id,
                                      f32 px, f32 py, f32 pz, ui8 type,
                                      f64 *spectrumE, f64 *spectrumCDF, ui32 nbins) {

    f32 phi = JKISS32(particles_data, id);
    f32 theta = JKISS32(particles_data, id);

    phi  *= gpu_twopi;
    theta = acosf(1.0f - 2.0f*theta);

    ui32 pos = binary_search(JKISS32(particles_data, id), spectrumCDF, nbins);

    // set photons
    particles_data.E[id] = spectrumE[pos];
    particles_data.dx[id] = cosf(phi)*sinf(theta);
    particles_data.dy[id] = sinf(phi)*sinf(theta);
    particles_data.dz[id] = cosf(theta);
    particles_data.px[id] = px;
    particles_data.py[id] = py;
    particles_data.pz[id] = pz;
    particles_data.tof[id] = 0.0f;
    particles_data.endsimu[id] = PARTICLE_ALIVE;
    particles_data.next_discrete_process[id] = NO_PROCESS;
    particles_data.next_interaction_distance[id] = 0.0;
    particles_data.level[id] = PRIMARY;
    particles_data.pname[id] = type;
    particles_data.geometry_id[id] = 0;
}

// Kernel to create new particles (sources manager)
__global__ void kernel_point_source(ParticlesData particles_data,
                                    f32 px, f32 py, f32 pz, ui8 type,
                                    f64 *spectrumE, f64 *spectrumCDF, ui32 nbins) {

    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particles_data.size) return;

    point_source(particles_data, id, px, py, pz, type,
                 spectrumE, spectrumCDF, nbins);

}

///////////////////////////////////////////////////////////////////////

// Constructor
PointSource::PointSource() {
    // Default parameters
    m_px = 0.0f; m_py = 0.0f; m_pz = 0.0f;
    m_nb_of_energy_bins = 0;
    m_spectrumE_h = NULL;
    m_spectrumE_d = NULL;
    m_spectrumCDF_h = NULL;
    m_spectrumCDF_d = NULL;
    m_particle_type = PHOTON;
}

// Destructor
PointSource::~PointSource() {
    free(m_spectrumE_h);
    free(m_spectrumCDF_h);
    hipFree(m_spectrumE_d);
    hipFree(m_spectrumCDF_d);
}

// Setting function
void PointSource::set_position(f32 vpx, f32 vpy, f32 vpz) {
    m_px=vpx; m_py=vpy; m_pz=vpz;
}

void PointSource::set_particle_type(std::string pname) {
    if (pname == "photon") {
        m_particle_type = PHOTON;
    } else if (pname == "electron") {
        m_particle_type = ELECTRON;
    } else if (pname == "positron") {
        m_particle_type = POSITRON;
    }
}

void PointSource::set_mono_energy(f32 valE) {
    m_spectrumE_h = (f64*)malloc(sizeof(f64));
    m_spectrumE_h[0] = valE;
    m_spectrumCDF_h = (f64*)malloc(sizeof(f64));
    m_spectrumCDF_h[0] = 1.0;
    m_nb_of_energy_bins = 1;
}

void PointSource::set_energy_spectrum(f64 *valE, f64 *hist, ui32 nb) {

    // Allocation
    m_spectrumE_h = (f64*)malloc(nb*sizeof(f64));
    m_spectrumCDF_h = (f64*)malloc(nb*sizeof(f64));
    m_nb_of_energy_bins = nb;

    // Get the sum
    f64 sum = 0;
    ui32 i = 0;
    while (i<nb) {
        sum += hist[i];
        ++i;
    }
    // Normalize
    i=0; while (i<nb) {
        m_spectrumCDF_h[i] = hist[i] / sum;
        // In the mean time copy energy value
        m_spectrumE_h[i] = valE[i];
        ++i;
    }
    // Get the final CDF
    i=1; while (i<nb) {
        m_spectrumCDF_h[i] += m_spectrumCDF_h[i-1];
        ++i;
    }
    // Watchdog
    m_spectrumCDF_h[nb-1] = 1.0f;
}

// Main function
bool PointSource::m_check_mandatory() {
    if (m_nb_of_energy_bins == 0) return false;
    else return true;
}

void PointSource::initialize(GlobalSimulationParameters params) {

    // Check if everything was set properly
    if ( !m_check_mandatory() ) {
        print_error("Missing parameters for the point source!");
        exit_simulation();
    }

    // Store global parameters
    m_params = params;

    // Handle GPU device
    if (m_params.data_h.device_target == GPU_DEVICE && m_nb_of_energy_bins > 1) {
        // GPU mem allocation
        HANDLE_ERROR( hipMalloc((void**) &m_spectrumE_d, m_nb_of_energy_bins*sizeof(f64)) );
        HANDLE_ERROR( hipMalloc((void**) &m_spectrumCDF_d, m_nb_of_energy_bins*sizeof(f64)) );
        // GPU mem copy
        HANDLE_ERROR( hipMemcpy(m_spectrumE_d, m_spectrumE_h,
                                 sizeof(f64)*m_nb_of_energy_bins, hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(m_spectrumCDF_d, m_spectrumCDF_h,
                                 sizeof(f64)*m_nb_of_energy_bins, hipMemcpyHostToDevice) );
    }

}

void PointSource::get_primaries_generator(Particles particles) {

    if (m_params.data_h.device_target == CPU_DEVICE) {

        ui32 id=0; while (id<particles.size) {
            point_source(particles.data_h, id, m_px, m_py, m_pz, m_particle_type,
                         m_spectrumE_d, m_spectrumCDF_d, m_nb_of_energy_bins);
            ++id;
        }

    } else if (m_params.data_h.device_target == GPU_DEVICE) {

        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = (particles.size + m_params.data_h.gpu_block_size - 1) / m_params.data_h.gpu_block_size;

        kernel_point_source<<<grid, threads>>>(particles.data_d, m_px, m_py, m_pz, m_particle_type,
                                               m_spectrumE_d, m_spectrumCDF_d, m_nb_of_energy_bins);
        cuda_error_check("Error ", " Kernel_point_source");

    }

}

#endif

















