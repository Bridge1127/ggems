#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef POINT_SOURCE_CU
#define POINT_SOURCE_CU

#include "point_source.cuh"

// External function
__host__ __device__ void point_source_primary_generator(ParticleStack particles, unsigned int id,
                                                        f32 px, f32 py, f32 pz, f32 energy,
                                                        unsigned char type, unsigned int geom_id) {

    f32 phi = JKISS32(particles, id);
    f32 theta = JKISS32(particles, id);

    phi  *= gpu_twopi;
    theta = acosf(1.0f - 2.0f*theta);

    // set photons
    particles.E[id] = energy;
    particles.dx[id] = cosf(phi)*sinf(theta);
    particles.dy[id] = sinf(phi)*sinf(theta);
    particles.dz[id] = cosf(theta);
    particles.px[id] = px;
    particles.py[id] = py;
    particles.pz[id] = pz;
    particles.tof[id] = 0.0f;
    particles.endsimu[id] = PARTICLE_ALIVE;
    particles.level[id] = PRIMARY;
    particles.pname[id] = type;
    particles.geometry_id[id] = geom_id;
}


PointSource::PointSource() {

    // Default parameters
    px = 0.0f; py = 0.0f; pz = 0.0f;
    energy = 60.0*keV;
    source_name = "Source01";
    seed = 10;
    geometry_id = 0;
}

// Setting function

void PointSource::set_position(f32 vpx, f32 vpy, f32 vpz) {
    px=vpx; py=vpy; pz=vpz;
}

void PointSource::set_energy(f32 venergy) {
    energy=venergy;
}

void PointSource::set_seed(unsigned int vseed) {
    seed=vseed;
}

void PointSource::set_in_geometry(unsigned int vgeometry_id) {
    geometry_id=vgeometry_id;
}

void PointSource::set_source_name(std::string vsource_name) {
    source_name=vsource_name;
}

#endif
