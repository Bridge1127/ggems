#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef GEOMETRY_BUILDER_CU
#define GEOMETRY_BUILDER_CU

#include "geometry_builder.cuh"

/////////////////////////////////////////////////////////////////////////////////////
///////// Host/Device functions /////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

// Function that return the material of a volume
unsigned int __host__ __device__ get_geometry_material(Scene geometry, unsigned int id_geom, float3 pos) {
    unsigned int adr_geom = geometry.ptr_objects[id_geom];
    unsigned int obj_type = (unsigned int)geometry.data_objects[adr_geom+ADR_OBJ_TYPE];

    if (obj_type != VOXELIZED) {
        return (unsigned int)geometry.data_objects[adr_geom+ADR_OBJ_MAT_ID];
    } else if (obj_type == VOXELIZED) {
        // Change particle frame (into voxelized volume)
        pos.x -= geometry.data_objects[adr_geom+ADR_AABB_XMIN]; // -= xmin
        pos.y -= geometry.data_objects[adr_geom+ADR_AABB_YMIN]; // -= ymin
        pos.z -= geometry.data_objects[adr_geom+ADR_AABB_ZMIN]; // -= zmin
        // Get the voxel index
        int3 ind;
        ind.x = (unsigned int)(pos.x / geometry.data_objects[adr_geom+ADR_VOXELIZED_SX]); // / sx
        ind.y = (unsigned int)(pos.y / geometry.data_objects[adr_geom+ADR_VOXELIZED_SY]); // / sy
        ind.z = (unsigned int)(pos.z / geometry.data_objects[adr_geom+ADR_VOXELIZED_SZ]); // / sz
//        printf("Vos ind %i %i %i aabb %f %f, %f %f, %f %f\n", ind.x, ind.y, ind.z,
//               geometry.data_objects[adr_geom+ADR_AABB_XMIN],
//               geometry.data_objects[adr_geom+ADR_AABB_XMAX],
//               geometry.data_objects[adr_geom+ADR_AABB_YMIN],
//               geometry.data_objects[adr_geom+ADR_AABB_YMAX],
//               geometry.data_objects[adr_geom+ADR_AABB_ZMIN],
//               geometry.data_objects[adr_geom+ADR_AABB_ZMAX]);
        // Return material
        unsigned int abs_ind = ind.z * (geometry.data_objects[adr_geom+ADR_VOXELIZED_NY]*geometry.data_objects[adr_geom+ADR_VOXELIZED_NX])
                                        + ind.y*geometry.data_objects[adr_geom+ADR_VOXELIZED_NX] + ind.x;
        //printf("Mat: %i\n", (unsigned int)geometry.data_objects[adr_geom+ADR_VOXELIZED_DATA+abs_ind]);
        return (unsigned int)geometry.data_objects[adr_geom+ADR_VOXELIZED_DATA+abs_ind];
    } else {
        return 0;
    }
}

// Get distance from an object
f32 __host__ __device__ get_distance_to_object(Scene geometry, unsigned int adr_geom,
                                                 unsigned int obj_type, float3 pos, float3 dir) {

    f32 distance = FLT_MAX;

    // AABB volume
    if (obj_type == AABB) {

        // Read first the bounding box
        f32 xmin = geometry.data_objects[adr_geom+ADR_AABB_XMIN];
        f32 xmax = geometry.data_objects[adr_geom+ADR_AABB_XMAX];
        f32 ymin = geometry.data_objects[adr_geom+ADR_AABB_YMIN];
        f32 ymax = geometry.data_objects[adr_geom+ADR_AABB_YMAX];
        f32 zmin = geometry.data_objects[adr_geom+ADR_AABB_ZMIN];
        f32 zmax = geometry.data_objects[adr_geom+ADR_AABB_ZMAX];

        distance = hit_ray_AABB(pos, dir, xmin, xmax, ymin, ymax, zmin, zmax);

    // Sphere volume
    } else if (obj_type == SPHERE) {

        // Read first sphere parameters
        float3 c = make_float3(geometry.data_objects[adr_geom+ADR_SPHERE_CX],
                               geometry.data_objects[adr_geom+ADR_SPHERE_CY],
                               geometry.data_objects[adr_geom+ADR_SPHERE_CZ]);
        f32 r = geometry.data_objects[adr_geom+ADR_SPHERE_RADIUS];

        distance = hit_ray_sphere(pos, dir, c, r);

    } else if (obj_type == VOXELIZED) {

        // Change particle frame (into voxelized volume)
        float3 posinvox;
        posinvox.x = pos.x - geometry.data_objects[adr_geom+ADR_AABB_XMIN]; // -= xmin
        posinvox.y = pos.y - geometry.data_objects[adr_geom+ADR_AABB_YMIN]; // -= ymin
        posinvox.z = pos.z - geometry.data_objects[adr_geom+ADR_AABB_ZMIN]; // -= zmin
        // Get spacing
        float3 s;
        s.x = geometry.data_objects[adr_geom+ADR_VOXELIZED_SX];
        s.y = geometry.data_objects[adr_geom+ADR_VOXELIZED_SY];
        s.z = geometry.data_objects[adr_geom+ADR_VOXELIZED_SZ];
        // Get the voxel index
        int3 ind;
        ind.x = (unsigned int)(posinvox.x / s.x);
        ind.y = (unsigned int)(posinvox.y / s.y);
        ind.z = (unsigned int)(posinvox.z / s.z);

        //printf("Ind %i %i %i\n", ind.x, ind.y, ind.z);

        // Then get the voxel bounding box
        f32 volxmin = geometry.data_objects[adr_geom+ADR_AABB_XMIN];
        f32 volymin = geometry.data_objects[adr_geom+ADR_AABB_YMIN];
        f32 volzmin = geometry.data_objects[adr_geom+ADR_AABB_ZMIN];

        f32 xmin, ymin, xmax, ymax, zmin, zmax;
        xmin = ind.x*s.x + volxmin; xmax = xmin+s.x;
        ymin = ind.y*s.y + volymin; ymax = ymin+s.y;
        zmin = ind.z*s.z + volzmin; zmax = zmin+s.z;

//        xmin = (dir.x > 0 && posinvox.x > (ind.x+1)*s.x-EPSILON3) ? (ind.x+1)*s.x+volxmin : ind.x*s.x+volxmin;
//        ymin = (dir.y > 0 && posinvox.y > (ind.y+1)*s.y-EPSILON3) ? (ind.y+1)*s.y+volymin : ind.y*s.y+volymin;
//        zmin = (dir.z > 0 && posinvox.z > (ind.z+1)*s.z-EPSILON3) ? (ind.z+1)*s.z+volzmin : ind.z*s.z+volzmin;
//        xmax = (dir.x < 0 && posinvox.x < xmin + EPSILON3) ? xmin-s.x : xmin+s.x;
//        ymax = (dir.y < 0 && posinvox.y < ymin + EPSILON3) ? ymin-s.y : ymin+s.y;
//        zmax = (dir.z < 0 && posinvox.z < zmin + EPSILON3) ? zmin-s.z : zmin+s.z;

        // Get the distance
        distance = hit_ray_AABB(pos, dir, xmin, xmax, ymin, ymax, zmin, zmax);

        if ((distance > -EPSILON6 && distance < EPSILON6) || distance > 100000) {

            printf("::::: Pos %f %f %f\n", pos.x, pos.y, pos.z);
            printf("::::: Org %f %f %f\n", geometry.data_objects[adr_geom+ADR_AABB_XMIN],
                   geometry.data_objects[adr_geom+ADR_AABB_YMIN],
                   geometry.data_objects[adr_geom+ADR_AABB_ZMIN]);
            printf("::::: RefPos %f %f %f\n", posinvox.x, posinvox.y, posinvox.z);
            printf("::::: Scl %f %f %f\n", s.x, s.y, s.z);
            printf("::::: Ind %i %i %i\n", ind.x, ind.y, ind.z);
            printf("::::: Vox %f %f, %f %f, %f %f\n", xmin, xmax, ymin, ymax, zmin, zmax);
            printf("::::: Dist %f\n", distance);
            f32 a = -8.000009;
            f32 b = 296.0;
            f32 c = a+b;
            printf("----- test %2.20f\n", c);
        }

    } else if (obj_type == MESHED) {

        unsigned int octree_type = geometry.data_objects[adr_geom+ADR_MESHED_OCTREE_TYPE];

        // Read first the bounding box
        f32 xmin = geometry.data_objects[adr_geom+ADR_AABB_XMIN];
        f32 xmax = geometry.data_objects[adr_geom+ADR_AABB_XMAX];
        f32 ymin = geometry.data_objects[adr_geom+ADR_AABB_YMIN];
        f32 ymax = geometry.data_objects[adr_geom+ADR_AABB_YMAX];
        f32 zmin = geometry.data_objects[adr_geom+ADR_AABB_ZMIN];
        f32 zmax = geometry.data_objects[adr_geom+ADR_AABB_ZMAX];

        // First check the bounding box that contains the mesh
        if (!test_ray_AABB(pos, dir, xmin, xmax, ymin, ymax, zmin, zmax)) return FLT_MAX;

        // If no octree first check every triangle
        distance = FLT_MAX;
        f32 tri_distance;
        if (octree_type == NO_OCTREE) {
            unsigned int nb_tri = geometry.data_objects[adr_geom+ADR_MESHED_NB_TRIANGLES];
            unsigned int i=0;
            while (i < nb_tri) {
                // Fetch a triangle
                unsigned int ptr_tri = adr_geom+ADR_MESHED_DATA+ i*9; // 3 vertices of float3
                float3 u = make_float3(geometry.data_objects[ptr_tri],
                                       geometry.data_objects[ptr_tri+1],
                                       geometry.data_objects[ptr_tri+2]);
                float3 v = make_float3(geometry.data_objects[ptr_tri+3],
                                       geometry.data_objects[ptr_tri+4],
                                       geometry.data_objects[ptr_tri+5]);
                float3 w = make_float3(geometry.data_objects[ptr_tri+6],
                                       geometry.data_objects[ptr_tri+7],
                                       geometry.data_objects[ptr_tri+8]);
                // Get distance to this triangle
                tri_distance = hit_ray_triangle(pos, dir, u, v, w);
                if (tri_distance < distance) distance = tri_distance;

                ++i;
            }
        // If regular octree
        } else if (octree_type == REG_OCTREE) {

            //// First get the octree index

            // Change particle frame (into voxelized volume)
            float3 localpos;
            localpos.x = pos.x - geometry.data_objects[adr_geom+ADR_AABB_XMIN]; // -= xmin
            localpos.y = pos.y - geometry.data_objects[adr_geom+ADR_AABB_YMIN]; // -= ymin
            localpos.z = pos.z - geometry.data_objects[adr_geom+ADR_AABB_ZMIN]; // -= zmin
            // Get spacing
            float3 s;
            s.x = geometry.data_objects[adr_geom+ADR_VOXELIZED_SX];
            s.y = geometry.data_objects[adr_geom+ADR_VOXELIZED_SY];
            s.z = geometry.data_objects[adr_geom+ADR_VOXELIZED_SZ];
            // Get the voxel index
            int3 ind;
            ind.x = (unsigned int)(localpos.x / s.x);
            ind.y = (unsigned int)(localpos.y / s.y);
            ind.z = (unsigned int)(localpos.z / s.z);

            // DDA algorithm

            float3 finc;
            finc.x = dir.x*s.x;
            finc.y = dir.y*s.y;
            finc.z = dir.z*s.z;
            float3 fpos;
            fpos.x = f32(ind.x);
            fpos.y = f32(ind.y);
            fpos.z = f32(ind.z);

            unsigned int nb_tri = geometry.data_objects[adr_geom+ADR_MESHED_NB_TRIANGLES];
            unsigned int nx = geometry.data_objects[adr_geom+ADR_MESHED_OCTREE_NX];
            unsigned int ny = geometry.data_objects[adr_geom+ADR_MESHED_OCTREE_NY];
            unsigned int nz = geometry.data_objects[adr_geom+ADR_MESHED_OCTREE_NZ];
            unsigned int adr_octree = adr_geom+ADR_MESHED_DATA+ 9*nb_tri; // 3 vertices of float3

            unsigned int index = ind.z*nx*ny + ind.y*nx + ind.x;

            // DDA until to find triangles on an octree cell
            while (geometry.data_objects[adr_octree+index] == 0) {
                ind.x = (unsigned int)fpos.x;
                ind.y = (unsigned int)fpos.y;
                ind.z = (unsigned int)fpos.z;

                // check boundary
                if (ind.x <0 && ind.x >= nx &&
                    ind.y <0 && ind.y >= ny &&
                    ind.z <0 && ind.z >= nz) {
                    break;
                }

                // new index
                index = ind.z*nx*ny + ind.y*nx + ind.x;
                // iterate DDA line
                fpos = f3_add(fpos, finc);
            }

            // if no triangle where found
            if (geometry.data_objects[adr_octree+index] == 0) {
                return FLT_MAX;
            // else check every triangle contain of the octree cell
            } else {
                unsigned int tri_per_cell = geometry.data_objects[adr_octree+index];
                unsigned int adr_to_cell = adr_octree + (nx*ny*nz) + index;
                unsigned int ptr_list_tri = adr_octree + 2*(nx*ny*nz) + geometry.data_objects[adr_to_cell];
                unsigned int i=0;
                while (i < tri_per_cell) {
                    unsigned int ptr_tri = geometry.data_objects[ptr_list_tri + i*9];

                    float3 u = make_float3(geometry.data_objects[ptr_tri],
                                           geometry.data_objects[ptr_tri+1],
                                           geometry.data_objects[ptr_tri+2]);
                    float3 v = make_float3(geometry.data_objects[ptr_tri+3],
                                           geometry.data_objects[ptr_tri+4],
                                           geometry.data_objects[ptr_tri+5]);
                    float3 w = make_float3(geometry.data_objects[ptr_tri+6],
                                           geometry.data_objects[ptr_tri+7],
                                           geometry.data_objects[ptr_tri+8]);

                    // Get distance to this triangle
                    tri_distance = hit_ray_triangle(pos, dir, u, v, w);
                    if (tri_distance < distance) distance = tri_distance;

                    ++i;
                } // while
            } // if triangle

        } // if regoctree

    } // if meshed

    return distance;
}

// Find the next geometry along the path of the particle
void __host__ __device__ get_next_geometry_boundary(Scene geometry, unsigned int cur_geom,
                                                     float3 pos, float3 dir,
                                                     f32 &interaction_distance,
                                                     unsigned int &geometry_volume) {

    geometry_volume = cur_geom;
    f32 distance;

    ////// Mother

    // First check the mother volume (particle escaping the volume)
    unsigned int adr_geom = geometry.ptr_objects[cur_geom];
    unsigned int obj_type = (unsigned int)geometry.data_objects[adr_geom+ADR_OBJ_TYPE];

    // Special case of voxelized volume where there are voxel boundary
    if (obj_type == VOXELIZED) {           
        // Volume bounding box
        f32 safety = get_distance_to_object(geometry, adr_geom, AABB, pos, dir);
        // Voxel boundary
        distance = get_distance_to_object(geometry, adr_geom, VOXELIZED, pos, dir);

        // If the safety is equal to distance (numerically very close espilon6) to the voxel
        // boundary it means, that the particle is escaping the volume.
        printf("         Safety %e vox distance %e pos %f %f %f\n", safety, distance, pos.x, pos.y, pos.z);
        if (fabs(distance-safety) < EPSILON3) {
            geometry_volume = geometry.mother_node[cur_geom];
        } else {
            // Distance < safety = Still inside the volume
            geometry_volume = cur_geom;
        }


    // Any other volumes
    } else {
        distance = get_distance_to_object(geometry, adr_geom, obj_type, pos, dir);
        geometry_volume = geometry.mother_node[cur_geom];
    }
    // First intersection distance given by the current volume
    interaction_distance = distance;// + EPSILON3; // overshoot

    ////// Children

    // Then check every child contains in this node
    unsigned int adr_node = geometry.ptr_nodes[cur_geom];
    unsigned int offset_node = 0;
    unsigned int id_child_geom;

    while (offset_node < geometry.size_of_nodes[cur_geom]) {

        // Child id
        id_child_geom = geometry.child_nodes[adr_node + offset_node];

        // Determine the type of the volume
        unsigned int adr_child_geom = geometry.ptr_objects[id_child_geom];
        obj_type = (unsigned int)geometry.data_objects[adr_child_geom+ADR_OBJ_TYPE];

        // Special case for voxelized volume (check the outter boundary)
        if (obj_type == VOXELIZED) {
            // Volume bounding box
            distance = get_distance_to_object(geometry, adr_child_geom, AABB, pos, dir);
        } else {
            // Any other volumes
            distance = get_distance_to_object(geometry, adr_child_geom, obj_type, pos, dir);
        }

        if (distance <= interaction_distance) {
            interaction_distance = distance;// + EPSILON3; // overshoot
            geometry_volume = id_child_geom;
        }

        //printf("Daughter %i dist %f id %i\n", obj_type, distance, id_child_geom);

        ++offset_node;
    }
}

/////////////////////////////////////////////////////////////////////////////////////
///////// Geometry Builder class ////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

GeometryBuilder::GeometryBuilder() {

    // Init the size of the structure Geometry
    world.ptr_objects_dim = 0;
    world.size_of_objects_dim = 0;
    world.data_objects_dim = 0;
    world.ptr_nodes_dim = 0;
    world.size_of_nodes_dim = 0;
    world.child_nodes_dim = 0;
    world.mother_node_dim = 0;

    // Init the first node id
    world.cur_node_id = 0;
}

///// Private ////////////////////////////////////////////////////

// Update the tree address
void GeometryBuilder::update_tree_address() {
    world.ptr_nodes[0] = 0;
    unsigned int i=1;
    while (i < world.ptr_nodes_dim) {
        world.ptr_nodes[i] = world.ptr_nodes[i-1] + world.size_of_nodes[i-1];
        ++i;
    }
}

// Search and return the material index for a given material name
unsigned int GeometryBuilder::get_material_index(std::string material_name) {

    // Check if this material is already used, if it is return the corresponding index
    unsigned int index = 0;
    while (index < materials_list.size()) {
        if (materials_list[index] == material_name) return index;
        ++index;
    }

    // If it is not, add a new entry into the material table
    index = materials_list.size();
    materials_list.push_back(material_name);

    return index;
}

///// Hierarchical structure of the geometry ////////////////////////

// Add the root
void GeometryBuilder::add_root() {

    array_push_back(&world.ptr_nodes, world.ptr_nodes_dim, 0);
    array_push_back(&world.size_of_nodes, world.size_of_nodes_dim, 0);
    array_push_back(&world.mother_node, world.mother_node_dim, 0);
    world.cur_node_id = 0;

}

// Add a node
void GeometryBuilder::add_node(unsigned int mother_id) {
    // New node ID
    world.cur_node_id++;

    // Insert this object into the tree
    array_insert(&world.child_nodes, world.child_nodes_dim,
                 world.ptr_nodes[mother_id]+world.size_of_nodes[mother_id], world.cur_node_id);

    // Update the tree
    world.size_of_nodes[mother_id]++;
    array_push_back(&world.size_of_nodes, world.size_of_nodes_dim, 0);
    array_push_back(&world.ptr_nodes, world.ptr_nodes_dim, world.cur_node_id);
    array_push_back(&world.mother_node, world.mother_node_dim, mother_id);

    // Update tree address
    update_tree_address();
}

// Print the tree structure of the geometry
void GeometryBuilder::print_tree() {
    // print each node
    unsigned int i = 0;
    unsigned int j = 0;
    while (i < world.size_of_nodes_dim) {
        printf("(mother: %i)--[node: %i]--(childs: ", world.mother_node[i], i);
        j=0; while (j < world.size_of_nodes[i]) {
            printf("%i,", world.child_nodes[world.ptr_nodes[i]+j]);
            ++j;
        }
        printf(")\n");
        ++i;
    }
    printf("\n");
}

///// Utils ////////////////////////////////////////////////////////////////////////////////

// Print the current world
void GeometryBuilder::print_geometry() {
    // Print out the tree structure
    print_tree();

    // Print out every object name
    unsigned int i;
    printf("List of object:\n");
    i=0; while (i < name_objects.size()) {
        printf("%i - %s\n", i, name_objects[i].c_str());
        ++i;
    }
    printf("\n");

    // Print out every material name
    printf("List of material:\n");
    i=0; while (i < materials_list.size()) {
        printf("%i - %s\n", i, materials_list[i].c_str());
        ++i;
    }
    printf("\n");

    // Print out each object contains on the tree
    i=0; while (i < world.ptr_objects_dim) {
        // Get obj address
        unsigned int address_obj = world.ptr_objects[i];

        // Object name
        printf("::: %s :::\n", name_objects[i].c_str());

        // Same header for everyone
        unsigned int type = (unsigned int)(world.data_objects[address_obj+ADR_OBJ_TYPE]);
        unsigned int mat = (unsigned int)(world.data_objects[address_obj+ADR_OBJ_MAT_ID]);
        f32 xmin = world.data_objects[address_obj+ADR_AABB_XMIN];
        f32 xmax = world.data_objects[address_obj+ADR_AABB_XMAX];
        f32 ymin = world.data_objects[address_obj+ADR_AABB_YMIN];
        f32 ymax = world.data_objects[address_obj+ADR_AABB_YMAX];
        f32 zmin = world.data_objects[address_obj+ADR_AABB_ZMIN];
        f32 zmax = world.data_objects[address_obj+ADR_AABB_ZMAX];

        // Print information0
        switch (type) {
        case AABB:
            printf("type: AABB\n"); break;
        case SPHERE:
            printf("type: SPHERE\n"); break;
        } // switch

        printf("material: %s\n", materials_list[mat].c_str());
        printf("xmin: %f xmax: %f ymin: %f ymax: %f zmin: %f zmax: %f\n\n",
                xmin, xmax, ymin, ymax, zmin, zmax);


        ++i;
    } // while
}



/*
// Print out the geometry raw data
void GeometryBuilder::print_raw() {

    // Print out every object name
    unsigned int i;
    printf("List of object [%lu]: ", World.name_objects.size());
    i=0; while (i < World.name_objects.size()) {
        printf("%s ", World.name_objects[i].c_str());
        ++i;
    }
    printf("\n\n");

    // Print out every material name
    printf("List of material [%lu]: ", World.materials_list.size());
    i=0; while (i < World.materials_list.size()) {
        printf("%s ", World.materials_list[i].c_str());
        ++i;
    }
    printf("\n\n");

    // Print out size of objects
    printf("Size of objects [%lu]: ", World.size_of_objects.size());
    i=0; while (i < World.size_of_objects.size()) {
        printf("%i ", World.size_of_objects[i]);
        ++i;
    }
    printf("\n\n");

    // Print out object addresses
    printf("Object addresses [%lu]: ", World.ptr_objects.size());
    i=0; while (i < World.ptr_objects.size()) {
        printf("%i ", World.ptr_objects[i]);
        ++i;
    }
    printf("\n\n");

    // Print out object data
    printf("Object data [%lu]: ", World.data_objects.size());
    i=0; while (i < World.data_objects.size()) {
        printf("%f ", World.data_objects[i]);
        ++i;
    }
    printf("\n\n");

}
*/

/*
// Save the world in order to share an use it later
void GeometryBuilder::save_ggems_geometry(std::string filename) {

    // check extension
    if (filename.size() < 10) {
        printf("Error, to export a ggems geometry, the exension must be '.ggems_geom'!\n");
        return;
    }
    std::string ext = filename.substr(filename.size()-10);
    if (ext!="ggems_geom") {
        printf("Error, to export a ggems geometry, the exension must be '.ggems_geom'!\n");
        return;
    }

    FILE *pfile = fopen(filename.c_str(), "wb");
    unsigned int i, nb, tmp;

    // .: Tree :.  -  First export the tree that structure the world

    // 1. ptr_nodes [N, data]
    nb = World.tree.ptr_nodes.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.tree.ptr_nodes.data(), nb, sizeof(unsigned int), pfile);

    // 2. size_of_nodes [N, data]
    nb = World.tree.size_of_nodes.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.tree.size_of_nodes.data(), nb, sizeof(unsigned int), pfile);

    // 3. child_nodes [N, data]
    nb = World.tree.child_nodes.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.tree.child_nodes.data(), nb, sizeof(unsigned int), pfile);

    // 4. mother_node [N, data]
    nb = World.tree.mother_node.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.tree.mother_node.data(), nb, sizeof(unsigned int), pfile);

    // 5. cur_node_id [val]
    fwrite(&World.tree.cur_node_id, 1, sizeof(unsigned int), pfile);

    // .: World :.  -  Then export the world

    // 6. name_objects [N, data]
    nb = World.name_objects.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    i=0; while (i < nb) {
        tmp = World.name_objects[i].size();
        fwrite(&tmp, 1, sizeof(unsigned int), pfile);
        fwrite(World.name_objects[i].c_str(), World.name_objects[i].size(), sizeof(char), pfile);
        ++i;
    }

    // 7. materials_list [N, data]
    nb = World.materials_list.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    i=0; while (i < nb) {
        tmp = World.materials_list[i].size();
        fwrite(&tmp, 1, sizeof(unsigned int), pfile);
        fwrite(World.materials_list[i].c_str(), World.materials_list[i].size(), sizeof(char), pfile);
        ++i;
    }

    // 8. ptr_objects [N, data]
    nb = World.ptr_objects.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.ptr_objects.data(), nb, sizeof(unsigned int), pfile);

    // 9. size_of_objects [N, data]
    nb = World.size_of_objects.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.size_of_objects.data(), nb, sizeof(unsigned int), pfile);

    // 10. data_objects [N, data] (the big one!!!)
    nb = World.data_objects.size();
    fwrite(&nb, 1, sizeof(unsigned int), pfile);
    fwrite(World.data_objects.data(), nb, sizeof(f32), pfile);


    fclose(pfile);
}
*/


////
////////////////////// Object management ///////////////////////////////////////////////////
////
//
// !!!! Convention of the head of any object written in the world structure !!!!
//
// Object Type
//  array_push_back(world.data_objects, world.data_objects_dim, (f32)AABB);
// Material index
//  array_push_back(world.data_objects, world.data_objects_dim, (f32)get_material_index(obj.material_name));
// AABB parameters
//  array_push_back(world.data_objects, world.data_objects_dim, obj.xmin);
//  array_push_back(world.data_objects, world.data_objects_dim, obj.xmax);
//  array_push_back(world.data_objects, world.data_objects_dim, obj.ymin);
//  array_push_back(world.data_objects, world.data_objects_dim, obj.ymax);
//  array_push_back(world.data_objects, world.data_objects_dim, obj.zmin);
//  array_push_back(world.data_objects, world.data_objects_dim, obj.zmax);

// Add the world
unsigned int GeometryBuilder::add_world(Aabb obj) {

    // Add the root tree
    add_root();

    // Put this object into buffer
    buffer_aabb[world.cur_node_id] = obj;
    buffer_obj_type[world.cur_node_id] = AABB;

    return world.cur_node_id;

}

// Add an AABB object into the world
unsigned int GeometryBuilder::add_object(Aabb obj, unsigned int mother_id) {

    // Add this object to the tree
    add_node(mother_id);

    // Put this object into buffer
    buffer_aabb[world.cur_node_id] = obj;
    buffer_obj_type[world.cur_node_id] = AABB;

    return world.cur_node_id;
}

// Add a Sphere object into the world
unsigned int GeometryBuilder::add_object(Sphere obj, unsigned int mother_id) {

    // Add this object to the tree
    add_node(mother_id);

    // Put this object into buffer
    buffer_sphere[world.cur_node_id] = obj;
    buffer_obj_type[world.cur_node_id] = SPHERE;

    return world.cur_node_id;
}

// Add a Voxelized object into the world
unsigned int GeometryBuilder::add_object(Voxelized obj, unsigned int mother_id) {

    // Add this object to the tree
    add_node(mother_id);

    // Put this object into buffer
    buffer_voxelized[world.cur_node_id] = obj;
    buffer_obj_type[world.cur_node_id] = VOXELIZED;

    return world.cur_node_id;
}

// Add a Meshed object into the world
unsigned int GeometryBuilder::add_object(Meshed obj, unsigned int mother_id) {

    // Add thid object to the tree
    add_node(mother_id);

    // Put this object into buffer
    buffer_meshed[world.cur_node_id] = obj;
    buffer_obj_type[world.cur_node_id] = MESHED;

    return world.cur_node_id;
}

////////////////////////////////////////////////////////////////////////

// Build AABB object into the scene structure
void GeometryBuilder::build_object(Aabb obj) {

    // Store the address to access to this object
    array_push_back(&world.ptr_objects, world.ptr_objects_dim, world.data_objects_dim);

    // Store the information of this object

    // Object Type
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)AABB);
    // Material index
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)get_material_index(obj.material_name));
     // AABB parameters
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmax);

    // Name of this object
    name_objects.push_back(obj.object_name);
    // Color of this object
    object_colors.push_back(obj.color);
    // Transparency of this object
    object_transparency.push_back(obj.transparency);
    // Wireframe option of this object
    object_wireframe.push_back(obj.wireframe);
    // Store the size of this object
    array_push_back(&world.size_of_objects, world.size_of_objects_dim, SIZE_AABB_OBJ);
}

// Build sphere object into the scene structure
void GeometryBuilder::build_object(Sphere obj) {
    // Store the address to access to this object
    array_push_back(&world.ptr_objects, world.ptr_objects_dim, world.data_objects_dim);

    // Store the information of this object

    // Object Type
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)SPHERE);
    // Material index
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)get_material_index(obj.material_name));
     // AABB parameters
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmax);
    // Sphere parameters
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cx);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cy);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cz);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.radius);

    // Name of this object
    name_objects.push_back(obj.object_name);
    // Color of this object
    object_colors.push_back(obj.color);
    // Transparency of this object
    object_transparency.push_back(obj.transparency);
    // Wireframe option of this object
    object_wireframe.push_back(obj.wireframe);
    // Store the size of this object
    array_push_back(&world.size_of_objects, world.size_of_objects_dim, SIZE_SPHERE_OBJ);
}

// Build voxelized object into the scene structure
void GeometryBuilder::build_object(Voxelized obj) {
    // TODO
    // If optimizer, every object contains within the voxelized volume must be identified
    // For instance when considering YVAN navigator (BVH must be stored on the world), each
    // voxel contain ID of the child volume

    ///// First step
    // We need to merge and update the material ID according the current list of materials
    // Build a LUT to convert the old IDs in new ones
    std::vector<unsigned int> new_id;
    unsigned int i = 0;
    while (i < obj.list_of_materials.size()) {
        new_id.push_back(get_material_index(obj.list_of_materials[i]));
        ++i;
    }

    // Now convert every material ID contains on the voxelized volume
    i=0; while (i < obj.number_of_voxels) {
        obj.data[i] = new_id[obj.data[i]];
        ++i;
    }
    /////

    // Store the address to access to this object
    array_push_back(&world.ptr_objects, world.ptr_objects_dim, world.data_objects_dim);

    // Store the information of this object

    // Object Type
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)VOXELIZED);
    // Material index
    array_push_back(&world.data_objects, world.data_objects_dim, -1.0f); // // Heterogeneous material
    // AABB parameters
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmax);
    // Parameters for this object
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_vox_x);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_vox_y);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_vox_z);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.spacing_x);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.spacing_y);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.spacing_z);
    // Finally append voxelized data into the world
    array_append_array(&world.data_objects, world.data_objects_dim, &(obj.data), obj.number_of_voxels);

    // Name of this object
    name_objects.push_back(obj.object_name);
    // Color of this object
    object_colors.push_back(obj.color);
    // Transparency of this object
    object_transparency.push_back(obj.transparency);
    // Wireframe option of this object
    object_wireframe.push_back(obj.wireframe);
    // Store the size of this object
    array_push_back(&world.size_of_objects, world.size_of_objects_dim, obj.number_of_voxels+SIZE_VOXELIZED_OBJ);

}

// Build meshed object into the scene structure
void GeometryBuilder::build_object(Meshed obj) {

    // Store the address to access to this object
    array_push_back(&world.ptr_objects, world.ptr_objects_dim, world.data_objects_dim);

    // Store the information of this object

    // Object Type
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)MESHED);
    // Material index
    array_push_back(&world.data_objects, world.data_objects_dim, (f32)get_material_index(obj.material_name));
    // AABB parameters
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.xmax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.ymax);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmin);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.zmax);
    // Parameters for this object
    array_push_back(&world.data_objects, world.data_objects_dim, obj.number_of_vertices);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.number_of_triangles);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.octree_type);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_cell_x);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_cell_y);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.nb_cell_z);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cell_size_x);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cell_size_y);
    array_push_back(&world.data_objects, world.data_objects_dim, obj.cell_size_z);

    // Append triangles into the world
    array_append_array(&world.data_objects, world.data_objects_dim, &obj.vertices, 3*obj.number_of_vertices); // xyz

    // Finally append the octree if defined
    if (obj.octree_type == REG_OCTREE) {
        // Append the number of objects per cell
        f32 *tmp = &obj.nb_objs_per_cell[0]; // create a pointer to append into the world
        array_append_array(&world.data_objects, world.data_objects_dim, &tmp, obj.nb_objs_per_cell.size());

        // Append the addr of each cell
        tmp = &obj.addr_to_cell[0];
        array_append_array(&world.data_objects, world.data_objects_dim, &tmp, obj.addr_to_cell.size());

        // Append the list of objects per cell
        tmp = &obj.list_objs_per_cell[0];
        array_append_array(&world.data_objects, world.data_objects_dim, &tmp, obj.list_objs_per_cell.size());
    }

    //////////////

    // Name of this object
    name_objects.push_back(obj.object_name);
    // Color of this object
    object_colors.push_back(obj.color);
    // Transparency of this object
    object_transparency.push_back(obj.transparency);
    // Wireframe option of this object
    object_wireframe.push_back(obj.wireframe);
    // Store the size of this object
    if (obj.octree_type == REG_OCTREE) {
        array_push_back(&world.size_of_objects, world.size_of_objects_dim, 3*obj.number_of_vertices + obj.nb_objs_per_cell.size() +
                                                                           obj.addr_to_cell.size() + obj.list_objs_per_cell.size() + SIZE_MESHED_OBJ);

    } else { // NO_OCTREE
        array_push_back(&world.size_of_objects, world.size_of_objects_dim, 3*obj.number_of_vertices+SIZE_MESHED_OBJ);
    }

    // Clear data of the octree
    obj.nb_objs_per_cell.clear();
    obj.addr_to_cell.clear();
    obj.list_objs_per_cell.clear();

}

// Build the complete scene
void GeometryBuilder::build_scene() {

    // Scan every object a build it to the scene structure

    unsigned int i = 0;
    while (i < world.ptr_nodes_dim) {

        // AABB
        if (buffer_obj_type[i] == AABB) {
            build_object(buffer_aabb[i]);
        // Sphere
        } else if (buffer_obj_type[i] == SPHERE) {
            build_object(buffer_sphere[i]);
        // Voxelized
        } else if (buffer_obj_type[i] == VOXELIZED) {
            build_object(buffer_voxelized[i]);
        // Meshed
        } else if (buffer_obj_type[i] == MESHED) {
            build_object(buffer_meshed[i]);
        }

        ++i;
    }

}

#endif
