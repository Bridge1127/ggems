#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// FIREwork is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// FIREwork is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with FIREwork.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef PHOTON_NAVIGATOR_CU
#define PHOTON_NAVIGATOR_CU

#include "photon_navigator.cuh"

// CPU photon navigator
__host__ void cpu_photon_navigator(ParticleStack &particles, ui32 part_id,
                          Scene geometry, MaterialsTable materials,
                          PhotonCrossSectionTable photon_CS_table,
                          GlobalSimulationParameters parameters,
                          Singles &singles,
                          HistoryBuilder &history) {

    // Read position
    f64xyz pos;
    pos.x = particles.px[part_id];
    pos.y = particles.py[part_id];
    pos.z = particles.pz[part_id];

    // Read direction
    f64xyz dir;
    dir.x = particles.dx[part_id];
    dir.y = particles.dy[part_id];
    dir.z = particles.dz[part_id];

    // Get the current volume containing the particle
    ui32 cur_id_geom = particles.geometry_id[part_id];

    // Get the material that compose this volume
    ui32 id_mat = get_geometry_material(geometry, cur_id_geom, pos);

#ifdef DEBUG
    printf("  begin %i\n", part_id);
    printf("     Cur id geom %i mat %i\n", cur_id_geom, id_mat);
    printf("     InitPos %f %f %f\n", pos.x, pos.y, pos.z);
    printf("     Energy %f\n", particles.E[part_id]);
#endif

    //// Find next discrete interaction ///////////////////////////////////////

    f64 next_interaction_distance = F64_MAX;
    ui8 next_discrete_process = 0;
    ui32 next_geometry_volume = cur_id_geom;
    f64 interaction_distance;
    f64 cross_section;

    // Search the energy index to read CS
    ui32 E_index = binary_search(particles.E[part_id], photon_CS_table.E_bins,
                                 photon_CS_table.nb_bins);

    // TODO if E_index = 0?
    assert(E_index != 0);
    /////////////////////

    //printf("Before CS\n");

    // If photoelectric
    if (parameters.physics_list[PHOTON_PHOTOELECTRIC]) {
        cross_section = get_CS_from_table(photon_CS_table.E_bins, photon_CS_table.Photoelectric_Std_CS,
                                          particles.E[part_id], E_index, id_mat, photon_CS_table.nb_bins);
        interaction_distance = -log( JKISS32(particles, part_id) ) / cross_section;
#ifdef DEBUG
        printf(" Photoelectric: CS %e dist %e\n", cross_section, interaction_distance);
#endif
        if (interaction_distance < next_interaction_distance) {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_PHOTOELECTRIC;
        }
        //if (cur_id_geom==1) printf("E %e CS %e\n", particles.E[part_id], cross_section);
    }

    // If Compton
    if (parameters.physics_list[PHOTON_COMPTON]) {
        cross_section = get_CS_from_table(photon_CS_table.E_bins, photon_CS_table.Compton_Std_CS,
                                          particles.E[part_id], E_index, id_mat, photon_CS_table.nb_bins);
        interaction_distance = -log( JKISS32(particles, part_id) ) / cross_section;
#ifdef DEBUG
        printf(" Compton: CS %e dist %e\n", cross_section, interaction_distance);
#endif
        if (interaction_distance < next_interaction_distance) {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_COMPTON;
        }
    }

    // If Rayleigh
    if (parameters.physics_list[PHOTON_RAYLEIGH]) {
        cross_section = get_CS_from_table(photon_CS_table.E_bins, photon_CS_table.Rayleigh_Lv_CS,
                                          particles.E[part_id], E_index, id_mat, photon_CS_table.nb_bins);
        interaction_distance = -log( JKISS32(particles, part_id) ) / cross_section;
#ifdef DEBUG
        printf(" Rayleigh: CS %e dist %e\n", cross_section, interaction_distance);
#endif
        if (interaction_distance < next_interaction_distance) {
            next_interaction_distance = interaction_distance;
            next_discrete_process = PHOTON_RAYLEIGH;
        }

    }

    //// Get the next distance boundary volume /////////////////////////////////

    //printf("Before geom\n");

    ui32 hit_id_geom = 0;
    get_next_geometry_boundary(geometry, cur_id_geom, pos, dir, interaction_distance, hit_id_geom);
#ifdef DEBUG
        printf(" Geom: dist %e\n", interaction_distance);
#endif
    if (interaction_distance <= next_interaction_distance) {
        next_interaction_distance = interaction_distance + EPSILON3; // Overshoot
        next_discrete_process = GEOMETRY_BOUNDARY;
        next_geometry_volume = hit_id_geom;        
    }

    //// Move particle //////////////////////////////////////////////////////

    //printf("Move particle\n");

    // TODO
    // Compute the energy deposit position randomly along the path
    //if (parameters.dose_flag) {
        //f32xyz pos_edep = add_vector(photon.pos, scale_vector(photon.dir, next_interaction_distance*prng()));
    //}

    // Move the particle
    pos = fxyz_add(pos, fxyz_scale(dir, next_interaction_distance));

    // TODO
    //particles.tof[id] += gpu_speed_of_light * next_interaction_distance;

    particles.px[part_id] = pos.x;
    particles.py[part_id] = pos.y;
    particles.pz[part_id] = pos.z;

    particles.geometry_id[part_id] = next_geometry_volume;

    // Check world boundary
    f64 xmin = geometry.data_objects[ADR_AABB_XMIN]; // adr_world_geom = 0
    f64 xmax = geometry.data_objects[ADR_AABB_XMAX];
    f64 ymin = geometry.data_objects[ADR_AABB_YMIN];
    f64 ymax = geometry.data_objects[ADR_AABB_YMAX];
    f64 zmin = geometry.data_objects[ADR_AABB_ZMIN];
    f64 zmax = geometry.data_objects[ADR_AABB_ZMAX];

    // Stop simulation if out of the world
    if (!test_point_AABB(pos, xmin, xmax, ymin, ymax, zmin, zmax)) {

        particles.endsimu[part_id] = PARTICLE_DEAD;

        // Record this step if required
        if (history.record_flag == ENABLED) {
            history.cpu_record_a_step(particles, part_id);
        }

        //if (particles.E[part_id] == 0.5) printf("No Interaction\n");

        return;
    }

//    // Stop simulation if out of the world
//    if (   pos.x <= xmin || pos.x >= xmax
//        || pos.y <= ymin || pos.y >= ymax
//        || pos.z <= zmin || pos.z >= zmax) {

//        particles.endsimu[part_id] = PARTICLE_DEAD;

//        // Record this step if required
//        if (history.record_flag == ENABLED) {
//            history.cpu_record_a_step(particles, part_id);
//        }

//        //if (particles.E[part_id] == 0.5) printf("No Interaction\n");

//        return;
//    }

    //// Apply discrete process //////////////////////////////////////////////////

    f32 discrete_loss = 0.0f;
    SecParticle electron;
    electron.E = 0;

#ifdef DEBUG
    printf("     Dist %f NextVol %i pos %f %f %f ", next_interaction_distance, next_geometry_volume, pos.x, pos.y, pos.z);
#endif

    if (next_discrete_process == PHOTON_COMPTON) {

        //   TODO: cutE = materials.electron_cut_energy[mat]                 cutE
        electron = Compton_SampleSecondaries_standard(particles, 0.0, part_id, parameters);



        // Debug
        //printf("id %i - pos %f %f %f - dir %f %f %f - Cmpt - geom cur %i hit %i\n", part_id, pos.x, pos.y, pos.z,
        //                                                                 dir.x, dir.y, dir.z,
        //                                                                 cur_id_geom, next_geometry_volume);
#ifdef DEBUG
        printf(" Compton\n");
#endif
    }

    if (next_discrete_process == PHOTON_PHOTOELECTRIC) {

        //   TODO: cutE = materials.electron_cut_energy[mat]                                               cutE
        electron = Photoelec_SampleSecondaries_standard(particles, materials, photon_CS_table,
                                                                    E_index, 0.0, id_mat, part_id, parameters);

        // Debug
        //printf("id %i - pos %f %f %f - dir %f %f %f - PE - geom cur %i hit %i\n", part_id, pos.x, pos.y, pos.z,
        //                                                               dir.x, dir.y, dir.z,
        //                                                               cur_id_geom, next_geometry_volume);
#ifdef DEBUG
        printf(" PE\n");
#endif
    }

    if (next_discrete_process == PHOTON_RAYLEIGH) {
        Rayleigh_SampleSecondaries_Livermore(particles, materials, photon_CS_table, E_index, id_mat, part_id);
        //printf("Rayleigh\n");
    }


    if (next_discrete_process == GEOMETRY_BOUNDARY) {
        // Debug
        //printf("id %i - pos %f %f %f - dir %f %f %f - Bnd - geom cur %i hit %i\n", part_id, pos.x, pos.y, pos.z,
        //                                                                 dir.x, dir.y, dir.z,
        //                                                                 cur_id_geom, next_geometry_volume);
#ifdef DEBUG
        printf(" Geom\n");
#endif
    }

    //// Get discrete energy lost

    // If e- is not tracking drop its energy
    if (electron.endsimu == PARTICLE_DEAD) {
        discrete_loss += electron.E;
    }
    // If gamma is absorbed drop its energy
    if (particles.endsimu[part_id] == PARTICLE_DEAD) {
        discrete_loss += particles.E[part_id];
    }

    //// Handle sensitive object and singles detection


    if (parameters.record_singles_flag &&
            get_geometry_is_sensitive(geometry, cur_id_geom) && discrete_loss > 0) {

        printf("ID %i Cur id %i flag %i Pos %f %f %f Eloss %f\n", part_id, cur_id_geom,
               get_geometry_is_sensitive(geometry, cur_id_geom), pos.x, pos.y, pos.z, discrete_loss);

        if (singles.nb_hits[part_id] == 0) {
            singles.px[part_id] = pos.x*discrete_loss;
            singles.py[part_id] = pos.y*discrete_loss;
            singles.pz[part_id] = pos.z*discrete_loss;
            singles.E[part_id] = discrete_loss;
            singles.nb_hits[part_id] += 1;
        } else {
            singles.px[part_id] += pos.x*discrete_loss;
            singles.py[part_id] += pos.y*discrete_loss;
            singles.pz[part_id] += pos.z*discrete_loss;
            singles.E[part_id] += discrete_loss;
            singles.nb_hits[part_id] += 1;
        }

    }

    //// This part is for debuging and vrml viewer

    // Record this step if required
    if (history.record_flag == ENABLED) {
        history.cpu_record_a_step(particles, part_id);
    }

/*
    // DEBUGING: phasespace
    if (next_geometry_volume == 0 && particles.endsimu[part_id] == PARTICLE_ALIVE) {
        printf("%e %e %e %e %e %e %e\n", particles.E[part_id], pos.x, pos.y, pos.z, dir.x, dir.y, dir.z);
        particles.endsimu[part_id] = PARTICLE_DEAD;
        return;
    }
*/



}













#endif
