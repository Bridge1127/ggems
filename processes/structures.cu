// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef STRUCTURES_CU
#define STRUCTURES_CU
#include "stuctures.h"

// Some error "checkers"
// comes from "cuda by example" book
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#ifndef HANDLE_ERROR
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#endif


// comes from "cuda programming" book
__host__ void cuda_error_check (const char * prefix, const char * postfix) {
    if(hipPeekAtLastError() != hipSuccess ) {
        printf("\n%s%s%s\n",prefix, hipGetErrorString(hipGetLastError()),postfix);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

}


// Stack device allocation
void _stack_device_malloc(ParticleStack &stackpart, int stack_size) {
    stackpart.size = stack_size;
    unsigned int mem_stackpart_float = stack_size * sizeof(float);
    unsigned int mem_stackpart_uint = stack_size * sizeof(unsigned int);
    unsigned int mem_stackpart_char = stack_size * sizeof(char);

    // property
    HANDLE_ERROR(hipMalloc((void**) &stackpart.E, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.dx, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.dy, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.dz, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.px, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.py, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.pz, mem_stackpart_float));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.tof, mem_stackpart_float));
    // PRNG
    HANDLE_ERROR(hipMalloc((void**) &stackpart.prng_state_1, mem_stackpart_uint));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.prng_state_2, mem_stackpart_uint));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.prng_state_3, mem_stackpart_uint));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.prng_state_4, mem_stackpart_uint));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.prng_state_5, mem_stackpart_uint));
    // simulation
    HANDLE_ERROR(hipMalloc((void**) &stackpart.endsimu, mem_stackpart_char));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.level, mem_stackpart_char));
    HANDLE_ERROR(hipMalloc((void**) &stackpart.pname, mem_stackpart_char));
}

// Init particle seeds with the main seed
void wrap_init_particle_seeds(ParticleStack &d_p, int seed) {
    unsigned int *state1 = (unsigned int*)malloc(sizeof(unsigned int)*d_p.size);
    unsigned int *state2 = (unsigned int*)malloc(sizeof(unsigned int)*d_p.size);
    unsigned int *state3 = (unsigned int*)malloc(sizeof(unsigned int)*d_p.size);
    unsigned int *state4 = (unsigned int*)malloc(sizeof(unsigned int)*d_p.size);
    unsigned int *state5 = (unsigned int*)malloc(sizeof(unsigned int)*d_p.size);

    srand(seed);
    int i=0;
    while (i<d_p.size) {
        // init random seed
        state1[i] = rand();
        state2[i] = rand();
        state3[i] = rand();
        state4[i] = rand();
        state5[i] = 0;      // carry
        ++i;
    }
// printf("%f %f %f %f %");
    HANDLE_ERROR(hipMemcpy(d_p.prng_state_1, state1, sizeof(unsigned int)*d_p.size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_p.prng_state_2, state2, sizeof(unsigned int)*d_p.size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_p.prng_state_3, state3, sizeof(unsigned int)*d_p.size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_p.prng_state_4, state4, sizeof(unsigned int)*d_p.size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_p.prng_state_5, state5, sizeof(unsigned int)*d_p.size, hipMemcpyHostToDevice));
}

// Copy electron cross section table to device
void  wrap_copy_crosssection_to_device (CrossSectionTableElectrons &h_etables,
                                        CrossSectionTableElectrons &d_etables,
                                        char *m_physics_list) {

    unsigned int mem_mat_bins_flt = h_etables.nb_mat*h_etables.nb_bins * sizeof(float);

    d_etables.E_min = h_etables.E_min;
    d_etables.E_max = h_etables.E_max;
    d_etables.nb_bins = h_etables.nb_bins;
    d_etables.nb_mat = h_etables.nb_mat;
    d_etables.cutEnergyElectron = h_etables.cutEnergyElectron;
    d_etables.cutEnergyGamma = h_etables.cutEnergyGamma;

    HANDLE_ERROR(hipMalloc((void**) &d_etables.E, mem_mat_bins_flt));
    HANDLE_ERROR(hipMalloc((void**) &d_etables.eRange, mem_mat_bins_flt));

    HANDLE_ERROR(hipMemcpy(d_etables.E, h_etables.E, mem_mat_bins_flt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_etables.eRange, h_etables.eRange, mem_mat_bins_flt, hipMemcpyHostToDevice));

    if(m_physics_list[ELECTRON_MSC] == 1) {
        HANDLE_ERROR(hipMalloc((void**) &d_etables.eMSC, mem_mat_bins_flt));

        HANDLE_ERROR(hipMemcpy(d_etables.eMSC, h_etables.eMSC, mem_mat_bins_flt, hipMemcpyHostToDevice));
    }

    if(m_physics_list[ELECTRON_BREMSSTRAHLUNG] == 1) {
        HANDLE_ERROR(hipMalloc((void**) &d_etables.eBremdedx, mem_mat_bins_flt));
        HANDLE_ERROR(hipMalloc((void**) &d_etables.eBremCS, mem_mat_bins_flt));

        HANDLE_ERROR(hipMemcpy(d_etables.eBremdedx, h_etables.eBremdedx, mem_mat_bins_flt, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(d_etables.eBremCS, h_etables.eBremCS, mem_mat_bins_flt, hipMemcpyHostToDevice));

    }

    if(m_physics_list[ELECTRON_IONISATION] == 1) {
        HANDLE_ERROR(hipMalloc((void**) &d_etables.eIonisationdedx, mem_mat_bins_flt));
        HANDLE_ERROR(hipMalloc((void**) &d_etables.eIonisationCS, mem_mat_bins_flt));

        HANDLE_ERROR(hipMemcpy(d_etables.eIonisationdedx, h_etables.eIonisationdedx, mem_mat_bins_flt, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(d_etables.eIonisationCS, h_etables.eIonisationCS, mem_mat_bins_flt, hipMemcpyHostToDevice));
    }


}

#endif
