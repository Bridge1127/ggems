#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file cross_sections.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 18 novembre 2015
 *
 *
 *
 */


#ifndef CROSS_SECTIONS_CU
#define CROSS_SECTIONS_CU
#include "cross_sections.cuh"

//// CrossSectionsManager class ////////////////////////////////////////////////////

CrossSections::CrossSections() {
    photon_CS.nb_bins = 0;
    photon_CS.nb_mat = 0;
    electronCSTable = new ElectronCrossSection;
    parameters = new GlobalSimulationParameters();
}

// Main function
bool CrossSections::m_check_mandatory() {
    if (photon_CS.nb_bins == 0 || photon_CS.nb_mat == 0) return false;
    else return true;
}

void CrossSections::initialize(Materials materials, GlobalSimulationParameters parameters) {

    ///WARNING fonction m_build_table deplacee au dessus de check. Non verifié
    

    // Build table on CPU side
    m_build_table(materials, parameters);

    // Check if everything was set properly
    if ( !m_check_mandatory() ) {
        print_error("CrossSectionsManager parameters error!");
        exit_simulation();
    }



    // Allocation and copy
    if (parameters.data_h.device_target == GPU_DEVICE) m_copy_cs_table_cpu2gpu();

}

// Build cross sections table according material, physics effects and particles
void CrossSections::m_build_table(Materials materials, GlobalSimulationParameters parameters) {

    // Read parameters
    ui32 nbin = parameters.data_h.cs_table_nbins;
    f32 min_E = parameters.data_h.cs_table_min_E;
    f32 max_E = parameters.data_h.cs_table_max_E;

    // First thing first, sample energy following the number of bins
    f32 slope = log(max_E / min_E);
    ui32 i = 0;
    photon_CS.data_h.E_bins = (f32*)malloc(nbin * sizeof(f32));
    while (i < nbin) {
        photon_CS.data_h.E_bins[i] = min_E * exp( slope * ((f32)i/((f32)nbin-1)) ) * MeV;
        ++i;
    }

    
    // Find if there are photon and electron in this simulation;
    there_is_photon = parameters.data_h.physics_list[PHOTON_COMPTON] ||
                         parameters.data_h.physics_list[PHOTON_PHOTOELECTRIC] ||
                         parameters.data_h.physics_list[PHOTON_RAYLEIGH];
                         
    there_is_electron = parameters.data_h.physics_list[ELECTRON_IONISATION] ||
                            parameters.data_h.physics_list[ELECTRON_BREMSSTRAHLUNG] ||
                            parameters.data_h.physics_list[ELECTRON_MSC];

    // Then init data
    ui32 tot_elt = materials.data_h.nb_materials*nbin;
    ui32 tot_elt_mem = tot_elt * sizeof(f32);

    // Photon CS table if need
    if (there_is_photon) {
        photon_CS.data_h.Compton_Std_CS = (f32*)malloc(tot_elt_mem);
        photon_CS.data_h.Photoelectric_Std_CS = (f32*)malloc(tot_elt_mem);
        photon_CS.data_h.Photoelectric_Std_xCS = (f32*)malloc(nbin * 101 * sizeof(f32)); // 100 Z elements,
                                                                                            // starting from index 1
        photon_CS.data_h.Rayleigh_Lv_CS = (f32*)malloc(tot_elt_mem);
        photon_CS.data_h.Rayleigh_Lv_SF = (f32*)malloc(nbin * 101 * sizeof(f32)); // 100 Z elements,
                                                                                     // starting from index 1
        photon_CS.data_h.Rayleigh_Lv_xCS = (f32*)malloc(nbin * 101 * sizeof(f32)); // 100 Z elements,
                                                                                     // starting from index 1       
        photon_CS.data_h.E_min = min_E;
        photon_CS.data_h.E_max = max_E;
        photon_CS.data_h.nb_bins = nbin;
        photon_CS.data_h.nb_mat = materials.data_h.nb_materials;
        photon_CS.nb_bins = nbin;
        photon_CS.nb_mat = materials.data_h.nb_materials;

        // Init value
        i=0; while (i < tot_elt) {            
            photon_CS.data_h.Compton_Std_CS[i] = 0.0f;
            photon_CS.data_h.Photoelectric_Std_CS[i] = 0.0f;
            photon_CS.data_h.Rayleigh_Lv_CS[i] = 0.0f;
            ++i;
        }
        i=0; while (i < (101*nbin)) { // 100 Z element starting from index 1
            photon_CS.data_h.Rayleigh_Lv_SF[i] = 0.0f;
            photon_CS.data_h.Rayleigh_Lv_xCS[i] = 0.0f;
            photon_CS.data_h.Photoelectric_Std_xCS[i] = 0.0f;
            ++i;
        }

    }
    
    // idem for e- table - TODO
    if (there_is_electron) 
    {
        electronCSTable->initialize(parameters,materials.data_h);
        electronCSTable->generateTable();
        G4cout<<"Init electrons OK "<<G4endl;
        // \DEBUG A ENLEVER 
        electronCSTable->printElectronTables("table/electronCSTable");
        
//         electron_CS
//         electron_CS.data_h = electronCSTable.get_data_h();
//         electron_CS.initialize(parameters,materials.data_h);
//         electron_CS.generateTable();
//         G4cout<<"Init electrons OK "<<G4endl;
//         // \DEBUG A ENLEVER 
//         electron_CS.printElectronTables("table2/electronCSTable");
        
    }
    // If Rayleigh scattering, load information once from G4 EM data library
    f32 *g4_ray_cs = NULL;
    f32 *g4_ray_sf = NULL;
    i8 *flag_Z = NULL;
    if (parameters.data_h.physics_list[PHOTON_RAYLEIGH]) {

        g4_ray_cs = Rayleigh_CS_Livermore_load_data();
        g4_ray_sf = Rayleigh_SF_Livermore_load_data();

        // use to flag is scatter factor are already defined for a given Z
        flag_Z = (i8*)malloc(101*sizeof(i8));
        i=0; while(i<101) {flag_Z[i]=0; ++i;}
    }
    
    // Get CS for each material, energy bin and phys effect
    ui32 imat=0;
    ui32 abs_index;
    while (imat < materials.data_h.nb_materials) {

        // for each energy bin
        i=0; while (i < nbin) {

            // absolute index to store data within the table
            abs_index = imat*nbin + i;

            // for each phys effect
            if (parameters.data_h.physics_list[PHOTON_COMPTON]) {
                photon_CS.data_h.Compton_Std_CS[abs_index] = Compton_CS_standard(materials.data_h, imat,
                                                                                 photon_CS.data_h.E_bins[i]);
            }
            if (parameters.data_h.physics_list[PHOTON_PHOTOELECTRIC]) {
                photon_CS.data_h.Photoelectric_Std_CS[abs_index] = Photoelec_CS_standard(materials.data_h, imat,
                                                                                         photon_CS.data_h.E_bins[i]);
            }
            if (parameters.data_h.physics_list[PHOTON_RAYLEIGH]) {
                photon_CS.data_h.Rayleigh_Lv_CS[abs_index] = Rayleigh_CS_Livermore(materials.data_h, g4_ray_cs,
                                                                                   imat, photon_CS.data_h.E_bins[i]);
            }

            // TODO
            // idem with Electron_CS_table

            ++i;
        } // i              

        // Special case for Photoelectric and Rayleigh where scatter factor and CS are needed for each Z
        if (parameters.data_h.physics_list[PHOTON_RAYLEIGH]) {
            ui32 iZ, Z;
            // This table compute scatter factor for each Z (only for Z which were not already defined)
            iZ=0; while (iZ < materials.data_h.nb_elements[imat]) {
                Z = materials.data_h.mixture[materials.data_h.index[imat]+iZ];

                f32 atom_num_dens = materials.data_h.atom_num_dens[materials.data_h.index[imat]+iZ];

                // If for this Z nothing was already calculated
                if (!flag_Z[Z]) {
                    flag_Z[Z] = 1;

                    // for each energy bin
                    i=0; while (i < nbin) {
                        // absolute index to store data within the table
                        abs_index = Z*nbin + i;
                        photon_CS.data_h.Rayleigh_Lv_SF[abs_index] = Rayleigh_SF_Livermore(g4_ray_sf,
                                                                                           photon_CS.data_h.E_bins[i],
                                                                                           Z);

                        photon_CS.data_h.Rayleigh_Lv_xCS[abs_index] = atom_num_dens *
                                                                      Rayleigh_CSPA_Livermore(g4_ray_cs, photon_CS.data_h.E_bins[i], Z);

                        photon_CS.data_h.Photoelectric_Std_xCS[abs_index] = atom_num_dens *
                                                                            Photoelec_CSPA_standard(photon_CS.data_h.E_bins[i], Z);

                        ++i;
                    } // i
                } // flag_Z
                ++iZ;

            } // iZ
        } // if

        ++imat;
    } // imat

    // Free mem
    free(flag_Z);
}

/*
// Print CS talbe (for debugging)
void CrossSectionsManager::print() {

    ui32 imat, iE, abs_index;


    printf("::::::::::::::::::::::::::::::::::::::::::::\n");
    printf("::::::::::::::::: Gamma ::::::::::::::::::::\n");
    printf("::::::::::::::::::::::::::::::::::::::::::::\n\n");

    printf("==== Compton Standard CS ====\n\n");

    imat=0; while (imat < photon_CS_table_h.nb_mat) {
        printf("## Material %i\n", imat);
        iE=0; while (iE < photon_CS_table_h.nb_bins) {
            abs_index = imat*photon_CS_table_h.nb_bins + iE;
            printf("E %e CS %e\n", photon_CS_table_h.E_bins[iE],
                                   photon_CS_table_h.Compton_Std_CS[abs_index]);
            ++iE;
        } // iE
        printf("\n");
        ++imat;
    } // imat
    printf("\n");

    printf("==== Photoelectric Standard CS ====\n");

    imat=0; while (imat < photon_CS_table_h.nb_mat) {
        printf("## Material %i\n", imat);
        iE=0; while (iE < photon_CS_table_h.nb_bins) {
            abs_index = imat*photon_CS_table_h.nb_bins + iE;
            printf("E %e CS %e\n", photon_CS_table_h.E_bins[iE],
                                   photon_CS_table_h.Photoelectric_Std_CS[abs_index]);
            ++iE;
        } // iE
        printf("\n");
        ++imat;
    } // imat
    printf("\n");

    imat=0; while (imat < 101) {
        printf("## Z %i\n", imat);
        iE=0; while (iE < photon_CS_table_h.nb_bins) {
            abs_index = imat*photon_CS_table_h.nb_bins + iE;
            printf("E %e CS %e\n", photon_CS_table_h.E_bins[iE],
                                   photon_CS_table_h.Photoelectric_Std_xCS[abs_index]);
            ++iE;
        } // iE
        printf("\n");
        ++imat;
    } // imat
    printf("\n");

    printf("==== Rayleigh Livermore CS ====\n");

    imat=0; while (imat < photon_CS_table_h.nb_mat) {
        printf("## Material %i\n", imat);
        iE=0; while (iE < photon_CS_table_h.nb_bins) {
            abs_index = imat*photon_CS_table_h.nb_bins + iE;
            printf("E %e CS %e\n", photon_CS_table_h.E_bins[iE],
                                   photon_CS_table_h.Rayleigh_Lv_CS[abs_index]);
            ++iE;
        } // iE
        printf("\n");
        ++imat;
    } // imat
    printf("\n");

    printf("==== Rayleigh Livermore SF ====\n");

    imat=0; while (imat < 101) {
        printf("## Z %i\n", imat);
        iE=0; while (iE < photon_CS_table_h.nb_bins) {
            abs_index = imat*photon_CS_table_h.nb_bins + iE;
            printf("E %e SF %e CS %e\n", photon_CS_table_h.E_bins[iE],
                                         photon_CS_table_h.Rayleigh_Lv_SF[abs_index],
                                         photon_CS_table_h.Rayleigh_Lv_xCS[abs_index]);
            ++iE;
        } // iE
        printf("\n");
        ++imat;
    } // imat
    printf("\n");


}
*/

// Copy CS table to the device
void CrossSections::m_copy_cs_table_cpu2gpu() {

    if(there_is_photon)
    {
        ui32 n = photon_CS.data_h.nb_bins;
        ui32 k = photon_CS.data_h.nb_mat;

        // Allocate GPU mem
        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.E_bins, n*sizeof(f32)) );

        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Compton_Std_CS, n*k*sizeof(f32)) );

        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Photoelectric_Std_CS, n*k*sizeof(f32)) );
        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Photoelectric_Std_xCS, n*101*sizeof(f32)) );

        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Rayleigh_Lv_CS, n*k*sizeof(f32)) );
        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Rayleigh_Lv_SF, n*101*sizeof(f32)) );
        HANDLE_ERROR( hipMalloc((void**) &photon_CS.data_d.Rayleigh_Lv_xCS, n*101*sizeof(f32)) );

        // Copy data to GPU
        photon_CS.data_d.nb_bins = n;
        photon_CS.data_d.nb_mat = k;
        photon_CS.data_d.E_min = photon_CS.data_h.E_min;
        photon_CS.data_d.E_max = photon_CS.data_h.E_max;

        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.E_bins, photon_CS.data_h.E_bins,
                                sizeof(f32)*n, hipMemcpyHostToDevice) );

        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Compton_Std_CS, photon_CS.data_h.Compton_Std_CS,
                                sizeof(f32)*n*k, hipMemcpyHostToDevice) );

        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Photoelectric_Std_CS, photon_CS.data_h.Photoelectric_Std_CS,
                                sizeof(f32)*n*k, hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Photoelectric_Std_xCS, photon_CS.data_h.Photoelectric_Std_xCS,
                                sizeof(f32)*n*101, hipMemcpyHostToDevice) );

        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Rayleigh_Lv_CS, photon_CS.data_h.Rayleigh_Lv_CS,
                                sizeof(f32)*n*k, hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Rayleigh_Lv_SF, photon_CS.data_h.Rayleigh_Lv_SF,
                                sizeof(f32)*n*101, hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(photon_CS.data_d.Rayleigh_Lv_xCS, photon_CS.data_h.Rayleigh_Lv_xCS,
                                sizeof(f32)*n*101, hipMemcpyHostToDevice) );

    }
    
    if (there_is_electron)
    {
    
    
    }
    
                             
}











#endif
