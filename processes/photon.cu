#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef PHOTON_CU
#define PHOTON_CU
#include "photon.cuh"


//////// Compton /////////////////////////////////////////////
// Model standard G4
//////////////////////////////////////////////////////////////

// Compton Cross Section Per Atom (Standard - Klein-Nishina)
__host__ __device__ float Compton_CSPA_standard(float E, unsigned short int Z) {
    float CrossSection = 0.0;
    if (Z<1 || E < 1e-4f) {return CrossSection;}

    float p1Z = Z*(2.7965e-23f + 1.9756e-27f*Z + -3.9178e-29f*Z*Z);
    float p2Z = Z*(-1.8300e-23f + -1.0205e-24f*Z + 6.8241e-27f*Z*Z);
    float p3Z = Z*(6.7527e-22f + -7.3913e-24f*Z + 6.0480e-27f*Z*Z);
    float p4Z = Z*(-1.9798e-21f + 2.7079e-24f*Z + 3.0274e-26f*Z*Z);
    float T0 = (Z < 1.5f)? 40.0e-3f : 15.0e-3f;
    float d1, d2, d3, d4, d5;

    d1 = fmaxf(E, T0) / 0.510998910f; 
    CrossSection = p1Z*logf(1.0f+2.0f*d1)/d1+(p2Z+p3Z*d1+p4Z*d1*d1)/(1.0f+20.0f*d1+230.0f*d1*d1+440.0f*d1*d1*d1);

    if (E < T0) {
        d1 = (T0+1.0e-3f) / 0.510998910f;
        d2 = p1Z*logf(1.0f+2.0f*d1)/d1+(p2Z+p3Z*d1+p4Z*d1*d1)/(1.0f+20.0f*d1+230.0f*d1*d1+440.0f*d1*d1*d1);
        d3 = (-T0 * (d2 - CrossSection)) / (CrossSection*1.0e-3f);
        d4 = (Z > 1.5f)? 0.375f-0.0556f*logf(Z) : 0.15f;
        d5 = logf(E / T0);
        CrossSection *= expf(-d5 * (d3 + d4*d5));
    }
    
    return CrossSection;
}

// Compute the total Compton cross section for a given material
__host__ __device__ float Compton_CS_standard(MaterialsTable materials, unsigned short int mat, float E) {
    float CS = 0.0f;
    int i;
    int index = materials.index[mat];
    // Model standard
    for (i = 0; i < materials.nb_elements[mat]; ++i) {
        CS += (materials.atom_num_dens[index+i] * 
               Compton_CSPA_standard(E, materials.mixture[index+i]));
    }
    return CS;
}


// Compton Scatter (Standard - Klein-Nishina) with secondary (e-)
__host__ __device__ SecParticle Compton_SampleSecondaries_standard(ParticleStack particles,
                                                                   float cutE,
                                                                   unsigned int id,
                                                                   GlobalSimulationParameters parameters) {

    float gamE0 = particles.E[id];
    float E0 = gamE0 / 0.510998910f;
    float3 gamDir0 = make_float3(particles.dx[id], particles.dy[id], particles.dz[id]);

    // sample the energy rate pf the scattered gamma

    float epszero = 1.0f / (1.0f + 2.0f * E0);
    float eps02 = epszero*epszero;
    float a1 = -logf(epszero);
    float a2 = a1 / (a1 + 0.5f*(1.0f-eps02));

    float greject, onecost, eps, eps2, sint2, cosTheta, sinTheta, phi;
    do {
        if (a2 > JKISS32(particles, id)) {
            eps = expf(-a1 * JKISS32(particles, id));
            eps2 = eps*eps;
        } else {
            eps2 = eps02 + (1.0f - eps02) * JKISS32(particles, id);
            eps = sqrt(eps2);
        }
        onecost = (1.0f - eps) / (eps * E0);
        sint2 = onecost * (2.0f - onecost);
        greject = 1.0f - eps * sint2 / (1.0f + eps2);
    } while (greject < JKISS32(particles, id));

    // scattered gamma angles

    if (sint2 < 0.0f) {sint2 = 0.0f;}
    cosTheta = 1.0f - onecost;
    sinTheta = sqrt(sint2);
    phi = JKISS32(particles, id) * gpu_twopi;

    // update the scattered gamma

    float3 gamDir1 = make_float3(sinTheta*cosf(phi), sinTheta*sinf(phi), cosTheta);
    gamDir1 = rotateUz(gamDir1, gamDir0);

    particles.dx[id] = gamDir1.x;
    particles.dy[id] = gamDir1.y;
    particles.dz[id] = gamDir1.z;
    float gamE1  = gamE0 * eps;
    if (gamE1 > 1.0e-06f) {particles.E[id] = gamE1;}
    else {
        particles.endsimu[id] = PARTICLE_DEAD;  // absorbed this particle
        particles.E[id] = gamE1;                // Local energy deposit
    }

    // kinematic of the scattered electron

    SecParticle electron;
    electron.pname = ELECTRON;
    electron.E = gamE0 - gamE1; // eKinE
    electron.dir = make_float3(0.0, 0.0, 0.0);
    electron.endsimu = PARTICLE_DEAD;

    //          DBL_MIN             cut production
    if (electron.E > 1.0e-38f && electron.E > cutE && parameters.secondaries_list[ELECTRON]) {
        electron.dir = f3_sub(f3_scale(gamDir0, gamE0), f3_scale(gamDir1, gamE1));
        electron.dir = f3_unit(electron.dir);
        electron.endsimu = PARTICLE_ALIVE;
    }

    //return e-
    return electron;
}

//////// Photoelectric ////////////////////////////////////////
// Model standard G4
//////////////////////////////////////////////////////////////

// PhotoElectric Cross Section Per Atom (Standard)
__host__ __device__ float PhotoElec_CSPA_standard(float E, unsigned short int Z) {
    // from Sandia data, the same for all Z
    float Emin = fmax(PhotoElec_std_IonizationPotentials(Z)*1e-6f, 0.01e-3f);
    if (E < Emin) {return 0.0f;}
    
    int start = PhotoElec_std_CumulIntervals(Z-1);
    int stop = start + PhotoElec_std_NbIntervals(Z);
    int pos=stop;
    while (E < PhotoElec_std_SandiaTable(pos, 0)*1.0e-3f){--pos;}
    float AoverAvo = 0.0103642688246f * ( (float)Z / PhotoElec_std_ZtoAratio(Z) );
    float rE = 1.0f / E;
    float rE2 = rE*rE;

    return rE * PhotoElec_std_SandiaTable(pos, 1) * AoverAvo * 0.160217648e-22f
        + rE2 * PhotoElec_std_SandiaTable(pos, 2) * AoverAvo * 0.160217648e-25f
        + rE * rE2 * PhotoElec_std_SandiaTable(pos, 3) * AoverAvo * 0.160217648e-28f
        + rE2 * rE2 * PhotoElec_std_SandiaTable(pos, 4) * AoverAvo * 0.160217648e-31f;
}

// Compute the total Compton cross section for a given material
__host__ __device__ float PhotoElec_CS_standard(MaterialsTable materials,
                                                unsigned short int mat, float E) {
    float CS = 0.0f;
    int i;
    int index = materials.index[mat];
    // Model standard
    for (i = 0; i < materials.nb_elements[mat]; ++i) {
        CS += (materials.atom_num_dens[index+i] * 
               PhotoElec_CSPA_standard(E, materials.mixture[index+i]));
    }
    return CS;
}

// Compute Theta distribution of the emitted electron, with respect to the incident Gamma
// The Sauter-Gavrila distribution for the K-shell is used
__host__ __device__ float PhotoElec_ElecCosThetaDistribution(ParticleStack part,
                                                             unsigned int id,
                                                             float kineEnergy) {
    float costeta = 1.0f;
    float gamma = kineEnergy * 1.9569513367f + 1.0f;  // 1/electron_mass_c2
    if (gamma > 5.0f) {return costeta;}
    float beta = sqrtf(gamma*gamma - 1.0f) / gamma;
    float b    = 0.5f*gamma*(gamma - 1.0f)*(gamma - 2.0f);

    float rndm, term, greject, grejsup;
    if (gamma < 2.0f) {grejsup = gamma*gamma*(1.0f + b - beta*b);}
    else              {grejsup = gamma*gamma*(1.0f + b + beta*b);}

    do {
        rndm = 1.0f - 2.0f*JKISS32(part,id);
        costeta = (rndm + beta) / (rndm*beta + 1.0f);
        term = 1.0f - beta*costeta;
        greject = ((1.0f - costeta*costeta)*(1.0f + b*term)) / (term*term);
    } while(greject < JKISS32(part,id)*grejsup);

    return costeta;
}

// PhotoElectric effect (standard) with secondary (e-)
__host__ __device__ SecParticle PhotoElec_SampleSecondaries_standard(ParticleStack particles,
                                                                     MaterialsTable mat,
                                                                     float cutE,
                                                                     unsigned short int matindex,
                                                                     unsigned int id,
                                                                     GlobalSimulationParameters parameters) {

    // Kill the photon without mercy
    particles.endsimu[id] = PARTICLE_DEAD;

    // Electron allocation
    SecParticle electron;
    electron.pname = ELECTRON;
    electron.endsimu = PARTICLE_DEAD;
    electron.E = 0.0f;

    // If no secondary required return a stillborn electron
    if (parameters.secondaries_list[ELECTRON] == DISABLED) return electron;

    //// Photo electron

    float energy = particles.E[id];
    //float cutE = mat.electron_cut_energy[matindex]; // TODO - JB
    float3 PhotonDirection = make_float3(particles.dx[id], particles.dy[id], particles.dz[id]);

    //*******************************************************************************
    // TODO - build a table of xSection into MaterialTable - JB

    // Select randomly one element constituing the material
    unsigned int n = mat.nb_elements[matindex]-1;
    unsigned int index = mat.index[matindex];
    unsigned int Z = mat.mixture[index+n];
    unsigned int i = 0;
    if (n > 0) {
        float x = JKISS32(particles,id) * PhotoElec_CS_standard(mat, matindex, energy);
        float xsec = 0.0f;
        while (i < n) {
            xsec += mat.atom_num_dens[index+i] * PhotoElec_CSPA_standard(energy, mat.mixture[index+i]);
            if (x <= xsec) {
                Z = mat.mixture[index+i];
                break;
            }
            ++i;
        }
    }
    //*******************************************************************************

    // Select atomic shell
    unsigned short int nShells = atom_NumberOfShells(Z);
    index = atom_IndexOfShells(Z);
    float bindingEnergy = atom_BindingEnergies(index) * eV; //1.0e-06f; // in eV
    i=0; while (i < nShells && energy < bindingEnergy) {       
        bindingEnergy = atom_BindingEnergies(index + i)* eV; //1.0e-06f; // in ev
        ++i;
    }
        
    // no shell available return stillborn electron
    if (i == nShells) {return electron;}
    float ElecKineEnergy = energy - bindingEnergy;

    float cosTeta = 0.0f;
    //                   1 eV                         cut production
    if (ElecKineEnergy > 1.0e-06f && ElecKineEnergy > cutE) {
        // direction of the photo electron
        cosTeta = PhotoElec_ElecCosThetaDistribution(particles, id, ElecKineEnergy);
        float sinTeta = sqrtf(1.0f - cosTeta*cosTeta);
        float Phi = gpu_twopi * JKISS32(particles, id);
        float3 ElecDirection = make_float3(sinTeta*cos(Phi), sinTeta*sin(Phi), cosTeta);
        ElecDirection = rotateUz(ElecDirection, PhotonDirection);
        // Configure the new electron
        electron.dir.x = ElecDirection.x;
        electron.dir.y = ElecDirection.y;
        electron.dir.z = ElecDirection.z;
        electron.E = ElecKineEnergy;
        electron.endsimu = PARTICLE_ALIVE;
        // gamma will depose energy given by the binding energy
        particles.E[id] = bindingEnergy;
    }
    
    // Return electron (dead or alive)
    return electron;

}


#endif
