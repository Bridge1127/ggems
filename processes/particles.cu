// GGEMS Copyright (C) 2015

/*!
 * \file particles.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 18 novembre 2015
 *
 *
 *
 */

#ifndef PARTICLES_CU
#define PARTICLES_CU
#include "particles.cuh"

/*
//// HistoryBuilder class ////////////////////////////////////////////////////

HistoryBuilder::HistoryBuilder() {
    current_particle_id = 0;
}

// Create a new particle track in the history
void HistoryBuilder::cpu_new_particle_track(ui32 a_pname) {

    // If need record the first position for the tracking history
    if (current_particle_id < max_nb_particles) {

        // new particle
        pname.push_back(a_pname);
        nb_steps.push_back(0);

        std::vector<OneParticleStep> NewParticleTrack;
        history_data.push_back(NewParticleTrack);

        current_particle_id++;

    }
}

// Reacord a step in a history track
void HistoryBuilder::cpu_record_a_step(ParticleStack particles, ui32 id_part) {

    // Absolute index is need to store particle history over different iteration
    ui32 abs_id_part = cur_iter*stack_size + id_part;

    OneParticleStep astep;

    astep.pos.x = particles.px[id_part];
    astep.pos.y = particles.py[id_part];
    astep.pos.z = particles.pz[id_part];
    astep.dir.x = particles.dx[id_part];
    astep.dir.y = particles.dy[id_part];
    astep.dir.z = particles.dz[id_part];
    astep.E = particles.E[id_part];

    // Add this step
    history_data[abs_id_part].push_back(astep);
    nb_steps[abs_id_part]++;

}
*/

//// ParticleManager class ///////////////////////////////////////////////////

ParticleManager::ParticleManager()
{
    particles.size = 0;
}

// Init stack
void ParticleManager::initialize ( GlobalSimulationParameters params )
{
    particles.size = params.data_h.size_of_particles_batch;
    particles.data_h.size = params.data_h.size_of_particles_batch;    

    m_params = params;

    // Check if everything was set properly
    if ( !m_check_mandatory() )
    {
        print_error ( "Stack allocation, is stack size set to zero?!" );
        exit_simulation();
    }

    // CPU allocation
    m_cpu_malloc_stack();

//    // Init seed
//    cpu_prng_init( particles.data_h.prng, particles.size, params.data_h.seed );

    m_cpu_init_stack_seed ( params.data_h.seed );

    // Init seeds
    if ( params.data_h.device_target == GPU_DEVICE )
    {
        // GPU allocation
        m_gpu_malloc_stack();

//        // Init seed on GPU side
//        gpu_prng_init( particles.data_d.prng, particles.size,
//                       params.data_h.seed, params.data_h.gpu_block_size );
/*
        srand(m_params.data_h.seed);
        prng_states *hostStates;
        hostStates = (prng_states*)malloc(particles.size * sizeof(prng_states));
        for (ui32 i=0; i<particles.size; i++)
        {
            prng_states aState;
            aState.state_1 = rand();
            aState.state_2 = rand();
            aState.state_3 = rand();
            aState.state_4 = rand();
            aState.state_5 = 0;

            hostStates[i] = aState;
        }
        hipMemcpy(particles.data_d.prng, hostStates, particles.size * sizeof(prng_states), hipMemcpyHostToDevice);
*/
        // Copy data to the GPU    - FIXME - seed on CPU side is not used - JB
        m_copy_seed_cpu2gpu();
    }

}

// Check mandatory
bool ParticleManager::m_check_mandatory()
{
    if ( particles.size == 0 ) return false;
    else return true;
}

// Memory allocation for this stack
void ParticleManager::m_cpu_malloc_stack()
{

    particles.data_h.E = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.dx = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.dy = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.dz = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.px = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.py = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.pz = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.tof = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );

    // scatter_order
    particles.data_h.scatter_order = (ui32*)malloc( particles.size * sizeof( ui32 ) );

    particles.data_h.prng_state_1 = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );
    particles.data_h.prng_state_2 = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );
    particles.data_h.prng_state_3 = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );
    particles.data_h.prng_state_4 = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );
    particles.data_h.prng_state_5 = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );

    //particles.data_h.prng = ( prng_states* ) malloc ( particles.size * sizeof ( prng_states ) );

    particles.data_h.geometry_id = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );
    particles.data_h.E_index = ( ui32* ) malloc ( particles.size * sizeof ( ui32 ) );

    particles.data_h.next_interaction_distance = ( f32* ) malloc ( particles.size * sizeof ( f32 ) );
    particles.data_h.next_discrete_process = ( ui8* ) malloc ( particles.size * sizeof ( ui8 ) );

    particles.data_h.endsimu = ( ui8* ) malloc ( particles.size * sizeof ( ui8 ) );
    particles.data_h.level = ( ui8* ) malloc ( particles.size * sizeof ( ui8 ) );
    particles.data_h.pname = ( ui8* ) malloc ( particles.size * sizeof ( ui8 ) );


    particles.data_h.sec_E =     ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_dx =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_dy =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_dz =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_px =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_py =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_pz =    ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_tof =   ( f32* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( f32 ) );
    particles.data_h.sec_pname = ( ui8* ) malloc ( particles.size * m_params.data_h.nb_of_secondaries * sizeof ( ui8 ) );

}

/*
void ParticleManager::m_cpu_free_stack() {

    free(stack_h.E);
    free(stack_h.dx);
    free(stack_h.dy);
    free(stack_h.dz);
    free(stack_h.px);
    free(stack_h.py);
    free(stack_h.pz);
    free(stack_h.tof);

    free(stack_h.prng_state_1);
    free(stack_h.prng_state_2);
    free(stack_h.prng_state_3);
    free(stack_h.prng_state_4);
    free(stack_h.prng_state_5);

    free(stack_h.geometry_id);

    free(stack_h.endsimu);
    free(stack_h.level);
    free(stack_h.pname);
}
*/

void ParticleManager::m_gpu_malloc_stack()
{    
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.E, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.dx, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.dy, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.dz, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.px, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.py, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.pz, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.tof, particles.size*sizeof ( f32 ) ) );

    // scatter_order
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.scatter_order, particles.size*sizeof ( ui32 ) ) );

    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng_state_1, particles.size*sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng_state_2, particles.size*sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng_state_3, particles.size*sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng_state_4, particles.size*sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng_state_5, particles.size*sizeof ( ui32 ) ) );


    //HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.prng, particles.size*sizeof ( prng_states ) ) );

    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.geometry_id, particles.size*sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.E_index, particles.size*sizeof ( ui32 ) ) );

    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.next_interaction_distance, particles.size*sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.next_discrete_process, particles.size*sizeof ( ui8 ) ) );

    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.endsimu, particles.size*sizeof ( ui8 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.level, particles.size*sizeof ( ui8 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.pname, particles.size*sizeof ( ui8 ) ) );

    particles.data_d.size = particles.data_h.size;

    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_E,     particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_dx,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_dy,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_dz,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_px,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_py,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_pz,    particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_tof,   particles.size * m_params.data_h.nb_of_secondaries *sizeof ( f32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &particles.data_d.sec_pname, particles.size * m_params.data_h.nb_of_secondaries *sizeof ( ui8 ) ) );

}


// Init particle seeds with the main seed
void ParticleManager::m_cpu_init_stack_seed ( ui32 seed )
{

    srand ( seed );
    ui32 i=0;
    while ( i<particles.size )
    {
        // init random seed
        particles.data_h.prng_state_1[i] = rand();
        particles.data_h.prng_state_2[i] = rand();
        particles.data_h.prng_state_3[i] = rand();
        particles.data_h.prng_state_4[i] = rand();
        particles.data_h.prng_state_5[i] = 0;      // carry

//         printf("%d %d %d %d %d \n",particles.data_h.prng_state_1[i],particles.data_h.prng_state_2[i],particles.data_h.prng_state_3[i],particles.data_h.prng_state_4[i],particles.data_h.prng_state_5[i]);

        ++i;
    }
}



void ParticleManager::m_copy_seed_cpu2gpu()
{

    // We consider that the CPU stack was previously initialized with seed
    HANDLE_ERROR ( hipMemcpy ( particles.data_d.prng_state_1, particles.data_h.prng_state_1,
                                sizeof ( ui32 ) *particles.size, hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( particles.data_d.prng_state_2, particles.data_h.prng_state_2,
                                sizeof ( ui32 ) *particles.size, hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( particles.data_d.prng_state_3, particles.data_h.prng_state_3,
                                sizeof ( ui32 ) *particles.size, hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( particles.data_d.prng_state_4, particles.data_h.prng_state_4,
                                sizeof ( ui32 ) *particles.size, hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( particles.data_d.prng_state_5, particles.data_h.prng_state_5,
                                sizeof ( ui32 ) *particles.size, hipMemcpyHostToDevice ) );

}


void ParticleManager::copy_gpu2cpu( Particles part )
{
    HANDLE_ERROR ( hipMemcpy ( part.data_h.E, part.data_d.E, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );

    HANDLE_ERROR ( hipMemcpy ( part.data_h.px, part.data_d.px, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( part.data_h.py, part.data_d.py, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( part.data_h.pz, part.data_d.pz, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );

    HANDLE_ERROR ( hipMemcpy ( part.data_h.dx, part.data_d.dx, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( part.data_h.dy, part.data_d.dy, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( part.data_h.dz, part.data_d.dz, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );

    HANDLE_ERROR ( hipMemcpy ( part.data_h.tof, part.data_d.tof, sizeof ( f32 ) *part.size, hipMemcpyDeviceToHost ) );

    HANDLE_ERROR ( hipMemcpy ( part.data_h.endsimu, part.data_d.endsimu, sizeof ( ui8 ) *part.size, hipMemcpyDeviceToHost ) );
}

void ParticleManager::print_stack( Particles part )
{
    std::vector< std::string > status;
    status.push_back("Alive");
    status.push_back("Dead");
    status.push_back("Freeze");

    ui32 i = 0; while ( i < part.size ) {
        printf("%i - E %f - p %f %f %f - d %f %f %f - tof %f - Status %s\n", i, part.data_h.E[i], part.data_h.px[i],
               part.data_h.py[i], part.data_h.pz[i], part.data_h.dx[i], part.data_h.dy[i], part.data_h.dz[i], part.data_h.tof[i],
               status[ part.data_h.endsimu[ i ] ].c_str() );
        ++i;
    }

}
















#endif
