#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2017

/*!
 * \file electron.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.2
 * \date 18 novembre 2015
 *
 * v0.2: JB - Change all structs and remove CPU exec
 *
 */

#ifndef ELECTRON_CU
#define ELECTRON_CU

#include "electron.cuh"

/// ElectronIonisation ///////////////////////////////////////////////////

// eIon de/dx
f32 ElectronIonisation_DEDX( const MaterialsData *h_materials, f32 Ekine, ui8 mat_id )
{
    f32 Dedx = 0.;
    f32 th = .25*sqrt ( h_materials->nb_electrons_per_vol[mat_id] / h_materials->nb_atoms_per_vol[mat_id] ) *keV;
    f32 lowLimit = .2*keV;
    f32 tmax, tkin;
    f32 eexc, eexc2, d, x;
    f32 tau, gamma, gamma2, beta2, bg2;

    tkin = Ekine;
    if ( Ekine < th ) tkin = th;
    tmax = tkin * .5;
    tau = tkin / electron_mass_c2;
    gamma = tau + 1.;
    gamma2 = gamma * gamma;
    beta2 = 1. - 1. / gamma2;
    bg2 = beta2 * gamma2;
    eexc = h_materials->electron_mean_excitation_energy[mat_id] / electron_mass_c2;
    eexc2 = eexc * eexc;
    d = std::min ( h_materials->electron_energy_cut[mat_id], tmax );
    d /= electron_mass_c2;

    Dedx = log ( 2.* ( tau+2. ) /eexc2 )-1.-beta2+log ( ( tau-d ) *d ) +tau/ ( tau-d )
            + ( .5*d*d+ ( 2.*tau+1. ) *log ( 1.-d/tau ) ) /gamma2;

    x = log ( bg2 ) / ( 2.*log ( 10. ) );

    //// DensCorrection
    f32 dens_correction = 0.0;

    if ( x < h_materials->fX0[mat_id] )
    {
        if ( h_materials->fD0[mat_id]>0. )
        {
            dens_correction = h_materials->fD0[mat_id] * pow ( 10.,2.* ( x-h_materials->fX0[mat_id] ) );
        }
    }
    else if ( x >= h_materials->fX1[mat_id] )
    {
        dens_correction = 2.*log ( 10. ) *x-h_materials->fC[mat_id];
    }
    else
    {
        dens_correction = 2.*log ( 10. ) * x-h_materials->fC[mat_id] + h_materials->fA[mat_id]
                *pow ( h_materials->fX1[mat_id]-x, h_materials->fM[mat_id] );
    }

    Dedx -= dens_correction;
    ////

    Dedx *= twopi_mc2_rcl2 * h_materials->nb_electrons_per_vol[mat_id]/beta2;

    if ( Dedx < 0. ) Dedx = 0.;

    if ( Ekine < th )
    {
        if ( Ekine >= lowLimit )
            Dedx *= sqrt ( tkin/Ekine );
        else
            Dedx *= sqrt ( tkin*Ekine ) /lowLimit;
    }

    //printf("E %e dE/dx %e     cut %e\n", Ekine, Dedx, h_materials->electron_energy_cut[mat_id]);

    return Dedx;

}

// Cross Section Per Electron
f32 ElectronIonisation_CSPE(f32 Ekine, f32 Ecut, ui16 Z)
{
    f32  Cross=0.;
    f32  tmax=std::min ( 1.*GeV,Ekine*.5 );
    f32  xmin,xmax,gamma,gamma2,beta2,g;

    if ( Ecut < tmax )
    {
        xmin = Ecut/Ekine;
        xmax = tmax/Ekine;
        gamma = Ekine/electron_mass_c2+1.;
        gamma2 = gamma*gamma;
        beta2 = 1.-1./gamma2;
        g = ( 2.*gamma-1. ) /gamma2;
        Cross = ( ( xmax-xmin ) * ( 1.-g+1./ ( xmin*xmax ) +1./ ( ( 1.-xmin ) * ( 1.-xmax ) ) )
                 -g*std::log ( xmax* ( 1.-xmin ) / ( xmin* ( 1.-xmax ) ) ) ) /beta2;

        Cross *= twopi_mc2_rcl2/Ekine;
    }

    return  Cross * ( f32 ) Z;
}

// Total cross section
f32 ElectronIonisation_CS(const MaterialsData *h_materials, f32 Ekine, ui16 mat_id)
{
    f32  CrossTotale=0.;
    ui32 index = h_materials->index[mat_id];

    for ( ui32 i=0; i<h_materials->nb_elements[mat_id]; ++i )
    {
        CrossTotale += ( h_materials->atom_num_dens[index+i] *
                ElectronIonisation_CSPE( Ekine, h_materials->electron_energy_cut[mat_id], h_materials->mixture[index+i] ) );
    }

    return  CrossTotale;
}

/////////////////////////////////////////////////////////////////////////////////////////////


/// Bremsstrahlung //////////////////////////////////////////////////////////////////////////



/// GPU
// Constant parameters for bremstrahlung table
__constant__ f32 gpu_ZZ[ 8 ] = { 2., 4., 6., 14., 26., 50., 82., 92. };
__constant__ f32 gpu_coefloss[ 8 ][ 11 ] =
{
    { .98916, .47564, -.2505, -.45186, .14462, .21307, -.013738, -.045689, -.0042914, .0034429, .00064189 },
    { 1.0626, .37662, -.23646, -.45188, .14295, .22906, -.011041, -.051398, -.0055123, .0039919, .00078003 },
    { 1.0954, .315, -.24011, -.43849, .15017, .23001, -.012846, -.052555, -.0055114, .0041283, .00080318 },
    { 1.1649, .18976, -.24972, -.30124, .1555, .13565, -.024765, -.027047, -.00059821, .0019373, .00027647 },
    { 1.2261, .14272, -.25672, -.28407, .13874, .13586, -.020562, -.026722, -.00089557, .0018665, .00026981 },
    { 1.3147, .020049, -.35543, -.13927, .17666, .073746, -.036076, -.013407, .0025727, .00084005, -1.4082e-05 },
    { 1.3986, -.10586, -.49187, -.0048846, .23621, .031652, -.052938, -.0076639, .0048181, .00056486, -.00011995 },
    { 1.4217, -.116, -.55497, -.044075, .27506, .081364, -.058143, -.023402, .0031322, .0020201, .00017519 }
};

__constant__ f32 gpu_coefsig[ 8 ][ 11 ] =
{
    { .4638, .37748, .32249, -.060362, -.065004, -.033457, -.004583, .011954, .0030404, -.0010077, -.00028131},
    { .50008, .33483, .34364, -.086262, -.055361, -.028168, -.0056172, .011129, .0027528, -.00092265, -.00024348},
    { .51587, .31095, .34996, -.11623, -.056167, -.0087154, .00053943, .0054092, .00077685, -.00039635, -6.7818e-05},
    { .55058, .25629, .35854, -.080656, -.054308, -.049933, -.00064246, .016597, .0021789, -.001327, -.00025983},
    { .5791, .26152, .38953, -.17104, -.099172, .024596, .023718, -.0039205, -.0036658, .00041749, .00023408},
    { .62085, .27045, .39073, -.37916, -.18878, .23905, .095028, -.068744, -.023809, .0062408, .0020407},
    { .66053, .24513, .35404, -.47275, -.22837, .35647, .13203, -.1049, -.034851, .0095046, .0030535},
    { .67143, .23079, .32256, -.46248, -.20013, .3506, .11779, -.1024, -.032013, .0092279, .0028592}
};


/// CPU
// Constant parameters for bremstrahlung table
const f32 cpu_ZZ[ 8 ] = { 2., 4., 6., 14., 26., 50., 82., 92. };
const f32 cpu_coefloss[ 8 ][ 11 ] =
{
    { .98916, .47564, -.2505, -.45186, .14462, .21307, -.013738, -.045689, -.0042914, .0034429, .00064189 },
    { 1.0626, .37662, -.23646, -.45188, .14295, .22906, -.011041, -.051398, -.0055123, .0039919, .00078003 },
    { 1.0954, .315, -.24011, -.43849, .15017, .23001, -.012846, -.052555, -.0055114, .0041283, .00080318 },
    { 1.1649, .18976, -.24972, -.30124, .1555, .13565, -.024765, -.027047, -.00059821, .0019373, .00027647 },
    { 1.2261, .14272, -.25672, -.28407, .13874, .13586, -.020562, -.026722, -.00089557, .0018665, .00026981 },
    { 1.3147, .020049, -.35543, -.13927, .17666, .073746, -.036076, -.013407, .0025727, .00084005, -1.4082e-05 },
    { 1.3986, -.10586, -.49187, -.0048846, .23621, .031652, -.052938, -.0076639, .0048181, .00056486, -.00011995 },
    { 1.4217, -.116, -.55497, -.044075, .27506, .081364, -.058143, -.023402, .0031322, .0020201, .00017519 }
};

#ifndef __CUDA_ARCH__
const f32 cpu_coefsig[ 8 ][ 11 ] =
{
    { .4638, .37748, .32249, -.060362, -.065004, -.033457, -.004583, .011954, .0030404, -.0010077, -.00028131},
    { .50008, .33483, .34364, -.086262, -.055361, -.028168, -.0056172, .011129, .0027528, -.00092265, -.00024348},
    { .51587, .31095, .34996, -.11623, -.056167, -.0087154, .00053943, .0054092, .00077685, -.00039635, -6.7818e-05},
    { .55058, .25629, .35854, -.080656, -.054308, -.049933, -.00064246, .016597, .0021789, -.001327, -.00025983},
    { .5791, .26152, .38953, -.17104, -.099172, .024596, .023718, -.0039205, -.0036658, .00041749, .00023408},
    { .62085, .27045, .39073, -.37916, -.18878, .23905, .095028, -.068744, -.023809, .0062408, .0020407},
    { .66053, .24513, .35404, -.47275, -.22837, .35647, .13203, -.1049, -.034851, .0095046, .0030535},
    { .67143, .23079, .32256, -.46248, -.20013, .3506, .11779, -.1024, -.032013, .0092279, .0028592}
};
#endif

__host__ __device__ f32 ZZ(ui8 pos)
{
#ifdef __CUDA_ARCH__
    return gpu_ZZ[ pos ];
#else
    return cpu_ZZ[ pos ];
#endif
}

__host__ __device__ f32 coefsig( ui8 i, ui8 j )
{
#ifdef __CUDA_ARCH__
    return gpu_coefsig[ i ][ j ];
#else
    return cpu_coefsig[ i ][ j ];
#endif
}

f32 ElectronBremmsstrahlung_loss ( f32 Z, f32 T, f32 Cut )
{
    ui32   i, j;
    ui32   NZ=8, Nloss=11, iz=0;
    f32    Loss;
    f32    dz, xx, yy, fl, E;
    f32    aaa=.414, bbb=.345, ccc=.460, delz=1.e6;
    f32    beta=1.0, ksi=2.0, clossh=.254, closslow=1./3., alosslow=1.;
    f32    Tlim=10.*MeV, xlim=1.2;

    for ( i=0; i<NZ; i++ )
    {
        dz =fabs ( Z-cpu_ZZ[i] );
        if ( dz<delz )
        {
            iz=i;
            delz=dz;
        }
    }
    xx = log10 ( T );
    fl = 1.;
    if ( xx<=xlim )
    {
        xx /= xlim;
        yy = 1.;
        fl = 0.;
        for ( j=0; j<Nloss; j++ )
        {
            fl += yy+cpu_coefloss[iz][j];
            yy *= xx;
        }
        if ( fl < .00001 )
            fl = .00001;
        else if ( fl > 1. )
            fl = 1.;
    }

    E = T+electron_mass_c2;
    Loss = Z* ( Z+ksi ) *E*E/ ( T+E ) *exp ( beta*log ( Cut/T ) ) * ( 2.-clossh*exp ( log ( Z ) /4. ) );
    if ( T<=Tlim )
        Loss /= exp ( closslow*log ( Tlim/T ) );
    if ( T<=Cut )
        Loss *= exp ( alosslow*log ( T/Cut ) );
    Loss *= ( aaa+bbb*T/Tlim ) / ( 1.+ccc*T/Tlim );
    Loss *=fl;
    Loss /=N_avogadro;

    return  Loss;
}

f32 ElectronBremsstrahlung_DEDX(const MaterialsData *h_materials, f32 Ekine, ui8 mat_id)
{

    ui32 i, n, nn, nmax;
    f32 Dedx;
    f32 totalEnergy, Z, natom, kp2, kmin, kmax, floss;
    f32 vmin,vmax,u,fac,c,v,dv;
    f32 thigh = 100.*GeV;
    f32 cut = std::min ( h_materials->photon_energy_cut[mat_id], Ekine );
    f32  /*rate,*/loss;
    //     f32  factorHigh=36./ ( 1450.*GeV );
    //     f32  coef1=-.5;
    //     f32  coef2=2./9.;
    f32 lowKinEnergy = 0.*eV;
    f32 highKinEnergy = 1.*GeV;
    f32 probsup = 1.;
    f32 MigdalConstant = elec_radius*hbarc*hbarc*4.*pi/ ( electron_mass_c2*electron_mass_c2 );

    totalEnergy = Ekine+electron_mass_c2;
    Dedx = 0.;

    if ( Ekine<lowKinEnergy ) return  0.;

    for ( i=0; i<h_materials->nb_elements[mat_id]; ++i ) // Check in each elt
    {
        int indexelt = i + h_materials->index[mat_id];
        Z = h_materials->mixture[indexelt];
        natom = h_materials->atom_num_dens[indexelt] / h_materials->nb_atoms_per_vol[mat_id];

        if ( Ekine<=thigh ) loss = ElectronBremmsstrahlung_loss ( Z, Ekine, cut );

        loss *= natom;
        kp2 = MigdalConstant * totalEnergy*totalEnergy * h_materials->nb_electrons_per_vol[mat_id];

        kmin = 1.*eV;
        kmax = cut;
        if ( kmax>kmin )
        {
            floss = 0.;
            nmax = 100;
            vmin =log ( kmin );
            vmax =log ( kmax );
            nn= ( int ) ( nmax* ( vmax-vmin ) / ( log ( highKinEnergy )-vmin ) ) ;
            if ( nn>0 )
            {
                dv = ( vmax-vmin ) /nn;
                v = vmin-dv;
                for ( n=0; n<=nn; n++ )
                {
                    v += dv;
                    u =exp ( v );
                    //fac=u*SupressionFunction(material,Ekine,u);   //LPM flag off
                    fac = u*1.;
                    fac *= probsup* ( u*u/ ( u*u+kp2 ) ) +1.-probsup;
                    if ( ( n==0 ) || ( n==nn ) )
                        c=.5;
                    else
                        c=1.;
                    fac*=c;
                    floss+=fac ;
                }
                floss *= dv/ ( kmax-kmin );
            }
            else
                floss=1.;
            if ( floss>1. )
                floss=1.;
            loss*=floss;
        }
        Dedx += loss;
    }

    if ( Dedx<0. ) Dedx=0.;

    Dedx *= h_materials->nb_atoms_per_vol[mat_id];

    return Dedx;  // I removed *mm2 => *1 - JB

}


__host__ __device__ f32 ElectronBremmsstrahlung_CSPA( f32 Z, f32 cut, f32 Ekine )
{
    i32 i,j,iz = 0, NZ = 8, Nsig = 11;
    f32 Cross = 0.;
    f32 ksi = 2., alfa = 1.;
    f32 csigh = .127, csiglow = .25, asiglow = .02*MeV;
    f32 Tlim = 10.*MeV;
    f32 xlim = 1.2, delz = 1.E6, absdelz;
    f32 xx, fs;

    if ( Ekine<1.*keV || Ekine<cut )
        return  Cross;

    for ( i=0; i<NZ; i++ )
    {
        absdelz = fabs ( Z-ZZ( i ) );
        if ( absdelz < delz )
        {
            iz = i;
            delz = absdelz;
        }
    }

    xx = log10f ( Ekine );
    fs = 1.;
    if ( xx <= xlim )
    {
        fs = coefsig( iz, Nsig-1 );
        for ( j = Nsig-2; j>=0; j-- )
            fs = fs*xx + coefsig( iz, j );
        if ( fs < 0. )
            fs = 0.;
    }
    Cross = Z* ( Z+ksi ) * ( 1.-csigh*expf ( logf ( Z ) /4. ) ) *powf ( logf ( Ekine/cut ), alfa );

    if ( Ekine <= Tlim )
        Cross *= expf ( csiglow*logf ( Tlim/Ekine ) ) * ( 1.+asiglow/ ( sqrtf ( Z ) *Ekine ) );
    Cross *= fs/N_avogadro;
    if ( Cross<0. )
        Cross=0.;

    return  Cross;
}


//                                                                                           v-- CS table max energy - JB
__host__ __device__ f32 ElectronBremmsstrahlung_CS(const MaterialsData *h_materials, f32 Ekine, f32 max_E, ui8 mat_id )
{
    i32 i, n, nn, nmax = 100;
    f32 Cross = 0.;
    f32 kmax, kmin, vmin, vmax, totalEnergy, kp2;
    f32 u, fac, c, v, dv, y;
    f32 tmax = fmin ( max_E, Ekine );
    f32 cut = fmax ( h_materials->photon_energy_cut[mat_id], 0.0001f ); // 0.1 keV
    if ( cut >= tmax ) return Cross;

    f32 fsig = 0.;
    f32 highKinEnergy = 1.*GeV;
    f32 probsup = 1.;
    f32 MigdalConstant = elec_radius*hbarc*hbarc*4.*pi / ( electron_mass_c2*electron_mass_c2 );

    ui32 index = h_materials->index[mat_id];

    for ( i=0; i<h_materials->nb_elements[mat_id]; i++ )
    {
        Cross += h_materials->atom_num_dens[index+i]
                 * ElectronBremmsstrahlung_CSPA(h_materials->mixture[index+i], cut, Ekine);

        if ( tmax < Ekine )
        {
            Cross -= h_materials->atom_num_dens[index+i]
                     * ElectronBremmsstrahlung_CSPA(h_materials->mixture[index+i], tmax, Ekine);
        }
    }

    kmax = tmax;
    kmin = cut;
    totalEnergy = Ekine+electron_mass_c2;
    kp2 = MigdalConstant * totalEnergy*totalEnergy * h_materials->nb_electrons_per_vol[mat_id];
    vmin = logf ( kmin );
    vmax = logf ( kmax );
    nn= ( i32 ) ( nmax* ( vmax-vmin ) / ( logf ( highKinEnergy )-vmin ) );

    if ( nn>0 )
    {
        dv = ( vmax-vmin ) /nn;
        v = vmin-dv;
        for ( n=0; n<=nn; n++ )
        {
            v += dv;
            u = expf ( v );
            //fac=SupressionFunction(material,Ekine,u);     //LPM flag is off
            fac = 1.;
            y = u/kmax;
            fac *= ( 4.-4.*y+3.*y*y ) /3.;
            fac *= probsup* ( u*u/ ( u*u+kp2 ) ) +1.-probsup;
            if ( ( n==0 ) || ( n==nn ) )
                c=.5;
            else
                c=1.;
            fac *= c;
            fsig += fac;
        }
        y = kmin/kmax;
        fsig *= dv/ ( -4.*logf ( y ) /3.-4.* ( 1.-y ) /3.+0.5* ( 1.-y*y ) );
    }
    else
        fsig = 1.;
    if ( fsig > 1. )
        fsig = 1.;
    Cross *= fsig;

    return Cross;  // I removed *mm2 => *1 - JB
}

/////////////////////////////////////////////////////////////////////////////////////////////

/// Electron Multiple Scattering ////////////////////////////////////////////////////////////

// constants for eMSC
const f32 Zdat[ 15 ] = { 4., 6., 13., 20., 26., 29., 32., 38., 47., 50., 56., 64., 74., 79., 82. };

const f32 Tdat[ 22 ] =
{
    100.*eV, 200.*eV, 400.*eV, 700.*eV, 1.*keV, 2.*keV, 4.*keV, 7.*keV,
    10.*keV, 20.*keV, 40.*keV, 70.*keV, 100.*keV, 200.*keV, 400.*keV, 700.*keV,
    1.*MeV, 2.*MeV, 4.*MeV, 7.*MeV, 10.*MeV, 20.*MeV
};

const f32 celectron[ 15 ][ 22 ] =
{
    { 1.125, 1.072, 1.051, 1.047, 1.047, 1.050, 1.052, 1.054, 1.054, 1.057, 1.062, 1.069, 1.075, 1.090, 1.105, 1.111, 1.112, 1.108, 1.100, 1.093, 1.089, 1.087 },
    { 1.408, 1.246, 1.143, 1.096, 1.077, 1.059, 1.053, 1.051, 1.052, 1.053, 1.058, 1.065, 1.072, 1.087, 1.101, 1.108, 1.109, 1.105, 1.097, 1.090, 1.086, 1.082 },
    { 2.833, 2.268, 1.861, 1.612, 1.486, 1.309, 1.204, 1.156, 1.136, 1.114, 1.106, 1.106, 1.109, 1.119, 1.129, 1.132, 1.131, 1.124, 1.113, 1.104, 1.099, 1.098 },
    { 3.879, 3.016, 2.380, 2.007, 1.818, 1.535, 1.340, 1.236, 1.190, 1.133, 1.107, 1.099, 1.098, 1.103, 1.110, 1.113, 1.112, 1.105, 1.096, 1.089, 1.085, 1.098 },
    { 6.937, 4.330, 2.886, 2.256, 1.987, 1.628, 1.395, 1.265, 1.203, 1.122, 1.080, 1.065, 1.061, 1.063, 1.070, 1.073, 1.073, 1.070, 1.064, 1.059, 1.056, 1.056 },
    { 9.616, 5.708, 3.424, 2.551, 2.204, 1.762, 1.485, 1.330, 1.256, 1.155, 1.099, 1.077, 1.070, 1.068, 1.072, 1.074, 1.074, 1.070, 1.063, 1.059, 1.056, 1.052 },
    { 11.72, 6.364, 3.811, 2.806, 2.401, 1.884, 1.564, 1.386, 1.300, 1.180, 1.112, 1.082, 1.073, 1.066, 1.068, 1.069, 1.068, 1.064, 1.059, 1.054, 1.051, 1.050 },
    { 18.08, 8.601, 4.569, 3.183, 2.662, 2.025, 1.646, 1.439, 1.339, 1.195, 1.108, 1.068, 1.053, 1.040, 1.039, 1.039, 1.039, 1.037, 1.034, 1.031, 1.030, 1.036 },
    { 18.22, 1.48, 5.333, 3.713, 3.115, 2.367, 1.898, 1.631, 1.498, 1.301, 1.171, 1.105, 1.077, 1.048, 1.036, 1.033, 1.031, 1.028, 1.024, 1.022, 1.021, 1.024 },
    { 14.14, 10.65, 5.710, 3.929, 3.266, 2.453, 1.951, 1.669, 1.528, 1.319, 1.178, 1.106, 1.075, 1.040, 1.027, 1.022, 1.020, 1.017, 1.015, 1.013, 1.013, 1.020 },
    { 14.11, 11.73, 6.312, 4.240, 3.478, 2.566, 2.022, 1.720, 1.569, 1.342, 1.186, 1.102, 1.065, 1.022, 1.003, 0.997, 0.995, 0.993, 0.993, 0.993, 0.993, 1.011 },
    { 22.76, 20.01, 8.835, 5.287, 4.144, 2.901, 2.219, 1.855, 1.677, 1.410, 1.224, 1.121, 1.073, 1.014, 0.986, 0.976, 0.974, 0.972, 0.973, 0.974, 0.975, 0.987 },
    { 50.77, 40.85, 14.13, 7.184, 5.284, 3.435, 2.520, 2.059, 1.837, 1.512, 1.283, 1.153, 1.091, 1.010, 0.969, 0.954, 0.950, 0.947, 0.949, 0.952, 0.954, 0.963 },
    { 65.87, 59.06, 15.87, 7.570, 5.567, 3.650, 2.682, 2.182, 1.939, 1.579, 1.325, 1.178, 1.108, 1.014, 0.965, 0.947, 0.941, 0.938, 0.940, 0.944, 0.946, 0.954 },
    { 55.60, 47.34, 15.92, 7.810, 5.755, 3.767, 2.760, 2.239, 1.985, 1.609, 1.343, 1.188, 1.113, 1.013, 0.960, 0.939, 0.933, 0.930, 0.933, 0.936, 0.939, 0.949 }
};

const f32 sig0[ 15 ] =
{
    .2672*barn, .5922*barn, 2.653*barn, 6.235*barn, 11.69*barn, 13.24*barn, 16.12*barn, 23.00*barn,
    35.13*barn, 39.95*barn, 50.85*barn, 67.19*barn, 91.15*barn, 104.4*barn, 113.1*barn
};

const f32 hecorr[ 15 ] =
{
    120.70, 117.50, 105.00, 92.92, 79.23, 74.510, 68.29, 57.39, 41.97, 36.14, 24.53, 10.21, -7.855, -16.84, -22.30
};

f32 ElectronMultipleScattering_CSPA( f32 Ekine,  ui8 Z )
{
    f32 AtomicNumber = ( f32 ) Z;

    i32 iZ = 14, iT = 21;
    f32  Cross = 0.;
    f32  eKin, eTot, T, E;
    f32  beta2, bg2, b2big, b2small, ratb2, Z23, tau, w;
    f32  Z1, Z2, ratZ;
    f32  c, c1, c2, cc1, cc2, corr;
    f32  Tlim = 10.*MeV;
    f32  sigmafactor = 2.*pi*elec_radius*elec_radius;
    f32  epsfactor = 2.*electron_mass_c2*electron_mass_c2*Bohr_radius*Bohr_radius/ ( hbarc*hbarc );
    f32  eps, epsmin = 1.e-4, epsmax = 1.e10;
    f32  beta2lim = Tlim* ( Tlim+2.*electron_mass_c2 ) / ( ( Tlim+electron_mass_c2 ) * ( Tlim+electron_mass_c2 ) );
    f32  bg2lim = Tlim* ( Tlim+2.*electron_mass_c2 ) / ( electron_mass_c2*electron_mass_c2 );

    Z23 = 2.*log ( AtomicNumber ) /3.;
    Z23 = exp ( Z23 );

    tau = Ekine/electron_mass_c2;
    c = electron_mass_c2*tau* ( tau+2. ) / ( electron_mass_c2* ( tau+1. ) ); // a simplifier
    w = c-2.;
    tau = .5* ( w+sqrt ( w*w+4.*c ) );
    eKin = electron_mass_c2*tau;

    eTot = eKin + electron_mass_c2;
    beta2 = eKin* ( eTot+electron_mass_c2 ) / ( eTot*eTot );
    bg2 = eKin* ( eTot+electron_mass_c2 ) / ( electron_mass_c2*electron_mass_c2 );
    eps = epsfactor*bg2/Z23;
    if ( eps<epsmin )
        Cross = 2.*eps*eps;
    else if ( eps<epsmax )
        Cross = log ( 1.+2.*eps )-2.*eps/ ( 1.+2.*eps );
    else
        Cross = log ( 2.*eps )-1.+1./eps;
    Cross *= AtomicNumber*AtomicNumber / ( beta2*bg2 );

    while ( ( iZ>=0 ) && ( Zdat[iZ] >= AtomicNumber ) )
        iZ -= 1;
    if ( iZ == 14 )
        iZ = 13;
    if ( iZ == -1 )
        iZ = 0;
    Z1 = Zdat[iZ];
    Z2 = Zdat[iZ+1];
    ratZ = ( AtomicNumber-Z1 ) * ( AtomicNumber+Z1 ) / ( ( Z2-Z1 ) * ( Z2+Z1 ) );

    if ( eKin<=Tlim )
    {
        while ( ( iT>=0 ) && ( Tdat[iT]>=eKin ) )
            iT -= 1;
        if ( iT == 21 )
            iT = 20;
        if ( iT == -1 )
            iT = 0;
        T = Tdat[iT];
        E = T+electron_mass_c2;
        b2small = T* ( E+electron_mass_c2 ) / ( E*E );
        T = Tdat[iT+1];
        E = T+electron_mass_c2;
        b2big = T* ( E+electron_mass_c2 ) / ( E*E );
        ratb2 = ( beta2-b2small ) / ( b2big-b2small );

        c1 = celectron[iZ][iT];
        c2 = celectron[iZ+1][iT];
        cc1 = c1+ratZ* ( c2-c1 );
        c1 = celectron[iZ][iT+1];
        c2 = celectron[iZ+1][iT+1];
        cc2 = c1+ratZ* ( c2-c1 );
        corr = cc1+ratb2* ( cc2-cc1 );
        Cross *= sigmafactor/corr;
    }
    else
    {
        c1 = bg2lim*sig0[iZ]* ( 1.+hecorr[iZ]* ( beta2-beta2lim ) ) /bg2;
        c2 = bg2lim*sig0[iZ+1]* ( 1.+hecorr[iZ+1]* ( beta2-beta2lim ) ) /bg2;
        if ( ( AtomicNumber>=Z1 ) && ( AtomicNumber<=Z2 ) )
            Cross = c1+ratZ* ( c2-c1 );
        else if ( AtomicNumber<Z1 )
            Cross = AtomicNumber*AtomicNumber*c1/ ( Z1*Z1 );
        else if ( AtomicNumber>Z2 )
            Cross = AtomicNumber*AtomicNumber*c2/ ( Z2*Z2 );
    }
    return  Cross;
}

f32 ElectronMultipleScattering_CS(const MaterialsData *h_material, f32 Ekine, ui8 mat_id)
{
    ui32 i;
    f32 CrossTotale = 0.;
    ui32 index = h_material->index[mat_id];

    for ( i=0; i<h_material->nb_elements[mat_id]; i++ )
    {
        CrossTotale += h_material->atom_num_dens[index+i]
                       *ElectronMultipleScattering_CSPA( Ekine, h_material->mixture[index+i] );
    }

    return  CrossTotale;
}

/////////////////////////////////////////////////////////////////////////////////////////////

#endif
