#include "hip/hip_runtime.h"
// This file is part of GGEMS
//
// GGEMS is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GGEMS is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GGEMS.  If not, see <http://www.gnu.org/licenses/>.
//
// GGEMS Copyright (C) 2013-2014 Julien Bert

#ifndef VOXELIZED_SOURCE_CU
#define VOXELIZED_SOURCE_CU

#include "voxelized_source.cuh"

//// External function
__host__ __device__ void voxelized_source_primary_generator(ParticleStack particles, ui32 id,
                                                            f32 *cdf_index, f32 *cdf_act, ui32 nb_acts,
                                                            f32 px, f32 py, f32 pz,
                                                            ui32 nb_vox_x, ui32 nb_vox_y, ui32 nb_vox_z,
                                                            f32 sx, f32 sy, f32 sz,
                                                            f32 energy, ui8 type, ui32 geom_id) {

    f32 jump = (f32)(nb_vox_x*nb_vox_y);
    f32 ind, x, y, z;

    // use cdf to find the next emission spot
    f32 rnd = JKISS32(particles, id);
    ui32 pos = binary_search(rnd, cdf_act, nb_acts);
    
    // convert position index to emitted position
    ind = cdf_index[pos];
    z = floor(ind / jump);
    ind -= (z*jump);
    y = floor(ind / (f32)nb_vox_x);
    x = ind - y*nb_vox_x;

    // random positon within the voxel
    x += JKISS32(particles, id);
    y += JKISS32(particles, id);
    z += JKISS32(particles, id);

    // Due to float operation aproximation: 1+(1-Epsilon) = 2
    // we need to check that x, y, z are not equal to the size of the vox source
    // x, y, z must be in [0, size[
    if (x == nb_vox_x) x -= EPSILON3;
    if (y == nb_vox_y) y -= EPSILON3;
    if (z == nb_vox_z) z -= EPSILON3;

    // convert in mm
    x *= sx;
    y *= sy;
    z *= sz;

    // shift according to center of phantom and translation
    x = x - nb_vox_x*sx*0.5 + px;
    y = y - nb_vox_y*sy*0.5 + py;
    z = z - nb_vox_z*sz*0.5 + pz;

    // random orientation
    f32 phi = JKISS32(particles, id);
    f32 theta = JKISS32(particles, id);
    phi *= gpu_twopi;
    theta = acosf(1.0f - 2.0f*theta);

    // compute direction vector
    f32 dx = cos(phi)*sin(theta);
    f32 dy = sin(phi)*sin(theta);
    f32 dz = cos(theta);

    // set particle stack 1
    particles.E[id] = energy;
    particles.dx[id] = dx;
    particles.dy[id] = dy;
    particles.dz[id] = dz;
    particles.px[id] = x;
    particles.py[id] = y;
    particles.pz[id] = z;
    particles.tof[id] = 0.0;
    particles.endsimu[id] = PARTICLE_ALIVE;
    particles.level[id] = PRIMARY;
    particles.pname[id] = type;
    particles.geometry_id[id] = geom_id;

    // set particle stack 2
    ++id;
    particles.E[id] = energy;
    particles.dx[id] = -dx;    // back2back
    particles.dy[id] = -dy;
    particles.dz[id] = -dz;
    particles.px[id] = x;
    particles.py[id] = y;
    particles.pz[id] = z;
    particles.tof[id] = 0.0;
    particles.endsimu[id] = PARTICLE_ALIVE;
    particles.level[id] = PRIMARY;
    particles.pname[id] = type;
    particles.geometry_id[id] = geom_id;
}

__host__ __device__ void voxelized_source_primary_mono_generator(ParticleStack particles, ui32 id,
                                                            f32 *cdf_index, f32 *cdf_act, ui32 nb_acts,
                                                            f32 px, f32 py, f32 pz,
                                                            ui32 nb_vox_x, ui32 nb_vox_y, ui32 nb_vox_z,
                                                            f32 sx, f32 sy, f32 sz,
                                                            f32 energy, ui8 type, ui32 geom_id) {

    f32 jump = (f32)(nb_vox_x*nb_vox_y);
    f32 ind, x, y, z;

    // use cdf to find the next emission spot
    f32 rnd = JKISS32(particles, id);
    ui32 pos = binary_search(rnd, cdf_act, nb_acts);

    // convert position index to emitted position
    ind = cdf_index[pos];
    z = floor(ind / jump);
    ind -= (z*jump);
    y = floor(ind / (f32)nb_vox_x);
    x = ind - y*nb_vox_x;
    
    // random positon within the voxel
    x += JKISS32(particles, id);
    y += JKISS32(particles, id);
    z += JKISS32(particles, id);

    // Due to float operation aproximation: 1+(1-Epsilon) = 2
    // we need to check that x, y, z are not equal to the size of the vox source
    // x, y, z must be in [0, size[
    if (x == nb_vox_x) x -= EPSILON3;
    if (y == nb_vox_y) y -= EPSILON3;
    if (z == nb_vox_z) z -= EPSILON3;

    // convert in mm
    x *= sx;
    y *= sy;
    z *= sz;

    // shift according to center of phantom and translation
    x = x - nb_vox_x*sx*0.5 + px;
    y = y - nb_vox_y*sy*0.5 + py;
    z = z - nb_vox_z*sz*0.5 + pz;

    // random orientation
    f32 phi = JKISS32(particles, id);
    f32 theta = JKISS32(particles, id);
    phi *= gpu_twopi;
    theta = acosf(1.0f - 2.0f*theta);

    // compute direction vector
    f32 dx = cos(phi)*sin(theta);
    f32 dy = sin(phi)*sin(theta);
    f32 dz = cos(theta);

    // set particle stack 1
    particles.E[id] = energy;
    particles.dx[id] = dx;
    particles.dy[id] = dy;
    particles.dz[id] = dz;
    particles.px[id] = x;
    particles.py[id] = y;
    particles.pz[id] = z;
    particles.tof[id] = 0.0;
    particles.endsimu[id] = PARTICLE_ALIVE;
    particles.level[id] = PRIMARY;
    particles.pname[id] = type;
    particles.geometry_id[id] = geom_id;
}

VoxelizedSource::VoxelizedSource() {
    // Default values
    seed=10;
    geometry_id=0;
    source_name="VoxSrc01";
    source_type="back2back";
    px=0.0; py=0.0; pz=0.0;
    energy=511*keV;

    // Init pointer
    activity_volume = NULL;
    activity_cdf = NULL;
    activity_index = NULL;
}

void VoxelizedSource::set_position(f32 vpx, f32 vpy, f32 vpz) {
    px = vpx; py = vpy; pz = vpz;
}

void VoxelizedSource::set_energy(f32 venergy) {
    energy = venergy;
}

void VoxelizedSource::set_histpoint(f32 venergy, f32 vpart) {
      energy_hist.push_back(venergy);
      partpdec.push_back(vpart);
}  

void VoxelizedSource::set_source_type(std::string vtype) {
    source_type = vtype;
}

void VoxelizedSource::set_seed(ui32 vseed) {
    seed = vseed;
}

void VoxelizedSource::set_in_geometry(ui32 vgeometry_id) {
    geometry_id = vgeometry_id;
}

void VoxelizedSource::set_source_name(std::string vsource_name) {
    source_name = vsource_name;
}

//// MHD //////////////////////////////////////////////////////:

// Skip comment starting with "#"
void VoxelizedSource::skip_comment(std::istream & is) {
    i8 c;
    i8 line[1024];
    if (is.eof()) return;
    is >> c;
    while (is && (c=='#')) {
        is.getline(line, 1024);
        is >> c;
        if (is.eof()) return;
    }
    is.unget();
}

// Remove all white space
std::string VoxelizedSource::remove_white_space(std::string txt) {
    txt.erase(remove_if(txt.begin(), txt.end(), isspace), txt.end());
    return txt;
}

// Read mhd key
std::string VoxelizedSource::read_mhd_key(std::string txt) {
    txt = txt.substr(0, txt.find("="));
    return remove_white_space(txt);
}

// Read string mhd arg
std::string VoxelizedSource::read_mhd_string_arg(std::string txt) {
    txt = txt.substr(txt.find("=")+1);
    return remove_white_space(txt);
}

// Read i32 mhd arg
i32 VoxelizedSource::read_mhd_int(std::string txt) {
    i32 res;
    txt = txt.substr(txt.find("=")+1);
    txt = remove_white_space(txt);
    std::stringstream(txt) >> res;
    return res;
}

// Read int mhd arg
i32 VoxelizedSource::read_mhd_int_atpos(std::string txt, i32 pos) {
    i32 res;
    txt = txt.substr(txt.find("=")+2);
    if (pos==0) {
        txt = txt.substr(0, txt.find(" "));
    }
    if (pos==1) {
        txt = txt.substr(txt.find(" ")+1);
        txt = txt.substr(0, txt.find(" "));
    }
    if (pos==2) {
        txt = txt.substr(txt.find(" ")+1);
        txt = txt.substr(txt.find(" ")+1);
    }
    std::stringstream(txt) >> res;
    return res;
}

// Read f32 mhd arg
f32 VoxelizedSource::read_mhd_f32_atpos(std::string txt, i32 pos) {
    f32 res;
    txt = txt.substr(txt.find("=")+2);
    if (pos==0) {
        txt = txt.substr(0, txt.find(" "));
    }
    if (pos==1) {
        txt = txt.substr(txt.find(" ")+1);
        txt = txt.substr(0, txt.find(" "));
    }
    if (pos==2) {
        txt = txt.substr(txt.find(" ")+1);
        txt = txt.substr(txt.find(" ")+1);
    }
    std::stringstream(txt) >> res;
    return res;
}

// Load activities from mhd file (only f32 data)
void VoxelizedSource::load_from_mhd(std::string filename) {

    /////////////// First read the MHD file //////////////////////

    std::string line, key;
    nb_vox_x=0, nb_vox_y=0, nb_vox_z=0;
    spacing_x=0, spacing_y=0, spacing_z=0;

    // Watchdog
    std::string ObjectType="", BinaryData="", BinaryDataByteOrderMSB="", CompressedData="",
                ElementType="", ElementDataFile="";
    i32 NDims=0;

    // Read range file
    std::ifstream file(filename.c_str());
    if(!file) { printf("Error, file %s not found \n", filename.c_str()); exit(EXIT_FAILURE);}
    while (file) {
        skip_comment(file);
        std::getline(file, line);

        if (file) {
            key = read_mhd_key(line);
            if (key=="ObjectType")              ObjectType = read_mhd_string_arg(line);
            if (key=="NDims")                   NDims = read_mhd_int(line);
            if (key=="BinaryData")              BinaryData = read_mhd_string_arg(line);
            if (key=="BinaryDataByteOrderMSB")  BinaryDataByteOrderMSB=read_mhd_string_arg(line);
            if (key=="CompressedData")          CompressedData = read_mhd_string_arg(line);
            //if (key=="TransformMatrix") printf("Matrix\n");
            //if (key=="Offset")  printf("Offset\n");
            //if (key=="CenterOfRotation") printf("CoR\n");
            if (key=="ElementSpacing") {
                                                spacing_x=read_mhd_f32_atpos(line, 0);
                                                spacing_y=read_mhd_f32_atpos(line, 1);
                                                spacing_z=read_mhd_f32_atpos(line, 2);
            }
            if (key=="DimSize") {
                                                nb_vox_x=read_mhd_int_atpos(line, 0);
                                                nb_vox_y=read_mhd_int_atpos(line, 1);
                                                nb_vox_z=read_mhd_int_atpos(line, 2);
            }

            //if (key=="AnatomicalOrientation") printf("Anato\n");
            if (key=="ElementType")             ElementType = read_mhd_string_arg(line);
            if (key=="ElementDataFile")         ElementDataFile = read_mhd_string_arg(line);
        }

    } // read file

    // Check header
    if (ObjectType != "Image") {
        printf("Error, mhd header: ObjectType = %s\n", ObjectType.c_str());
        exit(EXIT_FAILURE);
    }
    if (BinaryData != "True") {
        printf("Error, mhd header: BinaryData = %s\n", BinaryData.c_str());
        exit(EXIT_FAILURE);
    }
    if (BinaryDataByteOrderMSB != "False") {
        printf("Error, mhd header: BinaryDataByteOrderMSB = %s\n", BinaryDataByteOrderMSB.c_str());
        exit(EXIT_FAILURE);
    }
    if (CompressedData != "False") {
        printf("Error, mhd header: CompressedData = %s\n", CompressedData.c_str());
        exit(EXIT_FAILURE);
    }
    if (ElementType != "MET_FLOAT") {
        printf("Error, mhd header: ElementType = %s\n", ElementType.c_str());
        exit(EXIT_FAILURE);
    }
    if (ElementDataFile == "") {
        printf("Error, mhd header: ElementDataFile = %s\n", ElementDataFile.c_str());
        exit(EXIT_FAILURE);
    }
    if (NDims != 3) {
        printf("Error, mhd header: NDims = %i\n", NDims);
        exit(EXIT_FAILURE);
    }

    if (nb_vox_x == 0 || nb_vox_y == 0 || nb_vox_z == 0 ||
            spacing_x == 0 || spacing_y == 0 || spacing_z == 0) {
        printf("Error when loading mhd file (unknown dimension and spacing)\n");
        printf("   => dim %i %i %i - spacing %f %f %f\n", nb_vox_x, nb_vox_y, nb_vox_z,
                                                          spacing_x, spacing_y, spacing_z);
        exit(EXIT_FAILURE);
    }
    // Read data
    FILE *pfile = fopen(ElementDataFile.c_str(), "rb");
    if (!pfile) {
        std::string nameWithRelativePath = filename;
        i32 lastindex = nameWithRelativePath.find_last_of(".");
        nameWithRelativePath = nameWithRelativePath.substr(0, lastindex);
        nameWithRelativePath+=".raw";
        pfile = fopen(nameWithRelativePath.c_str(), "rb");
        if (!pfile) {
            printf("Error when loading mhd file: %s\n", ElementDataFile.c_str());
            exit(EXIT_FAILURE);
        }
    }

    number_of_voxels = nb_vox_x*nb_vox_y*nb_vox_z;

    activity_volume = (f32*)malloc(sizeof(f32) * number_of_voxels);
    fread(activity_volume, sizeof(f32), number_of_voxels, pfile);
    fclose(pfile);

    // Compute the associated CDF of the activities
    compute_cdf();

}

// Compute the CDF of the activities
void VoxelizedSource::compute_cdf() {

    // count nb of non zeros activities
    ui32 nb=0;
    ui32 i=0; while (i<number_of_voxels) {
        if (activity_volume[i] != 0.0f) ++nb;
        ++i;
    }
    activity_size = nb;

    // mem allocation
    activity_index = (f32*)malloc(nb*sizeof(f32));
    activity_cdf = (f32*)malloc(nb*sizeof(f32));

    // Buffer
    f64* cdf = new f64[nb];

    // fill array with non zeros values activity
    ui32 index = 0;
    f64 val;
    f64 sum = 0.0; // for the cdf
    i=0; while (i<number_of_voxels) {
        val = activity_volume[i];
        if (val != 0.0f) {
            activity_index[index] = i;
            cdf[index] = val;
            sum += val;
            //printf("cdf i %d val %lf \n", index, cdf[index]);
            ++index;
        }
        ++i;
    }
    tot_activity = sum;
    printf("tot_activity %lf \n", sum);
    
    // compute cummulative density function
    cdf[0] /= sum;
    activity_cdf[0] = cdf[0];
      
    i = 1; while (i<nb) {
       // printf("i %d test div %4.12lf \n", i, (cdf[i]/sum));
        cdf[i] = (cdf[i]/sum) + cdf[i-1];
        activity_cdf[i]= (f32) cdf[i];
       // printf("i %d test div %4.12lf \n", i, cdf[i]);
        ++i;
    }

    delete cdf;

}









#endif
