#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file ct_detector.cu
 * \brief
 * \author J. Bert <bert.jul@gmail.com>
 * \author Didier Benoit <didier.benoit13@gmail.com>
 * \version 0.1
 * \date 18 novembre 2015
 *
 *
 *
 */

#ifndef CT_DETECTOR_CU
#define CT_DETECTOR_CU

#define MAX_SCATTER_ORDER 3

#include <iomanip>
#include <sstream>

#include "ggems_detector.cuh"
#include "ct_detector.cuh"
#include "image_reader.cuh"

__host__ __device__ void ct_detector_track_to_in( ParticlesData &particles, ObbData detector_volume,  ui32 id )
{
    // If freeze (not dead), re-activate the current particle
    if( particles.endsimu[ id ] == PARTICLE_FREEZE )
    {
        particles.endsimu[ id ] = PARTICLE_ALIVE;
    }
    else if ( particles.endsimu[ id ] == PARTICLE_DEAD )
    {
        return;
    }

    // Read position
    f32xyz pos;
    pos.x = particles.px[ id ];
    pos.y = particles.py[ id ];
    pos.z = particles.pz[ id ];

    // Read direction
    f32xyz dir;
    dir.x = particles.dx[ id ];
    dir.y = particles.dy[ id ];
    dir.z = particles.dz[ id ];

    // Project particle to detector
    f32 dist = hit_ray_OBB( pos, dir,
                            detector_volume.xmin, detector_volume.xmax,
                            detector_volume.ymin, detector_volume.ymax,
                            detector_volume.zmin, detector_volume.zmax,
                            detector_volume.center,
                            detector_volume.u, detector_volume.v, detector_volume.w );

    if( dist == FLT_MAX )
    {
        particles.endsimu[id] = PARTICLE_DEAD;
        particles.E[ id ] = 0.0f;
        return;
    }
    else
    {
        // Check if the path of the particle cross the volume sufficiently
        f32 cross = dist_overlap_ray_OBB( pos, dir, detector_volume.xmin,
                                          detector_volume.xmax, detector_volume.ymin, detector_volume.ymax,
                                          detector_volume.zmin, detector_volume.zmax, detector_volume.center,
                                          detector_volume.u, detector_volume.v, detector_volume.w );

        if( cross < EPSILON3 )
        {
            particles.endsimu[id] = PARTICLE_DEAD;
            particles.E[ id ] = 0.0f;
            return;
        }

        // move the particle slightly inside the volume
        pos = fxyz_add( pos, fxyz_scale( dir, dist + EPSILON3 ) );
    }

    // Save particle position
    particles.px[ id ] = pos.x;
    particles.py[ id ] = pos.y;
    particles.pz[ id ] = pos.z;
}

// Digitizer
__host__ __device__ void ct_detector_digitizer( ParticlesData particles, f32 orbiting_angle, ObbData detector_volume,
                                                /*f32 pixel_size_x,*/ f32 pixel_size_y, f32 pixel_size_z,
                                                /*ui32 nb_pixel_x,*/ ui32 nb_pixel_y, ui32 nb_pixel_z,
                                                f32 threshold,
                                                ui32* projection, ui32* scatter_order,
                                                ui32 id )
{
    // Read position
    f32xyz pos;
    pos.x = particles.px[ id ];
    pos.y = particles.py[ id ];
    pos.z = particles.pz[ id ];

    //f32 rot_posx = pos.x * cosf( orbiting_angle ) + pos.y * sinf( orbiting_angle );  - This var is not used - JB
    f32 rot_posy = -pos.x * sinf( orbiting_angle ) + pos.y * cosf( orbiting_angle );

    // Calculate pixel id
    ui32 idx_xr = (ui32)( ( rot_posy - detector_volume.ymin ) / pixel_size_y );
    ui32 idx_yr = (ui32)( ( pos.z - detector_volume.zmin ) / pixel_size_z );

    if( idx_xr >= nb_pixel_y || idx_yr >= nb_pixel_z || particles.E[ id ] < threshold )
    {
        particles.endsimu[id] = PARTICLE_DEAD;
        particles.E[ id ] = 0.0f;
        return;
    }

    ggems_atomic_add( projection, idx_xr + idx_yr * nb_pixel_y, 1 );

    // Scatter increment index
    ui32 n_scatter_order = ( particles.scatter_order[ id ] < MAX_SCATTER_ORDER ) ?
                particles.scatter_order[ id ] : MAX_SCATTER_ORDER;

    // Increment the scatter
    if( n_scatter_order != 0 )
    {
        ui32 scatter_idx =
                ( n_scatter_order - 1 ) * nb_pixel_y * nb_pixel_z;
        ggems_atomic_add( scatter_order,
                          idx_xr + idx_yr * nb_pixel_y + scatter_idx, 1 );
    }
}


// Kernel that move particles to the voxelized volume boundary
__global__ void kernel_ct_detector_track_to_in( ParticlesData particles,
                                                ObbData detector_volume )
{

    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particles.size) return;

    ct_detector_track_to_in( particles, detector_volume, id);
}

// Kernel digitizer
__global__ void kernel_ct_detector_digitizer( ParticlesData particles, f32 orbiting_angle, ObbData detector_volume,
                                              /*f32 pixel_size_x,*/ f32 pixel_size_y, f32 pixel_size_z,
                                              /*ui32 nb_pixel_x,*/ ui32 nb_pixel_y, ui32 nb_pixel_z,
                                              f32 threshold,
                                              ui32* projection, ui32* scatter_order )
{

    const ui32 id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particles.size) return;

    ct_detector_digitizer( particles, orbiting_angle, detector_volume,
                           /*f32 pixel_size_x,*/ pixel_size_y, pixel_size_z,
                           /*ui32 nb_pixel_x,*/ nb_pixel_y, nb_pixel_z,
                           threshold,
                           projection, scatter_order, id );
}

void CTDetector::track_to_in( Particles particles )
{
    if( m_params.data_h.device_target == CPU_DEVICE )
    {
        ui32 id = 0;
        while( id < particles.size )
        {
            ct_detector_track_to_in( particles.data_h,
                                     m_detector_volume.volume.data_h,
                                     id );
            ++id;
        }
    }
    else if( m_params.data_h.device_target == GPU_DEVICE )
    {
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 )
                / m_params.data_h.gpu_block_size;

        kernel_ct_detector_track_to_in<<<grid, threads>>>( particles.data_d,
                                                           m_detector_volume.volume.data_d );
        cuda_error_check("Error ", " Kernel_ct_detector (track to in)");
        hipDeviceSynchronize();
    }
}

void CTDetector::digitizer( Particles particles )
{
    if( m_params.data_h.device_target == CPU_DEVICE )
    {
        ui32 id = 0;
        while( id < particles.size )
        {
            ct_detector_digitizer( particles.data_h, m_orbiting_angle, m_detector_volume.volume.data_h,
                                   /*f32 pixel_size_x,*/ m_pixel_size_y, m_pixel_size_z,
                                   /*ui32 nb_pixel_x,*/ m_nb_pixel_y, m_nb_pixel_z,
                                   m_threshold,
                                   m_projection_h, m_scatter_order_h, id );
            ++id;
        }
    }
    else if( m_params.data_h.device_target == GPU_DEVICE )
    {
        dim3 threads, grid;
        threads.x = m_params.data_h.gpu_block_size;
        grid.x = ( particles.size + m_params.data_h.gpu_block_size - 1 )
                / m_params.data_h.gpu_block_size;

        kernel_ct_detector_digitizer<<<grid, threads>>>( particles.data_d, m_orbiting_angle, m_detector_volume.volume.data_d,
                                                         /*f32 pixel_size_x,*/ m_pixel_size_y, m_pixel_size_z,
                                                         /*ui32 nb_pixel_x,*/ m_nb_pixel_y, m_nb_pixel_z,
                                                         m_threshold,
                                                         m_projection_d, m_scatter_order_d);
        cuda_error_check("Error ", " Kernel_ct_detector (digitizer)");
        hipDeviceSynchronize();
    }
}

CTDetector::CTDetector()
    : GGEMSDetector(),
      m_pixel_size_x( 0.0f ),
      m_pixel_size_y( 0.0f ),
      m_pixel_size_z( 0.0f ),
      m_nb_pixel_x( 0 ),
      m_nb_pixel_y( 0 ),
      m_nb_pixel_z( 0 ),
      m_posx( 0.0f ),
      m_posy( 0.0f ),
      m_posz( 0.0f ),
      m_threshold( 0.0f ),
      m_orbiting_angle( 0.0 ),
      m_projection_h( nullptr ),
      m_projection_d( nullptr ),
      m_scatter_order_h( nullptr ),
      m_scatter_order_d( nullptr )
{
    set_name( "ct_detector" );
}

CTDetector::~CTDetector()
{
    if( m_projection_h )
    {
        delete[] m_projection_h;
        m_projection_h = nullptr;
    }

    if( m_scatter_order_h )
    {
        delete[] m_scatter_order_h;
        m_scatter_order_h = nullptr;
    }

    if( m_params.data_h.device_target == GPU_DEVICE )
    {
        if( m_projection_d )
        {
            hipFree( m_projection_d );
        }

        if( m_scatter_order_d )
        {
            hipFree( m_scatter_order_d );
        }
    }
}

void CTDetector::set_dimension( f32 w, f32 h, f32 d )
{
    m_nb_pixel_x = w;
    m_nb_pixel_y = h;
    m_nb_pixel_z = d;
}

void CTDetector::set_pixel_size( f32 sx, f32 sy, f32 sz )
{
    m_pixel_size_x = sx;
    m_pixel_size_y = sy;
    m_pixel_size_z = sz;
}

void CTDetector::set_position( f32 x, f32 y, f32 z )
{
    m_posx = x;
    m_posy = y;
    m_posz = z;
}

void CTDetector::set_threshold( f32 threshold )
{
    m_threshold = threshold;
}

void CTDetector::set_orbiting( f32 orbiting_angle )
{
    m_orbiting_angle = orbiting_angle;
}

bool CTDetector::m_check_mandatory()
{
    if( m_pixel_size_x == 0.0f ||
            m_pixel_size_y == 0.0f ||
            m_pixel_size_z == 0.0f ||
            m_nb_pixel_x == 0 ||
            m_nb_pixel_y == 0 ||
            m_nb_pixel_z == 0 )
    {
        return false;
    }
    else
    {
        return true;
    }
}

void CTDetector::save_projection( std::string filename )
{
    // Check if CPU or GPU
    if( m_params.data_h.device_target == GPU_DEVICE )
    {
        HANDLE_ERROR( hipMemcpy( m_projection_h,
                                  m_projection_d,
                                  sizeof( ui32 ) * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z,
                                  hipMemcpyDeviceToHost ) );
    }

    ui16 *projection16 = new ui16[ m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z ];
    for( ui32 i = 0; i < m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z; ++i )
    {
        projection16[ i ] = m_projection_h[ i ];
    }

    // Global sinogram
    ImageReader::record3Dimage(
                filename,
                projection16,
                make_f32xyz( 0.0f, 0.0f, 0.0f ),
                make_f32xyz( m_pixel_size_x, m_pixel_size_y, m_pixel_size_z ),
                make_ui32xyz( m_nb_pixel_x, m_nb_pixel_y, m_nb_pixel_z ),
                false
                );

    delete[] projection16;
}

void CTDetector::save_scatter( std::string basename )
{
    if( m_params.data_h.device_target == GPU_DEVICE )
    {
        HANDLE_ERROR( hipMemcpy( m_scatter_order_h,
                                  m_scatter_order_d,
                                  sizeof( ui32 ) * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z
                                  * MAX_SCATTER_ORDER, hipMemcpyDeviceToHost ) );
    }

    ui16 *scatter16 = new ui16[ MAX_SCATTER_ORDER * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z ];

    // Loop over the scatter order
    for( ui32 i = 0; i < MAX_SCATTER_ORDER; ++i )
    {
        // Determine the filename
        std::ostringstream out( std::ostringstream::out );
        out << basename << "_" << std::setfill( '0' ) << std::setw( 3 ) << i
            << ".mhd";

        for( ui32 j = 0; j < m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z; ++j )
        {
            scatter16[ j + i * m_nb_pixel_x * m_nb_pixel_y ] = m_scatter_order_h[ j + i * m_nb_pixel_x * m_nb_pixel_y ];
        }

        // Save the scatter image for each order
        ImageReader::record3Dimage(
                    out.str(),
                    &scatter16[ i * m_nb_pixel_x * m_nb_pixel_y ],
                make_f32xyz( 0.0f, 0.0f, 0.0f ),
                make_f32xyz( m_pixel_size_x, m_pixel_size_y, m_pixel_size_z ),
                make_ui32xyz( m_nb_pixel_x, m_nb_pixel_y, m_nb_pixel_z ),
                false
                );

    }
    delete[] scatter16;
}

ui32 CTDetector::getDetectedParticles()
{
    ui32 count = 0;
    for( ui32 i = 0; i < m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z; ++i )
    {
        count += m_projection_h[ i ];
    }

    return count;
}

ui32 CTDetector::getScatterNumber( ui32 scatter_order )
{
    ui32 count = 0;
    for( ui32 i = 0; i < m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z; ++i )
    {
        count += m_scatter_order_h[ i + m_nb_pixel_x * m_nb_pixel_y
                * m_nb_pixel_z * scatter_order ];
    }

    return count;
}

void CTDetector::print_info_scatter()
{
    // Get the number of detected particles
    ui32 detected_particles = getDetectedParticles();

    // Get the number of scatter by order and the total scatter
    ui32 count_scatter[ MAX_SCATTER_ORDER ];
    ui32 total_scatter = 0;
    for( ui32 i = 0; i < MAX_SCATTER_ORDER; ++i )
    {
        count_scatter[ i ] = getScatterNumber( i );
        total_scatter += count_scatter[ i ];
    }

    // Direct particle
    ui32 direct_particles = detected_particles - total_scatter;

    std::cout << std::endl;
    GGcout << "------------------------------------------" << GGendl;
    GGcout << "Detected particles:            " << std::setfill( ' ' )
           << std::setw( 10 ) << detected_particles << GGendl;
    GGcout << "Direct particles:              " << std::setfill( ' ' )
           << std::setw( 10 ) << direct_particles << " [" << std::setfill( ' ' )
           << std::setw( 6 ) << std::setprecision( 2 ) << std::fixed
           << 100.0f * (float)direct_particles / detected_particles << " %]" << GGendl;
    for( ui32 i = 0; i < MAX_SCATTER_ORDER; ++i )
    {
        GGcout << "Scattered particles order " << std::setfill( ' ' )
               << std::setw( 2 ) << i + 1 << " : " << std::setfill( ' ' )
               << std::setw( 10 ) << count_scatter[ i ] << " [" << std::setfill( ' ' )
               << std::setw( 6 ) << std::setprecision( 2 ) << std::fixed
               << 100.0f * (float)count_scatter[ i ] / detected_particles << " %]"
               << GGendl;
    }
    std::cout << std::endl;

}

void CTDetector::m_copy_detector_cpu2gpu()
{
    m_detector_volume.volume.data_d.xmin = m_detector_volume.volume.data_h.xmin;
    m_detector_volume.volume.data_d.xmax = m_detector_volume.volume.data_h.xmax;

    m_detector_volume.volume.data_d.ymin = m_detector_volume.volume.data_h.ymin;
    m_detector_volume.volume.data_d.ymax = m_detector_volume.volume.data_h.ymax;

    m_detector_volume.volume.data_d.zmin = m_detector_volume.volume.data_h.zmin;
    m_detector_volume.volume.data_d.zmax = m_detector_volume.volume.data_h.zmax;

    m_detector_volume.volume.data_d.angle = m_detector_volume.volume.data_h.angle;
    m_detector_volume.volume.data_d.translate = m_detector_volume.volume.data_h.translate;
    m_detector_volume.volume.data_d.center = m_detector_volume.volume.data_h.center;

    m_detector_volume.volume.data_d.u = m_detector_volume.volume.data_h.u;
    m_detector_volume.volume.data_d.v = m_detector_volume.volume.data_h.v;
    m_detector_volume.volume.data_d.w = m_detector_volume.volume.data_h.w;

    m_detector_volume.volume.data_d.size = m_detector_volume.volume.data_h.size;
}

void CTDetector::initialize( GlobalSimulationParameters params )
{
    // Check the parameters
    if( !m_check_mandatory() )
    {
        print_error( "CTDetector: missing parameters!!!" );
        exit_simulation();
    }

    // Params
    m_params = params;

    // Fill the detector volume parameters
    m_detector_volume.set_size(
                m_pixel_size_x * m_nb_pixel_x,
                m_pixel_size_y * m_nb_pixel_y,
                m_pixel_size_z * m_nb_pixel_z
                );

    m_detector_volume.set_center_position( 0.0f, 0.0f, 0.0f );
    m_detector_volume.translate( m_posx, m_posy, m_posz );
    m_detector_volume.rotate( 0.0, 0.0, m_orbiting_angle );

    // Allocate
    m_projection_h = new ui32[ m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z ];
    memset( m_projection_h, 0.0f, m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z
            * sizeof( ui32 ) );

    // MAX_SCATTER_ORDER first scatter orders are only registered
    m_scatter_order_h =
            new ui32[ MAX_SCATTER_ORDER * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z ];
    memset( m_scatter_order_h, 0.0f, MAX_SCATTER_ORDER * m_nb_pixel_x
            * m_nb_pixel_y * m_nb_pixel_z * sizeof( ui32 ) );

    // Copy projection data to GPU
    if( m_params.data_h.device_target == GPU_DEVICE )
    {
        m_copy_detector_cpu2gpu();

        // GPU mem allocation
        HANDLE_ERROR( hipMalloc( (void**)&m_projection_d,
                                  m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z * sizeof( ui32 ) ) );
        // GPU mem copy
        HANDLE_ERROR( hipMemcpy( m_projection_d,
                                  m_projection_h,
                                  sizeof( ui32 ) * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z,
                                  hipMemcpyHostToDevice ) );

        // GPU mem allocation
        HANDLE_ERROR( hipMalloc( (void**)&m_scatter_order_d,
                                  MAX_SCATTER_ORDER * m_nb_pixel_x * m_nb_pixel_y * m_nb_pixel_z
                                  * sizeof( ui32 ) ) );
        // GPU mem copy
        HANDLE_ERROR( hipMemcpy( m_scatter_order_d,
                                  m_scatter_order_h,
                                  MAX_SCATTER_ORDER * sizeof( ui32 ) * m_nb_pixel_x * m_nb_pixel_y
                                  * m_nb_pixel_z, hipMemcpyHostToDevice ) );
    }
}

#endif

