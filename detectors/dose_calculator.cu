#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file dose_calculator.cu
 * \brief
 * \author Y. Lemaréchal
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.2
 * \date 02/12/2015
 * \date 26/02/2016, add volume of interest, change offset handling and fix many bugs - JB
 * \date 18/04/2016, change every things, use unified memory, improve code - JB
 *
 *
 */

#ifndef DOSE_CALCULATOR_CU
#define DOSE_CALCULATOR_CU

#include "dose_calculator.cuh"

/// CPU&GPU functions //////////////////////////////////////////////////////////

// Analog deposition
__host__ __device__ void dose_record_standard ( DoseData *dose, f32 Edep, f32 px, f32 py, f32 pz )
{

    if (px < dose->xmin + EPSILON3 || px > dose->xmax - EPSILON3) return;
    if (py < dose->ymin + EPSILON3 || py > dose->ymax - EPSILON3) return;
    if (pz < dose->zmin + EPSILON3 || pz > dose->zmax - EPSILON3) return;

    // Defined index phantom    
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( px + dose->offset.x ) * dose->inv_dosel_size.x );
    index_phantom.y = ui32 ( ( py + dose->offset.y ) * dose->inv_dosel_size.y );
    index_phantom.z = ui32 ( ( pz + dose->offset.z ) * dose->inv_dosel_size.z );
    index_phantom.w = index_phantom.z * dose->slice_nb_dosels + index_phantom.y * dose->nb_dosels.x + index_phantom.x;

    //printf("Edep %e  pos %e %e %e  index %i\n", Edep, px, py, pz, index_phantom.w);

#ifdef DEBUG

    if ( index_phantom.x >= dose->nb_dosels.x || index_phantom.y >= dose->nb_dosels.y || index_phantom.z >= dose->nb_dosels.z)
    {
        printf(" IndexX %i  NbDox %i  px %f  Off %f invDox %f\n", index_phantom.x, dose->nb_dosels.x, px, dose->offset.x, dose->inv_dosel_size.x);
        printf(" IndexY %i  NbDox %i  py %f  Off %f invDox %f\n", index_phantom.y, dose->nb_dosels.y, py, dose->offset.y, dose->inv_dosel_size.y);
        printf(" IndexZ %i  NbDox %i  pz %f  Off %f invDox %f\n", index_phantom.z, dose->nb_dosels.z, pz, dose->offset.z, dose->inv_dosel_size.z);
        //index_phantom.z = 0;
    }

    assert( index_phantom.x < dose->nb_dosels.x );
    assert( index_phantom.y < dose->nb_dosels.y );
    assert( index_phantom.z < dose->nb_dosels.z );
#endif

/*
#ifdef __CUDA_ARCH__
    atomicAdd(&dose.edep[index_phantom.w], Edep);
    atomicAdd(&dose.edep_squared[index_phantom.w], Edep*Edep);
    atomicAdd(&dose.number_of_hits[index_phantom.w], ui32(1));
#else
    dose.edep[index_phantom.w] += Edep;
    dose.edep_squared[index_phantom.w] += (Edep*Edep);
    dose.number_of_hits[index_phantom.w] += 1;
#endif
*/

    ggems_atomic_add_f64( dose->edep, index_phantom.w, f64( Edep ) );
    ggems_atomic_add_f64( dose->edep_squared, index_phantom.w, f64( Edep) * f64( Edep ) );
    ggems_atomic_add( dose->number_of_hits, index_phantom.w, ui32 ( 1 ) );                  // ui32, limited to 4.29e9 - JB


}

// TLE deposition
__host__ __device__ void dose_record_TLE( DoseData *dose, f32 Edep, f32 px, f32 py, f32 pz,
                                          f32 length, f32 mu_en)
{

    if (px < dose->xmin + EPSILON3 || px > dose->xmax - EPSILON3) return;
    if (py < dose->ymin + EPSILON3 || py > dose->ymax - EPSILON3) return;
    if (pz < dose->zmin + EPSILON3 || pz > dose->zmax - EPSILON3) return;

    // Defined index phantom
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( px + dose->offset.x ) * dose->inv_dosel_size.x );
    index_phantom.y = ui32 ( ( py + dose->offset.y ) * dose->inv_dosel_size.y );
    index_phantom.z = ui32 ( ( pz + dose->offset.z ) * dose->inv_dosel_size.z );
    index_phantom.w = index_phantom.z * dose->slice_nb_dosels + index_phantom.y * dose->nb_dosels.x + index_phantom.x;

#ifdef DEBUG

    if ( index_phantom.x >= dose->nb_dosels.x || index_phantom.y >= dose->nb_dosels.y || index_phantom.z >= dose->nb_dosels.z)
    {
        printf(" IndexX %i  NbDox %i  px %f  Off %f invDox %f\n", index_phantom.x, dose->nb_dosels.x, px, dose->offset.x, dose->inv_dosel_size.x);
        printf(" IndexY %i  NbDox %i  py %f  Off %f invDox %f\n", index_phantom.y, dose->nb_dosels.y, py, dose->offset.y, dose->inv_dosel_size.y);
        printf(" IndexZ %i  NbDox %i  pz %f  Off %f invDox %f\n", index_phantom.z, dose->nb_dosels.z, pz, dose->offset.z, dose->inv_dosel_size.z);
        //index_phantom.z = 0;
    }

    assert( index_phantom.x < dose->nb_dosels.x );
    assert( index_phantom.y < dose->nb_dosels.y );
    assert( index_phantom.z < dose->nb_dosels.z );
#endif

    // TLE
    f64 energy_dropped = Edep * mu_en * length * 0.1; // arbitrary factor (see in GATE)

/*
#ifdef __CUDA_ARCH__
    atomicAdd(&dose.edep[index_phantom.w], energy_dropped);
    atomicAdd(&dose.edep_squared[index_phantom.w], energy_dropped*energy_dropped);
    atomicAdd(&dose.number_of_hits[index_phantom.w], ui32(1));
#else
    dose.edep[index_phantom.w] += energy_dropped;
    dose.edep_squared[index_phantom.w] += (energy_dropped*energy_dropped);
    dose.number_of_hits[index_phantom.w] += 1;
#endif
*/

    ggems_atomic_add_f64( dose->edep, index_phantom.w, energy_dropped );
    ggems_atomic_add_f64( dose->edep_squared, index_phantom.w, energy_dropped * energy_dropped );
    ggems_atomic_add( dose->number_of_hits, index_phantom.w, ui32 ( 1 ) );                  // ui32, limited to 4.29e9 - JB

}

/// Private /////////////////////////////////////////////////////////////////////

bool DoseCalculator::m_check_mandatory()
{
    if ( !m_flag_materials || !m_flag_phantom ) return false;
    else return true;
}

void DoseCalculator::m_uncertainty_calculation( ui32 dosel_id_x, ui32 dosel_id_y, ui32 dosel_id_z )
{

    // Relative statistical uncertainty (from Ma et al. PMB 47 2002 p1671) - JB
    //              /                                    \ ^1/2
    //              |    N*Sum(Edep^2) - Sum(Edep)^2     |
    //  relError =  | __________________________________ |
    //              |                                    |
    //              \         (N-1)*Sum(Edep)^2          /
    //
    //   where Edep represents the energy deposit in one hit and N the number of energy deposits (hits)

    // The same without developing - JB (from Walters, Kawrakow and Rogers Med. Phys. 29 2002)
    //                  /                                   \
    //             1    | Sum(Edep^2)      / Sum(Edep) \^2  |
    //  var(x) = _____  | ___________  --  |___________|    |
    //                  |                  |           |    |
    //            N-1   \     N            \    N      /    /
    //
    //  s(x) = ( var )^1/2
    //
    //  relError = s(x) / Sum(Edep)/N
    //

    ui32 index = dosel_id_z * h_dose->slice_nb_dosels + dosel_id_y * h_dose->nb_dosels.x + dosel_id_x;

    f64 N = h_dose->number_of_hits[index];
    f64 sum_E = h_dose->edep[index];

    if ( N > 1 && sum_E != 0.0 )
    {
        f64 sum_E2 = h_dose->edep_squared[index];
        f64 sum2_E = sum_E * sum_E;
        f64 s = ( (N*sum_E2) - sum2_E ) / ( (N-1) * sum2_E );

#ifdef DEBUG
        //assert(s >= 0.0);
        if ( s < 0.0 ) s = 1.0;
#endif
        m_uncertainty_values[ index ] = powf( s, 0.5 );
    }
    else
    {
        m_uncertainty_values[ index ] = 1.0;
    }

}

void DoseCalculator::m_dose_to_water_calculation( ui32 dosel_id_x, ui32 dosel_id_y, ui32 dosel_id_z )
{

    f64 vox_vol = h_dose->dosel_size.x * h_dose->dosel_size.y * h_dose->dosel_size.z;
    f64 density = 1.0 * gram/cm3;
    ui32 index = dosel_id_z * h_dose->slice_nb_dosels + dosel_id_y * h_dose->nb_dosels.x + dosel_id_x;

    m_dose_values[ index ] = h_dose->edep[ index ] / density / vox_vol / gray;

}

void DoseCalculator::m_dose_to_phantom_calculation( ui32 dosel_id_x, ui32 dosel_id_y, ui32 dosel_id_z )
{

    f64 vox_vol = h_dose->dosel_size.x * h_dose->dosel_size.y * h_dose->dosel_size.z;

    // Convert doxel_id into position
    f32 pos_x = ( dosel_id_x * h_dose->dosel_size.x ) - h_dose->offset.x;
    f32 pos_y = ( dosel_id_y * h_dose->dosel_size.y ) - h_dose->offset.y;
    f32 pos_z = ( dosel_id_z * h_dose->dosel_size.z ) - h_dose->offset.z;

    // Convert position into phantom voxel index
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / m_phantom.h_volume->spacing_x;
    ivoxsize.y = 1.0 / m_phantom.h_volume->spacing_y;
    ivoxsize.z = 1.0 / m_phantom.h_volume->spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32 ( ( pos_x + m_phantom.h_volume->off_x ) * ivoxsize.x );
    index_phantom.y = ui32 ( ( pos_y + m_phantom.h_volume->off_y ) * ivoxsize.y );
    index_phantom.z = ui32 ( ( pos_z + m_phantom.h_volume->off_z ) * ivoxsize.z );
    index_phantom.w = index_phantom.z*m_phantom.h_volume->nb_vox_x*m_phantom.h_volume->nb_vox_y
                         + index_phantom.y*m_phantom.h_volume->nb_vox_x
                         + index_phantom.x; // linear index

#ifdef DEBUG
    assert( index_phantom.x < m_phantom.h_volume->nb_vox_x );
    assert( index_phantom.y < m_phantom.h_volume->nb_vox_y );
    assert( index_phantom.z < m_phantom.h_volume->nb_vox_z );
#endif

    // Get density for this voxel
    f64 density = m_materials.h_materials->density[ m_phantom.h_volume->values[ index_phantom.w ] ]; // density given by the material id

    // Compute the dose
    ui32 index = dosel_id_z * h_dose->slice_nb_dosels + dosel_id_y * h_dose->nb_dosels.x + dosel_id_x;

    if ( density > m_dose_min_density )
    {        
        m_dose_values[index] = h_dose->edep[ index ] / density / vox_vol / gray;
    }
    else
    {
        m_dose_values[index] = 0.0f;
    }
}

void DoseCalculator::m_copy_dosemap_to_gpu()
{
    ui32 n = h_dose->tot_nb_dosels;

    /// First, struct allocation
    HANDLE_ERROR( hipMalloc( (void**) &d_dose, sizeof( DoseData ) ) );

    /// Device pointers allocation
    // Data
    f64 *edep;
    HANDLE_ERROR( hipMalloc((void**) &edep, n*sizeof(f64)) );
    f64 *edep_squared;
    HANDLE_ERROR( hipMalloc((void**) &edep_squared, n*sizeof(f64)) );
    ui32 *number_of_hits;
    HANDLE_ERROR( hipMalloc((void**) &number_of_hits, n*sizeof(ui32)) );

    /// Copy host data to device
    HANDLE_ERROR( hipMemcpy( edep, h_dose->edep,
                              n*sizeof(f64), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( edep_squared, h_dose->edep_squared,
                              n*sizeof(f64), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( number_of_hits, h_dose->number_of_hits,
                              n*sizeof(ui32), hipMemcpyHostToDevice ) );

    /// Bind data to the struct
    HANDLE_ERROR( hipMemcpy( &(d_dose->edep), &edep,
                              sizeof(d_dose->edep), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->edep_squared), &edep_squared,
                              sizeof(d_dose->edep_squared), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->number_of_hits), &number_of_hits,
                              sizeof(d_dose->number_of_hits), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->nb_dosels), &(h_dose->nb_dosels),
                              sizeof(d_dose->nb_dosels), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->dosel_size), &(h_dose->dosel_size),
                              sizeof(d_dose->dosel_size), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->inv_dosel_size), &(h_dose->inv_dosel_size),
                              sizeof(d_dose->inv_dosel_size), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->offset), &(h_dose->offset),
                              sizeof(d_dose->offset), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->xmin), &(h_dose->xmin),
                              sizeof(d_dose->xmin), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->xmax), &(h_dose->xmax),
                              sizeof(d_dose->xmax), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->ymin), &(h_dose->ymin),
                              sizeof(d_dose->ymin), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->ymax), &(h_dose->ymax),
                              sizeof(d_dose->ymax), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->zmin), &(h_dose->zmin),
                              sizeof(d_dose->zmin), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->zmax), &(h_dose->zmax),
                              sizeof(d_dose->zmax), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( &(d_dose->tot_nb_dosels), &(h_dose->tot_nb_dosels),
                              sizeof(d_dose->tot_nb_dosels), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(d_dose->slice_nb_dosels), &(h_dose->slice_nb_dosels),
                              sizeof(d_dose->slice_nb_dosels), hipMemcpyHostToDevice ) );

}


void DoseCalculator::m_copy_dosemap_to_cpu()
{
    ui32 n = h_dose->tot_nb_dosels;

    /// Device pointers allocation
    f64 *edep;
    f64 *edep_squared;
    ui32 *number_of_hits;

    /// Bind data struct to new pointer
    HANDLE_ERROR( hipMemcpy( &edep, &(d_dose->edep),
                              sizeof(d_dose->edep), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( &edep_squared, &(d_dose->edep_squared),
                              sizeof(d_dose->edep_squared), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( &number_of_hits, &(d_dose->number_of_hits),
                              sizeof(d_dose->number_of_hits), hipMemcpyDeviceToHost ) );

    /// Copy device data to host
    HANDLE_ERROR( hipMemcpy( h_dose->edep, edep,
                              n*sizeof(f64), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( h_dose->edep_squared, edep_squared,
                              n*sizeof(f64), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( h_dose->number_of_hits, number_of_hits,
                              n*sizeof(ui32), hipMemcpyDeviceToHost ) );

}



/// Class
DoseCalculator::DoseCalculator()
{
    m_dosel_size.x = 0;
    m_dosel_size.y = 0;
    m_dosel_size.z = 0;

    m_offset.x = FLT_MAX;
    m_offset.y = FLT_MAX;
    m_offset.z = FLT_MAX;

    m_nb_of_dosels.x = 0;
    m_nb_of_dosels.y = 0;
    m_nb_of_dosels.z = 0;

    m_xmin = 0; m_xmax = 0;
    m_ymin = 0; m_ymax = 0;
    m_zmin = 0; m_zmax = 0;

    // Min density to compute the dose
    m_dose_min_density = 0.0;

    // Some flags
    m_flag_phantom = false;
    m_flag_materials = false;
    m_flag_dose_calculated = false;
    m_flag_uncertainty_calculated = false;

    // Init the struc
    h_dose = (DoseData*) malloc( sizeof(DoseData) );
    h_dose->edep = nullptr;
    h_dose->edep_squared = nullptr;
    h_dose->number_of_hits = nullptr;
    h_dose->nb_dosels.x = 0;
    h_dose->nb_dosels.y = 0;
    h_dose->nb_dosels.z = 0;
    h_dose->dosel_size.x = 0;
    h_dose->dosel_size.y = 0;
    h_dose->dosel_size.z = 0;
    h_dose->inv_dosel_size.x = 0;
    h_dose->inv_dosel_size.y = 0;
    h_dose->inv_dosel_size.z = 0;
    h_dose->offset.x = 0;
    h_dose->offset.y = 0;
    h_dose->offset.z = 0;
    h_dose->xmin = 0; h_dose->xmax = 0;
    h_dose->ymin = 0; h_dose->ymax = 0;
    h_dose->zmin = 0; h_dose->zmax = 0;
    h_dose->tot_nb_dosels = 0;
    h_dose->slice_nb_dosels = 0;

    // Dose and uncertainty values
    m_dose_values = NULL;
    m_uncertainty_values = NULL;

    // Init other struct
    mh_params = nullptr;
    d_dose = nullptr;

}

DoseCalculator::~DoseCalculator()
{

}

/// Setting

void DoseCalculator::set_dosel_size ( f32 sx, f32 sy, f32 sz )
{
    m_dosel_size = make_f32xyz( sx, sy, sz );
}

void DoseCalculator::set_voi( f32 xmin, f32 xmax, f32 ymin, f32 ymax, f32 zmin, f32 zmax )
{
    m_xmin = xmin; m_xmax = xmax;
    m_ymin = ymin; m_ymax = ymax;
    m_zmin = zmin; m_zmax = zmax;
}

/*
void DoseCalculator::set_offset ( f32 ox, f32 oy, f32 oz )
{
    dose.data_h.ox = ox;
    dose.data_h.oy = oy;
    dose.data_h.oz = oz;
}
*/

void DoseCalculator::set_voxelized_phantom ( VoxelizedPhantom aphantom )
{
    m_phantom = aphantom;
    m_flag_phantom = true;
}

void DoseCalculator::set_materials ( Materials materials )
{
    m_materials = materials;
    m_flag_materials = true;
}

// In g/cm3 ? TODO - JB
void DoseCalculator::set_min_density ( f32 min )
{
    m_dose_min_density = min;
}

VoxVolumeData<f32> * DoseCalculator::get_dose_map()
{
    if ( !m_flag_dose_calculated )
    {
        calculate_dose_to_water();
    }

    VoxVolumeData<f32> *dosemap = new VoxVolumeData<f32>;
    dosemap->nb_vox_x = h_dose->nb_dosels.x;
    dosemap->nb_vox_y = h_dose->nb_dosels.y;
    dosemap->nb_vox_z = h_dose->nb_dosels.z;

    dosemap->off_x = h_dose->offset.x;
    dosemap->off_y = h_dose->offset.y;
    dosemap->off_z = h_dose->offset.z;

    dosemap->spacing_x = h_dose->dosel_size.x;
    dosemap->spacing_y = h_dose->dosel_size.y;
    dosemap->spacing_z = h_dose->dosel_size.z;

    dosemap->number_of_voxels = h_dose->tot_nb_dosels;

    dosemap->xmin = h_dose->xmin;
    dosemap->xmax = h_dose->xmax;
    dosemap->ymin = h_dose->ymin;
    dosemap->ymax = h_dose->ymax;
    dosemap->zmin = h_dose->zmin;
    dosemap->zmax = h_dose->zmax;

    dosemap->values = m_dose_values;

    return dosemap;
}

/// Init
void DoseCalculator::initialize ( GlobalSimulationParametersData *h_params )
{
//     GGcout << " DoseCalculator initialize " << GGendl;
    
    // Check if everything was set properly
    if ( !m_check_mandatory() )
    {
        print_error ( "Dose calculator, phantom and materials are not set?!" );
        exit_simulation();
    }

    // Copy params
    mh_params = h_params;

    /// Compute dosemap parameters /////////////////////////////

    // Select a doxel size
    if ( m_dosel_size.x > 0.0 && m_dosel_size.y > 0.0 && m_dosel_size.z > 0.0 )
    {
        h_dose->dosel_size = m_dosel_size;
        h_dose->inv_dosel_size = fxyz_inv( m_dosel_size );
    }
    else
    {
        h_dose->dosel_size = make_f32xyz( m_phantom.h_volume->spacing_x,
                                       m_phantom.h_volume->spacing_y,
                                       m_phantom.h_volume->spacing_z );
        h_dose->inv_dosel_size = fxyz_inv( h_dose->dosel_size );
    }

/*
    // Compute min-max volume of interest
    f32xyz phan_size = make_f32xyz( m_phantom.h_volume->nb_vox_x * m_phantom.h_volume->spacing_x,
                                    m_phantom.h_volume->nb_vox_y * m_phantom.h_volume->spacing_y,
                                    m_phantom.h_volume->nb_vox_z * m_phantom.h_volume->spacing_z );
    f32xyz half_phan_size = fxyz_scale( phan_size, 0.5f );
    f32 phan_xmin = -half_phan_size.x; f32 phan_xmax = half_phan_size.x;
    f32 phan_ymin = -half_phan_size.y; f32 phan_ymax = half_phan_size.y;
    f32 phan_zmin = -half_phan_size.z; f32 phan_zmax = half_phan_size.z;
    */

    // Select a min-max VOI
    if ( !m_xmin && !m_xmax && !m_ymin && !m_ymax && !m_zmin && !m_zmax )
    {
        h_dose->xmin = m_phantom.h_volume->xmin;
        h_dose->xmax = m_phantom.h_volume->xmax;
        h_dose->ymin = m_phantom.h_volume->ymin;
        h_dose->ymax = m_phantom.h_volume->ymax;
        h_dose->zmin = m_phantom.h_volume->zmin;
        h_dose->zmax = m_phantom.h_volume->zmax;
    }
    else
    {
        h_dose->xmin = m_xmin;
        h_dose->xmax = m_xmax;
        h_dose->ymin = m_ymin;
        h_dose->ymax = m_ymax;
        h_dose->zmin = m_zmin;
        h_dose->zmax = m_zmax;
    }

    // Get the current dimension of the dose map
    f32xyz cur_dose_size = make_f32xyz( h_dose->xmax - h_dose->xmin,
                                        h_dose->ymax - h_dose->ymin,
                                        h_dose->zmax - h_dose->zmin );

    // New nb of voxels
    h_dose->nb_dosels.x = floor( cur_dose_size.x / h_dose->dosel_size.x );
    h_dose->nb_dosels.y = floor( cur_dose_size.y / h_dose->dosel_size.y );
    h_dose->nb_dosels.z = floor( cur_dose_size.z / h_dose->dosel_size.z );
    h_dose->slice_nb_dosels = h_dose->nb_dosels.x * h_dose->nb_dosels.y;
    h_dose->tot_nb_dosels = h_dose->slice_nb_dosels * h_dose->nb_dosels.z;

    // Compute the new size (due to integer nb of doxels)
    f32xyz new_dose_size = fxyz_mul( h_dose->dosel_size, cast_ui32xyz_to_f32xyz( h_dose->nb_dosels ) );

    if ( new_dose_size.x <= 0.0 || new_dose_size.y <= 0.0 || new_dose_size.z <= 0.0 )
    {
        GGcerr << "Dosemap dimension: "
               << new_dose_size.x << " "
               << new_dose_size.y << " "
               << new_dose_size.z << GGendl;
        exit_simulation();
    }

    // Compute new min and max after voxel alignment // TODO: Check here, offset is not considered? - JB
    f32xyz half_delta_size = fxyz_scale( fxyz_sub( cur_dose_size, new_dose_size ), 0.5f );

    h_dose->xmin += half_delta_size.x;
    h_dose->xmax -= half_delta_size.x;

    h_dose->ymin += half_delta_size.y;
    h_dose->ymax -= half_delta_size.y;

    h_dose->zmin += half_delta_size.z;
    h_dose->zmax -= half_delta_size.z;

    // Get the new offset
    h_dose->offset.x = m_phantom.h_volume->off_x - ( h_dose->xmin - m_phantom.h_volume->xmin );
    h_dose->offset.y = m_phantom.h_volume->off_y - ( h_dose->ymin - m_phantom.h_volume->ymin );
    h_dose->offset.z = m_phantom.h_volume->off_z - ( h_dose->zmin - m_phantom.h_volume->zmin );

    // Init dose map
    h_dose->edep = (f64*)malloc( h_dose->tot_nb_dosels*sizeof(f64) );
    h_dose->edep_squared = (f64*)malloc( h_dose->tot_nb_dosels*sizeof(f64) );
    h_dose->number_of_hits = (ui32*)malloc( h_dose->tot_nb_dosels*sizeof(ui32) );
    ui32 i=0; while (i < h_dose->tot_nb_dosels)
    {
        h_dose->edep[i] = 0.0;
        h_dose->edep_squared[i] = 0.0;
        h_dose->number_of_hits[i] = 0;
        ++i;
    }

    //////////////////////////////////////////////////////////

    // Host allocation
    m_dose_values = (f32*)malloc( h_dose->tot_nb_dosels * sizeof(f32) );
    m_uncertainty_values = (f32*)malloc( h_dose->tot_nb_dosels * sizeof(f32) );

    // Device allocation and copy
    m_copy_dosemap_to_gpu();

}

void DoseCalculator::calculate_dose_to_water()
{
    GGcout << "Compute dose to water" << GGendl;         

    m_copy_dosemap_to_cpu();

    // Calculate the dose to water and the uncertainty
    for ( ui32 iz=0; iz < h_dose->nb_dosels.z; iz++ )
    {
        for ( ui32 iy=0; iy < h_dose->nb_dosels.y; iy++ )
        {
            for ( ui32 ix=0; ix < h_dose->nb_dosels.x; ix++ )
            {
                m_dose_to_water_calculation( ix, iy, iz );
                m_uncertainty_calculation( ix, iy, iz );
            }
        }
    }

    m_flag_dose_calculated = true;
    m_flag_uncertainty_calculated = true;
}

void DoseCalculator::calculate_dose_to_medium()
{
    // Check if everything was set properly
    if ( !m_flag_materials || !m_flag_phantom )
    {
        GGcerr << "Dose calculator, phantom and materials data are required!" << GGendl;
        exit_simulation();
    }

    GGcout << "Compute dose to medium" << GGendl;

    m_copy_dosemap_to_cpu();

    // Calculate the dose to phantom and the uncertainty
    for ( ui32 iz=0; iz < h_dose->nb_dosels.z; iz++ )
    {
        for ( ui32 iy=0; iy < h_dose->nb_dosels.y; iy++ )
        {
            for ( ui32 ix=0; ix < h_dose->nb_dosels.x; ix++ )
            {
                m_dose_to_phantom_calculation( ix, iy, iz );
                m_uncertainty_calculation( ix, iy, iz );
            }
        }
    }
    
    m_flag_dose_calculated = true;
    m_flag_uncertainty_calculated = true;

}


/*
void DoseCalculator::m_cpu_malloc_dose()
{
    dose.data_h.edep = new f64[dose.data_h.tot_nb_doxels];
    dose.data_h.dose = new f64[dose.data_h.tot_nb_doxels];
    dose.data_h.edep_squared = new f64[dose.data_h.tot_nb_doxels];
    dose.data_h.number_of_hits = new ui32[dose.data_h.tot_nb_doxels];
    dose.data_h.uncertainty = new f64[dose.data_h.tot_nb_doxels];
}
*/
/*
void DoseCalculator::m_gpu_malloc_dose()
{
    // GPU allocation
    HANDLE_ERROR ( hipMalloc ( ( void** ) &dose.data_d.edep,           dose.data_h.tot_nb_doxels * sizeof ( f64 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &dose.data_d.dose,           dose.data_h.tot_nb_doxels * sizeof ( f64 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &dose.data_d.edep_squared,   dose.data_h.tot_nb_doxels * sizeof ( f64 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &dose.data_d.number_of_hits, dose.data_h.tot_nb_doxels * sizeof ( ui32 ) ) );
    HANDLE_ERROR ( hipMalloc ( ( void** ) &dose.data_d.uncertainty,    dose.data_h.tot_nb_doxels * sizeof ( f64 ) ) );
    
//     GGcout << "DoseCalculator : GPU allocation " << dose.data_h.nb_of_voxels << GGendl;
    
}
*/

/*
void DoseCalculator::m_copy_dose_cpu2gpu()
{

    dose.data_d.nb_doxels = dose.data_h.nb_doxels;
    dose.data_d.doxel_size = dose.data_h.doxel_size;
    dose.data_d.inv_doxel_size = dose.data_h.inv_doxel_size;
    dose.data_d.offset = dose.data_h.offset;
    dose.data_d.tot_nb_doxels = dose.data_h.tot_nb_doxels;
    dose.data_d.slice_nb_doxels = dose.data_h.slice_nb_doxels;

    dose.data_d.xmin = dose.data_h.xmin;
    dose.data_d.xmax = dose.data_h.xmax;
    dose.data_d.ymin = dose.data_h.ymin;
    dose.data_d.ymax = dose.data_h.ymax;
    dose.data_d.zmin = dose.data_h.zmin;
    dose.data_d.zmax = dose.data_h.zmax;

    // Copy values to GPU arrays
    HANDLE_ERROR ( hipMemcpy ( dose.data_d.edep,           dose.data_h.edep,           sizeof ( f64 ) *dose.data_h.tot_nb_doxels,  hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_d.dose,           dose.data_h.dose,           sizeof ( f64 ) *dose.data_h.tot_nb_doxels,  hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_d.edep_squared,   dose.data_h.edep_squared,   sizeof ( f64 ) *dose.data_h.tot_nb_doxels,  hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_d.number_of_hits, dose.data_h.number_of_hits, sizeof ( ui32 ) *dose.data_h.tot_nb_doxels, hipMemcpyHostToDevice ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_d.uncertainty,    dose.data_h.uncertainty,    sizeof ( f64 ) *dose.data_h.tot_nb_doxels,  hipMemcpyHostToDevice ) );
    
}
*/

/*
void DoseCalculator::m_copy_dose_gpu2cpu()
{
//     dose.data_h.nx = dose.data_d.nx;
//     dose.data_h.ny = dose.data_d.ny;
//     dose.data_h.nz = dose.data_d.nz;
// 
//     dose.data_h.spacing_x = dose.data_d.spacing_x;
//     dose.data_h.spacing_y = dose.data_d.spacing_y;
//     dose.data_h.spacing_z = dose.data_d.spacing_z;
// 
//     dose.data_h.ox = dose.data_d.ox;
//     dose.data_h.oy = dose.data_d.oy;
//     dose.data_h.oz = dose.data_d.oz;

//     dose.data_h.nb_of_voxels = dose.data_d.nb_of_voxels;

//     GGcout << "DoseCalculator : Copy to GPU " << dose.data_h.nb_of_voxels << GGendl;
    // Copy values to GPU arrays
    HANDLE_ERROR ( hipMemcpy ( dose.data_h.edep,           dose.data_d.edep,           sizeof ( f64  ) *dose.data_h.tot_nb_doxels,  hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_h.dose,           dose.data_d.dose,           sizeof ( f64  ) *dose.data_h.tot_nb_doxels,  hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_h.edep_squared,   dose.data_d.edep_squared,   sizeof ( f64  ) *dose.data_h.tot_nb_doxels,  hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_h.number_of_hits, dose.data_d.number_of_hits, sizeof ( ui32 ) *dose.data_h.tot_nb_doxels,  hipMemcpyDeviceToHost ) );
    HANDLE_ERROR ( hipMemcpy ( dose.data_h.uncertainty,    dose.data_d.uncertainty,    sizeof ( f64  ) *dose.data_h.tot_nb_doxels,  hipMemcpyDeviceToHost ) );

}
*/


void DoseCalculator::write ( std::string filename )
{
    // Update host dose data
    m_copy_dosemap_to_cpu();

    // Create an IO object
    ImageIO *im_io = new ImageIO;

    std::string format = im_io->get_extension( filename );
    filename = im_io->get_filename_without_extension( filename );

    // Convert Edep and EdepSquared from f64 to f32
    ui32 tot = h_dose->nb_dosels.x*h_dose->nb_dosels.y*h_dose->nb_dosels.z;
    f32 *f32edep = new f32[ tot ];
    f32 *f32edepSq = new f32[ tot ];
    ui32 i=0; while ( i < tot )
    {
        f32edep[ i ] = (f32)h_dose->edep[ i ];
        f32edepSq[ i ] = (f32)h_dose->edep_squared[ i ];
        ++i;
    }

    // Get output name
    std::string edep_out( filename + "-Edep." + format );
    std::string edep_squared_out( filename + "-EdepSquared." + format );
    std::string uncer_out( filename + "-Uncertainty." + format );
    std::string hit_out( filename + "-Hit." + format );
    std::string dose_out( filename + "-Dose." + format );

    // Export Edep and EdepSquared
    im_io->write_3D( edep_out, f32edep, h_dose->nb_dosels, h_dose->offset, h_dose->dosel_size );
    im_io->write_3D( edep_squared_out, f32edepSq, h_dose->nb_dosels, h_dose->offset, h_dose->dosel_size );

    // Export uncertainty
    if ( !m_flag_uncertainty_calculated )
    {
        // Calculate the dose to phantom and the uncertainty
        for ( ui32 iz=0; iz < h_dose->nb_dosels.z; iz++ )
        {
            for ( ui32 iy=0; iy < h_dose->nb_dosels.y; iy++ )
            {
                for ( ui32 ix=0; ix < h_dose->nb_dosels.x; ix++ )
                {
                    m_uncertainty_calculation( ix, iy, iz );
                }
            }
        }
    }

    // Export uncertainty and hits
    im_io->write_3D( uncer_out, m_uncertainty_values, h_dose->nb_dosels, h_dose->offset, h_dose->dosel_size );
    im_io->write_3D( hit_out, h_dose->number_of_hits, h_dose->nb_dosels, h_dose->offset, h_dose->dosel_size );

    // Export dose
    if ( m_flag_dose_calculated )
    {
        im_io->write_3D( dose_out, m_dose_values, h_dose->nb_dosels, h_dose->offset, h_dose->dosel_size );
    }

    delete im_io;
    delete f32edep, f32edepSq;

}



#endif
