#include "hip/hip_runtime.h"
// GGEMS Copyright (C) 2015

/*!
 * \file dose_calculator.cu
 * \brief
 * \author Y. Lemaréchal
 * \author J. Bert <bert.jul@gmail.com>
 * \version 0.1
 * \date 2 december 2015
 *
 *
 *
 */

#ifndef DOSE_CALCULATOR_CU
#define DOSE_CALCULATOR_CU

#include "dose_calculator.cuh"

/// CPU&GPU functions
__host__ __device__ void dose_record_standard(DoseData dose, f32 Edep, f32xyz pos) {

    // Defined index phantom
    f32xyz ivoxsize;
    ivoxsize.x = 1.0 / dose.spacing_x;
    ivoxsize.y = 1.0 / dose.spacing_y;
    ivoxsize.z = 1.0 / dose.spacing_z;
    ui32xyzw index_phantom;
    index_phantom.x = ui32( (pos.x+dose.ox) * ivoxsize.x );
    index_phantom.y = ui32( (pos.y+dose.oy) * ivoxsize.y );
    index_phantom.z = ui32( (pos.z+dose.oz) * ivoxsize.z );
    index_phantom.w = index_phantom.z*dose.nx*dose.ny
                      + index_phantom.y*dose.nx
                      + index_phantom.x; // linear index

    // Score dosemap
    ggems_atomic_add(dose.edep, index_phantom.w, Edep);
    ggems_atomic_add(dose.edep_squared, index_phantom.w, Edep*Edep);
    ggems_atomic_add(dose.number_of_hits, index_phantom.w, ui32(1));

}

/// Class
DoseCalculator::DoseCalculator()
{

    m_dose.data_h.nx = 0;
    m_dose.data_h.ny = 0;
    m_dose.data_h.nz = 0;
    
    // Voxel size per dimension
    m_dose.data_h.spacing_x = 0.0;
    m_dose.data_h.spacing_y = 0.0;
    m_dose.data_h.spacing_z = 0.0;
    
    // Offset
    m_dose.data_h.ox = 0.0;
    m_dose.data_h.oy = 0.0;
    m_dose.data_h.oz = 0.0;
        
    m_dose.data_h.edep = NULL;
    m_dose.data_h.dose = NULL;
    m_dose.data_h.edep_squared = NULL;
    m_dose.data_h.number_of_hits = NULL;
    
    m_dose.data_h.uncertainty = NULL;
}

DoseCalculator::~DoseCalculator()
{

    delete [] m_dose.data_h.edep ;
    delete [] m_dose.data_h.dose ;
    delete [] m_dose.data_h.edep_squared ;
    delete [] m_dose.data_h.number_of_hits ;
    delete [] m_dose.data_h.uncertainty ;

}

/// Setting
void DoseCalculator::set_size_in_voxel(ui32 nx, ui32 ny, ui32 nz) {
    m_dose.data_h.nx = nx;
    m_dose.data_h.ny = ny;
    m_dose.data_h.nz = nz;
}

void DoseCalculator::set_voxel_size(f32 sx, f32 sy, f32 sz) {
    m_dose.data_h.spacing_x = sx;
    m_dose.data_h.spacing_y = sy;
    m_dose.data_h.spacing_z = sz;
}

void DoseCalculator::set_offset(f32 ox, f32 oy, f32 oz) {
    m_dose.data_h.ox = ox;
    m_dose.data_h.oy = oy;
    m_dose.data_h.oz = oz;
}

/// Init
void DoseCalculator::initialize(GlobalSimulationParameters params)
{

    // Check if everything was set properly
    if ( !m_check_mandatory() ) {
        print_error("Dose calculator, size or spacing are set to zero?!");
        exit_simulation();
    }

    // Initi nb of voxels
    m_dose.data_h.nb_of_voxels = m_dose.data_h.nx*m_dose.data_h.ny*m_dose.data_h.nz;

    // CPU allocation
    m_cpu_malloc_dose();

    // Init values to 0 or 1
    for(int i = 0; i< m_dose.data_h.nb_of_voxels ; i++)
    {

        m_dose.data_h.edep[i] = 0.0;
        m_dose.data_h.dose[i] = 0.0;
        m_dose.data_h.edep_squared[i] = 0.0;
        m_dose.data_h.number_of_hits[i] = 0.0;
        m_dose.data_h.uncertainty[i] = 1.0;

    }

    // Copy to GPU if required
    if (params.data_h.device_target == GPU_DEVICE) {
        // GPU allocation
        m_gpu_malloc_dose();
        // Copy data to the GPU
        m_copy_dose_cpu2gpu();
    }

}


// __host__ __device__ void DoseCalculator::store_energy_and_energy2(ui32 voxel, f32 energy)
// {
// 
// #if defined(__CUDA_ARCH__)
//  ggems_atomic_add(dose_d.edep, voxel, energy);
//  ggems_atomic_add(dose_d.edep_squared, voxel, energy*energy);
// #else
//  ggems_atomic_add(dose_h.edep, voxel, energy);
//  ggems_atomic_add(dose_h.edep_squared, voxel, energy*energy);
// #endif
// 
// }



/*
void DoseCalculator::write_dosi(std::string histname)
{


ImageReader::record3Dimage(  histname,  
dose_h.edep,
make_f32xyz(dose_h.x0,dose_h.y0,dose_h.z0), 
make_f32xyz(dose_h.spacing_x,dose_h.spacing_y,dose_h.spacing_z),
make_i32xyz(dose_h.nx,dose_h.ny,dose_h.nz) ,
false);

}
*/

/// Private
bool DoseCalculator::m_check_mandatory() {
    if (m_dose.data_h.nx == 0 || m_dose.data_h.ny == 0 || m_dose.data_h.nz == 0 ||
        m_dose.data_h.spacing_x == 0 || m_dose.data_h.spacing_y == 0 || m_dose.data_h.spacing_z == 0) return false;
    else return true;
}

void DoseCalculator::m_cpu_malloc_dose() {
    m_dose.data_h.edep = new f32[m_dose.data_h.nb_of_voxels];
    m_dose.data_h.dose = new f32[m_dose.data_h.nb_of_voxels];
    m_dose.data_h.edep_squared = new f32[m_dose.data_h.nb_of_voxels];
    m_dose.data_h.number_of_hits = new ui32[m_dose.data_h.nb_of_voxels];
    m_dose.data_h.uncertainty = new f32[m_dose.data_h.nb_of_voxels];
}

void DoseCalculator::m_gpu_malloc_dose() {
    // GPU allocation
    HANDLE_ERROR( hipMalloc((void**) &m_dose.data_d.edep,           m_dose.data_h.nb_of_voxels * sizeof(f32)));
    HANDLE_ERROR( hipMalloc((void**) &m_dose.data_d.dose,           m_dose.data_h.nb_of_voxels * sizeof(f32)));
    HANDLE_ERROR( hipMalloc((void**) &m_dose.data_d.edep_squared,   m_dose.data_h.nb_of_voxels * sizeof(f32)));
    HANDLE_ERROR( hipMalloc((void**) &m_dose.data_d.number_of_hits, m_dose.data_h.nb_of_voxels * sizeof(ui32)));
    HANDLE_ERROR( hipMalloc((void**) &m_dose.data_d.uncertainty,    m_dose.data_h.nb_of_voxels * sizeof(f32)));
}

void DoseCalculator::m_copy_dose_cpu2gpu()
{
    m_dose.data_d.nx = m_dose.data_h.nx;
    m_dose.data_d.ny = m_dose.data_h.ny;
    m_dose.data_d.nz = m_dose.data_h.nz;

    m_dose.data_d.spacing_x = m_dose.data_h.spacing_x;
    m_dose.data_d.spacing_y = m_dose.data_h.spacing_y;
    m_dose.data_d.spacing_z = m_dose.data_h.spacing_z;

    m_dose.data_d.ox = m_dose.data_h.ox;
    m_dose.data_d.oy = m_dose.data_h.oy;
    m_dose.data_d.oz = m_dose.data_h.oz;

    m_dose.data_d.nb_of_voxels = m_dose.data_h.nb_of_voxels;

    // Copy values to GPU arrays
    HANDLE_ERROR( hipMemcpy(m_dose.data_d.edep,           m_dose.data_h.edep,           sizeof(f32)*m_dose.data_h.nb_of_voxels, hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(m_dose.data_d.dose,           m_dose.data_h.dose,           sizeof(f32)*m_dose.data_h.nb_of_voxels, hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(m_dose.data_d.edep_squared,   m_dose.data_h.edep_squared,   sizeof(f32)*m_dose.data_h.nb_of_voxels, hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(m_dose.data_d.number_of_hits, m_dose.data_h.number_of_hits, sizeof(ui32)*m_dose.data_h.nb_of_voxels, hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(m_dose.data_d.uncertainty,    m_dose.data_h.uncertainty,    sizeof(f32)*m_dose.data_h.nb_of_voxels, hipMemcpyHostToDevice));
}

















#endif
